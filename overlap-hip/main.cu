#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <hip/hip_runtime.h>

__global__
void incKernel(int *g_out, const int *g_in, int N, int inner_reps) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    for (int i = 0; i < inner_reps; ++i) {
      g_out[idx] = (i == 0 ? g_in[idx] : g_out[idx]) + 1;
    }
  }
}

#define STREAM_COUNT 4

int *h_data_in[STREAM_COUNT];
int *d_data_in[STREAM_COUNT];

int *h_data_out[STREAM_COUNT];
int *d_data_out[STREAM_COUNT];

hipStream_t stream[STREAM_COUNT];

int N = 1 << 22;
int nreps = 10;  // number of times each experiment is repeated
int inner_reps = 5;  // loop iterations in the GPU kernel

int memsize;

dim3 block (256, 1, 1);
dim3 grid (N/256, 1, 1);

float processWithStreams(int streams_used);
bool check();


int main(int argc, char *argv[]) {

  printf("Length of the array = %d\n", N);

  memsize = N * sizeof(int);

  // Allocate resources
  for (int i = 0; i < STREAM_COUNT; ++i) {
    hipHostMalloc((void**)&h_data_in[i], memsize, hipHostMallocDefault);
    hipHostMalloc((void**)&h_data_out[i], memsize, hipHostMallocDefault);

    hipMalloc(&d_data_in[i], memsize);
    hipMemset(d_data_in[i], 0, memsize);

    hipMalloc(&d_data_out[i], memsize);

    hipStreamCreate(&stream[i]);
  }

  // initialize host memory
  for (int i = 0; i < STREAM_COUNT; ++i) {
    memset(h_data_in[i], 0, memsize);
  }

  // Process pipelined work
  float serial_time = processWithStreams(1);
  float overlap_time = processWithStreams(STREAM_COUNT);

  printf("\nAverage measured timings over %d repetitions:\n", nreps);
  printf(" Avg. time when execution fully serialized\t: %f ms\n",
         serial_time / nreps);
  printf(" Avg. time when overlapped using %d streams\t: %f ms\n", STREAM_COUNT,
         overlap_time / nreps);
  printf(" Avg. speedup gained (serialized - overlapped)\t: %f\n",
         (serial_time - overlap_time) / nreps);

  printf("\nMeasured throughput:\n");
  printf(" Fully serialized execution\t\t: %f GB/s\n",
         (nreps * (memsize * 2e-6)) / serial_time);
  printf(" Overlapped using %d streams\t\t: %f GB/s\n", STREAM_COUNT,
         (nreps * (memsize * 2e-6)) / overlap_time);

  // Verify the results, we will use the results for final output
  bool bResults = check();
  printf("\n%s\n", bResults ? "PASS" : "FAIL");

  // Free resources
  for (int i = 0; i < STREAM_COUNT; ++i) {
    hipHostFree(h_data_in[i]);
    hipFree(d_data_in[i]);

    hipHostFree(h_data_out[i]);
    hipFree(d_data_out[i]);

    hipStreamDestroy(stream[i]);
  }

  // Test result
  exit(bResults ? EXIT_SUCCESS : EXIT_FAILURE);
}

float processWithStreams(int streams_used) {
  int current_stream = 0;

  auto start = std::chrono::steady_clock::now();

  // Do processing in a loop
  //
  // Note: All memory commands are processed in the order they are issued,
  // independent of the stream they are enqueued in. Hence the pattern by
  // which the copy and kernel commands are enqueued in the stream
  // has an influence on the achieved overlap.

  for (int i = 0; i < nreps; ++i) {
    int next_stream = (current_stream + 1) % streams_used;

    // Process current frame
    hipLaunchKernelGGL(incKernel, grid, block, 0, stream[current_stream], 
        d_data_out[current_stream], d_data_in[current_stream], N, inner_reps);

    // Upload next frame
    hipMemcpyAsync(d_data_in[next_stream], h_data_in[next_stream], memsize,
                    hipMemcpyHostToDevice, stream[next_stream]);

    // Download current frame
    hipMemcpyAsync(h_data_out[current_stream], d_data_out[current_stream], memsize,
                    hipMemcpyDeviceToHost, stream[current_stream]);

    current_stream = next_stream;
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

  return (time * 1e-6f); // milliseconds
}

bool check() {
  bool passed = true;

  for (int j = 0; j < STREAM_COUNT; ++j) {
    for (int i = 0; i < N; ++i) {
      passed &= (h_data_out[j][i] == inner_reps);
    }
  }
  return passed;
}
