#include "hip/hip_runtime.h"
//  Copyright 2013 Google Inc. All Rights Reserved.
//
//  Licensed under the Apache License, Version 2.0 (the "License");
//  you may not use this file except in compliance with the License.
//  You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <pthread.h>

#include "cbow.h"

const int vocab_hash_size = 30000000;  // Maximum 30 * 0.7 = 21M words in the vocabulary

// Precision of float numbers

struct vocab_word {
  int cn;
  int *point;
  char *word, *code, codelen;
};

char train_file[MAX_STRING], output_file[MAX_STRING];
char save_vocab_file[MAX_STRING], read_vocab_file[MAX_STRING];
struct vocab_word *vocab;
int binary = 0, cbow = 1, debug_mode = 2, window = 5, min_count = 5, num_threads = 12, min_reduce = 1;
int *vocab_hash;
int vocab_max_size = 1000, vocab_size = 0, layer1_size = 100,  layer1_size_aligned;;
long long train_words = 0, word_count_actual = 0, file_size = 0;
int iter = 5,  classes = 0;
real alpha = 0.025, starting_alpha, sample = 1e-3;
real *syn0;
int * sen;
clock_t start;

int hs = 0, negative = 5;
int table_size = 1e8;
int *table;

void InitUnigramTable() {
  int a, i;
  double train_words_pow = 0;
  double d1, power = 0.75;
  table = (int *)malloc(table_size * sizeof(int));
  for (a = 0; a < vocab_size; a++) train_words_pow += pow(vocab[a].cn, power);
  i = 0;
  d1 = pow(vocab[i].cn, power) / train_words_pow;
  for (a = 0; a < table_size; a++) {
    table[a] = i;
    if (a / (double)table_size > d1) {
      i++;
      d1 += pow(vocab[i].cn, power) / train_words_pow;
    }
    if (i >= vocab_size) i = vocab_size - 1;
  }
}

// Reads a single word from a file, assuming space + tab + EOL to be word boundaries
void ReadWord(char *word, FILE *fin) {
  int a = 0, ch;
  while (!feof(fin)) {
    ch = fgetc(fin);
    if (ch == 13) continue;
    if ((ch == ' ') || (ch == '\t') || (ch == '\n')) {
      if (a > 0) {
        if (ch == '\n') ungetc(ch, fin);
        break;
      }
      if (ch == '\n') {
        strcpy(word, (char *)"</s>");
        return;
      } else continue;
    }
    word[a] = ch;
    a++;
    if (a >= MAX_STRING - 1) a--;   // Truncate too long words
  }
  word[a] = 0;
}

// Returns hash value of a word
int GetWordHash(char *word) {
  unsigned int a, hash = 0;
  for (a = 0; a < strlen(word); a++) hash = hash * 257 + word[a];
  hash = hash % vocab_hash_size;
  return hash;
}

// Returns position of a word in the vocabulary; if the word is not found, returns -1
int SearchVocab(char *word) {
  unsigned int hash = GetWordHash(word);
  while (1) {
    if (vocab_hash[hash] == -1) return -1;
    if (!strcmp(word, vocab[vocab_hash[hash]].word)) return vocab_hash[hash];
    hash = (hash + 1) % vocab_hash_size;
  }
  return -1;
}

// Reads a word and returns its index in the vocabulary
int ReadWordIndex(FILE *fin) {
  char word[MAX_STRING];
  ReadWord(word, fin);
  if (feof(fin)) return -1;
  return SearchVocab(word);
}

// Adds a word to the vocabulary
int AddWordToVocab(char *word) {
  unsigned int hash, length = strlen(word) + 1;
  if (length > MAX_STRING) length = MAX_STRING;
  vocab[vocab_size].word = (char *)calloc(length, sizeof(char));
  strcpy(vocab[vocab_size].word, word);
  vocab[vocab_size].cn = 0;
  vocab_size++;
  // Reallocate memory if needed
  if (vocab_size + 2 >= vocab_max_size) {
    vocab_max_size += 1000;
    vocab = (struct vocab_word *)realloc(vocab, vocab_max_size * sizeof(struct vocab_word));
  }
  hash = GetWordHash(word);
  while (vocab_hash[hash] != -1) hash = (hash + 1) % vocab_hash_size;
  vocab_hash[hash] = vocab_size - 1;
  return vocab_size - 1;
}

// Used later for sorting by word counts
int VocabCompare(const void *a, const void *b) {
  return ((struct vocab_word *)b)->cn - ((struct vocab_word *)a)->cn;
}

// Sorts the vocabulary by frequency using word counts
void SortVocab() {
  int a, size;
  unsigned int hash;
  // Sort the vocabulary and keep </s> at the first position
  qsort(&vocab[1], vocab_size - 1, sizeof(struct vocab_word), VocabCompare);
  for (a = 0; a < vocab_hash_size; a++) vocab_hash[a] = -1;
  size = vocab_size;
  train_words = 0;
  for (a = 0; a < size; a++) {
    // Words occuring less than min_count times will be discarded from the vocab
    if ((vocab[a].cn < min_count) && (a != 0)) {
      vocab_size--;
      free(vocab[a].word);
    } else {
      // Hash will be re-computed, as after the sorting it is not actual
      hash=GetWordHash(vocab[a].word);
      while (vocab_hash[hash] != -1) hash = (hash + 1) % vocab_hash_size;
      vocab_hash[hash] = a;
      train_words += vocab[a].cn;
    }
  }
  vocab = (struct vocab_word *)realloc(vocab, (vocab_size + 1) * sizeof(struct vocab_word));
  // Allocate memory for the binary tree construction
  for (a = 0; a < vocab_size; a++) {
    vocab[a].code = (char *)calloc(MAX_CODE_LENGTH, sizeof(char));
    vocab[a].point = (int *)calloc(MAX_CODE_LENGTH, sizeof(int));
  }
}

// Reduces the vocabulary by removing infrequent tokens
void ReduceVocab() {
  int a, b = 0;
  unsigned int hash;
  for (a = 0; a < vocab_size; a++) if (vocab[a].cn > min_reduce) {
    vocab[b].cn = vocab[a].cn;
    vocab[b].word = vocab[a].word;
    b++;
  } else free(vocab[a].word);
  vocab_size = b;
  for (a = 0; a < vocab_hash_size; a++) vocab_hash[a] = -1;
  for (a = 0; a < vocab_size; a++) {
    // Hash will be re-computed, as it is not actual
    hash = GetWordHash(vocab[a].word);
    while (vocab_hash[hash] != -1) hash = (hash + 1) % vocab_hash_size;
    vocab_hash[hash] = a;
  }
  fflush(stdout);
  min_reduce++;
}

// Create binary Huffman tree using the word counts
// Frequent words will have short uniqe binary codes
void CreateBinaryTree() {
  int a, b, i, min1i, min2i, pos1, pos2, point[MAX_CODE_LENGTH];
  char code[MAX_CODE_LENGTH];
  int *count = (int *)calloc(vocab_size * 2 + 1, sizeof(int));
  int *binary = (int *)calloc(vocab_size * 2 + 1, sizeof(int));
  int *parent_node = (int *)calloc(vocab_size * 2 + 1, sizeof(int));
  for (a = 0; a < vocab_size; a++) count[a] = vocab[a].cn;
  for (a = vocab_size; a < vocab_size * 2; a++) count[a] = 1e9;
  pos1 = vocab_size - 1;
  pos2 = vocab_size;
  // Following algorithm constructs the Huffman tree by adding one node at a time
  for (a = 0; a < vocab_size - 1; a++) {
    // First, find two smallest nodes 'min1, min2'
    if (pos1 >= 0) {
      if (count[pos1] < count[pos2]) {
        min1i = pos1;
        pos1--;
      } else {
        min1i = pos2;
        pos2++;
      }
    } else {
      min1i = pos2;
      pos2++;
    }
    if (pos1 >= 0) {
      if (count[pos1] < count[pos2]) {
        min2i = pos1;
        pos1--;
      } else {
        min2i = pos2;
        pos2++;
      }
    } else {
      min2i = pos2;
      pos2++;
    }
    count[vocab_size + a] = count[min1i] + count[min2i];
    parent_node[min1i] = vocab_size + a;
    parent_node[min2i] = vocab_size + a;
    binary[min2i] = 1;
  }
  // Now assign binary code to each vocabulary word
  for (a = 0; a < vocab_size; a++) {
    b = a;
    i = 0;
    while (1) {
      code[i] = binary[b];
      point[i] = b;
      i++;
      b = parent_node[b];
      if (b == vocab_size * 2 - 2) break;
    }
    vocab[a].codelen = i;
    vocab[a].point[0] = vocab_size - 2;
    for (b = 0; b < i; b++) {
      vocab[a].code[i - b - 1] = code[b];
      vocab[a].point[i - b] = point[b] - vocab_size;
    }
  }
  free(count);
  free(binary);
  free(parent_node);
}

void LearnVocabFromTrainFile() {
  char word[MAX_STRING];
  FILE *fin;
  int a, i;
  for (a = 0; a < vocab_hash_size; a++) vocab_hash[a] = -1;
  fin = fopen(train_file, "rb");
  if (fin == NULL) {
    printf("ERROR: training data file not found!\n");
    exit(1);
  }
  vocab_size = 0;
  AddWordToVocab((char *)"</s>");
  while (1) {
    ReadWord(word, fin);
    if (feof(fin)) break;
    train_words++;
    if ((debug_mode > 1) && (train_words % 100000 == 0)) {
      printf("%lldK%c", train_words / 1000, 13);
      fflush(stdout);
    }
    i = SearchVocab(word);
    if (i == -1) {
      a = AddWordToVocab(word);
      vocab[a].cn = 1;
    } else vocab[i].cn++;
    if (vocab_size > vocab_hash_size * 0.7) ReduceVocab();
  }
  SortVocab();
  if (debug_mode > 0) {
    printf("Vocab size: %d\n", vocab_size);
    printf("Words in train file: %lld\n", train_words);
  }
  file_size = ftell(fin);
  fclose(fin);
}

void SaveVocab() {
  int i;
  FILE *fo = fopen(save_vocab_file, "wb");
  for (i = 0; i < vocab_size; i++) fprintf(fo, "%s %d\n", vocab[i].word, vocab[i].cn);
  fclose(fo);
}

void ReadVocab() {
  int a;
  char c;
  char word[MAX_STRING];
  FILE *fin = fopen(read_vocab_file, "rb");
  if (fin == NULL) {
    printf("Vocabulary file not found\n");
    exit(1);
  }
  for (a = 0; a < vocab_hash_size; a++) vocab_hash[a] = -1;
  vocab_size = 0;
  while (1) {
    ReadWord(word, fin);
    if (feof(fin)) break;
    a = AddWordToVocab(word);
    fscanf(fin, "%d%c", &vocab[a].cn, &c);
  }
  SortVocab();
  if (debug_mode > 0) {
    printf("Vocab size: %d\n", vocab_size);
    printf("Words in train file: %lld\n", train_words);
  }
  fin = fopen(train_file, "rb");
  if (fin == NULL) {
    printf("ERROR: training data file not found!\n");
    exit(1);
  }
  fseek(fin, 0, SEEK_END);
  file_size = ftell(fin);
  fclose(fin);
}

void InitNet() {
  int a, b;
  unsigned int next_random = 1;
  a = posix_memalign((void **)&syn0, 128, (int)vocab_size * layer1_size_aligned * sizeof(real));
  if (syn0 == NULL) {printf("Memory allocation failed\n"); exit(1);}

  for (a = 0; a < vocab_size; a++) for (b = 0; b < layer1_size; b++) {
    next_random = next_random * (unsigned int)1664525 + 1013904223;
    syn0[a * layer1_size_aligned + b] = (((next_random & 0xFFFF) / (real)65536) - 0.5) / layer1_size;
  }
  CreateBinaryTree();
}

void *TrainModelThread(void *id) {
  int   word, sentence_length = 0;
  long long word_count = 0, last_word_count = 0;
  int    local_iter = iter;
  unsigned int next_random = (long)id;
  int sentence_num;
  clock_t now;
  real * alpha_ptr = (float *) sen + MAX_SENTENCE_NUM * MAX_SENTENCE_LENGTH;
  FILE *fi = fopen(train_file, "rb");
  fseek(fi, file_size / (int)num_threads * (long)id, SEEK_SET);
  sentence_length = 0;
  sentence_num = 0;

  while (1) {
    if (word_count - last_word_count > 10000) {
      word_count_actual += word_count - last_word_count;
      last_word_count = word_count;
      if ((debug_mode > 1)) {
        now=clock();
        printf("%cAlpha: %f  Progress: %.2f%%  Words/thread/sec: %.2fk  ", 13, alpha,
            word_count_actual / (real)(iter * train_words + 1) * 100,
            word_count_actual / ((real)(now - start + 1) / (real)CLOCKS_PER_SEC * 1000));
        fflush(stdout);
      }
      alpha = starting_alpha * (1 - word_count_actual / (real)(iter * train_words + 1));
      if (alpha < starting_alpha * 0.0001) alpha = starting_alpha * 0.0001;
    }

    while (1) {
      word = ReadWordIndex(fi);
      if (feof(fi)) break;
      if (word == -1) continue;
      word_count++;
      if (word == 0) break;
      // The subsampling randomly discards frequent words while keeping the ranking same
      if (sample > 0) {
        real ran = (sqrt(vocab[word].cn / (sample * train_words)) + 1) * (sample * train_words) / vocab[word].cn;
        next_random = next_random * (unsigned int)1664525 + 1013904223;
        if (ran < (next_random & 0xFFFF) / (real)65536) continue;
      }
      sen[sentence_num * MAX_SENTENCE_LENGTH + sentence_length] = word;
      sentence_length++;
      if (sentence_length >= MAX_SENTENCE_LENGTH){
        alpha_ptr[sentence_num] = alpha;
        sentence_num ++;
        sentence_length = 0;
        if (sentence_num >= MAX_SENTENCE_NUM) break;
      }
    }

    //=====================================================
    // Do GPU training here
    //=====================================================
    TrainGPU(sentence_num);

    sentence_num = 0;
    sentence_length = 0;

    if (feof(fi) || (word_count > train_words / num_threads)) {
      word_count_actual += word_count - last_word_count;
      local_iter--;
      if (local_iter == 0) break;
      word_count = 0;
      last_word_count = 0;
      fseek(fi, file_size / (int)num_threads * (long)id, SEEK_SET);
    }
  }

  GetResultFromGPU();
  fclose(fi);
  pthread_exit(NULL);
}

void TrainModel() {

  // Check output file first
  if (output_file[0] == 0) return;
  FILE *fo = fopen(output_file, "wb");
  if (fo == NULL) {
    printf("ERROR: failed to open output file!\n");
    return;
  }

  long a, b, c, d;
  pthread_t *pt = (pthread_t *)malloc(num_threads * sizeof(pthread_t));
  printf("Starting training using file %s\n", train_file);
  starting_alpha = alpha;
  if (read_vocab_file[0] != 0) ReadVocab(); else LearnVocabFromTrainFile();
  if (save_vocab_file[0] != 0) SaveVocab();

  InitNet();

  if (negative > 0) InitUnigramTable();

  initializeGPU();

  start = clock();

  // Training on a GPU
  for (a = 0; a < num_threads; a++) pthread_create(&pt[a], NULL, TrainModelThread, (void *)a);

  // Training complete
  for (a = 0; a < num_threads; a++) pthread_join(pt[a], NULL);

  cleanUpGPU();

  if (classes == 0) {
    // Save the word vectors
    fprintf(fo, "%d %d\n", vocab_size, layer1_size);
    for (a = 0; a < vocab_size; a++) {
      fprintf(fo, "%s ", vocab[a].word);
      if (binary) for (b = 0; b < layer1_size; b++) fwrite(&syn0[a * layer1_size_aligned + b], sizeof(real), 1, fo);
      else for (b = 0; b < layer1_size; b++) fprintf(fo, "%lf ", syn0[a * layer1_size_aligned + b]);
      fprintf(fo, "\n");
    }
  } else {
    // Run K-means on the word vectors
    int clcn = classes, iter = 10, closeid;
    int *centcn = (int *)malloc(classes * sizeof(int));
    int *cl = (int *)calloc(vocab_size, sizeof(int));
    real closev, x;
    real *cent = (real *)calloc(classes * layer1_size, sizeof(real));
    for (a = 0; a < vocab_size; a++) cl[a] = a % clcn;
    for (a = 0; a < iter; a++) {
      for (b = 0; b < clcn * layer1_size; b++) cent[b] = 0;
      for (b = 0; b < clcn; b++) centcn[b] = 1;
      for (c = 0; c < vocab_size; c++) {
        for (d = 0; d < layer1_size; d++) cent[layer1_size * cl[c] + d] += syn0[c * layer1_size_aligned + d];
        centcn[cl[c]]++;
      }
      for (b = 0; b < clcn; b++) {
        closev = 0;
        for (c = 0; c < layer1_size; c++) {
          cent[layer1_size * b + c] /= centcn[b];
          closev += cent[layer1_size * b + c] * cent[layer1_size * b + c];
        }
        closev = sqrt(closev);
        for (c = 0; c < layer1_size; c++) cent[layer1_size * b + c] /= closev;
      }
      for (c = 0; c < vocab_size; c++) {
        closev = -10;
        closeid = 0;
        for (d = 0; d < clcn; d++) {
          x = 0;
          for (b = 0; b < layer1_size; b++) x += cent[layer1_size * d + b] * syn0[c * layer1_size_aligned + b];
          if (x > closev) {
            closev = x;
            closeid = d;
          }
        }
        cl[c] = closeid;
      }
    }
    // Save the K-means classes
    for (a = 0; a < vocab_size; a++) fprintf(fo, "%s %d\n", vocab[a].word, cl[a]);
    free(centcn);
    free(cent);
    free(cl);
  }
  free(pt);
  fclose(fo);
}

int ArgPos(char *str, int argc, char **argv) {
  int a;
  for (a = 1; a < argc; a++) if (!strcmp(str, argv[a])) {
    if (a == argc - 1) {
      printf("Argument missing for %s\n", str);
      exit(1);
    }
    return a;
  }
  return -1;
}

int main(int argc, char **argv) {
  int i;
  if (argc == 1) {
    printf("WORD VECTOR estimation toolkit v 0.1c\n\n");
    printf("Options:\n");
    printf("Parameters for training:\n");
    printf("\t-train <file>\n");
    printf("\t\tUse text data from <file> to train the model\n");
    printf("\t-output <file>\n");
    printf("\t\tUse <file> to save the resulting word vectors / word clusters\n");
    printf("\t-size <int>\n");
    printf("\t\tSet size of word vectors; default is 100\n");
    printf("\t-window <int>\n");
    printf("\t\tSet max skip length between words; default is 5\n");
    printf("\t-sample <float>\n");
    printf("\t\tSet threshold for occurrence of words. Those that appear with higher frequency in the training data\n");
    printf("\t\twill be randomly down-sampled; default is 1e-3, useful range is (0, 1e-5)\n");
    printf("\t-hs <int>\n");
    printf("\t\tUse Hierarchical Softmax; default is 0 (not used)\n");
    printf("\t-negative <int>\n");
    printf("\t\tNumber of negative examples; default is 5, common values are 3 - 10 (0 = not used)\n");
    printf("\t-threads <int>\n");
    printf("\t\tUse <int> threads (default 12)\n");
    printf("\t-iter <int>\n");
    printf("\t\tRun more training iterations (default 5)\n");
    printf("\t-min-count <int>\n");
    printf("\t\tThis will discard words that appear less than <int> times; default is 5\n");
    printf("\t-alpha <float>\n");
    printf("\t\tSet the starting learning rate; default is 0.025 for skip-gram and 0.05 for CBOW\n");
    printf("\t-classes <int>\n");
    printf("\t\tOutput word classes rather than word vectors; default number of classes is 0 (vectors are written)\n");
    printf("\t-debug <int>\n");
    printf("\t\tSet the debug mode (default = 2 = more info during training)\n");
    printf("\t-binary <int>\n");
    printf("\t\tSave the resulting vectors in binary moded; default is 0 (off)\n");
    printf("\t-save-vocab <file>\n");
    printf("\t\tThe vocabulary will be saved to <file>\n");
    printf("\t-read-vocab <file>\n");
    printf("\t\tThe vocabulary will be read from <file>, not constructed from the training data\n");
    printf("\t-cbow <int>\n");
    printf("\t\tUse the continuous bag of words model; default is 1 (use 0 for skip-gram model)\n");
    printf("\nExamples:\n");
    printf("./word2vec -train data.txt -output vec.txt -size 200 -window 5 -sample 1e-4 -negative 5 -hs 0 -binary 0 -cbow 1 -iter 3\n\n");
    return 0;
  }
  output_file[0] = 0;
  save_vocab_file[0] = 0;
  read_vocab_file[0] = 0;
  if ((i = ArgPos((char *)"-size", argc, argv)) > 0){
    layer1_size = atoi(argv[i + 1]);
    layer1_size_aligned = ((layer1_size - 1)/ALIGNMENT_FACTOR + 1) * ALIGNMENT_FACTOR;
  }
  if ((i = ArgPos((char *)"-train", argc, argv)) > 0) strcpy(train_file, argv[i + 1]);
  if ((i = ArgPos((char *)"-save-vocab", argc, argv)) > 0) strcpy(save_vocab_file, argv[i + 1]);
  if ((i = ArgPos((char *)"-read-vocab", argc, argv)) > 0) strcpy(read_vocab_file, argv[i + 1]);
  if ((i = ArgPos((char *)"-debug", argc, argv)) > 0) debug_mode = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-binary", argc, argv)) > 0) binary = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-cbow", argc, argv)) > 0) cbow = atoi(argv[i + 1]);
  if (cbow) alpha = 0.05;
  if ((i = ArgPos((char *)"-alpha", argc, argv)) > 0) alpha = atof(argv[i + 1]);
  if ((i = ArgPos((char *)"-output", argc, argv)) > 0) strcpy(output_file, argv[i + 1]);
  if ((i = ArgPos((char *)"-window", argc, argv)) > 0) window = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-sample", argc, argv)) > 0) sample = atof(argv[i + 1]);
  if ((i = ArgPos((char *)"-hs", argc, argv)) > 0) hs = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-negative", argc, argv)) > 0) negative = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-threads", argc, argv)) > 0) num_threads = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-iter", argc, argv)) > 0) iter = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-min-count", argc, argv)) > 0) min_count = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-classes", argc, argv)) > 0) classes = atoi(argv[i + 1]);
  vocab = (struct vocab_word *)calloc(vocab_max_size, sizeof(struct vocab_word));
  vocab_hash = (int *)calloc(vocab_hash_size, sizeof(int));

  TrainModel();

  free(vocab);
  free(vocab_hash);
  free(table);
  return 0;
}
