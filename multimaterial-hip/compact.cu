#include "hip/hip_runtime.h"
#include <chrono>
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

char *cp_to_device(char *from, size_t size) {
  char *tmp;
  hipMalloc((void**)&tmp, size);
  hipMemcpy(tmp, from, size, hipMemcpyHostToDevice);
  return tmp;
}

void cp_to_host(char *to, char*from, size_t size) {
  hipMemcpy(to, from, size, hipMemcpyDeviceToHost);
  hipFree(from);
}

__global__ void ccc_loop1(
  const int * __restrict__ imaterial,
  const int * __restrict__ nextfrac,
  const double * __restrict__ rho_compact,
  const double * __restrict__ rho_compact_list, 
  const double * __restrict__ Vf_compact_list,
  const double * __restrict__ V,
  double * __restrict__ rho_ave_compact,
  int sizex, int sizey,
  int * __restrict__ mmc_index)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if (i >= sizex || j >= sizey) return;
#ifdef FUSED
  double ave = 0.0;
  int ix = imaterial[i+sizex*j];

  if (ix <= 0) {
    // condition is 'ix >= 0', this is the equivalent of
    // 'until ix < 0' from the paper
#ifdef LINKED
    for (ix = -ix; ix >= 0; ix = nextfrac[ix]) {
      ave += rho_compact_list[ix] * Vf_compact_list[ix];
    }
#else
    for (int idx = mmc_index[-ix]; idx < mmc_index[-ix+1]; idx++) {
      ave += rho_compact_list[idx] * Vf_compact_list[idx];  
    }
#endif
    rho_ave_compact[i+sizex*j] = ave/V[i+sizex*j];
  }
  else {
#endif
    // We use a distinct output array for averages.
    // In case of a pure cell, the average density equals to the total.
    rho_ave_compact[i+sizex*j] = rho_compact[i+sizex*j] / V[i+sizex*j];
#ifdef FUSED
  }
#endif
}

__global__ void ccc_loop1_2(
  const double * __restrict__ rho_compact_list,
  const double * __restrict__  Vf_compact_list,
  const double * __restrict__  V,
  double * __restrict__ rho_ave_compact,
  const int * __restrict__ mmc_index,
  const int  mmc_cells,
  const int * __restrict__ mmc_i,
  const int * __restrict__ mmc_j,
  int sizex, int sizey)
{
  int c = threadIdx.x + blockIdx.x * blockDim.x;
  if (c >= mmc_cells) return;
  double ave = 0.0;
  for (int m = mmc_index[c]; m < mmc_index[c+1]; m++) {
    ave +=  rho_compact_list[m] * Vf_compact_list[m];
  }
  rho_ave_compact[mmc_i[c]+sizex*mmc_j[c]] = ave/V[mmc_i[c]+sizex*mmc_j[c]];
}

__global__ void ccc_loop2(
  const int * __restrict__ imaterial,
  const int * __restrict__ matids,
  const int * __restrict__ nextfrac,
  const double * __restrict__ rho_compact,
  const double * __restrict__ rho_compact_list, 
  const double * __restrict__ t_compact,
  const double * __restrict__ t_compact_list, 
  const double * __restrict__  Vf_compact_list,
  const double * __restrict__ n,
  double * __restrict__  p_compact,
  double * __restrict__ p_compact_list,
  int sizex, int sizey,
  int * __restrict__ mmc_index)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if (i >= sizex || j >= sizey) return;

  int ix = imaterial[i+sizex*j];
  if (ix <= 0) {
#ifdef FUSED
    // NOTE: I think the paper describes this algorithm (Alg. 9) wrong.
    // The solution below is what I believe to good.

    // condition is 'ix >= 0', this is the equivalent of
    // 'until ix < 0' from the paper
#ifdef LINKED
    for (ix = -ix; ix >= 0; ix = nextfrac[ix]) {
      double nm = n[matids[ix]];
      p_compact_list[ix] = (nm * rho_compact_list[ix] * t_compact_list[ix]) / Vf_compact_list[ix];
    }
#else
    for (int idx = mmc_index[-ix]; idx < mmc_index[-ix+1]; idx++) {
      double nm = n[matids[idx]];
      p_compact_list[idx] = (nm * rho_compact_list[idx] * t_compact_list[idx]) / Vf_compact_list[idx];
    }
#endif
#endif
  }
  else {
    // NOTE: HACK: we index materials from zero, but zero can be a list index
    int mat = ix - 1;
    // NOTE: There is no division by Vf here, because the fractional volume is 1.0 in the pure cell case.
    p_compact[i+sizex*j] = n[mat] * rho_compact[i+sizex*j] * t_compact[i+sizex*j];;
  }
}

__global__ void ccc_loop2_2(
  const int * __restrict__ matids,
  const double * __restrict__ rho_compact_list, 
  const double * __restrict__ t_compact_list,
  const double * __restrict__ Vf_compact_list,
  const double * __restrict__ n,
  double * __restrict__ p_compact_list,
  int * __restrict__ mmc_index,
  int mmc_cells)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= mmc_cells) return;
  double nm = n[matids[idx]];
  p_compact_list[idx] = (nm * rho_compact_list[idx] * t_compact_list[idx]) / Vf_compact_list[idx];
}

__global__ void ccc_loop3(
  const int * __restrict__ imaterial,
  const int * __restrict__ nextfrac,
  const int * __restrict__ matids,
  const double * __restrict__ rho_compact, 
  const double * __restrict__ rho_compact_list, 
  double * __restrict__ rho_mat_ave_compact, 
  double * __restrict__ rho_mat_ave_compact_list, 
  const double * __restrict__ x,
  const double * __restrict__ y,
  int sizex, int sizey,
  int * __restrict__ mmc_index)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if (i >= sizex-1 || j >= sizey-1 || i < 1 || j < 1) return;

  // o: outer
  double xo = x[i+sizex*j];
  double yo = y[i+sizex*j];

  // There are at most 9 neighbours in 2D case.
  double dsqr[9];

  // for all neighbours
  for (int nj = -1; nj <= 1; nj++) {

    for (int ni = -1; ni <= 1; ni++) {

      dsqr[(nj+1)*3 + (ni+1)] = 0.0;

      // i: inner
      double xi = x[(i+ni)+sizex*(j+nj)];
      double yi = y[(i+ni)+sizex*(j+nj)];

      dsqr[(nj+1)*3 + (ni+1)] += (xo - xi) * (xo - xi);
      dsqr[(nj+1)*3 + (ni+1)] += (yo - yi) * (yo - yi);
    }
  }

  int ix = imaterial[i+sizex*j];

  if (ix <= 0) {
    // condition is 'ix >= 0', this is the equivalent of
    // 'until ix < 0' from the paper
#ifdef LINKED
    for (ix = -ix; ix >= 0; ix = nextfrac[ix]) {
#else
      for (int ix = mmc_index[-imaterial[i+sizex*j]]; ix < mmc_index[-imaterial[i+sizex*j]+1]; ix++) {
#endif
        int mat = matids[ix];
        double rho_sum = 0.0;
        int Nn = 0;

        // for all neighbours
        for (int nj = -1; nj <= 1; nj++) {

          for (int ni = -1; ni <= 1; ni++) {

            int ci = i+ni, cj = j+nj;
            int jx = imaterial[ci+sizex*cj];

            if (jx <= 0) {
              // condition is 'jx >= 0', this is the equivalent of
              // 'until jx < 0' from the paper
#ifdef LINKED
              for (jx = -jx; jx >= 0; jx = nextfrac[jx]) {
#else
                for (int jx = mmc_index[-imaterial[ci+sizex*cj]]; jx < mmc_index[-imaterial[ci+sizex*cj]+1]; jx++) {
#endif
                  if (matids[jx] == mat) {
                    rho_sum += rho_compact_list[jx] / dsqr[(nj+1)*3 + (ni+1)];
                    Nn += 1;

                    // The loop has an extra condition: "and not found".
                    // This makes sense, if the material is found, there won't be any more of the same.
                    break;
                  }
                }
              }
              else {
                // NOTE: In this case, the neighbour is a pure cell, its material index is in jx.
                // In contrast, Algorithm 10 loads matids[jx] which I think is wrong.

                // NOTE: HACK: we index materials from zero, but zero can be a list index
                int mat_neighbour = jx - 1;
                if (mat == mat_neighbour) {
                  rho_sum += rho_compact[ci+sizex*cj] / dsqr[(nj+1)*3 + (ni+1)];
                  Nn += 1;
                }
              } // end if (jx <= 0)
            } // end for (int ni)
          } // end for (int nj)

          rho_mat_ave_compact_list[ix] = rho_sum / Nn;
        } // end for (ix = -ix)
      } // end if (ix <= 0)
      else {
        // NOTE: In this case, the cell is a pure cell, its material index is in ix.
        // In contrast, Algorithm 10 loads matids[ix] which I think is wrong.

        // NOTE: HACK: we index materials from zero, but zero can be a list index
        int mat = ix - 1;

        double rho_sum = 0.0;
        int Nn = 0;

        // for all neighbours
        for (int nj = -1; nj <= 1; nj++) {
          if ((j + nj < 0) || (j + nj >= sizey)) // TODO: better way?
            continue;

          for (int ni = -1; ni <= 1; ni++) {
            if ((i + ni < 0) || (i + ni >= sizex)) // TODO: better way?
              continue;

            int ci = i+ni, cj = j+nj;
            int jx = imaterial[ci+sizex*cj];

            if (jx <= 0) {
              // condition is 'jx >= 0', this is the equivalent of
              // 'until jx < 0' from the paper
#ifdef LINKED
              for (jx = -jx; jx >= 0; jx = nextfrac[jx]) {
#else
                for (int jx = mmc_index[-imaterial[ci+sizex*cj]]; jx < mmc_index[-imaterial[ci+sizex*cj]+1]; jx++) {
#endif
                  if (matids[jx] == mat) {
                    rho_sum += rho_compact_list[jx] / dsqr[(nj+1)*3 + (ni+1)];
                    Nn += 1;

                    // The loop has an extra condition: "and not found".
                    // This makes sense, if the material is found, there won't be any more of the same.
                    break;
                  }
                }
              }
              else {
                // NOTE: In this case, the neighbour is a pure cell, its material index is in jx.
                // In contrast, Algorithm 10 loads matids[jx] which I think is wrong.

                // NOTE: HACK: we index materials from zero, but zero can be a list index
                int mat_neighbour = jx - 1;
                if (mat == mat_neighbour) {
                  rho_sum += rho_compact[ci+sizex*cj] / dsqr[(nj+1)*3 + (ni+1)];
                  Nn += 1;
                }
              } // end if (jx <= 0)
            } // end for (int ni)
          } // end for (int nj)

          rho_mat_ave_compact[i+sizex*j] = rho_sum / Nn;
        } // end else
      }

      struct full_data
      {
        int sizex;
        int sizey;
        int Nmats;
        double * __restrict__ rho;
        double * __restrict__ rho_mat_ave;
        double * __restrict__ p;
        double * __restrict__ Vf;
        double * __restrict__ t;
        double * __restrict__ V;
        double * __restrict__ x;
        double * __restrict__ y;
        double * __restrict__ n;
        double * __restrict__ rho_ave;
      };

      struct compact_data
      {
        int sizex;
        int sizey;
        int Nmats;
        double * __restrict__ rho_compact;
        double * __restrict__ rho_compact_list;
        double * __restrict__ rho_mat_ave_compact;
        double * __restrict__ rho_mat_ave_compact_list;
        double * __restrict__ p_compact;
        double * __restrict__ p_compact_list;
        double * __restrict__ Vf_compact_list;
        double * __restrict__ t_compact;
        double * __restrict__ t_compact_list;
        double * __restrict__ V;
        double * __restrict__ x;
        double * __restrict__ y;
        double * __restrict__ n;
        double * __restrict__ rho_ave_compact;
        int * __restrict__ imaterial;
        int * __restrict__ matids;
        int * __restrict__ nextfrac;
        int * __restrict__ mmc_index;
        int * __restrict__ mmc_i;
        int * __restrict__ mmc_j;
        int mm_len;
        int mmc_cells;
      };

      void compact_cell_centric(full_data cc, compact_data ccc, int argc, char** argv)
      {
        int sizex = cc.sizex;
        int sizey = cc.sizey;
        int Nmats = cc.Nmats;
        int mmc_cells = ccc.mmc_cells;
        int mm_len = ccc.mm_len;

        int    *d_imaterial = (int *)cp_to_device((char*)ccc.imaterial, sizex*sizey*sizeof(int));
        int    *d_matids = (int *)cp_to_device((char*)ccc.matids, mm_len*sizeof(int));
        int    *d_nextfrac = (int *)cp_to_device((char*)ccc.nextfrac, mm_len*sizeof(int));
        int    *d_mmc_index = (int *)cp_to_device((char*)ccc.mmc_index, (mmc_cells+1)*sizeof(int));
        int    *d_mmc_i = (int *)cp_to_device((char*)ccc.mmc_i, (mmc_cells)*sizeof(int));
        int    *d_mmc_j = (int *)cp_to_device((char*)ccc.mmc_j, (mmc_cells)*sizeof(int));
        double *d_x = (double *)cp_to_device((char*)ccc.x, sizex*sizey*sizeof(double));
        double *d_y = (double *)cp_to_device((char*)ccc.y, sizex*sizey*sizeof(double));
        double *d_rho_compact = (double *)cp_to_device((char*)ccc.rho_compact, sizex*sizey*sizeof(double));
        double *d_rho_compact_list = (double *)cp_to_device((char*)ccc.rho_compact_list,mm_len*sizeof(double));
        double *d_rho_mat_ave_compact = (double *)cp_to_device((char*)ccc.rho_mat_ave_compact, sizex*sizey*sizeof(double));
        double *d_rho_mat_ave_compact_list = (double *)cp_to_device((char*)ccc.rho_mat_ave_compact_list,mm_len*sizeof(double));
        double *d_p_compact = (double *)cp_to_device((char*)ccc.p_compact, sizex*sizey*sizeof(double));
        double *d_p_compact_list = (double *)cp_to_device((char*)ccc.p_compact_list,mm_len*sizeof(double));
        double *d_t_compact = (double *)cp_to_device((char*)ccc.t_compact, sizex*sizey*sizeof(double));
        double *d_t_compact_list = (double *)cp_to_device((char*)ccc.t_compact_list,mm_len*sizeof(double));
        double *d_Vf_compact_list = (double *)cp_to_device((char*)ccc.Vf_compact_list, mm_len*sizeof(double));
        double *d_V = (double *)cp_to_device((char*)ccc.V, sizex*sizey*sizeof(double));
        double *d_n = (double *)cp_to_device((char*)ccc.n, Nmats*sizeof(double));
        double *d_rho_ave_compact = (double *)cp_to_device((char*)ccc.rho_ave_compact, sizex*sizey*sizeof(double));

        int thx = 32;
        int thy = 4;
        dim3 threads(thx,thy,1);
        dim3 blocks((sizex-1)/thx+1, (sizey-1)/thy+1, 1);

        // Cell-centric algorithms
        // Computational loop 1 - average density in cell
        hipDeviceSynchronize();
        auto t0 = std::chrono::system_clock::now();
        ccc_loop1 <<< dim3(blocks), dim3(threads) >>> (d_imaterial, d_nextfrac, d_rho_compact, d_rho_compact_list, d_Vf_compact_list, d_V, d_rho_ave_compact, sizex, sizey, d_mmc_index);
#ifndef FUSED
        ccc_loop1_2 <<< dim3((mmc_cells-1)/(thx*thy)+1), dim3((thx*thy)) >>> (d_rho_compact_list, d_Vf_compact_list, d_V, d_rho_ave_compact, d_mmc_index, mmc_cells, d_mmc_i, d_mmc_j, sizex, sizey);
#endif
        hipDeviceSynchronize();
        std::chrono::duration<double> t1 = std::chrono::system_clock::now() - t0;
        printf("Compact matrix, cell centric, alg 1: %g msec\n", t1.count() * 1000);

        // Computational loop 2 - Pressure for each cell and each material
        t0 = std::chrono::system_clock::now();
        ccc_loop2 <<< dim3(blocks), dim3(threads) >>> (d_imaterial, d_matids,d_nextfrac, d_rho_compact, d_rho_compact_list, d_t_compact, d_t_compact_list, d_Vf_compact_list, d_n, d_p_compact, d_p_compact_list, sizex, sizey, d_mmc_index);
#ifndef FUSED
        ccc_loop2_2 <<< dim3((mm_len-1)/(thx*thy)+1), dim3((thx*thy)) >>> (d_matids, d_rho_compact_list, d_t_compact_list, d_Vf_compact_list, d_n, d_p_compact_list, d_mmc_index, mm_len);
#endif
        hipDeviceSynchronize();
        std::chrono::duration<double> t2 = std::chrono::system_clock::now() - t0;
        printf("Compact matrix, cell centric, alg 2: %g msec\n", t2.count() * 1000);

        // Computational loop 3 - Average density of each material over neighborhood of each cell
        t0 = std::chrono::system_clock::now();
        ccc_loop3 <<< dim3(blocks), dim3(threads) >>> (d_imaterial,d_nextfrac, d_matids, d_rho_compact, d_rho_compact_list, d_rho_mat_ave_compact, d_rho_mat_ave_compact_list, d_x, d_y, sizex, sizey, d_mmc_index);  
        hipDeviceSynchronize();
        std::chrono::duration<double> t3 = std::chrono::system_clock::now() - t0;
        printf("Compact matrix, cell centric, alg 3: %g msec\n", t3.count() * 1000);

        cp_to_host((char*)ccc.x, (char*)d_x, sizex*sizey*sizeof(double));
        cp_to_host((char*)ccc.y, (char*)d_y, sizex*sizey*sizeof(double));
        cp_to_host((char*)ccc.rho_compact, (char*)d_rho_compact, sizex*sizey*sizeof(double));
        cp_to_host((char*)ccc.rho_compact_list, (char*)d_rho_compact_list, mm_len*sizeof(double));
        cp_to_host((char*)ccc.rho_mat_ave_compact, (char*)d_rho_mat_ave_compact, sizex*sizey*sizeof(double));
        cp_to_host((char*)ccc.rho_mat_ave_compact_list, (char*)d_rho_mat_ave_compact_list, mm_len*sizeof(double));
        cp_to_host((char*)ccc.p_compact, (char*)d_p_compact, sizex*sizey*sizeof(double));
        cp_to_host((char*)ccc.p_compact_list, (char*)d_p_compact_list, mm_len*sizeof(double));
        cp_to_host((char*)ccc.t_compact, (char*)d_t_compact, sizex*sizey*sizeof(double));
        cp_to_host((char*)ccc.t_compact_list, (char*)d_t_compact_list, mm_len*sizeof(double));
        cp_to_host((char*)ccc.Vf_compact_list, (char*)d_Vf_compact_list, mm_len*sizeof(double));
        cp_to_host((char*)ccc.V, (char*)d_V, sizex*sizey*sizeof(double));
        cp_to_host((char*)ccc.n, (char*)d_n, Nmats*sizeof(double));
        cp_to_host((char*)ccc.rho_ave_compact, (char*)d_rho_ave_compact, sizex*sizey*sizeof(double));

        // hipFree are included in cp_to_host
      }

      bool compact_check_results(full_data cc, compact_data ccc)
      {
        int sizex = cc.sizex;
        int sizey = cc.sizey;
        int Nmats = cc.Nmats;
        //int mmc_cells = ccc.mmc_cells;
        //int mm_len = ccc.mm_len;

        printf("Checking results of compact representation... ");

        for (int j = 0; j < sizey; j++) {
          for (int i = 0; i < sizex; i++) {
            if (fabs(cc.rho_ave[i+sizex*j] - ccc.rho_ave_compact[i+sizex*j]) > 0.0001) {
              printf("1. full matrix and compact cell-centric values are not equal! (%f, %f, %d, %d)\n",
                  cc.rho_ave[i+sizex*j], ccc.rho_ave_compact[i+sizex*j], i, j);
              return false;
            }
            int ix = ccc.imaterial[i+sizex*j];
            if (ix <= 0) {
#ifdef LINKED
              for (ix = -ix; ix >= 0; ix = ccc.nextfrac[ix]) {
#else
              for (int ix = ccc.mmc_index[-ccc.imaterial[i+sizex*j]]; ix < ccc.mmc_index[-ccc.imaterial[i+sizex*j]+1]; ix++) {
#endif
                int mat = ccc.matids[ix];
                if (fabs(cc.p[(i+sizex*j)*Nmats+mat] - ccc.p_compact_list[ix]) > 0.0001) {
                  printf("2. full matrix and compact cell-centric values are not equal! (%f, %f, %d, %d, %d)\n",
                      cc.p[(i+sizex*j)*Nmats+mat], ccc.p_compact_list[ix], i, j, mat);
                  return false;
                }

                if (fabs(cc.rho_mat_ave[(i+sizex*j)*Nmats+mat] - ccc.rho_mat_ave_compact_list[ix]) > 0.0001) {
                  printf("3. full matrix and compact cell-centric values are not equal! (%f, %f, %d, %d, %d)\n",
                      cc.rho_mat_ave[(i+sizex*j)*Nmats+mat], ccc.rho_mat_ave_compact_list[ix], i, j, mat);
                  return false;
                }
              }
            }
            else {
              // NOTE: HACK: we index materials from zero, but zero can be a list index
              int mat = ix - 1;
              if (fabs(cc.p[(i+sizex*j)*Nmats+mat] - ccc.p_compact[i+sizex*j]) > 0.0001) {
                printf("2. full matrix and compact cell-centric values are not equal! (%f, %f, %d, %d, %d)\n",
                    cc.p[(i+sizex*j)*Nmats+mat], ccc.p_compact[i+sizex*j], i, j, mat);
                return false;
              }

              if (fabs(cc.rho_mat_ave[(i+sizex*j)*Nmats+mat] - ccc.rho_mat_ave_compact[i+sizex*j]) > 0.0001) {
                printf("3. full matrix and compact cell-centric values are not equal! (%f, %f, %d, %d, %d)\n",
                    cc.rho_mat_ave[(i+sizex*j)*Nmats+mat], ccc.rho_mat_ave_compact[i+sizex*j], i, j, mat);
                return false;
              }
            }
          }
        }
        printf("All tests passed!\n");
        return true;
      }
