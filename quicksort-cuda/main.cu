#include "hip/hip_runtime.h"
/*
   Copyright (c) 2014-2019, Intel Corporation
   Redistribution and use in source and binary forms, with or without
   modification, are permitted provided that the following conditions
   are met:
 * Redistributions of source code must retain the above copyright
 notice, this list of conditions and the following disclaimer.
 * Redistributions in binary form must reproduce the above
 copyright notice, this list of conditions and the following
 disclaimer in the documentation and/or other materials provided
 with the distribution.
 * Neither the name of Intel Corporation nor the names of its
 contributors may be used to endorse or promote products
 derived from this software without specific prior written
 permission.
 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 POSSIBILITY OF SUCH DAMAGE.
 */

// QuicksortMain.cpp : Defines the entry point for the console application.
//

#include <stdio.h>
#include <assert.h>
#include <string.h>
#include <limits.h>
#include <math.h>
#include <iostream>
#include <algorithm>
#include <iterator>
#include <random>
#include <vector>
#include <map>
#include <hip/hip_runtime.h>

#define RUN_CPU_SORTS
//#define GET_DETAILED_PERFORMANCE

#define gpucheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

// Types:
typedef unsigned int uint;
#ifdef min
#undef min
#endif
#ifdef max
#undef max
#endif

/// return a timestamp with sub-second precision
/** QueryPerformanceCounter and clock_gettime have an undefined starting point (null/zero)
 *  and can wrap around, i.e. be nulled again. **/
double seconds() {
  struct timespec now;
  clock_gettime(CLOCK_MONOTONIC, &now);
  return now.tv_sec + now.tv_nsec / 1000000000.0;
}


bool parseArgs(int argc, char** argv, unsigned int* test_iterations, unsigned int* widthReSz, unsigned int* heightReSz)
{
  const char sUsageString[512] = "Usage: Quicksort [num test iterations] [SurfWidth(^2 only)] [SurfHeight(^2 only)]";

  if (argc != 4)
  {
    printf(sUsageString);
    return false;
  }
  else
  {
    *test_iterations  = atoi (argv[1]);
    *widthReSz  = atoi (argv[2]);
    *heightReSz  = atoi (argv[3]);
    return true;
  }
}


#include "Quicksort.h"
#include "QuicksortKernels.cuh"


template <class T>
T* partition(T* left, T* right, T pivot) {
  // move pivot to the end
  T temp = *right;
  *right = pivot;
  *left = temp;

  T* store = left;

  for(T* p = left; p != right; p++) {
    if (*p < pivot) {
      temp = *store;
      *store = *p;
      *p = temp;
      store++;
    }
  }

  temp = *store;
  *store = pivot;
  *right = temp;

  return store;
}

  template <class T>
void quicksort(T* data, int left, int right)
{
  T* store = partition(data + left, data + right, data[left]);
  int nright = store-data;
  int nleft = nright+1;

  if (left < nright) {
    if (nright - left > 32) {
      quicksort(data, left, nright);
    } else
      std::sort(data + left, data + nright + 1);
  }

  if (nleft < right) {
    if (right - nleft > 32)  {
      quicksort(data, nleft, right);
    } else {
      std::sort(data + nleft, data + right + 1);
    }
  }
}

template <class T>
void gqsort(T *db,
    T *dnb,
    std::vector<block_record<T>>& blocks,
    std::vector<parent_record>& parents,
    std::vector<work_record<T>>& news,
    bool reset) {

  news.resize(blocks.size()*2);

#ifdef GET_DETAILED_PERFORMANCE
  static double absoluteTotal = 0.0;
  static uint count = 0;

  if (reset) {
    absoluteTotal = 0.0;
    count = 0;
  }

  double beginClock, endClock;
  beginClock = seconds();
#endif

  block_record<T> *blocksb;
  parent_record *parentsb;
  work_record<T> *newsb;
  gpucheck(hipMalloc((void**)&blocksb, sizeof(block_record<T>)*blocks.size()));
  gpucheck(hipMalloc((void**)&parentsb, sizeof(parent_record)*parents.size()));
  gpucheck(hipMalloc((void**)&newsb, sizeof(work_record<T>)*news.size()));
  gpucheck(hipMemcpy(blocksb, blocks.data(), sizeof(block_record<T>)*blocks.size(), hipMemcpyHostToDevice));
  gpucheck(hipMemcpy(parentsb, parents.data(), sizeof(parent_record)*parents.size(), hipMemcpyHostToDevice));
  gpucheck(hipMemcpy(newsb, news.data(), sizeof(work_record<T>)*news.size(), hipMemcpyHostToDevice));

  gqsort_kernel<<<dim3(blocks.size()), dim3(GQSORT_LOCAL_WORKGROUP_SIZE)>>>(
      db, dnb, blocksb, parentsb, newsb);

  gpucheck( hipPeekAtLastError() );
  gpucheck( hipDeviceSynchronize() );
  gpucheck(hipMemcpy(news.data(), newsb, sizeof(work_record<T>)*news.size(), hipMemcpyDeviceToHost));

  gpucheck(hipFree(blocksb));
  gpucheck(hipFree(parentsb));
  gpucheck(hipFree(newsb));

#ifdef GET_DETAILED_PERFORMANCE
  endClock = seconds();
  double totalTime = endClock - beginClock;
  absoluteTotal += totalTime;
  std::cout << ++count << ": gqsort time " << absoluteTotal * 1000 << " ms" << std::endl;
#endif

#ifdef DEBUG
  printf("\noutput news\n");
  for (int i = 0; i < news.size(); i++) {
    printf("%u %u %u %u\n", news[i].start, news[i].end, news[i].pivot, news[i].direction);
  }
#endif
}

template <class T>
void lqsort(T *db, T *dnb, std::vector<work_record<T>>& done) {

#ifdef GET_DETAILED_PERFORMANCE
  double beginClock, endClock;
  beginClock = seconds();
#endif
  work_record<T>* doneb;
  //std::cout << "done size is " << done.size() << std::endl;

  gpucheck(hipMalloc((void**)&doneb, sizeof(work_record<T>)*done.size()));
  gpucheck(hipMemcpy(doneb, done.data(), sizeof(work_record<T>)*done.size(), hipMemcpyHostToDevice));

  lqsort_kernel<<<dim3(done.size()), dim3(LQSORT_LOCAL_WORKGROUP_SIZE)>>>(db, dnb, doneb);
  gpucheck( hipPeekAtLastError() );
  gpucheck( hipDeviceSynchronize() );

  // Lets do phase 2 pass
  gpucheck(hipFree(doneb));
#ifdef GET_DETAILED_PERFORMANCE
  endClock = seconds();
  double totalTime = endClock - beginClock;
  std::cout << "lqsort time " << totalTime * 1000 << " ms" << std::endl;
#endif
}

size_t optp(size_t s, double k, size_t m) {
  return (size_t)pow(2, floor(log(s*k + m)/log(2.0) + 0.5));
}

template <class T>
void GPUQSort(size_t size, T* d, T* dn)  {

  // allocate buffers
  T *db, *dnb;
  hipMalloc((void**)&db, ((sizeof(T)*size)/64 + 1)*64);
  hipMemcpy(db, d, ((sizeof(T)*size)/64 + 1)*64, hipMemcpyHostToDevice);
  hipMalloc((void**)&dnb, ((sizeof(T)*size)/64 + 1)*64);
  hipMemcpy(dnb, dn, ((sizeof(T)*size)/64 + 1)*64, hipMemcpyHostToDevice);

  const size_t MAXSEQ = optp(size, 0.00009516, 203);
  const size_t MAX_SIZE = 12*std::max(MAXSEQ, (size_t)QUICKSORT_BLOCK_SIZE);
  //std::cout << "MAXSEQ = " << MAXSEQ << std::endl;
  uint startpivot = median_host(d[0], d[size/2], d[size-1]);
  std::vector<work_record<T>> work, done, news;
  work.reserve(MAX_SIZE);
  done.reserve(MAX_SIZE);
  news.reserve(MAX_SIZE);
  std::vector<parent_record> parent_records;
  parent_records.reserve(MAX_SIZE);
  std::vector<block_record<T>> blocks;
  blocks.reserve(MAX_SIZE);

  work.push_back(work_record<T>(0, size, startpivot, 1));

  bool reset = true;

  while(!work.empty() /*&& work.size() + done.size() < MAXSEQ*/) {
    size_t blocksize = 0;

    for(auto it = work.begin(); it != work.end(); ++it) {
      blocksize += std::max((it->end - it->start)/MAXSEQ, (size_t)1);
    }
    for(auto it = work.begin(); it != work.end(); ++it) {
      uint start = it->start;
      uint end   = it->end;
      uint pivot = it->pivot;
      uint direction = it->direction;
      uint blockcount = (end - start + blocksize - 1)/blocksize;
      parent_record prnt(start, end, start, end, blockcount-1);
      parent_records.push_back(prnt);

      for(uint i = 0; i < blockcount - 1; i++) {
        uint bstart = start + blocksize*i;
        block_record<T> br(bstart, bstart+blocksize, pivot, direction, parent_records.size()-1);
        blocks.push_back(br);
      }
      block_record<T> br(start + blocksize*(blockcount - 1), end, pivot, direction, parent_records.size()-1);
      blocks.push_back(br);
    }
    //std::cout << " blocks = " << blocks.size() << " parent records = " << parent_records.size() << " news = " << news.size() << std::endl;

    gqsort<T>(db, dnb, blocks, parent_records, news, reset);

    reset = false;
    work.clear();
    parent_records.clear();
    blocks.clear();
    for(auto it = news.begin(); it != news.end(); ++it) {
      if (it->direction != EMPTY_RECORD) {
        if (it->end - it->start <= QUICKSORT_BLOCK_SIZE /*size/MAXSEQ*/) {
          if (it->end - it->start > 0)
            done.push_back(*it);
        } else {
          work.push_back(*it);
        }
      }
    }
    news.clear();
  }
  for(auto it = work.begin(); it != work.end(); ++it) {
    if (it->end - it->start > 0)
      done.push_back(*it);
  }

  if (done.size() > 0)
    lqsort<T>(db, dnb, done);

  hipMemcpy(d, db, ((sizeof(T)*size)/64 + 1)*64, hipMemcpyDeviceToHost);
  hipFree(db);
  hipFree(dnb);
}

template <class T>
int test(uint arraySize, unsigned int  NUM_ITERATIONS,
         const std::string& type_name)
{
  double totalTime, quickSortTime, stdSortTime;
  double beginClock, endClock;

  printf("\n\n\n--------------------------------------------------------------------\n");
  printf("Allocating array size of %d (data type: %s)\n", arraySize, type_name.c_str());
  T* pArray = (T*)aligned_alloc (4096, ((arraySize*sizeof(T))/64 + 1)*64);
  T* pArrayCopy = (T*)aligned_alloc (4096, ((arraySize*sizeof(T))/64 + 1)*64);
  std::generate(pArray, pArray + arraySize, [](){static T i = 0; return ++i; });
  std::shuffle(pArray, pArray + arraySize, std::mt19937(19937));

#ifdef RUN_CPU_SORTS
  std::cout << "Sorting the regular way..." << std::endl;
  std::copy(pArray, pArray + arraySize, pArrayCopy);

  beginClock = seconds();
  std::sort(pArrayCopy, pArrayCopy + arraySize);
  endClock = seconds();
  totalTime = endClock - beginClock;
  std::cout << "Time to sort: " << totalTime * 1000 << " ms" << std::endl;
  stdSortTime = totalTime;

  std::cout << "quicksort on the cpu: " << std::endl;
  std::copy(pArray, pArray + arraySize, pArrayCopy);

  beginClock = seconds();
  quicksort(pArrayCopy, 0, arraySize-1);
  endClock = seconds();
  totalTime = endClock - beginClock;
  std::cout << "Time to sort: " << totalTime * 1000 << " ms" << std::endl;
  quickSortTime = totalTime;
#ifdef TRUST_BUT_VERIFY
  {
    std::vector<uint> verify(arraySize);
    std::copy(pArray, pArray + arraySize, verify.begin());

    std::cout << "verifying: ";
    std::sort(verify.begin(), verify.end());
    bool correct = std::equal(verify.begin(), verify.end(), pArrayCopy);
    unsigned int num_discrepancies = 0;
    if (!correct) {
      for(size_t i = 0; i < arraySize; i++) {
        if (verify[i] != pArrayCopy[i]) {
          //std:: cout << "discrepancy at " << i << " " << pArrayCopy[i] << " expected " << verify[i] << std::endl;
          num_discrepancies++;
        }
      }
    }
    std::cout << std::boolalpha << correct << std::endl;
    if (!correct) {
      char y;
      std::cout << "num_discrepancies: " << num_discrepancies << std::endl;
      std::cin >> y;
    }
  }
#endif
#endif // RUN_CPU_SORTS

  std::cout << "Sorting with GPU quicksort: " << std::endl;
  std::vector<uint> original(arraySize);
  std::copy(pArray, pArray + arraySize, original.begin());

  std::vector<double> times;
  times.resize(NUM_ITERATIONS);
  double AverageTime = 0.0;
  uint num_failures = 0;
  for(uint k = 0; k < NUM_ITERATIONS; k++) {
    std::copy(original.begin(), original.end(), pArray);
    std::vector<uint> seqs;
    std::vector<uint> verify(arraySize);
    std::copy(pArray, pArray + arraySize, verify.begin());

    beginClock = seconds();
    GPUQSort(arraySize, pArray, pArrayCopy);
    endClock = seconds();
    totalTime = endClock - beginClock;
    std::cout << "Time to sort: " << totalTime * 1000 << " ms" << std::endl;
    times[k] = totalTime;
    AverageTime += totalTime;
#ifdef TRUST_BUT_VERIFY
    std::cout << "verifying: ";
    std::sort(verify.begin(), verify.end());
    bool correct = std::equal(verify.begin(), verify.end(), pArray);
    unsigned int num_discrepancies = 0;
    if (!correct) {
      for(size_t i = 0; i < arraySize; i++) {
        if (verify[i] != pArray[i]) {
          std:: cout << "discrepancy at " << i << " " << pArray[i] << " expected " << verify[i] << std::endl;
          num_discrepancies++;
        }
      }
    }
    std::cout << std::boolalpha << correct << std::endl;
    if (!correct) {
      std::cout << "num_discrepancies: " << num_discrepancies << std::endl;
      num_failures ++;
    }
#endif
  }
  std::cout << " Number of failures: " << num_failures << " out of " << NUM_ITERATIONS << std::endl;
  AverageTime = AverageTime/NUM_ITERATIONS;
  std::cout << "Average Time: " << AverageTime * 1000 << " ms" << std::endl;
  double stdDev = 0.0, minTime = 1000000.0, maxTime = 0.0;
  for(uint k = 0; k < NUM_ITERATIONS; k++)
  {
    stdDev += (AverageTime - times[k])*(AverageTime - times[k]);
    minTime = std::min(minTime, times[k]);
    maxTime = std::max(maxTime, times[k]);
  }

  if (NUM_ITERATIONS > 1) {
    stdDev = sqrt(stdDev/(NUM_ITERATIONS - 1));
    std::cout << "Standard Deviation: " << stdDev * 1000 << std::endl;
    std::cout << "%error (3*stdDev)/Average: " << 3*stdDev / AverageTime * 100 << "%" << std::endl;
    std::cout << "min time: " << minTime * 1000 << " ms" << std::endl;
    std::cout << "max time: " << maxTime * 1000 << " ms" << std::endl;
  }

#ifdef RUN_CPU_SORTS
  std::cout << "Average speedup over CPU quicksort: " << quickSortTime/AverageTime << std::endl;
  std::cout << "Average speedup over CPU std::sort: " << stdSortTime/AverageTime << std::endl;
#endif // RUN_CPU_SORTS

  printf("-------done--------------------------------------------------------\n");
  free(pArray);
  free(pArrayCopy);

  return 0;
}


int main(int argc, char** argv)
{
  unsigned int  NUM_ITERATIONS;
  uint      heightReSz, widthReSz;


  bool success = parseArgs (argc, argv, &NUM_ITERATIONS, &widthReSz, &heightReSz);
  if (!success) return -1;
  uint arraySize = widthReSz*heightReSz;
  test<uint>(arraySize, NUM_ITERATIONS, "uint");
  test<float>(arraySize, NUM_ITERATIONS, "float");
  test<double>(arraySize, NUM_ITERATIONS, "double");
  return 0;
}



