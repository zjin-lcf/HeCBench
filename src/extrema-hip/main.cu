// Copyright (c) 2019-2020, NVIDIA CORPORATION.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "kernels.h"
#include "reference.h"

template <typename T>
long test_1D (const int length, const int order, const bool clip,
              const int repeat, const char* type) 
{
  T* x = (T*) malloc (sizeof(T)*length);
  for (int i = 0; i < length; i++)
    x[i] = rand() % length;
  
  bool* cpu_r = (bool*) malloc (sizeof(bool)*length);
  bool* gpu_r = (bool*) malloc (sizeof(bool)*length);

  T* d_x;
  bool *d_result;
  hipMalloc((void**)&d_x, length*sizeof(T));
  hipMemcpy(d_x, x, length*sizeof(T), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_result, length*sizeof(bool));

  dim3 grids ((length+255)/256);
  dim3 threads (256);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int n = 0; n < repeat; n++)
    hipLaunchKernelGGL(HIP_KERNEL_NAME(relextrema_1D<T>), grids, threads, 0, 0, length, order, clip, d_x, d_result);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average 1D kernel (type = %s, order = %d, clip = %d) execution time %f (s)\n", 
         type, order, clip, (time * 1e-9f) / repeat);

  hipMemcpy(gpu_r, d_result, length*sizeof(bool), hipMemcpyDeviceToHost);

  cpu_relextrema_1D<T>(length, order, clip, x, cpu_r);

  int error = 0;
  for (int i = 0; i < length; i++)
    if (cpu_r[i] != gpu_r[i]) {
      error = 1; 
      break;
    }

  hipFree(d_x);
  hipFree(d_result);
  free(x);
  free(cpu_r);
  free(gpu_r);
  if (error) printf("1D test: FAILED\n");
  return time;
}

template <typename T>
long test_2D (const int length_x, const int length_y, const int order,
              const bool clip, const int axis, const int repeat, const char* type) 
{
  const int length = length_x * length_y;
  T* x = (T*) malloc (sizeof(T)*length);
  for (int i = 0; i < length; i++)
    x[i] = rand() % length;
  
  bool* cpu_r = (bool*) malloc (sizeof(bool)*length);
  bool* gpu_r = (bool*) malloc (sizeof(bool)*length);

  T* d_x;
  bool *d_result;
  hipMalloc((void**)&d_x, length*sizeof(T));
  hipMemcpy(d_x, x, length*sizeof(T), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_result, length*sizeof(bool));

  dim3 grids ((length_x+15)/16, (length_y+15)/16);
  dim3 threads (16, 16);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int n = 0; n < repeat; n++)
    hipLaunchKernelGGL(relextrema_2D, grids, threads, 0, 0, length_x, length_y, order, clip, axis, d_x, d_result);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

  printf("Average 2D kernel (type = %s, order = %d, clip = %d, axis = %d) execution time %f (s)\n", 
         type, order, clip, axis, (time * 1e-9f) / repeat);

  hipMemcpy(gpu_r, d_result, length*sizeof(bool), hipMemcpyDeviceToHost);

  cpu_relextrema_2D(length_x, length_y, order, clip, axis, x, cpu_r);

  int error = 0;
  for (int i = 0; i < length; i++)
    if (cpu_r[i] != gpu_r[i]) {
      error = 1; 
      break;
    }

  hipFree(d_x);
  hipFree(d_result);
  free(x);
  free(cpu_r);
  free(gpu_r);
  if (error) printf("2D test: FAILED\n");
  return time;
}

int main(int argc, char* argv[]) {
  if (argc != 2) {
    printf("Usage ./%s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  long time = 0;

  for (int order = 1; order <= 128; order = order * 2) {
    time += test_1D<   int>(1000000, order, true, repeat, "int");
    time += test_1D<  long>(1000000, order, true, repeat, "long");
    time += test_1D< float>(1000000, order, true, repeat, "float");
    time += test_1D<double>(1000000, order, true, repeat, "double");
  }

  for (int order = 1; order <= 128; order = order * 2) {
    time += test_2D<   int>(1000, 1000, order, true, 1, repeat, "int");
    time += test_2D<  long>(1000, 1000, order, true, 1, repeat, "long");
    time += test_2D< float>(1000, 1000, order, true, 1, repeat, "float");
    time += test_2D<double>(1000, 1000, order, true, 1, repeat, "double");
  }

  for (int order = 1; order <= 128; order = order * 2) {
    time += test_2D<   int>(1000, 1000, order, true, 0, repeat, "int");
    time += test_2D<  long>(1000, 1000, order, true, 0, repeat, "long");
    time += test_2D< float>(1000, 1000, order, true, 0, repeat, "float");
    time += test_2D<double>(1000, 1000, order, true, 0, repeat, "double");
  }

  printf("\n-----------------------------------------------\n");
  printf("Total kernel execution time: %lf (s)", time * 1e-9);
  printf("\n-----------------------------------------------\n");

  return 0;
}
