#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "DCT8x8.h"

void DCT8x8(
    float* d_Dst,
    const float* d_Src,
    unsigned int stride,
    unsigned int imageH,
    unsigned int imageW,
    int dir
);

int Verify(const float* h_OutputGPU, 
                  float* h_OutputCPU, 
            const float* h_Input, 
            const unsigned int stride,
            const unsigned int imageH,
            const unsigned int imageW,
            const int dir )
{
  printf("Comparing against Host/C++ computation...\n"); 
  DCT8x8CPU(h_OutputCPU, h_Input, stride, imageH, imageW, dir);
  double sum = 0, delta = 0;
  double L2norm;
  for(unsigned int i = 0; i < imageH; i++)
    for(unsigned int j = 0; j < imageW; j++){
      sum += h_OutputCPU[i * stride + j] * h_OutputCPU[i * stride + j];
      delta += (h_OutputGPU[i * stride + j] - h_OutputCPU[i * stride + j]) * 
               (h_OutputGPU[i * stride + j] - h_OutputCPU[i * stride + j]);
    }
  L2norm = sqrt(delta / sum);
  printf("Relative L2 norm: %.3e\n\n", L2norm);
  if (L2norm < 1E-6) {
    printf("PASS\n"); 
    return 0;
  } else {
    printf("FAIL\n");
    return 1;
  }
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
  if (argc != 4) {
    printf("Usage: %s <image width> <image height> <repeat>\n", argv[0]);
    return 1;
  }
  const unsigned int imageW = atoi(argv[1]);
  const unsigned int imageH = atoi(argv[2]);
  const int numIterations = atoi(argv[3]);
  const unsigned int stride = imageW;

  float *h_Input, *h_OutputCPU, *h_OutputGPU;

  printf("Allocating and initializing host memory...\n");
  h_Input     = (float *)malloc(imageH * stride * sizeof(float));
  h_OutputCPU = (float *)malloc(imageH * stride * sizeof(float));
  h_OutputGPU = (float *)malloc(imageH * stride * sizeof(float));

  srand(2009);
  for(unsigned int i = 0; i < imageH; i++)
    for(unsigned int j = 0; j < imageW; j++)
      h_Input[i * stride + j] = (float)rand() / (float)RAND_MAX;

  float* d_Input;
  hipMalloc((void**)&d_Input, sizeof(float) * imageH * stride);
  hipMemcpy(d_Input, h_Input, sizeof(float) * imageH * stride, hipMemcpyHostToDevice);

  float* d_Output;
  hipMalloc((void**)&d_Output, sizeof(float) * imageH * stride);

  printf("Performing Forward DCT8x8 of %u x %u image on the device\n\n", imageH, imageW);

  int dir = DCT_FORWARD;

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for(int iter = 0; iter < numIterations; iter++)
    DCT8x8(
        d_Output,
        d_Input,
        stride,
        imageH,
        imageW,
        dir );

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average DCT8x8 kernel execution time %f (s)\n", (time * 1e-9f) / numIterations);

  hipMemcpy(h_OutputGPU, d_Output, sizeof(float) * imageH * stride, hipMemcpyDeviceToHost);

  if (Verify(h_OutputGPU, h_OutputCPU, h_Input, stride, imageH, imageW, dir) != 0)
    exit(1);

  printf("Performing Inverse DCT8x8 of %u x %u image on the device\n\n", imageH, imageW);

  dir = DCT_INVERSE;

  start = std::chrono::steady_clock::now();

  for(int iter = 0; iter < numIterations; iter++)
    DCT8x8(
        d_Output,
        d_Input,
        stride,
        imageH,
        imageW,
        dir );

  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average IDCT8x8 kernel execution time %f (s)\n", (time * 1e-9f) / numIterations);

  hipMemcpy(h_OutputGPU, d_Output, sizeof(float) * imageH * stride, hipMemcpyDeviceToHost);

  if (Verify(h_OutputGPU, h_OutputCPU, h_Input, stride, imageH, imageW, dir) != 0)
    exit(1);

  hipFree(d_Input);
  hipFree(d_Output);
  free(h_OutputGPU);
  free(h_OutputCPU);
  free(h_Input);

  return 0;
}
