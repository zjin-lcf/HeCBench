#include "hip/hip_runtime.h"
/**********************************************************************
  Copyright �2013 Advanced Micro Devices, Inc. All rights reserved.

  Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

  �   Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
  �   Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or
  other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY
  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS
  OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ********************************************************************/


#include "sobel.h"
#include "SDKBitMap.h"
#include "kernels.cu"

static bool compare(const float *refData, const float *data,
                    const int length, const float epsilon = 1e-6f)
{
  float error = 0.0f;
  float ref = 0.0f;
  for(int i = 1; i < length; ++i)
  {
    float diff = refData[i] - data[i];
    // if (diff != 0) printf("mismatch @%d: %f %f\n", i, refData[i] , data[i]);
    error += diff * diff;
    ref += refData[i] * refData[i];
  }
  float normRef = sqrtf((float) ref);
  if (fabs((float) ref) < 1e-7f)
  {
    return false;
  }
  float normError = sqrtf((float) error);
  error = normError / normRef;
  return error < epsilon;
}

int main(int argc, char * argv[])
{
  if (argc != 3) {
    printf("Usage: %s <path to file> <repeat>\n", argv[0]);
    return 1;
  }
  const char* filePath = argv[1];
  const int iterations = atoi(argv[2]);

  // load input bitmap image
  SDKBitMap inputBitmap;

  inputBitmap.load(filePath);

  // error if image did not load
  if(!inputBitmap.isLoaded())
  {
    printf("Failed to load input image!");
    return SDK_FAILURE;
  }

  // get width and height of input image
  const int height = inputBitmap.getHeight();
  const int width = inputBitmap.getWidth();
  const int pixelSize = sizeof(uchar4);
  const int imageSize = width * height * pixelSize;
  printf("Image height = %d and width = %d\n", height, width);

  // allocate memory for input image data
  uchar4 *inputImageData  = (uchar4*) malloc (imageSize);
  if (inputImageData == NULL)
    printf("Failed to allocate memory! (inputImageData)");

  // allocate memory for output image data
  uchar4 *outputImageData = (uchar4*) malloc (imageSize);
  if (outputImageData == NULL) 
    printf("Failed to allocate memory! (outputImageData)");

  // initialize the output
  memset(outputImageData, 0, imageSize);

  // get the pointer to pixel data
  uchar4 *pixelData = inputBitmap.getPixels();
  if(pixelData == NULL)
    printf("Failed to read pixel Data!");

  // Copy pixel data into inputImageData
  memcpy(inputImageData, pixelData, imageSize);

  // allocate memory for verification output
  uchar4* verificationOutput = (uchar4*) malloc (imageSize);
  if (verificationOutput == NULL) 
    printf("verificationOutput heap allocation failed!");

  // initialize the output
  memset(verificationOutput, 0, imageSize);

  // Create memory object for input Image
  uchar4 *inputImageBuffer;
  hipMalloc((void**)&inputImageBuffer, imageSize); 
  hipMemcpy(inputImageBuffer, inputImageData, imageSize, hipMemcpyHostToDevice);

  uchar4 *outputImageBuffer;
  hipMalloc((void**)&outputImageBuffer, imageSize); 
  hipMemset(outputImageBuffer, 0, imageSize); 

  // Enqueue a kernel run call.
  const int blockSizeX = 16;
  const int blockSizeY = 16;
  dim3 grid (width/blockSizeX, height/blockSizeY);
  dim3 block (blockSizeX, blockSizeY);

  printf("Executing kernel for %d iterations", iterations);
  printf("-------------------------------------------\n");

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for(int i = 0; i < iterations; i++)
  {
    sobel_filter<<<grid, block>>>(inputImageBuffer, outputImageBuffer, width, height);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time: %f (us)\n", (time * 1e-3f) / iterations);

  hipMemcpy(outputImageData, outputImageBuffer, imageSize, hipMemcpyDeviceToHost);
  hipFree(inputImageBuffer);
  hipFree(outputImageBuffer);

  // reference implementation
  reference (verificationOutput, inputImageData, width, height, pixelSize);

  float *outputDevice = (float*) malloc (imageSize * sizeof(float));
  if (outputDevice == NULL)
    printf("Failed to allocate host memory! (outputDevice)");

  float *outputReference = (float*) malloc (imageSize * sizeof(float));

  if (outputReference == NULL)
    printf("Failed to allocate host memory!" "(outputReference)");

  // copy uchar data to float array
  for(int i = 0; i < width * height; i++)
  {
    outputDevice[i * 4 + 0] = outputImageData[i].x;
    outputDevice[i * 4 + 1] = outputImageData[i].y;
    outputDevice[i * 4 + 2] = outputImageData[i].z;
    outputDevice[i * 4 + 3] = outputImageData[i].w;

    outputReference[i * 4 + 0] = verificationOutput[i].x;
    outputReference[i * 4 + 1] = verificationOutput[i].y;
    outputReference[i * 4 + 2] = verificationOutput[i].z;
    outputReference[i * 4 + 3] = verificationOutput[i].w;
  }

  // compare the results and see if they match
  bool ok = compare(outputReference, outputDevice, imageSize);
  if(ok)
    printf("PASS\n");
  else
    printf("FAIL\n");

  free(outputDevice);
  free(outputReference);
  free(verificationOutput);
  free(inputImageData);
  free(outputImageData);
  return ok ? 0 : 1;
}
