#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <chrono>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

#include "reference.h"

static void CheckError( hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
  }
}
#define CHECK_ERROR( err ) (CheckError( err, __FILE__, __LINE__ ))


template <typename T>
__global__ void BlockRangeAtomicOnGlobalMem(T* data, int n)
{
  unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  for ( unsigned int i = tid; i < n; i += blockDim.x*gridDim.x){
    atomicAdd(data+threadIdx.x, (T)1);  //arbitrary number to add
  }
}

template <typename T>
__global__ void WarpRangeAtomicOnGlobalMem(T* data, int n)
{
  unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  for ( unsigned int i = tid; i < n; i += blockDim.x*gridDim.x){
    atomicAdd(data+(i & 0x1F), (T)1); //arbitrary number to add
  }
}

template <typename T>
__global__ void SingleRangeAtomicOnGlobalMem(T* data, int offset, int n)
{
  unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  for ( unsigned int i = tid; i < n; i += blockDim.x*gridDim.x){
    atomicAdd(data+offset, (T)1);    //arbitrary number to add
  }
}

template <typename T>
__global__ void BlockRangeAtomicOnSharedMem(T* data, int n)
{
  __shared__ T smem_data[BLOCK_SIZE];
  unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  for ( unsigned int i = tid; i < n; i += blockDim.x*gridDim.x){
    atomicAdd(smem_data+threadIdx.x, (T)1);
  }
  if (blockIdx.x == gridDim.x)
    data[threadIdx.x] = smem_data[threadIdx.x];
}

template <typename T>
__global__ void WarpRangeAtomicOnSharedMem(T* data, int n)
{
  __shared__ T smem_data[32];
  unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  for ( unsigned int i = tid; i < n; i += blockDim.x*gridDim.x){
    atomicAdd(smem_data+(i & 0x1F), (T)1);
  }
  if (blockIdx.x == gridDim.x && threadIdx.x < 0x1F)
    data[threadIdx.x] = smem_data[threadIdx.x];
}

template <typename T>
__global__ void SingleRangeAtomicOnSharedMem(T* data, int offset, int n)
{
  __shared__ T smem_data[BLOCK_SIZE];
  unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  for ( unsigned int i = tid; i < n; i += blockDim.x*gridDim.x){
    atomicAdd(smem_data + offset, (T)1);
  }
  if (blockIdx.x == gridDim.x && threadIdx.x == 0)
    data[threadIdx.x] = smem_data[threadIdx.x];
}

template <typename T>
void atomicPerf (int n, int t, int repeat)
{
  size_t data_size = sizeof(T) * t;

  T* data = (T*) malloc (data_size);
  T* h_data = (T*) malloc (data_size);
  T* r_data = (T*) malloc (data_size);
  int fail;

  for(int i=0; i<t; i++) {
    data[i] = i%1024+1;
  }

  T* d_data;
  CHECK_ERROR( hipMalloc((void **)&d_data, data_size) );

  dim3 block (BLOCK_SIZE);
  dim3 grid (n / BLOCK_SIZE);

  CHECK_ERROR( hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice) );
  CHECK_ERROR( hipDeviceSynchronize() );
  auto start = std::chrono::steady_clock::now();
  for(int i=0; i<repeat; i++)
  {
    BlockRangeAtomicOnGlobalMem<T><<<grid, block>>>(d_data, n);
  }
  CHECK_ERROR( hipDeviceSynchronize() );
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of BlockRangeAtomicOnGlobalMem: %f (us)\n",
          time * 1e-3f / repeat);

  CHECK_ERROR( hipMemcpy(h_data, d_data, data_size, hipMemcpyDeviceToHost) );
  memcpy(r_data, data, data_size);
  for(int i=0; i<repeat; i++)
    BlockRangeAtomicOnGlobalMem_ref<T>(r_data, n);
  fail = memcmp(h_data, r_data, data_size);
  printf("%s\n", fail ? "FAIL" : "PASS");
  
  CHECK_ERROR( hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice) );
  CHECK_ERROR( hipDeviceSynchronize() );
  start = std::chrono::steady_clock::now();
  for(int i=0; i<repeat; i++)
  {
    WarpRangeAtomicOnGlobalMem<T><<<grid, block>>>(d_data, n);
  }
  CHECK_ERROR( hipDeviceSynchronize() );
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of WarpRangeAtomicOnGlobalMem: %f (us)\n",
          time * 1e-3f / repeat);

  CHECK_ERROR( hipMemcpy(h_data, d_data, data_size, hipMemcpyDeviceToHost) );
  memcpy(r_data, data, data_size);
  for(int i=0; i<repeat; i++)
    WarpRangeAtomicOnGlobalMem_ref<T>(r_data, n);
  fail = memcmp(h_data, r_data, data_size);
  printf("%s\n", fail ? "FAIL" : "PASS");

  CHECK_ERROR( hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice) );
  CHECK_ERROR( hipDeviceSynchronize() );
  start = std::chrono::steady_clock::now();
  for(int i=0; i<repeat; i++)
  {
    SingleRangeAtomicOnGlobalMem<T><<<grid, block>>>(d_data, i % BLOCK_SIZE, n);
  }
  CHECK_ERROR( hipDeviceSynchronize() );
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of SingleRangeAtomicOnGlobalMem: %f (us)\n",
          time * 1e-3f / repeat);

  CHECK_ERROR( hipMemcpy(h_data, d_data, data_size, hipMemcpyDeviceToHost) );
  memcpy(r_data, data, data_size);
  for(int i=0; i<repeat; i++)
    SingleRangeAtomicOnGlobalMem_ref<T>(r_data, i % BLOCK_SIZE, n);
  fail = memcmp(h_data, r_data, data_size);
  printf("%s\n", fail ? "FAIL" : "PASS");

  CHECK_ERROR( hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice) );
  CHECK_ERROR( hipDeviceSynchronize() );
  start = std::chrono::steady_clock::now();
  for(int i=0; i<repeat; i++)
  {
    BlockRangeAtomicOnSharedMem<T><<<grid, block>>>(d_data, n);
  }
  CHECK_ERROR( hipDeviceSynchronize() );
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of BlockRangeAtomicOnSharedMem: %f (us)\n",
          time * 1e-3f / repeat);

  CHECK_ERROR( hipMemcpy(h_data, d_data, data_size, hipMemcpyDeviceToHost) );
  fail = memcmp(h_data, data, data_size);
  printf("%s\n", fail ? "FAIL" : "PASS");

  CHECK_ERROR( hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice) );
  CHECK_ERROR( hipDeviceSynchronize() );
  start = std::chrono::steady_clock::now();
  for(int i=0; i<repeat; i++)
  {
    WarpRangeAtomicOnSharedMem<T><<<grid, block>>>(d_data, n);
  }
  CHECK_ERROR( hipDeviceSynchronize() );
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of WarpRangeAtomicOnSharedMem: %f (us)\n",
          time * 1e-3f / repeat);

  CHECK_ERROR( hipMemcpy(h_data, d_data, data_size, hipMemcpyDeviceToHost) );
  fail = memcmp(h_data, data, data_size);
  printf("%s\n", fail ? "FAIL" : "PASS");

  CHECK_ERROR( hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice) );
  CHECK_ERROR( hipDeviceSynchronize() );
  start = std::chrono::steady_clock::now();
  for(int i=0; i<repeat; i++)
  {
    SingleRangeAtomicOnSharedMem<T><<<grid, block>>>(d_data, i % BLOCK_SIZE, n);
  }
  CHECK_ERROR( hipDeviceSynchronize() );
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of SingleRangeAtomicOnSharedMem: %f (us)\n",
          time * 1e-3f / repeat);

  CHECK_ERROR( hipMemcpy(h_data, d_data, data_size, hipMemcpyDeviceToHost) );
  fail = memcmp(h_data, data, data_size);
  printf("%s\n", fail ? "FAIL" : "PASS");

  free(data);
  free(h_data);
  free(r_data);
  hipFree(d_data); 
}

int main(int argc, char* argv[])
{
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  const int n = 3*4*7*8*9*256; // number of threads
  const int len = 1024; // data array length
  
  printf("\nFP64 atomic add\n");
  atomicPerf<double>(n, len, repeat); 

  printf("\nINT32 atomic add\n");
  atomicPerf<int>(n, len, repeat); 

  printf("\nFP32 atomic add\n");
  atomicPerf<float>(n, len, repeat); 

  return 0;
}
