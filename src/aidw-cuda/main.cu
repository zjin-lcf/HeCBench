#include "hip/hip_runtime.h"
/*
 * GPU-accelerated AIDW interpolation algorithm 
 *
 * Implemented with / without Shared Memory
 *
 * By Dr.Gang Mei
 *
 * Created on 2015.11.06, China University of Geosciences, 
 *                        gang.mei@cugb.edu.cn
 * Revised on 2015.12.14, China University of Geosciences, 
 *                        gang.mei@cugb.edu.cn
 * 
 * Related publications:
 *  1) "Evaluating the Power of GPU Acceleration for IDW Interpolation Algorithm"
 *     http://www.hindawi.com/journals/tswj/2014/171574/
 *  2) "Accelerating Adaptive IDW Interpolation Algorithm on a Single GPU"
 *     http://arxiv.org/abs/1511.02186
 *
 * License: http://creativecommons.org/licenses/by/4.0/
 */

#include <cstdio>
#include <cstdlib>     
#include <vector>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>
#include "reference.h"

// Calculate the power parameter, and then weighted interpolating
// Without using shared memory
__global__
void AIDW_Kernel(
    const float *__restrict dx, 
    const float *__restrict dy,
    const float *__restrict dz,
    const int dnum,
    const float *__restrict ix,
    const float *__restrict iy,
          float *__restrict iz,
    const int inum,
    const float area,
    const float *__restrict avg_dist) 

{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < inum) {
    float sum = 0.f, dist = 0.f, t = 0.f, z = 0.f, alpha = 1.f;

    float r_obs = avg_dist[tid];                // The observed average nearest neighbor distance
    float r_exp = 0.5f / sqrtf(dnum / area);    // The expected nearest neighbor distance for a random pattern
    float R_S0 = r_obs / r_exp;                 // The nearest neighbor statistic

    // Normalize the R(S0) measure such that it is bounded by 0 and 1 by a fuzzy membership function 
    float u_R = 0.f;
    if(R_S0 >= R_min) u_R = 0.5f-0.5f * cosf(3.1415926f / R_max * (R_S0 - R_min));
    if(R_S0 >= R_max) u_R = 1.f;

    // Determine the appropriate distance-decay parameter alpha by a triangular membership function
    // Adaptive power parameter: a (alpha)
    if(u_R>= 0.f && u_R<=0.1f)  alpha = a1; 
    if(u_R>0.1f && u_R<=0.3f)  alpha = a1*(1.f-5.f*(u_R-0.1f)) + a2*5.f*(u_R-0.1f);
    if(u_R>0.3f && u_R<=0.5f)  alpha = a3*5.f*(u_R-0.3f) + a1*(1.f-5.f*(u_R-0.3f));
    if(u_R>0.5f && u_R<=0.7f)  alpha = a3*(1.f-5.f*(u_R-0.5f)) + a4*5.f*(u_R-0.5f);
    if(u_R>0.7f && u_R<=0.9f)  alpha = a5*5.f*(u_R-0.7f) + a4*(1.f-5.f*(u_R-0.7f));
    if(u_R>0.9f && u_R<=1.f)  alpha = a5;
    alpha *= 0.5f; // Half of the power

    // Weighted average
    for(int j = 0; j < dnum; j++) {
      dist = (ix[tid] - dx[j]) * (ix[tid] - dx[j]) + (iy[tid] - dy[j]) * (iy[tid] - dy[j]) ;
      t = 1.f / powf(dist, alpha);  sum += t;  z += dz[j] * t;
    }
    iz[tid] = z / sum;
  }
}

// Calculate the power parameter, and then weighted interpolating
// With using shared memory (Tiled version of the stage 2)
__global__
void AIDW_Kernel_Tiled(
    const float *__restrict dx, 
    const float *__restrict dy,
    const float *__restrict dz,
    const int dnum,
    const float *__restrict ix,
    const float *__restrict iy,
          float *__restrict iz,
    const int inum,
    const float area,
    const float *__restrict avg_dist)
{
  // Shared Memory
  __shared__ float sdx[BLOCK_SIZE];
  __shared__ float sdy[BLOCK_SIZE];
  __shared__ float sdz[BLOCK_SIZE];

  int tid = threadIdx.x + blockIdx.x * blockDim.x; 
  if (tid >= inum) return;

  float dist = 0.f, t = 0.f, alpha = 0.f;

  int part = (dnum - 1) / BLOCK_SIZE;
  int m, e;

  float sum_up = 0.f;
  float sum_dn = 0.f;   
  float six_s, siy_s;

  float r_obs = avg_dist[tid];               //The observed average nearest neighbor distance
  float r_exp = 0.5f / sqrtf(dnum / area); // The expected nearest neighbor distance for a random pattern
  float R_S0 = r_obs / r_exp;                //The nearest neighbor statistic

  float u_R = 0.f;
  if(R_S0 >= R_min) u_R = 0.5f-0.5f * cosf(3.1415926f / R_max * (R_S0 - R_min));
  if(R_S0 >= R_max) u_R = 1.f;

  // Determine the appropriate distance-decay parameter alpha by a triangular membership function
  // Adaptive power parameter: a (alpha)
  if(u_R>= 0.f && u_R<=0.1f)  alpha = a1; 
  if(u_R>0.1f && u_R<=0.3f)  alpha = a1*(1.f-5.f*(u_R-0.1f)) + a2*5.f*(u_R-0.1f);
  if(u_R>0.3f && u_R<=0.5f)  alpha = a3*5.f*(u_R-0.3f) + a1*(1.f-5.f*(u_R-0.3f));
  if(u_R>0.5f && u_R<=0.7f)  alpha = a3*(1.f-5.f*(u_R-0.5f)) + a4*5.f*(u_R-0.5f);
  if(u_R>0.7f && u_R<=0.9f)  alpha = a5*5.f*(u_R-0.7f) + a4*(1.f-5.f*(u_R-0.7f));
  if(u_R>0.9f && u_R<=1.f)  alpha = a5;
  alpha *= 0.5f; // Half of the power

  float six_t = ix[tid];
  float siy_t = iy[tid];
  int lid = threadIdx.x;
  for(m = 0; m <= part; m++) {  // Weighted Sum  
    int num_threads = min(BLOCK_SIZE, dnum - BLOCK_SIZE *m);
    if (lid < num_threads) {
      sdx[lid] = dx[lid + BLOCK_SIZE * m];
      sdy[lid] = dy[lid + BLOCK_SIZE * m];
      sdz[lid] = dz[lid + BLOCK_SIZE * m];
    }
    __syncthreads();

    for(e = 0; e < BLOCK_SIZE; e++) {
      six_s = six_t - sdx[e];
      siy_s = siy_t - sdy[e];
      dist = (six_s * six_s + siy_s * siy_s);
      t = 1.f / powf(dist, alpha);  sum_dn += t;  sum_up += t * sdz[e];
    }
  
  __syncthreads();

  }
  iz[tid] = sum_up / sum_dn;
}

int main(int argc, char *argv[])
{
  if (argc != 4) {
    printf("Usage: %s <pts> <check> <iterations>\n", argv[0]);
    printf("pts: number of points (unit: 1K)\n");
    printf("check: enable verification when the value is 1\n");
    return 1;
  }

  const int numk = atoi(argv[1]); // number of points (unit: 1K)
  const int check = atoi(argv[2]); // do check for small problem sizes
  const int iterations = atoi(argv[3]); // repeat kernel execution

  const int dnum = numk * 1024;
  const int inum = dnum;
  const size_t dnum_size = dnum * sizeof(float);
  const size_t inum_size = inum * sizeof(float);

  // Area of planar region
  const float width = 2000, height = 2000;
  const float area = width * height;

  std::vector<float> dx(dnum), dy(dnum), dz(dnum);
  std::vector<float> avg_dist(dnum);
  std::vector<float> ix(inum), iy(inum), iz(inum);
  std::vector<float> h_iz(inum);

  srand(123);
  for(int i = 0; i < dnum; i++)
  {
    dx[i] = rand()/(float)RAND_MAX * 1000;
    dy[i] = rand()/(float)RAND_MAX * 1000;
    dz[i] = rand()/(float)RAND_MAX * 1000;
  }

  for(int i = 0; i < inum; i++)
  {
    ix[i] = rand()/(float)RAND_MAX * 1000;
    iy[i] = rand()/(float)RAND_MAX * 1000;
    iz[i] = 0.f;
  }

  for(int i = 0; i < dnum; i++)
  {
    avg_dist[i] = rand()/(float)RAND_MAX * 3;
  }

  printf("Size = : %d K \n", numk);
  printf("dnum = : %d\ninum = : %d\n", dnum, inum);

  if (check) {
    printf("Verification enabled\n");
    reference (dx.data(), dy.data(), dz.data(), dnum, ix.data(), 
               iy.data(), h_iz.data(), inum, area, avg_dist.data());
  } else {
    printf("Verification disabled\n");
  }

  float *d_dx, *d_dy, *d_dz;
  float *d_avg_dist;
  float *d_ix, *d_iy, *d_iz;

  hipMalloc((void**)&d_dx, dnum_size); 
  hipMalloc((void**)&d_dy, dnum_size); 
  hipMalloc((void**)&d_dz, dnum_size); 
  hipMalloc((void**)&d_avg_dist, dnum_size); 
  hipMalloc((void**)&d_ix, inum_size); 
  hipMalloc((void**)&d_iy, inum_size); 
  hipMalloc((void**)&d_iz, inum_size); 

  hipMemcpy(d_dx, dx.data(), dnum_size, hipMemcpyHostToDevice); 
  hipMemcpy(d_dy, dy.data(), dnum_size, hipMemcpyHostToDevice); 
  hipMemcpy(d_dz, dz.data(), dnum_size, hipMemcpyHostToDevice); 
  hipMemcpy(d_avg_dist, avg_dist.data(), dnum_size, hipMemcpyHostToDevice); 
  hipMemcpy(d_ix, ix.data(), inum_size, hipMemcpyHostToDevice); 
  hipMemcpy(d_iy, iy.data(), inum_size, hipMemcpyHostToDevice); 

  dim3 threadsPerBlock (BLOCK_SIZE);
  dim3 blocksPerGrid ((inum + BLOCK_SIZE - 1) / BLOCK_SIZE);

  // Weighted Interpolate using AIDW

  AIDW_Kernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_dx, d_dy, d_dz, dnum, d_ix, d_iy, d_iz, inum, area, d_avg_dist);
  hipMemcpy(iz.data(), d_iz, inum_size, hipMemcpyDeviceToHost); 

  if (check) {
    bool ok = verify (iz.data(), h_iz.data(), inum, EPS);
    printf("%s\n", ok ? "PASS" : "FAIL");
  }

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < iterations; i++)
    AIDW_Kernel_Tiled<<<blocksPerGrid, threadsPerBlock>>>(
      d_dx, d_dy, d_dz, dnum, d_ix, d_iy, d_iz, inum, area, d_avg_dist);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time %f (s)\n", (time * 1e-9f) / iterations);

  hipFree(d_dx);
  hipFree(d_dy);
  hipFree(d_dz);
  hipFree(d_ix);
  hipFree(d_iy);
  hipFree(d_iz);
  hipFree(d_avg_dist);
  return 0;
}
