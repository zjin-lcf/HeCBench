#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

// A C model derived from the OpenCL kernel 
void softMax_cpu(const int numSlice, const int sliceSize, const float* src, float* dest) {
  for (int i = 0; i < numSlice; i++) {
    float max_ = src[i * sliceSize];
    for (int j = 1; j < sliceSize; j++) {
      max_ = (max_ < src[i * sliceSize + j]) ? src[i * sliceSize + j] : max_;
    }
    float sum = 0;
    for (int j = 0; j < sliceSize; j++) {
      float e = expf(src[i * sliceSize + j] - max_);
      sum += e;
      dest[i * sliceSize + j] = e;
    }
    for (int j = 0; j < sliceSize; j++) {
      dest[i * sliceSize + j] /= sum;
    }
  }
}

__global__
void softMax (const int numSlice, const int sliceSize,
              const float* src, float* dest)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= numSlice) return;
  float max_ = src[i * sliceSize];
  for (int j = 1; j < sliceSize; j++) {
    max_ = max(max_, src[i * sliceSize + j]);
  }
  float sum = 0;
  for (int j = 0; j < sliceSize; j++) {
    sum += expf(src[i * sliceSize + j] - max_);
  }
  for (int j = 0; j < sliceSize; j++) {
    dest[i * sliceSize + j] = expf(src[i * sliceSize + j] - max_) / sum;
  }
}

int main(int argc, char* argv[]) {
  if (argc != 4) {
    printf("Usage: %s <number of slices> <slice size> <repeat>\n", argv[0]);
    return 1;
  }
   
  int numSlice = atoi(argv[1]);
  int sliceSize = atoi(argv[2]);
  int repeat = atoi(argv[3]);
  int numElem = numSlice * sliceSize;

  float* input = (float*) aligned_alloc(1024, sizeof(float) * numElem);
  float* output_gpu = (float*) aligned_alloc(1024, sizeof(float) * numElem);
  float* output_cpu = (float*) aligned_alloc(1024, sizeof(float) * numElem);

  srand(2);
  for (int i = 0; i < numSlice; i++)
    for (int j = 0; j < sliceSize; j++)
      input[i*sliceSize+j] = rand() % 13; 

  float *d_input, *d_output;
  hipMalloc((void**)&d_input, sizeof(float) * numElem);
  hipMalloc((void**)&d_output, sizeof(float) * numElem);
  hipMemcpy(d_input, input, sizeof(float) * numElem, hipMemcpyHostToDevice);

  dim3 global_work_size ((numSlice+BLOCK_SIZE-1)/BLOCK_SIZE);
  dim3 local_work_size (BLOCK_SIZE);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int n = 0; n < repeat; n++) {
    hipLaunchKernelGGL(softMax, global_work_size, local_work_size, 0, 0,
                       numSlice, sliceSize, d_input, d_output);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time: %f (s)\n", (time * 1e-9f) / repeat);

  hipMemcpy(output_gpu, d_output, sizeof(float) * numElem, hipMemcpyDeviceToHost);

  // verification
  bool ok = true;
  softMax_cpu(numSlice, sliceSize, input, output_cpu);
  for (int i = 0; i < numElem; i++) {
    if (fabsf(output_cpu[i] - output_gpu[i]) > 1e-3) {
      printf("@index %d host: %f device: %f\n", i, output_cpu[i], output_gpu[i]);
      ok = false;
      break;
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");

  free(input);
  free(output_cpu);
  free(output_gpu);
  hipFree(d_input);
  hipFree(d_output);
  return ok ? 0 : 1;
}
