#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <stdint.h>
#include <chrono>
#include <utility>  // std::swap
#include <hip/hip_runtime.h>
#include "utils.h"
#include "kernels.h"
#include "kernels_wrapper.h"

//#define STBI_ONLY_BMP
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define HIP_CHECK(call)                                                    \
do {                                                                       \
    hipError_t err_ = call;                                                \
    if (err_ != hipSuccess) {                                              \
        fprintf(stderr, "HIP error at %s:%d code=%d(%s) \"%s\" \n",        \
                __FILE__, __LINE__, err_, hipGetErrorString(err_), #call); \
        exit(EXIT_FAILURE);                                                \
    }                                                                      \
} while (0)


int main(int argc, char **argv) {
  if(argc < 3){
    printf("Usage: %s <file> <number of seams to remove> [options]\n"
           "valid options:\n-u\tupdate costs instead of recomputing them\n"
           "-a\tapproximate computation\n", argv[0]);
    return 1;
  }

  char *check;
  long seams_to_remove = strtol(argv[2], &check, 10);  //10 specifies base-10
  if (check == argv[2]){   //if no characters were converted pointers are equal
    printf("ERROR: can't convert string to number, exiting.\n");
    return 1;
  }

  int w, h, ncomp;
  unsigned char* imgv = stbi_load(argv[1], &w, &h, &ncomp, 0);
  if(imgv == NULL){
    printf("ERROR: can't load image \"%s\", exiting.\n", argv[1]);
    printf("Reason: %s\n", stbi_failure_reason());
    return 1;
  }

  if(ncomp != 3){
    printf("ERROR: image does not have 3 components (RGB), exiting.\n");
    return 1;
  }

  if(seams_to_remove < 0 || seams_to_remove >= w){
    printf("ERROR: number of seams to remove is invalid, exiting.\n");
    return 1;
  }

  seam_carver_mode mode = SEAM_CARVER_STANDARD_MODE;

  if(argc >= 4){
    if(strcmp(argv[3],"-u") == 0){
      mode = SEAM_CARVER_UPDATE_MODE;
      printf("update mode selected.\n");
    }
    else if(strcmp(argv[3],"-a") == 0){
      mode = SEAM_CARVER_APPROX_MODE;
      printf("approximation mode selected.\n");
    }
    else{
      printf("an invalid option was specified and will be ignored. Valid options are: -u, -a.\n");
    }
  }

  printf("Image loaded. Resizing...\n");

  int current_w = w;
  uchar4 *h_pixels = build_pixels(imgv, w, h);
  const size_t img_bytes = (size_t)w * h * sizeof(uchar4);
  const int cost_bytes = w * h * sizeof(short);
  const int h_bytes = h * sizeof(int);
  const int w_bytes = w * sizeof(int);

  uchar4 *d_pixels;
  uchar4 *d_pixels_swap;
  short *d_costs_left, *d_costs_swap_left;
  short *d_costs_up, *d_costs_swap_up;
  short *d_costs_right, *d_costs_swap_right;
  int *d_index_map;
  int *d_offset_map;
  int *d_indices_ref;
  int *d_indices;
  int *d_seam;
  int *reduce_row; //M row to consider for reduce
  int *d_M;

  if(mode != SEAM_CARVER_APPROX_MODE) {
    HIP_CHECK(hipMalloc((void**)&d_costs_left, cost_bytes));
    HIP_CHECK(hipMalloc((void**)&d_costs_up, cost_bytes));
    HIP_CHECK(hipMalloc((void**)&d_costs_right, cost_bytes));
  }
  if(mode == SEAM_CARVER_UPDATE_MODE){
    HIP_CHECK(hipMalloc((void**)&d_costs_swap_left, cost_bytes));
    HIP_CHECK(hipMalloc((void**)&d_costs_swap_up, cost_bytes));
    HIP_CHECK(hipMalloc((void**)&d_costs_swap_right, cost_bytes));
  }
  //sum map in approx mode
  HIP_CHECK(hipMalloc((void**)&d_M, img_bytes));

  // rows to consider for reduce
  if(mode == SEAM_CARVER_APPROX_MODE)
    reduce_row = d_M; //first row
  else
    reduce_row = d_M + w*(h-1); //last row

  if(mode == SEAM_CARVER_APPROX_MODE){
    HIP_CHECK(hipMalloc((void**)&d_index_map, img_bytes));
    HIP_CHECK(hipMalloc((void**)&d_offset_map, img_bytes));
  }

  int* indices = (int*)malloc(w_bytes);
  for(int i = 0; i < w; i++) indices[i] = i;

  HIP_CHECK(hipMalloc((void**)&d_indices, w_bytes));

  HIP_CHECK(hipMalloc((void**)&d_indices_ref, w_bytes));
  HIP_CHECK(hipMemcpy(d_indices_ref, indices, w_bytes, hipMemcpyHostToDevice));
  HIP_CHECK(hipMalloc((void**)&d_seam, h_bytes));

  HIP_CHECK(hipMalloc((void**)&d_pixels, img_bytes));
  HIP_CHECK(hipMemcpy(d_pixels, h_pixels, img_bytes, hipMemcpyHostToDevice));
  HIP_CHECK(hipMalloc((void**)&d_pixels_swap, img_bytes));

  if(mode == SEAM_CARVER_UPDATE_MODE)
    compute_costs(current_w, w, h, d_pixels, d_costs_left, d_costs_up, d_costs_right);

  int num_iterations = 0;

  auto start = std::chrono::steady_clock::now();

  while(num_iterations < (int)seams_to_remove){

    if(mode == SEAM_CARVER_STANDARD_MODE)
      compute_costs(current_w, w, h, d_pixels, d_costs_left, d_costs_up, d_costs_right);

    if(mode != SEAM_CARVER_APPROX_MODE){
      compute_M(current_w, w, h, d_M, d_costs_left, d_costs_up, d_costs_right);
      find_min_index(current_w, d_indices_ref, d_indices, reduce_row);
      find_seam(current_w, w, h, d_M, d_indices, d_seam);
    }
    else{
      approx_setup(current_w, w, h, d_pixels, d_index_map, d_offset_map, d_M);
      approx_M(current_w, w, h, d_offset_map,  d_M);
      find_min_index(current_w, d_indices_ref, d_indices, reduce_row);
      approx_seam(w, h, d_index_map, d_indices, d_seam);
    }

    remove_seam(current_w, w, h, d_M, d_pixels, d_pixels_swap, d_seam);
    std::swap(d_pixels, d_pixels_swap);

    if(mode == SEAM_CARVER_UPDATE_MODE){
      update_costs(current_w, w, h, d_M, d_pixels,
                   d_costs_left, d_costs_up, d_costs_right,
                   d_costs_swap_left, d_costs_swap_up, d_costs_swap_right, d_seam );
      std::swap(d_costs_left, d_costs_swap_left);
      std::swap(d_costs_up, d_costs_swap_up);
      std::swap(d_costs_right, d_costs_swap_right);
    }

    current_w--;
    num_iterations++;
  }

  HIP_CHECK(hipDeviceSynchronize());
  auto end = std::chrono::steady_clock::now();
  float time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Execution time of seam carver kernels: %f (ms)\n", time * 1e-6f);

  HIP_CHECK(hipMemcpy(h_pixels, d_pixels, img_bytes, hipMemcpyDeviceToHost));
  unsigned char* output = flatten_pixels(h_pixels, w, h, current_w);
  printf("Image resized\n");

  printf("Saving in resized.bmp...\n");
  int success = stbi_write_bmp("resized.bmp", current_w, h, 3, output);
  printf("%s\n", success ? "Success" : "Failed");

  HIP_CHECK(hipFree(d_pixels));
  HIP_CHECK(hipFree(d_pixels_swap));
  if(mode != SEAM_CARVER_APPROX_MODE){
    HIP_CHECK(hipFree(d_costs_left));
    HIP_CHECK(hipFree(d_costs_up));
    HIP_CHECK(hipFree(d_costs_right));
  }
  if(mode == SEAM_CARVER_UPDATE_MODE){
    HIP_CHECK(hipFree(d_costs_swap_left));
    HIP_CHECK(hipFree(d_costs_swap_up));
    HIP_CHECK(hipFree(d_costs_swap_right));
  }
  HIP_CHECK(hipFree(d_M));
  HIP_CHECK(hipFree(d_indices));
  HIP_CHECK(hipFree(d_indices_ref));
  HIP_CHECK(hipFree(d_seam));
  if(mode == SEAM_CARVER_APPROX_MODE){
    HIP_CHECK(hipFree(d_index_map));
    HIP_CHECK(hipFree(d_offset_map));
  }
  free(h_pixels);
  free(output);
  free(indices);
  free(imgv);
  return 0;
}
