#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

typedef unsigned int T;
typedef uint4 VECTYPE;

// kernels
#include "sort_reduce.h"
#include "sort_top_scan.h"
#include "sort_bottom_scan.h"

void verifySort(const T *keys, const size_t size)
{
  bool passed = true;
  for (size_t i = 0; i < size - 1; i++)
  {
    if (keys[i] > keys[i + 1])
    {
      passed = false;
#ifdef VERBOSE_OUTPUT
      std::cout << "Idx: " << i;
      std::cout << " Key: " << keys[i] << "\n";
#endif
      break;
    }
  }
  if (passed)
    std::cout << "PASS" << std::endl;
  else {
    std::cout << "FAIL" << std::endl;
    exit(1);
  }
}

int main(int argc, char** argv) 
{
  if (argc != 3) 
  {
    printf("Usage: %s <problem size> <number of passes>\n.", argv[0]);
    return -1;
  }

  int select = atoi(argv[1]);
  int passes = atoi(argv[2]);

  // Problem Sizes
  int probSizes[4] = { 1, 8, 32, 64 };
  size_t size = probSizes[select];

  // Convert to MiB
  size = (size * 1024 * 1024) / sizeof(T);

  // Create input data on CPU
  unsigned int bytes = size * sizeof(T);

  T* h_idata = (T*) malloc (bytes); 
  T* h_odata = (T*) malloc (bytes); 

  // Initialize host memory
  std::cout << "Initializing host memory." << std::endl;
  for (unsigned int i = 0; i < size; i++)
  {
    h_idata[i] = i % 16; // Fill with some pattern
    h_odata[i] = size - i;
  }

  std::cout << "Running benchmark with input array length " << size << std::endl;

  // Number of local work items per group
  const size_t local_wsize  = 256;
  // Number of global work items
  const size_t global_wsize = 16384; 
  // 64 work groups
  const size_t num_work_groups = global_wsize / local_wsize;

  // The radix width in bits
  const int radix_width = 4; // Changing this requires major kernel updates
  //const int num_digits = (int)pow((double)2, radix_width); // n possible digits
  const int num_digits = 16;

  T* d_idata;
  T* d_odata;
  T* d_isums;

  hipMalloc((void**)&d_idata, size * sizeof(T));
  hipMemcpyAsync(d_idata, h_idata, size * sizeof(T), hipMemcpyHostToDevice, 0);
  hipMalloc((void**)&d_odata, size * sizeof(T));
  hipMalloc((void**)&d_isums, num_work_groups * num_digits * sizeof(T));

  T* d_in;
  T* d_out;

  double time = 0.0;

  for (int k = 0; k < passes; k++)
  {
    hipDeviceSynchronize();
    auto start = std::chrono::steady_clock::now();

    // Assuming an 8 bit byte.
    // shift is uint because Computecpp compiler has no operator>>(unsigned int, int);
    for (unsigned int shift = 0; shift < sizeof(T)*8; shift += radix_width)
    {
      // Like scan, we use a reduce-then-scan approach
      // But before proceeding, update the shift appropriately
      // for each kernel. This is how many bits to shift to the
      // right used in binning.
      // Also, the sort is not in place, so swap the input and output
      // buffers on each pass.
      bool even = ((shift / radix_width) % 2 == 0) ? true : false;
      d_in = even ? d_idata : d_odata;
      d_out = even ? d_odata : d_idata;

      hipLaunchKernelGGL(reduce, num_work_groups, local_wsize, 0, 0, d_in, d_isums, size, shift);
      hipLaunchKernelGGL(top_scan, 1, local_wsize, 0, 0, d_isums, num_work_groups);
      hipLaunchKernelGGL(bottom_scan, num_work_groups, local_wsize, 0, 0, d_out, d_in, d_isums, size, shift);
    }

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    time += std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  }  // passes

  printf("Average elapsed time per pass %lf (s)\n", time * 1e-9 / passes);

  hipMemcpy(h_odata, d_out, size * sizeof(T), hipMemcpyDeviceToHost);
  hipFree(d_idata);
  hipFree(d_odata);
  hipFree(d_isums);

  verifySort(h_odata, size);

  free(h_idata);
  free(h_odata);
  return 0;
}
