#include "hip/hip_runtime.h"
/***************************************************************************
Copyright 2020 Intel Corporation

Permission is hereby granted, free of charge, to any person obtaining a copy of
this software and associated documentation files (the "Software"), to deal in
the Software without restriction, including without limitation the rights to
use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
the Software, and to permit persons to whom the Software is furnished to do so,
subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

 
  Description:
    This code sample will implement a simple example of a Monte Carlo
    simulation of the diffusion of water molecules in tissue.
 
**************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <chrono>
#include <cmath>
#include <ctime>
#include <iomanip>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include "reference.h"

// Helper functions

// This function displays correct usage and parameters
void usage(std::string programName) {
  std::cout << " Incorrect number of parameters " << std::endl;
  std::cout << " Usage: ";
  std::cout << programName << " <Number of iterations within the kernel> ";
  std::cout << "<Kernel execution count>\n\n";
}

// This function prints a 2D matrix
template <typename T>
void print_matrix(T** matrix, size_t size_X, size_t size_Y) {
  std::cout << std::endl;
  for (size_t i = 0; i < size_X; ++i) {
    for (size_t j = 0; j < size_Y; ++j) {
      std::cout << std::setw(3) << matrix[i][j] << " ";
    }
    std::cout << std::endl;
  }
}

// This function prints a vector
template <typename T>
void print_vector(T* vector, size_t n) {
  std::cout << std::endl;
  for (size_t i = 0; i < n; ++i) {
    std::cout << vector[i] << " ";
  }
  std::cout << std::endl;
}

__global__
void Simulation(float*__restrict__ a_particleX,
                float*__restrict__ a_particleY,
		const float*__restrict__ a_randomX,
                const float*__restrict__ a_randomY, 
		size_t *__restrict__ a_map,
                const size_t n_particles,
                unsigned int nIterations,
                int grid_size,
                float radius)
{
  size_t ii = blockDim.x * blockIdx.x + threadIdx.x;
  if (ii >= n_particles) return;
  // Start iterations
  // Each iteration:
  //  1. Updates the position of all water molecules
  //  2. Checks if water molecule is inside a cell or not.
  //  3. Updates counter in cells array
  size_t iter = 0;
  float pX = a_particleX[ii];
  float pY = a_particleY[ii];
  size_t map_base = ii * grid_size * grid_size;
  while (iter < nIterations) {
    // Computes random displacement for each molecule
    // This example shows random distances between
    // -0.05 units and 0.05 units in both X and Y directions
    // Moves each water molecule by a random vector

    float randnumX = a_randomX[iter * n_particles + ii];
    float randnumY = a_randomY[iter * n_particles + ii];

    // Transform the scaled random numbers into small displacements
    float displacementX = randnumX / 1000.0f - 0.0495f;
    float displacementY = randnumY / 1000.0f - 0.0495f;

    // Move particles using random displacements
    pX += displacementX;
    pY += displacementY;

    // Compute distances from particle position to grid point
    float dX = pX - truncf(pX);
    float dY = pY - truncf(pY);

    // Compute grid point indices
    int iX = floorf(pX);
    int iY = floorf(pY);

    // Check if particle is still in computation grid
    if ((pX < grid_size) && (pY < grid_size) && (pX >= 0) && (pY >= 0)) {
      // Check if particle is (or remained) inside cell.
      // Increment cell counter in map array if so
      if ((dX * dX + dY * dY <= radius * radius))
        // The map array is organized as (particle, y, x)
        a_map[map_base + iY * grid_size + iX]++;
    }

    iter++;

  }  // Next iteration

  a_particleX[ii] = pX;
  a_particleY[ii] = pY;
}

// This function distributes simulation work across workers
void motion_device(float* particleX, float* particleY,
                   float* randomX, float* randomY, int** grid, size_t grid_size,
                   size_t n_particles, int nIterations, float radius,
                   size_t* map, int nRepeat) {

  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, 0);

  std::cout << " Running on " << devProp.name << std::endl;
  std::cout << " The device max work-group size is " << devProp.maxThreadsPerBlock << std::endl;
  std::cout << " The number of iterations is " << nIterations << std::endl;
  std::cout << " The number of kernel execution is " << nRepeat << std::endl;
  std::cout << " The number of particles is " << n_particles << std::endl;
  
  float *d_randomX;
  float *d_randomY;
  float *d_particleX;
  float *d_particleY;
  size_t *d_map;
  hipMalloc((void**)&d_randomX, sizeof(float) * n_particles * nIterations);
  hipMalloc((void**)&d_randomY, sizeof(float) * n_particles * nIterations);
  hipMalloc((void**)&d_particleX, sizeof(float) * n_particles);
  hipMalloc((void**)&d_particleY, sizeof(float) * n_particles);
  size_t MAP_SIZE = n_particles * grid_size * grid_size;
  hipMalloc((void**)&d_map, sizeof(size_t) * MAP_SIZE);

  hipMemcpy(d_randomX, randomX, sizeof(float) * n_particles * nIterations, hipMemcpyHostToDevice);
  hipMemcpy(d_randomY, randomY, sizeof(float) * n_particles * nIterations, hipMemcpyHostToDevice);

  double time_total = 0.0;

  for (int i = 0; i < nRepeat; i++) {
    // reset the device data
    hipMemcpy(d_particleX, particleX, sizeof(float) * n_particles, hipMemcpyHostToDevice);
    hipMemcpy(d_particleY, particleY, sizeof(float) * n_particles, hipMemcpyHostToDevice);
    hipMemcpy(d_map, map, sizeof(size_t) * MAP_SIZE, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    auto start = std::chrono::steady_clock::now();

    Simulation<<< dim3((n_particles + 255) / 256), dim3(256) >>> (
      d_particleX, 
      d_particleY, 
      d_randomX, 
      d_randomY, 
      d_map, 
      n_particles,
      nIterations,
      grid_size,
      radius);

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    time_total += time;
  }

  std::cout << std::endl;
  std::cout << "Average kernel execution time: " << (time_total * 1e-9) / nRepeat << " (s)";
  std::cout << std::endl;

  hipMemcpy(map, d_map, sizeof(size_t) * MAP_SIZE, hipMemcpyDeviceToHost);

  // For every cell in the grid, add all the counters from different
  // particles (workers) which are stored in the 3rd dimension of the 'map'
  // array
  for (size_t i = 0; i < n_particles; ++i) {
    for (size_t y = 0; y < grid_size; y++) {
      for (size_t x = 0; x < grid_size; x++) {
        if (map[i * grid_size * grid_size + y * grid_size + x] > 0) {
          grid[y][x] += map[i * grid_size * grid_size + y * grid_size + x];
        }
      }
    }
  }  // End loop for number of particles

  hipFree(d_randomX);
  hipFree(d_randomY);
  hipFree(d_particleX);
  hipFree(d_particleY);
  hipFree(d_map);
}  // End of function motion_device()


int main(int argc, char* argv[]) {
  if (argc != 3) {
    usage(argv[0]);
    return 1;
  }

  // Read command-line arguments
  int nIterations = std::stoi(argv[1]);
  int nRepeat = std::stoi(argv[2]);

  // Cell and Particle parameters
  const size_t grid_size = 21;    // Size of square grid
  const size_t n_particles = 147456;  // Number of particles
  const float radius = 0.5;       // Cell radius = 0.5*(grid spacing)

  // Stores a grid of cells
  int** grid = new int*[grid_size];
  for (size_t i = 0; i < grid_size; i++) grid[i] = new int[grid_size];

  // Stores all random numbers to be used in the simulation
  float* randomX = new float[n_particles * nIterations];
  float* randomY = new float[n_particles * nIterations];

  // Stores X and Y position of particles in the cell grid
  float* particleX = new float[n_particles];
  float* particleY = new float[n_particles];

  // 'map' array replicates grid to be used by each particle
  size_t MAP_SIZE = n_particles * grid_size * grid_size;
  size_t* map = new size_t[MAP_SIZE];
  size_t* map_ref = new size_t[MAP_SIZE];

  // Initialize arrays
  for (size_t i = 0; i < n_particles; i++) {
    // Initial position of particles in cell grid
    particleX[i] = 10.0;
    particleY[i] = 10.0;

    for (size_t y = 0; y < grid_size; y++) {
      for (size_t x = 0; x < grid_size; x++) {
        map[i * grid_size * grid_size + y * grid_size + x] = 0;
        map_ref[i * grid_size * grid_size + y * grid_size + x] = 0;
      }
    }
  }

  for (size_t y = 0; y < grid_size; y++) {
    for (size_t x = 0; x < grid_size; x++) {
      grid[y][x] = 0;
    }
  }

  // Compute vectors of random values for X and Y directions
  // Set the seed for rand() function.
  // Use a fixed seed for reproducibility/debugging
  srand(17);
  const size_t scale = 100; // Scale of random numbers
  for (size_t i = 0; i < n_particles * nIterations; i++) {
    randomX[i] = rand() % scale;
    randomY[i] = rand() % scale;
  }

  // Start timers
  auto start = std::chrono::steady_clock::now();

  // Call simulation function
  motion_device(particleX, particleY, randomX, randomY, grid, grid_size,
                n_particles, nIterations, radius, map, nRepeat);

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << std::endl;
  std::cout << "Simulation time: " << time * 1e-9 << " (s) ";
  std::cout << std::endl;

  motion_host(particleX, particleY, randomX, randomY, grid, grid_size,
              n_particles, nIterations, radius, map_ref, nRepeat);

  size_t count = 0;
  for (size_t i = 0; i < MAP_SIZE; i++) {
    if (map[i] != map_ref[i]) count++; 
  }
  std::cout << (count <= 2 ? "PASS" : "FAIL") << std::endl;

#ifdef DEBUG
  // Displays final grid only if grid small.
  if (grid_size <= 64) {
    std::cout << "\n ********************** OUTPUT GRID: " << std::endl;
    print_matrix<int>(grid, grid_size, grid_size);
  }
#endif

  // Cleanup
  for (size_t i = 0; i < grid_size; i++) delete grid[i];

  delete[] grid;
  delete[] particleX;
  delete[] particleY;
  delete[] randomX;
  delete[] randomY;
  delete[] map;
  delete[] map_ref;

  return 0;
}
