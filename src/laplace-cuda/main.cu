#include "hip/hip_runtime.h"
/** GPU Laplace solver using optimized red-black Gauss Seidel with SOR solver
 *
 * author Kyle E. Niemeyer
 * date 09/21/2012
 *
 * Solves Laplace's equation in 2D (e.g., heat conduction in a rectangular plate)
 * on GPU using CUDA with the red-black Gauss Seidel with sucessive overrelaxation
 * (SOR) that has been "optimized". This means that the red and black kernels
 * only loop over their respective cells, instead of over all cells and skipping
 * even/odd cells. This requires separate arrays for red and black cells.
 *
 * Boundary conditions:
 * T = 0 at x = 0, x = L, y = 0
 * T = TN at y = H
 */

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include "timer.h"
#include "kernels.h"
#include "reference.h"


/** Function to evaluate coefficient matrix and right-hand side vector.
 *
 * \param[in]   rowmax   number of rows
 * \param[in]   colmax   number of columns
 * \param[in]   th_cond  thermal conductivity
 * \param[in]   dx       grid size in x dimension (uniform)
 * \param[in]   dy       grid size in y dimension (uniform)
 * \param[in]   width    width of plate (z dimension)
 * \param[in]   TN       temperature at top boundary
 * \param[out]  aP       array of self coefficients
 * \param[out]  aW       array of west neighbor coefficients
 * \param[out]  aE       array of east neighbor coefficients
 * \param[out]  aS       array of south neighbor coefficients
 * \param[out]  aN       array of north neighbor coefficients
 * \param[out]  b        right-hand side array
 */
void fill_coeffs (int rowmax, int colmax, Real th_cond, Real dx, Real dy,
    Real width, Real TN, Real * aP, Real * aW, Real * aE,
    Real * aS, Real * aN, Real * b)
{
  int col, row;
  for (col = 0; col < colmax; ++col) {
    for (row = 0; row < rowmax; ++row) {
      int ind = col * rowmax + row;

      b[ind] = ZERO;
      Real SP = ZERO;

      if (col == 0) {
        // left BC: temp = 0
        aW[ind] = ZERO;
        SP = -TWO * th_cond * width * dy / dx;
      } else {
        aW[ind] = th_cond * width * dy / dx;
      }

      if (col == colmax - 1) {
        // right BC: temp = 0
        aE[ind] = ZERO;
        SP = -TWO * th_cond * width * dy / dx;
      } else {
        aE[ind] = th_cond * width * dy / dx;
      }

      if (row == 0) {
        // bottom BC: temp = 0
        aS[ind] = ZERO;
        SP = -TWO * th_cond * width * dx / dy;
      } else {
        aS[ind] = th_cond * width * dx / dy;
      }

      if (row == rowmax - 1) {
        // top BC: temp = TN
        aN[ind] = ZERO;
        b[ind] = TWO * th_cond * width * dx * TN / dy;
        SP = -TWO * th_cond * width * dx / dy;
      } else {
        aN[ind] = th_cond * width * dx / dy;
      }

      aP[ind] = aW[ind] + aE[ind] + aS[ind] + aN[ind] - SP;
    } // end for row
  } // end for col
} // end fill_coeffs


/** Main function that solves Laplace's equation in 2D (heat conduction in plate)
 *
 * Contains iteration loop for red-black Gauss-Seidel with SOR GPU kernels
 */
int main (void) {

  // size of plate
  Real L = 1.0;
  Real H = 1.0;
  Real width = 0.01;

  // thermal conductivity
  Real th_cond = 1.0;

  // temperature at top boundary
  Real TN = 1.0;

  // SOR iteration tolerance
  Real tol = 1.e-6;

  // number of cells in x and y directions
  // including unused boundary cells
  int num_rows = (NUM / 2) + 2;
  int num_cols = NUM + 2;
  int size_temp = num_rows * num_cols;
  int size = NUM * NUM;

  // size of cells
  Real dx = L / NUM;
  Real dy = H / NUM;

  // iterations for Red-Black Gauss-Seidel with SOR
  int iter;
  int it_max = 1e6;

  // allocate memory
  Real *aP, *aW, *aE, *aS, *aN, *b;
  Real *temp_red, *temp_black;
  Real *temp_red_ref, *temp_black_ref;

  // arrays of coefficients
  aP = (Real *) calloc (size, sizeof(Real));
  aW = (Real *) calloc (size, sizeof(Real));
  aE = (Real *) calloc (size, sizeof(Real));
  aS = (Real *) calloc (size, sizeof(Real));
  aN = (Real *) calloc (size, sizeof(Real));

  // RHS
  b = (Real *) calloc (size, sizeof(Real));

  // temperature arrays
  temp_red = (Real *) calloc (size_temp, sizeof(Real));
  temp_black = (Real *) calloc (size_temp, sizeof(Real));
  temp_red_ref = (Real *) calloc (size_temp, sizeof(Real));
  temp_black_ref = (Real *) calloc (size_temp, sizeof(Real));

  // set coefficients
  fill_coeffs (NUM, NUM, th_cond, dx, dy, width, TN, aP, aW, aE, aS, aN, b);

  int i;
  for (i = 0; i < size_temp; ++i) {
    temp_red[i] = ZERO;
    temp_black[i] = ZERO;
  }

  // block and grid dimensions
  dim3 dimBlock (BLOCK_SIZE, 2);
  dim3 dimGrid (NUM / (2 * BLOCK_SIZE), NUM/2);

  // print problem info
  printf("Problem size: %d x %d \n", NUM, NUM);

  // allocate device memory
  Real *aP_d, *aW_d, *aE_d, *aS_d, *aN_d, *b_d;
  Real *temp_red_d;
  Real *temp_black_d;
  Real *bl_norm_L2_d;
  Real *norm_L2_d;

  // residual
  // one for each temperature value
  int size_norm = size_temp;
  hipMalloc ((void**) &bl_norm_L2_d, size_norm * sizeof(Real));
  hipMemset (bl_norm_L2_d, 0, size_norm * sizeof(Real));

  // total residual
  hipMalloc ((void**) &norm_L2_d, sizeof(Real));

  hipMalloc ((void**) &aP_d, size * sizeof(Real));
  hipMalloc ((void**) &aW_d, size * sizeof(Real));
  hipMalloc ((void**) &aE_d, size * sizeof(Real));
  hipMalloc ((void**) &aS_d, size * sizeof(Real));
  hipMalloc ((void**) &aN_d, size * sizeof(Real));
  hipMalloc ((void**) &b_d, size * sizeof(Real));
  hipMalloc ((void**) &temp_red_d, size_temp * sizeof(Real));
  hipMalloc ((void**) &temp_black_d, size_temp * sizeof(Real));

  // copy to device memory
  hipMemcpy (aP_d, aP, size * sizeof(Real), hipMemcpyHostToDevice);
  hipMemcpy (aW_d, aW, size * sizeof(Real), hipMemcpyHostToDevice);
  hipMemcpy (aE_d, aE, size * sizeof(Real), hipMemcpyHostToDevice);
  hipMemcpy (aS_d, aS, size * sizeof(Real), hipMemcpyHostToDevice);
  hipMemcpy (aN_d, aN, size * sizeof(Real), hipMemcpyHostToDevice);
  hipMemcpy (b_d, b, size * sizeof(Real), hipMemcpyHostToDevice);
  hipMemcpy (temp_red_d, temp_red, size_temp * sizeof(Real), hipMemcpyHostToDevice);
  hipMemcpy (temp_black_d, temp_black, size_temp * sizeof(Real), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  StartTimer();

  // Determine temporary device storage requirements 
  void     *d_temp_storage = nullptr;
  size_t   temp_storage_bytes = 0;
  hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                         bl_norm_L2_d, norm_L2_d, size_norm);

  // Allocate temporary storage
  if (temp_storage_bytes != 0)
    hipMalloc(&d_temp_storage, temp_storage_bytes);

  // iteration loop
  for (iter = 1; iter <= it_max; ++iter) {

    Real norm_L2 = ZERO;

    red_kernel <<<dimGrid, dimBlock>>> (aP_d, aW_d, aE_d, aS_d, aN_d, b_d, temp_black_d, temp_red_d, bl_norm_L2_d);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, bl_norm_L2_d, norm_L2_d, size_norm);
    hipMemcpy(&norm_L2, norm_L2_d, sizeof(Real), hipMemcpyDeviceToHost);

    black_kernel <<<dimGrid, dimBlock>>> (aP_d, aW_d, aE_d, aS_d, aN_d, b_d, temp_red_d, temp_black_d, bl_norm_L2_d);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, bl_norm_L2_d, norm_L2_d, size_norm);
    Real temp = norm_L2;
    hipMemcpy(&norm_L2, norm_L2_d, sizeof(Real), hipMemcpyDeviceToHost);
    norm_L2 += temp;

    // calculate residual
    norm_L2 = sqrt(norm_L2 / ((Real)size));

    if (iter % 1000 == 0) printf("%5d, %0.6f\n", iter, norm_L2);

    // if tolerance has been reached, end SOR iterations
    if (norm_L2 < tol) break;
  }

  if (d_temp_storage != nullptr) hipFree(d_temp_storage);

  double runtime = GetTimer();
  printf("Total time for %i iterations: %f s\n", iter, runtime / 1000.0);

  // transfer final temperature values back
  hipMemcpy (temp_red, temp_red_d, size_temp * sizeof(Real), hipMemcpyDeviceToHost);
  hipMemcpy (temp_black, temp_black_d, size_temp * sizeof(Real), hipMemcpyDeviceToHost);

  // Reference
  int count = 0;

  for (iter = 1; iter <= it_max; ++iter) {
    Real norm_L2;
    norm_L2 = red_ref(aP, aW, aE, aS, aN, b, temp_black_ref, temp_red_ref);
    norm_L2 += black_ref (aP, aW, aE, aS, aN, b, temp_red_ref, temp_black_ref);
    norm_L2 = sqrt(norm_L2 / ((Real)size));
    if (norm_L2 < tol) break;
  }

  // print temperature data to file
  FILE * pfile;
  pfile = fopen("temperature.dat", "w");

  if (pfile != NULL) {
    fprintf(pfile, "#x\ty\ttemp(K)\n");

    int row, col;
    for (row = 1; row < NUM + 1; ++row) {
      for (col = 1; col < NUM + 1; ++col) {
        Real x_pos = (col - 1) * dx + (dx / 2);
        Real y_pos = (row - 1) * dy + (dy / 2);

        if ((row + col) % 2 == 0) {
          // even, so red cell
          int ind = col * num_rows + (row + (col % 2)) / 2;
          if ((temp_red[ind] - temp_red_ref[ind]) >= 1e-3f) count++;
          fprintf(pfile, "%f\t%f\t%f\n", x_pos, y_pos, temp_red[ind]);
        } else {
          // odd, so black cell
          int ind = col * num_rows + (row + ((col + 1) % 2)) / 2;
          if ((temp_black[ind] - temp_black_ref[ind]) >= 1e-3f) count++;
          fprintf(pfile, "%f\t%f\t%f\n", x_pos, y_pos, temp_black[ind]);
        }
      }
      fprintf(pfile, "\n");
    }
  }

  fclose(pfile);
  printf("%s\n", count == 0 ? "PASS" : "FAIL");

  hipFree(aP_d);
  hipFree(aW_d);
  hipFree(aE_d);
  hipFree(aS_d);
  hipFree(aN_d);
  hipFree(b_d);
  hipFree(temp_red_d);
  hipFree(temp_black_d);
  hipFree(bl_norm_L2_d);
  hipFree(norm_L2_d);

  free(aP);
  free(aW);
  free(aE);
  free(aS);
  free(aN);
  free(b);
  free(temp_red);
  free(temp_black);
  free(temp_red_ref);
  free(temp_black_ref);

  return 0;
}
