#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <stdint.h>
#include <chrono>
#include <utility>  // std::swap
#include <hip/hip_runtime.h>
#include "utils.h"
#include "kernels.h"
#include "kernels_wrapper.h"

//#define STBI_ONLY_BMP
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

int main(int argc, char **argv) {
  if(argc < 3){
    printf("Usage: %s <file> <number of seams to remove> [options]\n"
           "valid options:\n-u\tupdate costs instead of recomputing them\n"
           "-a\tapproximate computation\n", argv[0]);
    return 1;
  }

  char *check;
  long seams_to_remove = strtol(argv[2], &check, 10);  //10 specifies base-10
  if (check == argv[2]){   //if no characters were converted pointers are equal
    printf("ERROR: can't convert string to number, exiting.\n");
    return 1;
  }

  int w, h, ncomp;
  unsigned char* imgv = stbi_load(argv[1], &w, &h, &ncomp, 0);
  if(imgv == NULL){
    printf("ERROR: can't load image \"%s\", exiting.\n", argv[1]);
    printf("Reason: %s\n", stbi_failure_reason());
    return 1;
  }

  if(ncomp != 3){
    printf("ERROR: image does not have 3 components (RGB), exiting.\n");
    return 1;
  }

  if(seams_to_remove < 0 || seams_to_remove >= w){
    printf("ERROR: number of seams to remove is invalid, exiting.\n");
    return 1;
  }

  seam_carver_mode mode = SEAM_CARVER_STANDARD_MODE;

  if(argc >= 4){
    if(strcmp(argv[3],"-u") == 0){
      mode = SEAM_CARVER_UPDATE_MODE;
      printf("update mode selected.\n");
    }
    else if(strcmp(argv[3],"-a") == 0){
      mode = SEAM_CARVER_APPROX_MODE;
      printf("approximation mode selected.\n");
    }
    else{    
      printf("an invalid option was specified and will be ignored. Valid options are: -u, -a.\n");
    }
  }

  printf("Image loaded. Resizing...\n");

  int current_w = w;
  uchar4 *h_pixels = build_pixels(imgv, w, h);
  const int img_bytes = w * h * sizeof(uchar4);
  const int cost_bytes = w * h * sizeof(short);
  const int h_bytes = h * sizeof(int);
  const int w_bytes = w * sizeof(int);

  uchar4 *d_pixels;
  uchar4 *d_pixels_swap;
  short *d_costs_left, *d_costs_swap_left;
  short *d_costs_up, *d_costs_swap_up;
  short *d_costs_right, *d_costs_swap_right;
  int *d_index_map;
  int *d_offset_map;
  int *d_indices_ref;
  int *d_indices;
  int *d_seam;
  int *reduce_row; //M row to consider for reduce
  int *d_M;

  if(mode != SEAM_CARVER_APPROX_MODE) {
    hipMalloc((void**)&d_costs_left, cost_bytes); 
    hipMalloc((void**)&d_costs_up, cost_bytes); 
    hipMalloc((void**)&d_costs_right, cost_bytes); 
  }
  if(mode == SEAM_CARVER_UPDATE_MODE){
    hipMalloc((void**)&d_costs_swap_left, cost_bytes); 
    hipMalloc((void**)&d_costs_swap_up, cost_bytes); 
    hipMalloc((void**)&d_costs_swap_right, cost_bytes);
  }
  //sum map in approx mode
  hipMalloc((void**)&d_M, img_bytes); 

  // rows to consider for reduce
  if(mode == SEAM_CARVER_APPROX_MODE)
    reduce_row = d_M; //first row
  else
    reduce_row = d_M + w*(h-1); //last row

  if(mode == SEAM_CARVER_APPROX_MODE){
    hipMalloc((void**)&d_index_map, img_bytes);
    hipMalloc((void**)&d_offset_map, img_bytes);
  }

  int* indices = (int*)malloc(w_bytes);
  for(int i = 0; i < w; i++) indices[i] = i;

  hipMalloc((void**)&d_indices, w_bytes); 

  hipMalloc((void**)&d_indices_ref, w_bytes); 
  hipMemcpy(d_indices_ref, indices, w_bytes, hipMemcpyHostToDevice);   
  hipMalloc((void**)&d_seam, h_bytes);    

  hipMalloc((void**)&d_pixels, img_bytes);
  hipMemcpy(d_pixels, h_pixels, img_bytes, hipMemcpyHostToDevice);       
  hipMalloc((void**)&d_pixels_swap, img_bytes);

  if(mode == SEAM_CARVER_UPDATE_MODE)
    compute_costs(current_w, w, h, d_pixels, d_costs_left, d_costs_up, d_costs_right);

  int num_iterations = 0;

  auto start = std::chrono::steady_clock::now();

  while(num_iterations < (int)seams_to_remove){

    if(mode == SEAM_CARVER_STANDARD_MODE)
      compute_costs(current_w, w, h, d_pixels, d_costs_left, d_costs_up, d_costs_right);

    if(mode != SEAM_CARVER_APPROX_MODE){
      compute_M(current_w, w, h, d_M, d_costs_left, d_costs_up, d_costs_right);
      find_min_index(current_w, d_indices_ref, d_indices, reduce_row);
      find_seam(current_w, w, h, d_M, d_indices, d_seam);
    }
    else{
      approx_setup(current_w, w, h, d_pixels, d_index_map, d_offset_map, d_M);
      approx_M(current_w, w, h, d_offset_map,  d_M);
      find_min_index(current_w, d_indices_ref, d_indices, reduce_row);
      approx_seam(w, h, d_index_map, d_indices, d_seam);
    }

    remove_seam(current_w, w, h, d_M, d_pixels, d_pixels_swap, d_seam);
    std::swap(d_pixels, d_pixels_swap);

    if(mode == SEAM_CARVER_UPDATE_MODE){
      update_costs(current_w, w, h, d_M, d_pixels, 
                   d_costs_left, d_costs_up, d_costs_right,
                   d_costs_swap_left, d_costs_swap_up, d_costs_swap_right, d_seam );
      std::swap(d_costs_left, d_costs_swap_left);
      std::swap(d_costs_up, d_costs_swap_up);
      std::swap(d_costs_right, d_costs_swap_right);
    }

    current_w--;
    num_iterations++;
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  float time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Execution time of seam carver kernels: %f (ms)\n", time * 1e-6f);

  hipMemcpy(h_pixels, d_pixels, img_bytes, hipMemcpyDeviceToHost);
  unsigned char* output = flatten_pixels(h_pixels, w, h, current_w); 
  printf("Image resized\n");

  printf("Saving in resized.bmp...\n");
  int success = stbi_write_bmp("resized.bmp", current_w, h, 3, output);
  printf("%s\n", success ? "Success" : "Failed");

  hipFree(d_pixels);
  hipFree(d_pixels_swap);
  if(mode != SEAM_CARVER_APPROX_MODE){
    hipFree(d_costs_left);
    hipFree(d_costs_up);
    hipFree(d_costs_right);
  }
  if(mode == SEAM_CARVER_UPDATE_MODE){
    hipFree(d_costs_swap_left);
    hipFree(d_costs_swap_up);
    hipFree(d_costs_swap_right);
  }
  hipFree(d_M); 
  hipFree(d_indices); 
  hipFree(d_indices_ref); 
  hipFree(d_seam);
  if(mode == SEAM_CARVER_APPROX_MODE){
    hipFree(d_index_map);
    hipFree(d_offset_map);
  }
  free(h_pixels);
  free(output);   
  free(indices);
  return 0;
}
