/**
 * This file is the modified read-only mixbench GPU micro-benchmark suite.
 *
 **/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>

#define VECTOR_SIZE (8*1024*1024)
#define granularity (8)
#define fusion_degree (4)
#define seed 0.1f

__global__ void benchmark_func(float *g_data, const int blockdim,
                               const int compute_iterations)
{
  const unsigned int blockSize = blockdim;
  const int stride = blockSize;
  int idx = blockIdx.x*blockSize*granularity + threadIdx.x;
  const int big_stride = gridDim.x*blockSize*granularity;

  float tmps[granularity];
  for(int k=0; k<fusion_degree; k++) {
    #pragma unroll
    for(int j=0; j<granularity; j++) {
      // Load elements (memory intensive part)
      tmps[j] = g_data[idx+j*stride+k*big_stride];

      // Perform computations (compute intensive part)
      for(int i=0; i<compute_iterations; i++)
        tmps[j] = tmps[j]*tmps[j]+seed;
    }

    __syncthreads();

    // Multiply add reduction
    float sum = 0.f;
    #pragma unroll
    for(int j=0; j<granularity; j+=2)
      sum += tmps[j]*tmps[j+1];

    #pragma unroll
    for(int j=0; j<granularity; j++)
      g_data[idx+k*big_stride] = sum;

    __syncthreads();
  }
}

void mixbenchGPU(long size, int repeat) {
  const char *benchtype = "compute with global memory (block strided)";
  printf("Trade-off type:%s\n", benchtype);
  float *cd = (float*) malloc (size*sizeof(float));
  for (int i = 0; i < size; i++) cd[i] = 0;

  const long reduced_grid_size = size/granularity/128;
  const int block_dim = 256;
  const int grid_dim = reduced_grid_size/block_dim;

  float *d_cd;
  hipMalloc((void**)&d_cd, size*sizeof(float));
  hipMemcpy(d_cd, cd,  size*sizeof(float), hipMemcpyHostToDevice);

  // warmup
  for (int i = 0; i < repeat; i++) {
    benchmark_func<<<grid_dim, block_dim>>>(d_cd, block_dim, i);
  }

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    benchmark_func<<<grid_dim, block_dim>>>(d_cd, block_dim, i);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Total kernel execution time: %f (s)\n", time * 1e-9f);

  hipMemcpy(cd, d_cd, size*sizeof(float), hipMemcpyDeviceToHost);

  // verification
  bool ok = true;
  for (int i = 0; i < size; i++) {
    if (cd[i] != 0) {
      if (fabsf(cd[i] - 0.050807f) > 1e-6f) {
        ok = false;
        printf("Verification failed at index %d: %f\n", i, cd[i]);
        break;
      }
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");

  free(cd);
  hipFree(d_cd);
}

int main(int argc, char* argv[]) {
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  unsigned int datasize = VECTOR_SIZE*sizeof(float);

  printf("Buffer size: %dMB\n", datasize/(1024*1024));

  mixbenchGPU(VECTOR_SIZE, repeat);

  return 0;
}
