// NOTICE
// This file was stored by David Sanchez in 2023, based on a recovered hard-drive.  The code almost certainly dates to 2010 (earlier/later).
// I think I wrote this based on Matlab code by Grady Wright and Greg Barnett, under the supervision of Dave Yuen (and advice + comments from many).
// That said, I do not recall exactly when or under what conditions this was written and many parts may have been given to me or taken from other academic projects.
// In whole, in part, or in derivative this code forms the basis of some papers, but I've lost track of which ones.  I don't even know whether this
// is the most up-to-date such code.
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <chrono>
#include <hipblas.h>

// Parameters.  These may change from run-to-run
#define M         (500)         // vertical dimension of the temperature array
#define RA        (100000000.f) // Rayleigh number
#define XF        (2)           // Aspect ratio for the temperature array
#define STARTSTEP (0)           // First step

// Constants.  Stability cannot be assured if these values are altered.
//#define N (XF*(M-1)+1) // horizontal dimension of temperature array
#define N          (1000)
#define DX         (1.f/(M - 1.f)) // x and z-dimensional mesh spacing
#define DX2        (DX*DX) // DX^2, for saving a few computations.
#define OMEGACOEFF (-((DX*DX)*(DX*DX))*RA) // Used on every timestep

#define PI       3.1415926535897932384626433832795028841968 // Precision here is arbitrary and may be traded
                  // for performance if context allows
#define DT_START 0.000000000000005 //timestep width.  Needs to be small to
           //guide the model through high Ra runs.


#define FRAMESIZE (DX2/4.f) // How many iterations between saves
          // flatten a 2D grid with 1D blocks into a vector.  The functionality
          // could be extended to perform pointer arithmetic, but that's not necessary
          // in this code.
          //
          //  Invoked grid geometry
          //  (2D grid with 1D blocks)
          //
          //  |---|---|---|
          //  |1,1|1,2|1,3|
          //  |---|---|---|
          //  |2,1|2,2|2,3|  ==>  1,1 1,2 1,3 2,1 2,2 ...
          //  |---|---|---|
          //  |3,1|3,2|3,3|
          //  |---|---|---|


#define THREADID (((gridDim.x)*(blockIdx.y) + (blockIdx.x))*(blockDim.x) + (threadIdx.x))

// It is possible to alternate between 2D row-major and column-major formats by
// taking transposes.
#define TPOSE(i,j,ld) ((j)*(ld) + (i))

// Simplify calling G, since many of the arguments are assured
#define SHORTG(input, compute, save, frames) \
  G(h, input, d_Tbuff, d_DxT, d_y, d_u, d_v, d_psi, d_omega, d_dsc, \
    d_dsr, d_ei, d_dt, save, h_T, compute, frames, tstep)

//=============================================================================
//                  KERNELS
//=============================================================================

//=============================================================================
//                 ElementMultOmega
//=============================================================================
// Performs elementwise matrix multiplication on matrices shaped like omega,
// returning the result in A.
__global__ void ElemMultOmega(float* A, const float* B) {
  if(THREADID < (M-2)*(N-2) ) {
    A[THREADID] = A[THREADID]*B[THREADID];
  }
}

//=============================================================================
//                 ElementMultT
//=============================================================================
// Performs elementwise matrix multiplication on matrices shaped like T,
// putting the result in A.
__global__ void ElemMultT(float* A, const float* B) {
  if(THREADID < (M-2)*(N)) {
    A[THREADID] = A[THREADID]*B[THREADID];
  }
}

//=============================================================================
//                 ElementMultNu
//=============================================================================
// Performs elementwise matrix multiplication on matrices shaped like d_nutop,
// putting the result in A.
__global__ void ElemMultNu(float* A, const float* B) {
  if(THREADID < N) {
    A[THREADID] = A[THREADID]*B[THREADID];
  }
}

//=============================================================================
//                    SubOne
//=============================================================================
// Subtracts 1.f from every element in a vector (floats) of length N
__global__ void SubOne(float* A) {
  if(THREADID < N) {
    A[THREADID] = A[THREADID] - 1.f;
  }
}

//=============================================================================
//                    AddOne
//=============================================================================
// Adds 1.f from every element in a vector (floats) of length N
__global__ void AddOne(float* A) {
  if(THREADID < N) {
    A[THREADID] = A[THREADID] + 1.f;
  }
}

//=============================================================================
//                    AddX
//=============================================================================
// Adds x from every element in a vector (floats) of length (M-2)*N
__global__ void AddX(float* A, const float x) {
  if(THREADID < (M-2)*N) {
    A[THREADID] = A[THREADID] + x;
  }
}

//=============================================================================
//                   Updatedt
//=============================================================================
// Adaptive update rule for dt.  d_dt (a device-side one-element array) should
// be passed as dt, whereas ptru and ptrv point (1-indexed) to vectors u and v.
// The current value of dt[0] will be overwritten. Can be called with a 1D
// grid containing a single 1D block with one thread.
__global__ void Updatedt(int ptru, const float*__restrict__ u, int ptrv,
                         const float*__restrict__ v, float* dt) {
  dt[0] = max(abs(u[ptru - 1]),abs(v[ptrv - 1]));
  dt[0] = min(DX/dt[0],DX2/4.f);
}


//=============================================================================
//                  NusseltCompute
//=============================================================================
// Returns the Nusselt number of the array T, which is pointed to in GPU space
float NusseltCompute(hipblasHandle_t &h,
                     float* T, float* nutop, float* ztop,
                     float* zbot,float* nubot, float* trnu)
{
  float topsum, botsum;
  // Calculate the Nusselt number along the top of the array.
  // d_nutop is the last three rows of T, in inverse order,
  // with all 0s along the bottom.

  // Copy the last three rows of T into the first three rows of nutop
  hipblasScopy(h, N, (T + (M-5)*N), 1, (nutop), 1);
  hipblasScopy(h, N, (T + (M-4)*N), 1, (nutop + (N)), 1);
  hipblasScopy(h, N, (T + (M-3)*N), 1, (nutop + (2*N)), 1);

  float alpha = 0.f;
  float result;

  // Set the last row of nutop = 0.
  hipblasSscal(h, N, &alpha, (nutop + (3*N)), 1);

  // nutop += -( 1 - ztop)
  // => nutop += ztop; nutop -= 1
  alpha = 1.f;
  hipblasSaxpy(h, 4*N, &alpha, ztop, 1, nutop, 1);
  // Subtract 1 from every element in the array.  SubOne works on rows.
  SubOne<<<floorf(N/256.f) + 1, 256>>>(nutop);
  SubOne<<<floorf(N/256.f) + 1, 256>>>(nutop + N);
  SubOne<<<floorf(N/256.f) + 1, 256>>>(nutop + 2*N);
  SubOne<<<floorf(N/256.f) + 1, 256>>>(nutop + 3*N);

  // -(2/3)*row0 + 3*row1 - 6*row2 + (11/3)*row3
  // accumulate in the 0th row
  // scale the 0th row by -(2/3)
  alpha = -(2.f/3.f);
  hipblasSscal(h, N, &alpha, nutop, 1);
  // Add 3*row1
  alpha = 3.f;
  hipblasSaxpy(h, N, &alpha, (nutop + N), 1, nutop, 1);
  // Add - 6*row2
  alpha = -6.f;
  hipblasSaxpy(h, N, &alpha, (nutop + (2*N)), 1, nutop, 1);
  // Add (11/3)*row3
  alpha = (11.f/3.f);
  hipblasSaxpy(h, N, &alpha, (nutop + (3*N)), 1, nutop, 1);
  // Divide the array by 2*DX
  alpha = 1.f/(2.f*DX);
  hipblasSscal(h, 4*N, &alpha, nutop, 1);
  // Elementwise multiplication with trnu
  ElemMultNu<<<floorf(N/256.f) + 1, 256>>>(nutop, trnu);
  // Sum up the elements of row0, by performing a dot product with
  // a row that has been altered to be all 1s.
  // Empty row1, then add 1 to all its elements
  alpha = 0.f;
  hipblasSscal(h, N, &alpha, (nutop + N), 1);
  AddOne<<<floorf(N/256.f) + 1, 256>>>(nutop + N);

  hipblasSdot(h, N, nutop, 1, (nutop + N), 1, &result);
  topsum = result /(-XF);

  // Calculate the Nusselt number along the bottom of the array.
  // d_nubot's first row is all 1, and ith row is the i-1th row of d_T
  // Put the first row of T in nubot, then subtract to get 0, then AddOne
  hipblasSscal(h, N, &alpha, nubot, 1);
  AddOne<<<floorf(N/256.f) + 1, 256>>>(nubot);
  hipblasScopy(h, N, T, 1, (nubot + N), 1);
  hipblasScopy(h, N, (T + N), 1, (nubot + 2*N), 1);
  hipblasScopy(h, N, (T + 2*N), 1, (nubot + 3*N), 1);

  // nubot += -( 1 - zbot)
  // => nubot += zbot; nubot -= 1
  alpha = 1.f;
  hipblasSaxpy(h, 4*N, &alpha, zbot, 1, nubot, 1);
  // Subtract 1 from every element in the array.  SubOne works on rows.
  SubOne<<<floorf(N/256.f) + 1, 256>>>(nubot);
  SubOne<<<floorf(N/256.f) + 1, 256>>>(nubot + N);
  SubOne<<<floorf(N/256.f) + 1, 256>>>(nubot + 2*N);
  SubOne<<<floorf(N/256.f) + 1, 256>>>(nubot + 3*N);

  // -(11/3)*row0 + 6*row1 - 3*row2 + (2/3)*row3
  // accumulate in the 0th row
  // scale the 0th row by -(11/3)
  alpha = -(11.f/3.f);
  hipblasSscal(h, N, &alpha, nubot, 1);
  // Add 6*row1
  alpha = 6.f;
  hipblasSaxpy(h, N, &alpha, (nubot + N), 1, nubot, 1);
  // Add -3*row2
  alpha = -3.f;
  hipblasSaxpy(h, N, &alpha, (nubot + (2*N)), 1, nubot, 1);
  // Add (2/3)*row3
  alpha = 2.f/3.f;
  hipblasSaxpy(h, N, &alpha, (nubot + (3*N)), 1, nubot, 1);
  // Divide the array by 2*DX
  alpha = 1.f/(2.f*DX);
  hipblasSscal(h, 4*N, &alpha, nubot, 1);
  // Elementwise multiplication with trnu
  ElemMultNu<<<floorf(N/256.f) + 1, 256>>>(nubot, trnu);
  // Sum up the elements of row0, by performing a dot product with
  // a row that has been altered to be all 1s.
  // The second row of nutop has already been set up for this.
  hipblasSdot(h, N, nubot, 1, (nutop + N), 1, &result);
  botsum = result /(-XF);

  return(topsum);
}
//=============================================================================
//                    Dz
//=============================================================================
// Finite-difference approximation to the first derivative with respect to z of
// a matrix shaped like T.  Execution forks if f is known to be psi.  Uses only
// row manipulations and the subtraction of 1 from each element of a vector. To
// extract a row, cublas routines are used.  The elements of the first column
// are separated in memory by N elements, so the initial elements of each row
// are likewise separated.  The individual elements of a single row are
// separated in memory by 1 element.

void Dz(hipblasHandle_t &h, float* f, int is_it_psi, float* y) {
  // yrows[i] = frows[i + 1] - frows[i - 1]
  // Move all but the first two rows of f into the interior rows of y.
  // The end of one row is one element away from the beginning of the next,
  // so adjacent rows are laid out in memory identically to a vector.
  hipblasScopy(h, N*(M-4), (f + (2*N)), 1, (y + N), 1);
  // Subtract all but the last two rows of f.
  const float alpha = -1.f;
  const float alpha2 = 1.f/(2.f*DX);
  hipblasSaxpy(h, N*(M-4), &alpha, f, 1, (y + N), 1);

  if(is_it_psi == 1) {
    // yrows[0] = frows[1]
    // Move the second row of f into the first row of y
    hipblasScopy(h, N, (f + N), 1, y, 1);
  }
  else {
    // yrows[0] = frows[1] - 1
    // Move the second row of f into the first row of y
    hipblasScopy(h, N, (f + N), 1, y, 1);
    // Subtract 1 from every element in the first row of y.
    SubOne<<<floorf(N/256.f) + 1, 256>>>(y);
  }

  // yrows[M-3] = -frows[M-4]
  // Copy the second-to-last row of f into the last row of y
  hipblasScopy(h, N, (f + ((M - 4)*N)), 1, (y + (M - 3)*N), 1);
  // Scale by -1.f
  hipblasSscal(h, N, &alpha, (y + (M - 3)*N), 1);
  // Scale y by 1/(2*DX)
  hipblasSscal(h, N*(M-2), &alpha2, y, 1);
}

//=============================================================================
//                    Dzz
//=============================================================================
// Finite-difference approximation to the second derivative with respect to z
// of a  T-shaped array.  Uses only row manipulations and the addition of 1 to
// each element of a vector.  To extract a row, cublas routines are used.  The
// elements of the first column are separated in memory by N elements, so the
// initial elements of each row are likewise separated.  The individual
// elements of a single row are separated in memory by 1 element.

void Dzz(hipblasHandle_t &h, float* f, float* y) {
  // yrows[i] = frows[i - 1] - 2*frows[i] + frows[i + 1]
  // Move all but the last two rows of f into the interior rows of y.
  hipblasScopy(h, N*(M-4), f, 1, (y + N), 1);

  const float alpha = -2.f;
  const float alpha2 = 1.f;
  const float alpha3 = 1.f/DX2;

  // Subtract 2* the interior rows of f
  hipblasSaxpy(h, N*(M-4), &alpha, (f + N), 1, (y + N), 1);

  // Add all but the first two rows of f.
  hipblasSaxpy(h, N*(M-4), &alpha2, (f + (2*N)), 1, (y + N), 1);

  // yrows[0] = 1 - 2*frows[0] + frows[1]
  // Copy the first row of f into the first row of y
  hipblasScopy(h, N, f, 1, y, 1);

  // scale by -2
  hipblasSscal(h, N, &alpha, y, 1);

  // add the second row of f
  hipblasSaxpy(h, N, &alpha2, (f + N), 1, y, 1);

  // Add 1 to every element in the first row of y.
  AddOne<<<floorf(N/256.f) + 1, 256>>>(y);

  // yrows[M-3] = frows[M-4] - 2*frows[M-3]
  // move the second-to-last row of f into the last row of y
  hipblasScopy(h, N, (f + (M - 4)*N), 1, (y + (M - 3)*N), 1);
  // subtract -2* the last row of f
  hipblasSaxpy(h, N, &alpha, (f + (M - 3)*N), 1, (y + (M - 3)*N), 1);
  // Scale y by 1/DX2
  hipblasSscal(h, (M-2)*N, &alpha3, y, 1);
}

//=============================================================================
//                    Dx
//=============================================================================
// Finite difference approximation to the first derivative with
// respect to x of a T-shaped matrix.  Forks if f is known to be psi. Uses only
// column manipulations and assumes all matrices are in row-major.  To extract
// a column, cublas routines are used.  If the beginning of an array is at f,
// then the elements of the first row (start of each column) are separated by
// one element, and each element within a column is separated by the length
// of a row, N.

void Dx(hipblasHandle_t &h, float* f, int is_it_psi, float* y) {
  // ycols[i] = fcols[i+1] - fcols[i-1], interior cols
  // Copy all but the first two columns of f into the interior columns of y.
  // Copy row-by-row instead of column-by-column, since Dcopy is optimized
  // for longer vectors.

  for(int i = 0; i < M - 2; i++) {
    hipblasScopy(h, (N-2), (f + i*N) + 2, 1, (y + i*N) + 1, 1);
  }
  // Subtract the block corresponding to all but the last two columns of f.
  float alpha = -1.f;
  for(int i = 0; i < M - 2; i++) {
    hipblasSaxpy(h, (N-2), &alpha, (f + i*N), 1, (y + i*N) + 1, 1);
  }

  if(is_it_psi == 1) {
    float alpha = 6.f;
    float alpha2 = -3.f;
    float alpha3 = 2.f/3.f;
    // ycols[0] = 6*fcols[1] - 3*fcols[2] + (2/3)*fcols[3]
    // Begin by copying the second column of f into the first column of y
    hipblasScopy(h, (M-2), (f + 1), N, y, N);
    // Scale it by a factor of 6
    hipblasSscal(h, (M-2), &alpha, y, N);
    // Subtract the third column of 3*f
    hipblasSaxpy(h, (M-2), &alpha2, (f + 2), N, y, N);
    // Add the fourth column of (2/3)*f
    hipblasSaxpy(h, (M-2), &alpha3, (f + 3), N, y, N);

    alpha = -alpha;
    alpha2 = -alpha2;
    alpha3 = -alpha3;
    //ycols[N-1] = -6*fcols[N-2] + 3*fcols[N-3] - (2/3)*fcols[N-4]
    // Copy the second-to-last column of f into the last column of y
    hipblasScopy(h, (M-2), (f + (N - 2)), N, (y + (N - 1)), N);
    // Scale it by a factor of -6
    hipblasSscal(h, (M-2), &alpha, (y + (N - 1)), N);
    // Add the third-to-last column of 3*fcols[N-3]
    hipblasSaxpy(h, (M-2), &alpha2, (f + (N - 3)), N, (y + (N - 1)), N);
    // Subtract the fourth-to-last column of (2/3)*f[N-4]
    hipblasSaxpy(h, (M-2), &alpha3, (f + (N - 4)), N, (y + (N - 1)), N);
  }
  else {
    // outside columns = 0
    const float alpha = 0.f;
    hipblasSscal(h, (M-2), &alpha, y, N);
    hipblasSscal(h, (M-2), &alpha, (y + (N - 1)), N);

  }

  // Scale y by (1/(2*DX))
  alpha = (1.f/(2.f*DX));
  hipblasSscal(h, N*(M-2), &alpha, y, 1);
}

//=============================================================================
//                    Dxx
//=============================================================================
// Finite-difference approximation to the second derivative with
// respect to x of a T-shaped matrix.  The input is always going to be a temp-
// erature array.  Uses only column manipulations and assumes all matrices are
// in row-major.  To extract a column, cublas routines are used.  If the
// beginning of an array is at f, then the elements of the first row (start of
// each column) are separated by one element, and each element within a column
// is separated by the length of a row, N.

void Dxx(hipblasHandle_t &h, float* f, float* y) {
  const float alpha = -2.f;
  const float alpha2 = 2.f;
  const float alpha3 = 1.f;
  const float alpha4 = 1.f/(DX2);

  // ycols[i] = fcols[i-1] - 2*fcols[i] + fcols[i+1], interior columns
  // copy f into y
  hipblasScopy(h, N*(M-2), f, 1, y, 1);
  // scale by -2
  hipblasSscal(h, N*(M-2), &alpha, y, 1);
  // Add the block corresponding to all but the last two columns of f.
  for(int i = 0; i < M-2; i++) {
    hipblasSaxpy(h, (N-2), &alpha3, (f + i*N), 1, (y + i*N) + 1, 1);
  }
  // Add the block corresponding to all but the first two columns of f.
  for(int i = 0; i < M-2; i++) {
    hipblasSaxpy(h, (N-2), &alpha3, (f + i*N) + 2, 1, (y + i*N) + 1, 1);
  }

  // ycols[0] = -2*fcols[0] + 2*fcols[1]
  // Copy the first column of f into the first column of y.
  hipblasScopy(h, (M-2), f, N, y, N);
  // Scale the first column of y by -2.f
  hipblasSscal(h, (M-2), &alpha, y, N);
  // Add 2* the second column of f.
  hipblasSaxpy(h, (M-2), &alpha2, (f + 1), N, y, N);

  // ycols[N-1] = -2*fcols[N-1] + 2*fcols[N-2]
  // Move the last column of f into the last column of y
  hipblasScopy(h, (M-2), (f + (N - 1)), N, (y + (N - 1)), N);
  // Scale by -2.f
  hipblasSscal(h, (M-2), &alpha, (y + (N - 1)), N);
  // add 2* the second-to-last column of f.
  hipblasSaxpy(h, (M-2), &alpha2, (f + (N - 2)), N, (y + (N - 1)), N);

  // Scale y by 1/(DX^2)
  hipblasSscal(h, N*(M-2), &alpha4, y, 1);
}

//=============================================================================
//                     G
//=============================================================================
// Computes the RK1 approximation using finite difference method, storing the
// result in output
void G(hipblasHandle_t &h,
       float* f,
       float* Tbuff,
       float* DxT,
       float* y,
       float* u,
       float* v,
       float* psi,
       float* omega,
       float* dsc,
       float* dsr,
       float* ei,
       float* dt,
       float* output,
       float* h_T,
       int compute_velocity,
       int frames,
       int tstep) 
{
  // Define the grid dimensions
  dim3 grid(floorf(N/16.f) + 1, floorf((M-2)/16.f) + 1), block(256);

  // Define omega to be the interior columns of Dxf
  // Save Dx of f in DxT for later
  hipblasScopy(h, N*(M-2), f, 1, DxT, 1);
  Dx(h, f, 0, DxT);

  // Copy the interior columns of DxT to omega
  for(int i = 0; i < M-2; i ++) {
    hipblasScopy(h, (N-2), (DxT + i*N)+1, 1, (omega + i*(N-2)), 1);
  }

  // Perform some matrix multiplications.  cublas assumes everything is in
  // column-major, so while we want to perform:
  // omega = dsc*omega
  // omega = omega*dsr
  // omega = omega.*ei
  // omega = dsc*omega
  // omega = omega*dsr
  // we observe that Transpose(A*B) = Transpose(B)*Transpose(A) to perform
  // these same manipulations while preserving row-major storage.
  // Omega has dimensions (M-2)xN, but cublas thinks this is Nx(M-2).
  // dsc and dsr are square..
  // Perform Transpose(omega) = Transpose(omega)*Transpose(dsc):
  //
  // The call for cublasSgemm is (CUBLAS_OP_N, CUBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb,
  // beta, C, ldc), where A is m-by-k, B is k-by-n, and C is m-by-n.
  // Since A = C, k = n, so B is k-by-k.  B = dsc, so k=n=M-2 and m = N-2.
  const float alpha = 1.f;
  const float alpha2 = -1.f;
  const float beta = 0.f;

  hipblasSgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, N-2, M-2, M-2, &alpha, omega, N-2, dsc, M-2, &beta, Tbuff, N-2 );
  hipblasScopy(h, (N-2)*(M-2), Tbuff, 1, omega, 1);

  // Perform Tranpose(dsr)*Transpose(omega), store in omega.
  // since A is square, M = k = N-2, so n must be M-2.
  hipblasSgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, N-2, M-2, N-2, &alpha, dsr, N-2, omega, N-2, &beta, Tbuff, N-2);
  hipblasScopy(h, (N-2)*(M-2), Tbuff, 1, omega, 1);

  // elementwise matrix multiplication, storing the result in omega
  //DEBUG
  ElemMultOmega<<<grid, block>>>(omega, ei);

  // same Transpose(omega)*Transpose(dsc) operation as before
  hipblasSgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, N-2, M-2, M-2, &alpha, omega, N-2, dsc, M-2, &beta, Tbuff, N-2);
  hipblasScopy(h, (N-2)*(M-2), Tbuff, 1, omega, 1);


  // same Transpose(dsr)*Transpose(omega) operation as before
  hipblasSgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, N-2, M-2, N-2, &alpha, dsr, N-2, omega, N-2, &beta, Tbuff, N-2);
  hipblasScopy(h, (N-2)*(M-2), Tbuff, 1, omega, 1);

  // Scale omega by -(DX^4)*(RA) = OMEGACOEFF
  const float omegacoeff = OMEGACOEFF;
  hipblasSscal(h, (N-2)*(M-2), &omegacoeff, omega, 1);

  // interior columns of psi = (RA*DX^4)*omega
  // copy omega into the interior columns of psi
  // omega has rows of length N-2 instead of N

  for(int i = 0; i < M-2; i ++) {
    hipblasScopy(h, (N-2), (omega + i*(N-2)), 1, (psi + i*N)+1, 1);
  }

  // Velocity in the x-direction
  hipblasScopy(h, N*(M-2), f, 1, u, 1);
  Dz(h, psi, 1, u);

  // v is -Dxpsi, velocity in the z direction.
  // Place Dxpsi into v
  hipblasScopy(h, N*(M-2), f, 1, v, 1);
  Dx(h, psi, 1, v);

  // Change the sign of v.
  hipblasSscal(h, (M-2)*N, &alpha2, v, 1);
  // Store v in the z velocity file

  // If compute_velocity = 1, we need to update dt
  if(compute_velocity == 1) {
    // CublasIdamax returns 1-indexed pointers into the max element of a
    // float-precision vector
    int iu, iv;
    hipblasIsamax(h, N*(M-2), u, 1, &iu);
    hipblasIsamax(h, N*(M-2), v, 1, &iv);
    Updatedt<<<1,1>>>(iu, u, iv, v, dt);
  }

  // Place Dxxf into y
  hipblasScopy(h, N*(M-2), f, 1, y, 1);
  Dxx(h, f, y);

  // y = y + Dzzf
  // place Dzzf into Tbuff
  hipblasScopy(h, N*(M-2), f, 1, Tbuff, 1);
  Dzz(h, f, Tbuff);

  // Add the elements of y and Tbuff, storing in y.
  hipblasSaxpy(h, N*(M-2), &alpha, Tbuff, 1, y, 1);

  // u = u.*DxT, where .* denotes elementwise multiplication
  // Perform the elentwise multiplication, storing in u
  ElemMultT<<<grid, block>>>(u,DxT);

  // y = y + u
  // Add y and u, storing the result in y
  hipblasSaxpy(h, N*(M-2), &alpha, u, 1, y, 1);

  // u = DzT
  hipblasScopy(h, N*(M-2), f, 1, u, 1);
  Dz(h, f, 0, u);

  // u = v.*u, where .* denotes elementwise multiplication.
  ElemMultT<<<grid, block>>>(u,v);

  // y = y + u
  hipblasSaxpy(h, N*(M-2), &alpha, u, 1, y, 1);

  // copy into output
  hipblasScopy(h, N*(M-2), y, 1, output, 1);
}


//=============================================================================
//                 ENTRY POINT
//=============================================================================

int main(int argc, char* argv[]) {
  if (argc != 2) {
    printf("Usage: %s <timesteps>\n", argv[0]);
    return 1;
  }
  const int ENDSTEP = atoi(argv[1]); // Last timestep to compute

  printf("M = %d. ", M);   // vertical dimension of the temperature array
  printf("N = %d. ", N);   // horizontal dimension of temperature array
  printf("DX = %E. ", DX); // x and z-dimensional mesh spacing
  printf("Ra = %E.\n", RA);// Rayleigh number

  printf("\nInitialization\n");
  hipblasHandle_t h;
  hipblasStatus_t stat = hipblasCreate(&h);
  if(stat != HIPBLAS_STATUS_SUCCESS) {
    printf("Failed to initialize CUBLAS\n");
    return EXIT_FAILURE;
  }

  printf("Configuring host and device memory.\n");
  // initialize dt to the chosen start parameter
  float dt = DT_START;
  float h_dt = dt;

  float* d_dt; // Device-side shadow of dt, as a one-element array.
  hipMalloc((void**)&d_dt, sizeof(float));
  hipblasSetVector(1, sizeof(float), &h_dt, 1, d_dt, 1);

  // Define T
  float* d_T;
  hipMalloc((void**)&d_T, N*(M-2) * sizeof(float));

  float* h_X = (float*)malloc(N*sizeof(float));
  float* h_Z = (float*)malloc((M-2)*sizeof(float));
  float* h_T = (float*)malloc(N*(M-2)*sizeof(float));

  // Set the values of X and Z
  for(int i = 0; i < N; i++) {
    h_X[i] = (i*XF + 0.f)/(N - 1.f);
  }

  // Some wonky indexing here to accomodate the fact that T has its top and
  // bottom rows chopped off.
  for(int i = 1; i < (M - 1); i++) {
    h_Z[i - 1] = (i + 0.f)/(M - 1.f);
  }
  // Initialize T, perturbing it slightly.
  for(int i = 0; i < M-2; i++) {
    for(int j = 0; j < N; j++) {
      h_T[i*N + j] = 1 - h_Z[i] + 0.01*sin(PI*h_Z[i])*cos((PI/XF)*h_X[j]);
      // For debugging purposes
      //h_T[i*N + j] = i + 1;
    }
  }

  // Define the arrays necessary to calculate the Nusselt number.
  // These will be transferred to the GPU.
  float* h_ztop = (float*)malloc(4*N*sizeof(float));
  float* h_zbot = (float*)malloc(4*N*sizeof(float));


  for(int i = 0; i < 4*N; i++) {
    h_ztop[i] = 0.f;
    h_zbot[i] = 0.f;
  }

  for(int i = 0; i < N; i++ ) {
    // First row of h_ztop
    h_ztop[i] = 1-3*DX;
    // Second row of h_ztop
    h_ztop[i + N] = 1-2*DX;
    // Third row of h_ztop
    h_ztop[i + 2*N] = 1-DX;

    // Second row of h_zbot
    h_zbot[i + N] = h_Z[0];
    // Third row of h_zbot
    h_zbot[i + 2*N] = h_Z[1];
    // Fourth row of h_zbot
    h_zbot[i + 3*N] = h_Z[2];
  }

  // The bottom row of ztop is 1.f and the top row of zbot is 0.f
  for(int i = 3*N; i < 4*N; i++) {
    h_ztop[i] = 1.f;
  }

  // Shadow T in GPU memory.  Although cublasSetMatrix assumes its input is
  // in column-major,
  hipblasSetVector(N*(M-2), sizeof(float), h_T, 1, d_T, 1);

  float* d_omega;
  hipMalloc((void**)&d_omega, (M-2)*(N-2) * sizeof(float));

  float* d_psi;
  hipMalloc((void**)&d_psi, (M-2)*N * sizeof(float));

  float* d_dsc;
  hipMalloc((void**)&d_dsc, (M-2)*(M-2) * sizeof(float));

  float* d_dsr;
  hipMalloc((void**)&d_dsr, (N-2)*(N-2) * sizeof(float));

  float* d_ei;
  hipMalloc((void**)&d_ei, (N-2)*(M-2) * sizeof(float));

  float* d_tr;
  hipMalloc((void**)&d_tr, N*M * sizeof(float));

  float* d_trnu;
  hipMalloc((void**)&d_trnu, N * sizeof(float));

  float* d_ztop, *d_zbot, *d_nutop, *d_nubot;
  hipMalloc((void**)&d_ztop, 4*N * sizeof(float));
  hipMalloc((void**)&d_zbot, 4*N * sizeof(float));
  hipMalloc((void**)&d_nutop, 4*N * sizeof(float));
  hipMalloc((void**)&d_nubot, 4*N * sizeof(float));

  // Initialize dsr, dsc, lambda, mu, ei, and trNu and copy them over
  float* h_dsc = (float*)malloc((M-2)*(M-2)*sizeof(float));
  float* h_dsr = (float*)malloc((N-2)*(N-2)*sizeof(float));
  float* h_lambda = (float*)malloc((M-2)*sizeof(float));
  float* h_mu = (float*)malloc((N-2)*sizeof(float));
  float* h_ei = (float*)malloc((M-2)*(N-2)*sizeof(float));
  float* h_tr = (float*)malloc(M*N*sizeof(float));
  float* h_trnu = (float*)malloc(N*sizeof(float));
  // Set the value of h_dsc
  for(int i = 0; i < M-2; i++) {
    for(int j = 0; j < M-2; j++) {
      h_dsc[(M-2)*i + j] = sqrtf(2.f/(M-1.f))*sin((i+1.f)*(j+1.f)*PI/(M-1.f));
      // For debugging purposes.
      //if(i == j) h_dsc[(M-2)*i + j] = 1;
      //else h_dsc[(M-2)*i + j] = 0;
    }
  }

  // Set the value of h_dsr.
  for(int i = 0; i < N-2; i++) {
    for(int j = 0; j < N-2; j++) {
      h_dsr[(N-2)*i + j] = sqrtf(2.f/(N-1.f))*sin((i+1.f)*(j+1.f)*PI/(N-1.f));
      // For debugging purposes.
      //if(i == j) h_dsr[(N-2)*i + j] = 1;
      //else h_dsr[(N-2)*i + j] = 0;
    }
  }

  // Initialize lambda and mu, which are used to compute ei.
  for(int i = 0; i < M-2; i++) {
    h_lambda[i] = 2.f*cos((i + 1.f)*PI/(M - 1.f)) - 2.f;
  }

  for(int i = 0; i < N-2; i++) {
    h_mu[i] = 2.f*cos((i + 1.f)*PI/(N - 1.f)) - 2.f;
  }
  // Compute ei from lambda and mu.
  // The elements of ei are inverted on the last step to replace later
  // divisions by multiplications.
  for(int i = 0; i < M-2; i++) {
    for(int j = 0; j < N-2; j++) {
      h_ei[(N-2)*i + j] = h_lambda[i] + h_mu[j];
      h_ei[(N-2)*i + j] = (h_ei[(N-2)*i + j])*(h_ei[(N-2)*i + j]);
      h_ei[(N-2)*i + j] = 1.f/(h_ei[(N-2)*i + j]);
    }
  }
  // Compute tr
  for(int i = 0; i < M; i++) {
    for(int j = 0; j < N; j++) {
      h_tr[N*i + j] = DX*DX/4.f;
      if(j>0 && j<(M-1)) h_tr[N*i + j] = DX2/2.f;
      if(i>0 && i<(N-1)) h_tr[N*i + j] = DX2/2.f;
      if(j>0 && j<(M-1) && i>0 && i<(N-1)) h_tr[N*i + j] = DX2;
    }
  }

  // Compute trnu
  for(int i = 1; i < N-1; i++) {
    h_trnu[i] = DX;
  }
  h_trnu[0] = DX/2.f;
  h_trnu[N-1] = DX/2.f;

  // Copy the completed data over.
  hipblasSetVector((M-2)*(M-2), sizeof(float), h_dsc, 1, d_dsc, 1);
  hipblasSetVector((N-2)*(N-2), sizeof(float), h_dsr, 1, d_dsr, 1);
  hipblasSetVector((M-2)*(N-2), sizeof(float), h_ei, 1, d_ei, 1);
  hipblasSetVector(M*N, sizeof(float), h_tr, 1, d_tr, 1);
  hipblasSetVector(N, sizeof(float), h_trnu, 1, d_trnu, 1);
  hipblasSetVector(4*N, sizeof(float), h_ztop, 1, d_ztop, 1);
  hipblasSetVector(4*N, sizeof(float), h_ztop, 1, d_nutop, 1);
  hipblasSetVector(4*N, sizeof(float), h_zbot, 1, d_zbot, 1);
  hipblasSetVector(4*N, sizeof(float), h_zbot, 1, d_nubot, 1);

  float* d_u;
  float* d_v;
  float* d_xrk3;
  float* d_yrk3;
  float* d_zrk3;
  float* d_y;
  float* d_Tbuff;
  float* d_DxT;

  hipMalloc((void**)&d_u, N*(M-2) * sizeof(float));
  hipMalloc((void**)&d_v, N*(M-2) * sizeof(float));

  hipMalloc((void**)&d_xrk3, N*(M-2) * sizeof(float));
                            
  hipMalloc((void**)&d_yrk3, N*(M-2) * sizeof(float));
                            
  hipMalloc((void**)&d_zrk3, N*(M-2) * sizeof(float));
  hipMalloc((void**)&d_y, N*(M-2) * sizeof(float));

  hipMalloc((void**)&d_Tbuff, N*(M-2) * sizeof(float));

  hipMalloc((void**)&d_DxT, N*(M-2) * sizeof(float));

  // use d_T to define d_psi, d_u, d_v, and d_y = 0.
  float alpha = -1.f;
  hipblasScopy(h, N*(M-2), d_T, 1, d_psi, 1);
  hipblasSaxpy(h, N*(M-2), &alpha, d_psi, 1, d_psi, 1);
  hipblasScopy(h, N*(M-2), d_psi, 1, d_u, 1);
  hipblasScopy(h, N*(M-2), d_psi, 1, d_v, 1);
  hipblasScopy(h, N*(M-2), d_psi, 1, d_y, 1);

  //initialize all intermediate matrices to T;
  hipblasScopy(h, N*(M-2), d_T, 1, d_xrk3, 1);
  hipblasScopy(h, N*(M-2), d_T, 1, d_yrk3, 1);
  hipblasScopy(h, N*(M-2), d_T, 1, d_zrk3, 1);
  hipblasScopy(h, N*(M-2), d_T, 1, d_Tbuff, 1);
  hipblasScopy(h, N*(M-2), d_T, 1, d_DxT, 1);


  // DEBUG
  // temporary buffer for use in d_*rk3 stuff
  float* d_temp;
  hipMalloc((void**)&d_temp, N*(M-2) * sizeof(float));
  hipblasScopy(h, N*(M-2), d_T, 1, d_temp, 1);

  printf("Begin computation: \n");

  //=============================================================================
  //                Computation
  //=============================================================================

  // Begin timestep computation.
  float frames = 0.f;
  int tstep = 0;

  // Variable to store timing information
  auto start = std::chrono::steady_clock::now();

  for(int c = STARTSTEP; c < ENDSTEP; c++) {
    // Use SHORTG macro to call g succinctly.
    // x = g(T,0)
    // z = g(T + (dt/3)*x, 0)
    // z = g(T + (2*dt/3)*z, 1)
    // T = T + (dt/4)*(x + 3z)
    // Store the first part of RK3 in d_xrk3
    SHORTG(d_T, 1, d_xrk3, c);

    // add (dt/3)*d_xrk3 to T, store the result in T temporarily.
    //    cublasSaxpy(h, N*(M-2), (dt/3.f), d_xrk3, 1, d_T, 1);
    //DEBUG
    hipblasScopy(h, N*(M-2), d_T, 1, d_temp,  1);
    alpha = dt/3.f;
    hipblasSaxpy(h, N*(M-2), &alpha, d_xrk3, 1, d_temp, 1);
    // Compute d_yrk3 = g(T + (dt/3)*d_xrk3, 0) by using the updated T.
    //    SHORTG(d_T, 0, d_yrk3);

    //DEBUG
    SHORTG(d_temp, 0, d_yrk3, 0);

    // return d_T to its original state by subtracting (dt/3)*x
    //    cublasSaxpy(h, N*(M-2), (-(dt/3.f)), d_xrk3, 1, d_T, 1);
    // Add (2*dt/3)*d_yrk3 to T, store the result in T temporarily.
    //    cublasSaxpy(h, N*(M-2), (2.f*(dt/3)), d_yrk3, 1, d_T, 1);
    //DEBUG
    hipblasScopy(h, N*(M-2), d_T, 1, d_temp, 1);
    alpha = 2.f*(dt/3.f);
    hipblasSaxpy(h, N*(M-2), &alpha, d_yrk3, 1, d_temp, 1);
    // Compute d_zrk3 = g(T + (2*dt/3)*d_yrk3) by using the updated T.
    //    SHORTG(d_T, 1, d_zrk3);

    //DEBUG
    SHORTG(d_temp, 0, d_zrk3, 0);

    // return d_T to its original state by subtracting (2*dt/3)*d_yrk3
    //    cublasSaxpy(h, N*(M-2), (-(2.f*(dt/3))), d_yrk3, 1, d_T, 1);
    // T+= (dt/4)*(x + 3z)
    // Add (dt/4)*d_xrk3 to d_T
    //    cublasSaxpy(h, N*(M-2), (dt/4.f), d_xrk3, 1, d_T, 1);
    // Add 3*(dt/4)*d_zrk3 to d_T
    //    cublasSaxpy(h, N*(M-2), (3.f*(dt/4.f)), d_zrk3, 1, d_T, 1);
    //DEBUG
    alpha = dt/4.f;
    hipblasSaxpy(h, N*(M-2), &alpha, d_xrk3, 1, d_T, 1);
    alpha = 3.f * alpha;
    hipblasSaxpy(h, N*(M-2), &alpha, d_zrk3, 1, d_T, 1);

    // update the value of dt (in host) from d_dt (in device)
    hipblasGetVector(1, sizeof(float), d_dt, 1, &h_dt, 1);
    dt = h_dt;
    frames += dt;

    // keeps track of several Nu samples.
    if(frames > FRAMESIZE) {
      tstep++;
      // Calculate the nusselt number throughout the array and save
      float nunum = NusseltCompute(h, d_T, d_nutop, d_ztop, d_zbot, d_nubot, d_trnu);
      printf("Nusselt number: %.1f\n", nunum);
      frames = 0.f;
    }
  }

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  auto time_sec = time * 1e-9f;
  printf("Total compute time: %f (s)\n", time_sec);
  printf("Average compute time per step: %f (s)\n", time_sec / ENDSTEP);

  free(h_X);
  free(h_Z);
  free(h_T);

  free(h_dsr);
  free(h_dsc);
  free(h_lambda);
  free(h_mu);
  free(h_ei);
  free(h_tr);
  free(h_trnu);
  free(h_ztop);
  free(h_zbot);

  hipFree(d_temp);
  hipFree(d_T);
  hipFree(d_dt);
  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_xrk3);
  hipFree(d_yrk3);
  hipFree(d_zrk3);
  hipFree(d_y);
  hipFree(d_Tbuff);
  hipFree(d_DxT);
  hipFree(d_omega);
  hipFree(d_psi);
  hipFree(d_dsc);
  hipFree(d_dsr);
  hipFree(d_ei);
  hipFree(d_tr);
  hipFree(d_trnu);
  hipFree(d_ztop);
  hipFree(d_zbot);
  hipFree(d_nutop);
  hipFree(d_nubot);

  hipblasDestroy(h);
  printf("Done.\n");
  return 0;
}
