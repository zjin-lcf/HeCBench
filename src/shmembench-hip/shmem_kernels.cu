#include "hip/hip_runtime.h"
/**
 * shmem_kernels.cu: This file is part of the gpumembench suite.
 *
 * Contact: Elias Konstantinidis <ekondis@gmail.com>
 **/

#include <chrono> // timing
#include <stdio.h>
#include "hip/hip_runtime.h"

using namespace std::chrono;

#define TOTAL_ITERATIONS (1024)
#define BLOCK_SIZE 256

// shared memory swap operation
__device__ void shmem_swap(float4 *v1, float4 *v2){
  float4 tmp;
  tmp = *v2;
  *v2 = *v1;
  *v1 = tmp;
}

__device__ float4 init_val(int i){
  return make_float4(i, i+11, i+19, i+23);
}

__device__ float4 reduce_vector(float4 v1, float4 v2, float4 v3, float4 v4, float4 v5, float4 v6){
  return (v1 + v2 + v3 + v4 + v5 + v6);
}

__device__ void set_vector(float4 *target, int offset, float4 v){
  target[offset] = v;
}


__global__ void benchmark_shmem(float4 *g_data){

  __shared__ float4 shm_buffer[BLOCK_SIZE*6];

  int tid = threadIdx.x; 
  int globaltid = blockIdx.x*blockDim.x + tid;
  set_vector(shm_buffer, tid+0*blockDim.x, init_val(tid));
  set_vector(shm_buffer, tid+1*blockDim.x, init_val(tid+1));
  set_vector(shm_buffer, tid+2*blockDim.x, init_val(tid+3));
  set_vector(shm_buffer, tid+3*blockDim.x, init_val(tid+7));
  set_vector(shm_buffer, tid+4*blockDim.x, init_val(tid+13));
  set_vector(shm_buffer, tid+5*blockDim.x, init_val(tid+17));

  __syncthreads();  // __threadfence_block() is faster though

  #pragma unroll 32
  for(int j=0; j<TOTAL_ITERATIONS; j++){
    shmem_swap(shm_buffer+tid+0*blockDim.x, shm_buffer+tid+1*blockDim.x);
    shmem_swap(shm_buffer+tid+2*blockDim.x, shm_buffer+tid+3*blockDim.x);
    shmem_swap(shm_buffer+tid+4*blockDim.x, shm_buffer+tid+5*blockDim.x);

    __syncthreads();

    shmem_swap(shm_buffer+tid+1*blockDim.x, shm_buffer+tid+2*blockDim.x);
    shmem_swap(shm_buffer+tid+3*blockDim.x, shm_buffer+tid+4*blockDim.x);

    __syncthreads();
  }

  g_data[globaltid] = reduce_vector(shm_buffer[tid+0*blockDim.x], 
                                    shm_buffer[tid+1*blockDim.x],
                                    shm_buffer[tid+2*blockDim.x],
                                    shm_buffer[tid+3*blockDim.x],
                                    shm_buffer[tid+4*blockDim.x],
                                    shm_buffer[tid+5*blockDim.x]);
}

void shmembenchGPU(double *c, const long size, const int n) {
  const int TOTAL_BLOCKS = size/(BLOCK_SIZE);

  double *cd;
  hipMalloc((void**)&cd, size*sizeof(double));

  dim3 dimBlock(BLOCK_SIZE, 1, 1);
  dim3 dimGrid_f4(TOTAL_BLOCKS/4, 1, 1);

  auto start = high_resolution_clock::now();

  for (int i = 0; i < n; i++)
    benchmark_shmem<<< dimGrid_f4, dimBlock >>>((float4*)cd);

  hipDeviceSynchronize();
  auto end = high_resolution_clock::now();
  auto time_shmem_128b = duration_cast<nanoseconds>(end - start).count() / (double)n;
  printf("Average kernel execution time : %f (ms)\n", time_shmem_128b * 1e-6);

  // Copy results back to host memory
  hipMemcpy(c, cd, size*sizeof(double), hipMemcpyDeviceToHost);
  hipFree(cd);

  // simple checksum
  double sum = 0;
  for (long i = 0; i < size; i++) sum += c[i];
  if (sum != 21256458760384741137729978368.00)
    printf("checksum failed\n");
  
  printf("Memory throughput\n");
  const long long operations_bytes  = (6LL+4*5*TOTAL_ITERATIONS+6)*size*sizeof(float);
  const long long operations_128bit = (6LL+4*5*TOTAL_ITERATIONS+6)*size/4;

  printf("\tusing 128bit operations : %8.2f GB/sec (%6.2f billion accesses/sec)\n", 
    (double)operations_bytes / time_shmem_128b,
    (double)operations_128bit / time_shmem_128b);
}
