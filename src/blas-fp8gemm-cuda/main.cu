#include "hip/hip_runtime.h"
#include <chrono>
#include "helper.h"

/// Sample wrapper executing fp8 matmul with hipblasLtMatmul, with addition of per-tensor scaling, amax calculations, and
/// the workspace to support split-K algorithms.
///
/// pointer mode is for alpha and beta is always host, to change it configure the appropriate matmul descriptor
/// attribute matmul is not using cublas handle's configuration of math mode, here tensor ops are implicitly allowed; to
/// change this configure appropriate attribute in the preference handle
void LtFp8Matmul(const int repeat,
                 hipblasLtHandle_t ltHandle,
                 int m,
                 int n,
                 int k,
                 const float *alpha, /* host pointer */
                 const float *beta, /* host pointer */
                 const float *a_scale, /* device pointer */
                 const __hip_fp8_e4m3_fnuz *A,
                 int lda,
                 const float *b_scale, /* device pointer */
                 const __hip_fp8_e4m3_fnuz *B,
                 int ldb,
                 const float *c_scale, /* device pointer */
                 const __hip_bfloat16 *C,
                 int ldc,
                 const float *d_scale, /* device pointer */
                 __hip_fp8_e4m3_fnuz *D,
                 //__hip_bfloat16 *D,
                 float *amax_d, /* device pointer */
                 void *workspace,
                 size_t workspaceSize) {
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL, Ddesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;

    // A must be transposed and B non-transposed
    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    int returnedResults                             = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // set scaling factors
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &a_scale, sizeof(a_scale)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &b_scale, sizeof(b_scale)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_C_SCALE_POINTER, &c_scale, sizeof(c_scale)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER, &d_scale, sizeof(d_scale)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_AMAX_D_POINTER, &amax_d, sizeof(amax_d)));

    // set fast accumulation
    int8_t fast_accum = 1;
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, CUBLASLT_MATMUL_DESC_FAST_ACCUM, &fast_accum, sizeof(fast_accum)));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    // table of supported type combinations can be found in the documentation: https://docs.nvidia.com/cuda/cublas/index.html#cublasltmatmul
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8F_E4M3, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8F_E4M3, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_16BF, m, n, ldc));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_8F_E4M3, m, n, ldc));
    //checkCublasStatus(hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_16BF, m, n, ldc));

    // create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
    // will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
    // directly come from hipMalloc)
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    // we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
    // is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Ddesc, preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0) {
        printf("no heuristic function available for current configuration\n");
        return;
    }

    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < repeat; i++) {
      checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     operationDesc,
                                     alpha, A, Adesc,
                                     B, Bdesc, beta,
                                     C, Cdesc,
                                     D, Ddesc,
                                     &heuristicResult.algo,
                                     workspace,
                                     workspaceSize,
                                     0));
    }

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    auto ns_fp8 = (time / repeat);
    printf("Average hipblasLtMatmul execution time %10.3f (us) | ", ns_fp8 * 1e-3f);
    printf("Average hipblasLtMatmul performance %.1f (TFLOPS)\n", 2.f * m * k * n / ns_fp8 * 1e-3f);

    // descriptors are no longer needed as all GPU work was already enqueued
    if (preference) checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    if (Ddesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Ddesc));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
}


int main(int argc, char *argv[])
{
   if (argc != 2) {
     printf("Usage: %s <repeat>\n", argv[0]);
     return 1;
   }
   const int repeat = atoi(argv[1]);

   const int shapes[5][3] = {{16384, 8192, 1280},
                             {16384, 1024, 8192},
                             {16384, 8192, 7168},
                             {16384, 3584, 8192},
                             {8192, 8192, 8192}};

   for (int i = 0; i < 5; i++) {

     int m = shapes[i][0], n = shapes[i][1], k = shapes[i][2];
     printf("Matrix dimension (M, N, K) = (%d, %d, %d)\n", m, n, k);

     TestBench<__hip_fp8_e4m3_fnuz,
               __hip_bfloat16, // C type
               __hip_fp8_e4m3_fnuz, // output type
               //__hip_bfloat16, // output type
               float> props(m, n, k, 1.0f, 0.0f, 32ULL * 1024 * 1024);

     props.run([&props, repeat] {
          LtFp8Matmul(repeat,
                      props.ltHandle,
                      props.m,
                      props.n,
                      props.k,
                      &props.alpha,
                      &props.beta,
                      props.AscaleDev, props.Adev, props.k, // mxk
                      props.BscaleDev, props.Bdev, props.k, // kxn
                      props.CscaleDev, props.Cdev, props.m, // mxn
                      props.DscaleDev, props.Ddev,
                      props.DamaxDev,
                      props.workspace,
                      props.workspaceSize);
      });
    }

    return 0;
}
