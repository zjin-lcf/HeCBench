/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <algorithm>
#include <cmath>
#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

#define MAX_KERNEL_THREADS 256
#define mask 0xFFFFFFFF

// float or double
typedef float vtype;
typedef vector<vector<vtype>> matrix;

template<typename T>
__device__
T parallel_prefix_sum(const int n, const int *ind, const T *w)
{

  T sum = 0.0;
  T last;

  int mn =(((n+blockDim.x-1)/blockDim.x)*blockDim.x); //n in multiple of blockDim.x
  for (int i=threadIdx.x; i<mn; i+=blockDim.x) {
    //All threads (especially the last one) must always participate
    //in the shfl instruction, otherwise their sum will be undefined.
    //So, the loop stopping condition is based on multiple of n in loop increments,
    //so that all threads enter into the loop and inside we make sure we do not
    //read out of bounds memory checking for the actual size n.

    //check if the thread is valid
    bool valid  = i<n;

    //Notice that the last thread is used to propagate the prefix sum.
    //For all the threads, in the first iteration the last is 0, in the following
    //iterations it is the value at the last thread of the previous iterations.

    //get the value of the last thread
    last = __shfl_sync(mask, sum, blockDim.x-1, blockDim.x);

    //if you are valid read the value from memory, otherwise set your value to 0
    sum = (valid) ? w[ind[i]] : 0.0;

    //do prefix sum (of size warpSize=blockDim.x =< 32)
    for (int j=1; j<blockDim.x; j*=2) {
      T v = __shfl_up_sync(mask, sum, j, blockDim.x);
      if (threadIdx.x >= j) sum += v;
    }
    //shift by last
    sum += last;
    //notice that no __threadfence or __syncthreads are needed in this implementation
  }
  //get the value of the last thread (to all threads)
  last = __shfl_sync(mask, sum, blockDim.x-1, blockDim.x);

  return last;
}

// Volume of neighboors (*weight_s)
template<bool weighted, typename T>
__global__ void
jaccard_row_sum(const int n,
                const int *__restrict__ csrPtr,
                const int *__restrict__ csrInd,
                const T *__restrict__ w,
                      T *__restrict__ work)
{
  for (int row=threadIdx.y+blockIdx.y*blockDim.y; row<n; row+=gridDim.y*blockDim.y) {
    int start = csrPtr[row];
    int end   = csrPtr[row+1];
    int length= end-start;
    //compute row sums
    if (weighted) {
      T sum = parallel_prefix_sum(length, csrInd + start, w);
      if (threadIdx.x == 0) work[row] = sum;
    }
    else {
      work[row] = (T)length;
    }
  }
}

// Volume of intersections (*weight_i) and cumulated volume of neighboors (*weight_s)
// Note the number of columns is constrained by the number of rows
template<bool weighted, typename T>
__global__ void
jaccard_is(const int n, const int e,
           const int *__restrict__ csrPtr,
           const int *__restrict__ csrInd,
           const T *__restrict__ v,
           const T *__restrict__ work,
                 T *__restrict__ weight_i,
                 T *__restrict__ weight_s)
{
  for (int row=threadIdx.z+blockIdx.z*blockDim.z; row<n; row+=gridDim.z*blockDim.z) {
    for (int j=csrPtr[row]+threadIdx.y+blockIdx.y*blockDim.y;
             j<csrPtr[row+1]; j+=gridDim.y*blockDim.y) {
      int col = csrInd[j];
      //find which row has least elements (and call it reference row)
      int Ni = csrPtr[row+1] - csrPtr[row];
      int Nj = csrPtr[col+1] - csrPtr[col];
      int ref= (Ni < Nj) ? row : col;
      int cur= (Ni < Nj) ? col : row;

      //compute new sum weights
      weight_s[j] = work[row] + work[col];

      //compute new intersection weights
      //search for the element with the same column index in the reference row
      for (int i=csrPtr[ref]+threadIdx.x+blockIdx.x*blockDim.x; i<csrPtr[ref+1]; i+=gridDim.x*blockDim.x) {
        int match  =-1;
        int ref_col = csrInd[i];
        T ref_val = weighted ? v[ref_col] : (T)1.0;

        //binary search (column indices are sorted within each row)
        int left = csrPtr[cur];
        int right= csrPtr[cur+1]-1;
        while(left <= right){
          int middle = (left+right)>>1;
          int cur_col= csrInd[middle];
          if (cur_col > ref_col) {
            right=middle-1;
          }
          else if (cur_col < ref_col) {
            left=middle+1;
          }
          else {
            match = middle;
            break;
          }
        }

        //if the element with the same column index in the reference row has been found
        if (match != -1){
          atomicAdd(&weight_i[j],ref_val);
        }
      }
    }
  }
}

// Reference https://github.com/SPEAR-UIC/CodeGreen/tree/main/lassi_solutions
template<bool weighted, typename T>
__global__ void
jaccard_is_opt(const int n, const int e,
               const int *__restrict__ csrPtr,
               const int *__restrict__ csrInd,
               const T *__restrict__ v,
               const T *__restrict__ work,
                     T *__restrict__ weight_i,
                     T *__restrict__ weight_s)
{
  for (int row=threadIdx.z+blockIdx.z*blockDim.z; row<n; row+=gridDim.z*blockDim.z) {
    for (int j=csrPtr[row]+threadIdx.y+blockIdx.y*blockDim.y;
             j<csrPtr[row+1]; j+=gridDim.y*blockDim.y) {
      int col = csrInd[j];
      //find which row has least elements (and call it reference row)
      int Ni = csrPtr[row+1] - csrPtr[row];
      int Nj = csrPtr[col+1] - csrPtr[col];
      int ref= (Ni < Nj) ? row : col;
      int cur= (Ni < Nj) ? col : row;

      //compute new sum weights
      weight_s[j] = work[row] + work[col];

      //compute new intersection weights
      //search for the element with the same column index in the reference row
      if (threadIdx.x == 0) {
        T local_sum = 0;
        int i_ptr = csrPtr[ref];      // pointer in reference row
        int j_ptr = csrPtr[cur];        // pointer in current row
        int ref_end = csrPtr[ref+1];
        int cur_end = csrPtr[cur+1];

        // Two-pointer merge for intersection of the two sorted lists
        while (i_ptr < ref_end && j_ptr < cur_end) {
          int ref_col = csrInd[i_ptr];
          int cur_col = csrInd[j_ptr];
          if (ref_col == cur_col) {
            T ref_val = weighted ? v[ref_col] : (T)1.0;
            local_sum += ref_val;
            i_ptr++;
            j_ptr++;
          } else if (ref_col < cur_col) {
            i_ptr++;
          } else {
            j_ptr++;
          }
        }
        // perform a single atomic update per this j index
        if (local_sum != 0)
          atomicAdd(&weight_i[j], local_sum);
      }
    }
  }
}

template<bool weighted, typename T>
__global__ void
jaccard_jw(const int e,
    const T *__restrict__ csrVal,
    const T gamma,
    const T *__restrict__ weight_i,
    const T *__restrict__ weight_s,
          T *__restrict__ weight_j)
{
  for (int j=threadIdx.x+blockIdx.x*blockDim.x; j<e; j+=gridDim.x*blockDim.x) {
    T Wi =  weight_i[j];
    T Ws =  weight_s[j];
    weight_j[j] = (gamma*csrVal[j])* (Wi/(Ws-Wi));
  }
}

template <bool weighted, typename T>
__global__ void
fill(const int e, T* w, const T value)
{
  for (int j=threadIdx.x+blockIdx.x*blockDim.x; j<e; j+=gridDim.x*blockDim.x) {
    // e.g. w[0] is the weight of a non-zeron element when csr_ind[i] equals 0.
    // So multiple non-zero elements on different rows of a matrix may share
    // the same weight value
    w[j] = weighted ? (T)(j+1)/e : value;
  }
}

template <bool weighted, typename T>
void jaccard_weight (const int iteration, const int n, const int e,
    int* csr_ptr, int* csr_ind, T* csr_val)
{
  const T gamma = (T)0.46;  // arbitrary

  T *d_weight_i,
    *d_weight_s,
    *d_weight_j,
    *d_work;
  int *d_csrInd;
  int *d_csrPtr;
  T *d_csrVal;

#ifdef DEBUG
  T* weight_i = (T*) malloc (sizeof(T) * e);
  T* weight_s = (T*) malloc (sizeof(T) * e);
  T* work = (T*) malloc (sizeof(T) * n);
#endif
  T* weight_j = (T*) malloc (sizeof(T) * e);

  hipMalloc ((void**)&d_work, sizeof(T) * n);
  hipMalloc ((void**)&d_weight_i, sizeof(T) * e);
  hipMalloc ((void**)&d_weight_s, sizeof(T) * e);
  hipMalloc ((void**)&d_weight_j, sizeof(T) * e);
  hipMalloc ((void**)&d_csrVal, sizeof(T) * e);
  hipMalloc ((void**)&d_csrPtr, sizeof(int) * (n+1));
  hipMalloc ((void**)&d_csrInd, sizeof(int) * e);

  hipMemcpy(d_csrPtr, csr_ptr, sizeof(int) * (n+1), hipMemcpyHostToDevice);
  hipMemcpy(d_csrInd, csr_ind, sizeof(int) * e, hipMemcpyHostToDevice);
  hipMemcpy(d_csrVal, csr_val, sizeof(T) * e, hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < iteration; i++) {
    dim3 nthreads, nblocks; // reuse for multiple kernels

    nthreads.x = MAX_KERNEL_THREADS;
    nthreads.y = 1;
    nthreads.z = 1;
    nblocks.x  = (e+MAX_KERNEL_THREADS-1) / MAX_KERNEL_THREADS;
    nblocks.y  = 1;
    nblocks.z  = 1;

    fill<weighted, T><<<nblocks, nthreads>>>(e, d_weight_j, (T)1.0);
#ifdef DEBUG
    cudaMemcpy(weight_j, d_weight_j, sizeof(T) * e, cudaMemcpyDeviceToHost);
    for (int i = 0; i < e; i++) printf("wj: %d %f\n", i, weight_j[i]);
#endif

    // initialize volume of intersections
    fill<false, T><<<nblocks, nthreads>>>(e, d_weight_i, (T)0.0);

    // compute row sum with prefix sum
    const int y = 4;
    nthreads.x = 64/y;
    nthreads.y = y;
    nthreads.z = 1;
    nblocks.x  = 1;
    nblocks.y  = (n + nthreads.y - 1) / nthreads.y;  // less than MAX CUDA BLOCKs
    nblocks.z  = 1;
    jaccard_row_sum<weighted,T><<<nblocks,nthreads>>>(n, d_csrPtr, d_csrInd, d_weight_j, d_work);

#ifdef DEBUG
    cudaMemcpy(work, d_work, sizeof(T) * n, cudaMemcpyDeviceToHost);
    for (int i = 0; i < n; i++) printf("work: %d %f\n", i, work[i]);
#endif

    // compute volume of intersections (*weight_i) and cumulated volume of neighboors (*weight_s)
    // nthreads.x * nthreads.y * nthreads.z <= 256
    nthreads.x = 32/y;
    nthreads.y = y;
    nthreads.z = 8;
    nblocks.x  = 1;
    nblocks.y  = 1;
    nblocks.z  = (n + nthreads.z - 1)/nthreads.z; // less than CUDA_MAX_BLOCKS);
    jaccard_is_opt<weighted,T><<<nblocks,nthreads>>>(n, e, d_csrPtr,
        d_csrInd, d_weight_j, d_work, d_weight_i, d_weight_s);

#ifdef DEBUG
    cudaMemcpy(weight_i, d_weight_i, sizeof(T) * e, cudaMemcpyDeviceToHost);
    cudaMemcpy(weight_s, d_weight_s, sizeof(T) * e, cudaMemcpyDeviceToHost);
    for (int i = 0; i < e; i++) printf("wi: %d %f\n", i, weight_i[i]);
    for (int i = 0; i < e; i++) printf("ws: %d %f\n", i, weight_s[i]);
#endif

    // compute jaccard weights
    nthreads.x = std::min(e, MAX_KERNEL_THREADS);
    nthreads.y = 1;
    nthreads.z = 1;
    nblocks.x  = (e + nthreads.x - 1)/nthreads.x;  // less than MAX CUDA BLOCKs
    nblocks.y  = 1;
    nblocks.z  = 1;
    jaccard_jw<weighted,T><<<nblocks,nthreads>>>(e,
        d_csrVal, gamma, d_weight_i, d_weight_s, d_weight_j);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  cout << "Average execution time of kernels: " << (time * 1e-9f) / iteration << " (s)\n";

  hipMemcpy(weight_j, d_weight_j, sizeof(T) * e, hipMemcpyDeviceToHost);
#ifdef DEBUG
  // verify using known values when weighted is true
  float error;

  if (weighted)
    error = std::fabs(weight_j[0] - 0.306667) +
            std::fabs(weight_j[1] - 0.000000) +
            std::fabs(weight_j[2] - 3.680000) +
            std::fabs(weight_j[3] - 1.380000) +
            std::fabs(weight_j[4] - 0.788571) +
            std::fabs(weight_j[5] - 0.460000);

  else
    error = std::fabs(weight_j[0] - 0.230000) +
            std::fabs(weight_j[1] - 0.000000) +
            std::fabs(weight_j[2] - 3.680000) +
            std::fabs(weight_j[3] - 1.380000) +
            std::fabs(weight_j[4] - 0.920000) +
            std::fabs(weight_j[5] - 0.460000);

  if (error > 1e-5) {
    for (int i = 0; i < e; i++) printf("wj: %d %f\n", i, weight_j[i]);
    printf("FAIL");
  } else {
    printf("PASS");
  }
  printf("\n");
#endif

  hipFree (d_work);
  hipFree (d_weight_i);
  hipFree (d_weight_s);
  hipFree (d_weight_j);
  hipFree (d_csrInd);
  hipFree (d_csrVal);
  hipFree (d_csrPtr);
  free(weight_j);
#ifdef DEBUG
  free(weight_i);
  free(weight_s);
  free(work);
#endif
}

// Utilities
void printMatrix(const matrix& M)
{
  int m = M.size();
  int n = M[0].size();
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++)
      cout << M[i][j] << " ";
    cout << endl;
  }
}

template <typename T>
void printVector(const vector<T>& V, char* msg)
{
  cout << msg << "[ ";
  for_each(V.begin(), V.end(), [](int a) { cout << a << " "; });
  cout << "]" << endl;
}

// Reference: https://www.geeksforgeeks.org/sparse-matrix-representations-set-3-csr/
int main(int argc, char** argv)
{
  int iteration = 10;

#ifdef DEBUG
  matrix M  = {
    { 0, 0, 0, 1},
    { 5, 8, 0, 0},
    { 0, 0, 3, 0},
    { 0, 6, 0, 1}
  };
#else

  int numRow = atoi(argv[1]);
  int numCol = atoi(argv[2]);
  iteration = atoi(argv[3]);

  srand(2);

  matrix M;
  vector<vtype> rowElems(numCol);
  for (int r = 0; r < numRow; r++) {
    for (int c = 0; c < numCol; c++)
      rowElems[c] = rand() % 10;
    M.push_back(rowElems);
  }
#endif

  int row = M.size();
  int col = M[0].size();
  printf("Number of matrix rows and cols: %d %d\n", row, col);
  vector<vtype> csr_val;
  vector<int> csr_ptr = { 0 }; // require -std=c++11
  vector<int> csr_ind;
  int nnz = 0; // count Number of non-zero elements in each row

  for (int i = 0; i < row; i++) {
    for (int j = 0; j < col; j++) {
      if (M[i][j] != (vtype)0) {
        csr_val.push_back(M[i][j]);
        csr_ind.push_back(j);
        nnz++;
      }
    }
    csr_ptr.push_back(nnz);
  }

  // print when the matrix is small
  if (row <= 16 && col <= 16) {
    printMatrix(M);
    printVector(csr_val, (char*)"values = ");
    printVector(csr_ptr, (char*)"row pointer = ");
    printVector(csr_ind, (char*)"col indices = ");
  }

  jaccard_weight<true, vtype>(iteration, row, nnz, csr_ptr.data(), csr_ind.data(), csr_val.data());
  jaccard_weight<false, vtype>(iteration, row, nnz, csr_ptr.data(), csr_ind.data(), csr_val.data());

  return 0;
}

