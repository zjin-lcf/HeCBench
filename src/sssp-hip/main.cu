#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016 University of Cordoba and University of Illinois
 * All rights reserved.
 *
 * Developed by:    IMPACT Research Group
 *                  University of Cordoba and University of Illinois
 *                  http://impact.crhc.illinois.edu/
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * with the Software without restriction, including without limitation the
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 *      > Redistributions of source code must retain the above copyright notice,
 *        this list of conditions and the following disclaimers.
 *      > Redistributions in binary form must reproduce the above copyright
 *        notice, this list of conditions and the following disclaimers in the
 *        documentation and/or other materials provided with the distribution.
 *      > Neither the names of IMPACT Research Group, University of Cordoba,
 *        University of Illinois nor the names of its contributors may be used
 *        to endorse or promote products derived from this Software without
 *        specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS WITH
 * THE SOFTWARE.
 *
 */

#include <unistd.h>
#include <thread>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
#include "support/common.h"
#include "support/timer.h"
#include "support/verify.h"


// Params
struct Params {

  int         device;
  int         n_gpu_threads;
  int         n_gpu_blocks;
  int         n_threads;
  int         n_warmup;
  int         n_reps;
  const char *file_name;
  const char *comparison_file;
  int         switching_limit;

  Params(int argc, char **argv) {
    device          = 0;
    n_gpu_threads   = 256;
    n_gpu_blocks    = 8;
    n_threads       = 2;
    n_warmup        = 1;
    n_reps          = 1;
    file_name       = "input/NYR_input.dat";
    comparison_file = "output/NYR_bfs.out";
    switching_limit = 128;
    int opt;
    while((opt = getopt(argc, argv, "hd:i:g:t:w:r:f:c:l:")) >= 0) {
      switch(opt) {
        case 'h':
          usage();
          exit(0);
          break;
        case 'd': device          = atoi(optarg); break;
        case 'i': n_gpu_threads   = atoi(optarg); break;
        case 'g': n_gpu_blocks    = atoi(optarg); break;
        case 't': n_threads       = atoi(optarg); break;
        case 'w': n_warmup        = atoi(optarg); break;
        case 'r': n_reps          = atoi(optarg); break;
        case 'f': file_name       = optarg; break;
        case 'c': comparison_file = optarg; break;
        case 'l': switching_limit = atoi(optarg); break;
        default:
            fprintf(stderr, "\nUnrecognized option!\n");
            usage();
            exit(0);
      }
    }
    assert(n_gpu_threads > 0 && "Invalid # of device threads!");
    assert(n_gpu_blocks > 0 && "Invalid # of device blocks!");
    assert(n_threads > 0 && "Invalid # of host threads!");
  }

  void usage() {
    fprintf(stderr,
        "\nUsage:  ./sssp [options]"
        "\n"
        "\nGeneral options:"
        "\n    -h        help"
        "\n    -d <D>    GPU device ID (default=0)"
        "\n    -i <I>    # of device threads per block (default=256)"
        "\n    -g <G>    # of device blocks (default=8)"
        "\n    -t <T>    # of host threads (default=2)"
        "\n    -w <W>    # of untimed warmup iterations (default=1)"
        "\n    -r <R>    # of timed repetition iterations (default=1)"
        "\n"
        "\nBenchmark-specific options:"
        "\n    -f <F>    name of input file with control points (default=input/NYR_input.dat)"
        "\n    -c <C>    comparison file (default=output/NYR_bfs_BFS.out)"
        "\n    -l <L>    switching limit (default=128)"
        "\n");
  }
};

// Input Data
int read_input_size(int &n_nodes, int &n_edges, const Params &p) {
  FILE *fp = fopen(p.file_name, "r");
  if (fp == NULL) {
    printf("Error: failed to read file %s. Exit\n", p.file_name);
    return -1;
  }
    
  fscanf(fp, "%d", &n_nodes);
  fscanf(fp, "%d", &n_edges);
  if(fp) fclose(fp);
  return 0;
}

void read_input(int &source, Node *&h_nodes, Edge *&h_edges, const Params &p) {

  int   start, edgeno;
  int   n_nodes, n_edges;
  int   id, cost;
  FILE *fp = fopen(p.file_name, "r");

  fscanf(fp, "%d", &n_nodes);
  fscanf(fp, "%d", &n_edges);
  fscanf(fp, "%d", &source);
  printf("Number of nodes = %d\t", n_nodes);
  printf("Number of edges = %d\t", n_edges);

  // initalize the memory: Nodes
  for(int i = 0; i < n_nodes; i++) {
    fscanf(fp, "%d %d", &start, &edgeno);
    h_nodes[i].x = start;
    h_nodes[i].y = edgeno;
  }
#if PRINT_ALL
  for(int i = 0; i < n_nodes; i++) {
    printf("%d, %d\n", h_nodes[i].x, h_nodes[i].y);
  }
#endif

  // initalize the memory: Edges
  for(int i = 0; i < n_edges; i++) {
    fscanf(fp, "%d", &id);
    fscanf(fp, "%d", &cost);
    h_edges[i].x = id;
    h_edges[i].y = -cost;
  }
  if(fp)
    fclose(fp);
}

// GPU kernel
__global__ void SSSP_gpu(
    const Node *__restrict__ graph_nodes_av,
    const Edge *__restrict__ graph_edges_av,
    int *__restrict__ cost,
    int *__restrict__ color,
    const int *__restrict__ q1,
          int *__restrict__ q2,
    const int *__restrict__ n_t,
    int *__restrict__ head,
    int *__restrict__ tail,
    int *__restrict__ overflow,
    const int *__restrict__ gray_shade,
    int *__restrict__ iter)
{
  __shared__ int l_mem[W_QUEUE_SIZE+2];
  __shared__ int tail_bin;
  int* l_q2 = l_mem;
  int* shift = l_mem + W_QUEUE_SIZE;
  int* base = l_mem + W_QUEUE_SIZE + 1;

  const int tid     = threadIdx.x;
  const int gtid    = blockIdx.x * blockDim.x + threadIdx.x;
  const int WG_SIZE = blockDim.x;

  int n_t_local = *n_t; // atomicAdd(n_t, 0);
  int gray_shade_local = *gray_shade; // atomicAdd(&gray_shade[0], 0);

  if(tid == 0) {
    // Reset queue
    tail_bin = 0;
  }

  // Fetch frontier elements from the queue
  if(tid == 0)
    *base = atomicAdd(&head[0], WG_SIZE);
  __syncthreads();

  int my_base = *base;
  while(my_base < n_t_local) {

    // If local queue might overflow
    if(tail_bin >= W_QUEUE_SIZE / 2) {
      if(tid == 0) {
        // Add local tail_bin to tail
        *shift = atomicAdd(&tail[0], tail_bin);
      }
      __syncthreads();
      int local_shift = tid;
      while(local_shift < tail_bin) {
        q2[*shift + local_shift] = l_q2[local_shift];
        // Multiple threads are copying elements at the same time, so we shift by multiple elements for next iteration
        local_shift += WG_SIZE;
      }
      __syncthreads();
      if(tid == 0) {
        // Reset local queue
        tail_bin = 0;
      }
      __syncthreads();
    }

    if(my_base + tid < n_t_local && *overflow == 0) {
      // Visit a node from the current frontier
      int pid = q1[my_base + tid];
      //////////////// Visit node ///////////////////////////
      atomicExch(&color[pid], BLACK); // Node visited
      int  cur_cost = cost[pid]; // atomicAdd(&cost[pid], 0); // Look up shortest-path distance to this node
      Node cur_node;
      cur_node.x = graph_nodes_av[pid].x;
      cur_node.y = graph_nodes_av[pid].y;
      Edge cur_edge;
      // For each outgoing edge
      for(int i = cur_node.x; i < cur_node.y + cur_node.x; i++) {
        cur_edge.x = graph_edges_av[i].x;
        cur_edge.y = graph_edges_av[i].y;
        int id     = cur_edge.x;
        int cost_local   = cur_edge.y;
        cost_local += cur_cost;
        int orig_cost = atomicMax(&cost[id], cost_local);
        if(orig_cost < cost_local) {
          int old_color = atomicMax(&color[id], gray_shade_local);
          if(old_color != gray_shade_local) {
            // Push to the queue
            int tail_index = atomicAdd(&tail_bin, 1);
            if(tail_index >= W_QUEUE_SIZE) {
              *overflow = 1;
            } else
              l_q2[tail_index] = id;
          }
        }
      }
    }

    if(tid == 0)
      *base = atomicAdd(&head[0], WG_SIZE); // Fetch more frontier elements from the queue
    __syncthreads();
    my_base = *base;
  }
  /////////////////////////////////////////////////////////
  // Compute size of the output and allocate space in the global queue
  if(tid == 0) {
    *shift = atomicAdd(&tail[0], tail_bin);
  }
  __syncthreads();
  ///////////////////// CONCATENATE INTO GLOBAL MEMORY /////////////////////
  int local_shift = tid;
  while(local_shift < tail_bin) {
    q2[*shift + local_shift] = l_q2[local_shift];
    // Multiple threads are copying elements at the same time, so we shift by multiple elements for next iteration
    local_shift += WG_SIZE;
  }
  //////////////////////////////////////////////////////////////////////////

  if(gtid == 0) {
    atomicAdd(&iter[0], 1);
  }
}

// Main
int main(int argc, char **argv) {

  const Params p(argc, argv);
  Timer        timer;

  // Allocate
  int n_nodes, n_edges;
  int status = read_input_size(n_nodes, n_edges, p);
  if (status == -1) return 1;

  timer.start("Host/Device Allocation");
  Node * h_nodes = (Node *)malloc(sizeof(Node) * n_nodes);
  Node * d_nodes;
  hipMalloc((void**)&d_nodes, sizeof(Node) * n_nodes);
  Edge * h_edges = (Edge *)malloc(sizeof(Edge) * n_edges);
  Edge * d_edges;
  hipMalloc((void**)&d_edges, sizeof(Edge) * n_edges);
  std::atomic_int *h_color = (std::atomic_int *)malloc(sizeof(std::atomic_int) * n_nodes);
  int * d_color;
  hipMalloc((void**)&d_color, sizeof(int) * n_nodes);
  std::atomic_int *h_cost  = (std::atomic_int *)malloc(sizeof(std::atomic_int) * n_nodes);
  int * d_cost;
  hipMalloc((void**)&d_cost, sizeof(int) * n_nodes);
  int *            h_q1    = (int *)malloc(n_nodes * sizeof(int));
  int * d_q1;
  hipMalloc((void**)&d_q1, sizeof(int) * n_nodes);
  int *            h_q2    = (int *)malloc(n_nodes * sizeof(int));
  int * d_q2;
  hipMalloc((void**)&d_q2, sizeof(int) * n_nodes);
  std::atomic_int  h_head[1];
  int * d_head;
  hipMalloc((void**)&d_head, sizeof(int));
  std::atomic_int  h_tail[1];
  int * d_tail;
  hipMalloc((void**)&d_tail, sizeof(int));
  std::atomic_int  h_threads_end[1];
  int * d_threads_end;
  hipMalloc((void**)&d_threads_end, sizeof(int));
  std::atomic_int  h_threads_run[1];
  int * d_threads_run;
  hipMalloc((void**)&d_threads_run, sizeof(int));
  int              h_num_t[1];
  int * d_num_t;
  hipMalloc((void**)&d_num_t, sizeof(int));
  int              h_overflow[1];
  int * d_overflow;
  hipMalloc((void**)&d_overflow, sizeof(int));
  std::atomic_int  h_gray_shade[1];
  int * d_gray_shade;
  hipMalloc((void**)&d_gray_shade, sizeof(int));
  std::atomic_int  h_iter[1];
  int * d_iter;
  hipMalloc((void**)&d_iter, sizeof(int));
  hipDeviceSynchronize();
  //CUDA_ERR();
  //ALLOC_ERR(h_nodes, h_edges, h_color, h_cost, h_q1, h_q2);
  timer.stop("Host/Device Allocation");

  // Initialize
  timer.start("Initialization");
  const int max_gpu_threads = 256;
  int source;
  read_input(source, h_nodes, h_edges, p);
  for(int i = 0; i < n_nodes; i++) {
    h_cost[i].store(INF);
  }
  h_cost[source].store(0);
  for(int i = 0; i < n_nodes; i++) {
    h_color[i].store(WHITE);
  }
  h_tail[0].store(0);
  h_head[0].store(0);
  h_threads_end[0].store(0);
  h_threads_run[0].store(0);
  h_q1[0] = source;
  h_iter[0].store(0);
  h_overflow[0] = 0;
  h_gray_shade[0].store(GRAY0);
  timer.stop("Initialization");
  timer.print("Initialization", 1);

  // Copy to device
  timer.start("Copy To Device");
  hipMemcpy(d_nodes, h_nodes, sizeof(Node) * n_nodes, hipMemcpyHostToDevice);
  hipMemcpy(d_edges, h_edges, sizeof(Edge) * n_edges, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  //CUDA_ERR();
  timer.stop("Copy To Device");

  for(int rep = 0; rep < p.n_reps + p.n_warmup; rep++) {

    // Reset
    for(int i = 0; i < n_nodes; i++) {
      h_cost[i].store(INF);
    }
    h_cost[source].store(0);
    for(int i = 0; i < n_nodes; i++) {
      h_color[i].store(WHITE);
    }
    h_tail[0].store(0);
    h_head[0].store(0);
    h_threads_end[0].store(0);
    h_threads_run[0].store(0);
    h_q1[0] = source;
    h_iter[0].store(0);
    h_overflow[0] = 0;
    h_gray_shade[0].store(GRAY0);

    if(rep >= p.n_warmup)
      timer.start("Kernel");

    // Run first iteration in master CPU thread
    h_num_t[0] = 1;
    int pid;
    int index_i, index_o;
    for(index_i = 0; index_i < h_num_t[0]; index_i++) {
      pid = h_q1[index_i];
      h_color[pid].store(BLACK);
      int cur_cost = h_cost[pid].load();
      for(int i = h_nodes[pid].x; i < (h_nodes[pid].y + h_nodes[pid].x); i++) {
        int id   = h_edges[i].x;
        int cost = h_edges[i].y;
        cost += cur_cost;
        h_cost[id].store(cost);
        h_color[id].store(GRAY0);
        index_o       = h_tail[0].fetch_add(1);
        h_q2[index_o] = id;
      }
    }
    h_num_t[0] = h_tail[0].load();
    h_tail[0].store(0);
    h_threads_run[0].fetch_add(1);
    h_gray_shade[0].store(GRAY1);
    h_iter[0].fetch_add(1);
    if(rep >= p.n_warmup)
      timer.stop("Kernel");

    // Pointers to input and output queues
    int * h_qin  = h_q2;
    int * h_qout = h_q1;
    int * d_qin; //  = d_q2;
    int * d_qout; // = d_q1;

    const int CPU_EXEC = (p.n_threads > 0) ? 1 : 0;
    const int GPU_EXEC = (p.n_gpu_blocks > 0 && p.n_gpu_threads > 0) ? 1 : 0;

    // Run subsequent iterations on CPU or GPU until number of input queue elements is 0
    while(*h_num_t != 0) {

      if((*h_num_t < p.switching_limit || GPU_EXEC == 0) &&
          CPU_EXEC == 1) { // If the number of input queue elements is lower than switching_limit

        if(rep >= p.n_warmup)
          timer.start("Kernel on Host");

        // Continue until switching_limit condition is not satisfied
        while((*h_num_t != 0) && (*h_num_t < p.switching_limit || GPU_EXEC == 0) && CPU_EXEC == 1) {

          // Swap queues
          if(h_iter[0] % 2 == 0) {
            h_qin  = h_q1;
            h_qout = h_q2;
          } else {
            h_qin  = h_q2;
            h_qout = h_q1;
          }

          std::thread main_thread(run_cpu_threads, h_nodes, h_edges, h_cost, h_color, h_qin, h_qout, h_num_t,
              h_head, h_tail, h_threads_end, h_threads_run, h_gray_shade, h_iter, p.n_threads,
              p.switching_limit, GPU_EXEC);
          main_thread.join();

          h_num_t[0] = h_tail[0].load(); // Number of elements in output queue
          h_tail[0].store(0);
          h_head[0].store(0);
          if(h_iter[0].load() % 2 == 0)
            h_gray_shade[0].store(GRAY0);
          else
            h_gray_shade[0].store(GRAY1);
        }

        if(rep >= p.n_warmup)
          timer.stop("Kernel on Host");

      } else if((*h_num_t >= p.switching_limit || CPU_EXEC == 0) &&
          GPU_EXEC ==
          1) { // If the number of input queue elements is higher than or equal to switching_limit

        if(rep >= p.n_warmup)
          timer.start("Copy To Device");
        hipMemcpy(
            d_cost, h_cost, sizeof(int) * n_nodes, hipMemcpyHostToDevice);
        hipMemcpy(
            d_color, h_color, sizeof(int) * n_nodes, hipMemcpyHostToDevice);
        hipMemcpy(
            d_threads_run, h_threads_run, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(
            d_threads_end, h_threads_end, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(
            d_overflow, h_overflow, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(
            d_q1, h_q1, sizeof(int) * n_nodes, hipMemcpyHostToDevice);
        hipMemcpy(
            d_q2, h_q2, sizeof(int) * n_nodes, hipMemcpyHostToDevice);
        hipMemcpy(
            d_iter, h_iter, sizeof(int), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        //CUDA_ERR();
        if(rep >= p.n_warmup)
          timer.stop("Copy To Device");

        // Continue until switching_limit condition is not satisfied
        while((*h_num_t != 0) && (*h_num_t >= p.switching_limit || CPU_EXEC == 0) && GPU_EXEC == 1) {
          //printf("h_iter %d\n", h_iter[0].load());

          // Swap queues
          if(h_iter[0] % 2 == 0) {
            d_qin  = d_q1;
            d_qout = d_q2;
          } else {
            d_qin  = d_q2;
            d_qout = d_q1;
          }

          if(rep >= p.n_warmup)
            timer.start("Copy To Device");
          hipMemcpy(
              d_num_t, h_num_t, sizeof(int), hipMemcpyHostToDevice);
          hipMemcpy(
              d_tail, h_tail, sizeof(int), hipMemcpyHostToDevice);
          hipMemcpy(
              d_head, h_head, sizeof(int), hipMemcpyHostToDevice);
          hipMemcpy(
              d_gray_shade, h_gray_shade, sizeof(int), hipMemcpyHostToDevice);
          hipDeviceSynchronize();
          //CUDA_ERR();
          if(rep >= p.n_warmup)
            timer.stop("Copy To Device");

          assert(p.n_gpu_threads <= max_gpu_threads && 
              "The thread block size is greater than the maximum thread block size that can be used on this device");

          dim3 dimGrid(p.n_gpu_blocks);
          dim3 dimBlock(p.n_gpu_threads);

          if(rep >= p.n_warmup)
            timer.start("Kernel on Device");

          hipLaunchKernelGGL(SSSP_gpu, dimGrid, dimBlock, 0, 0, d_nodes, d_edges, d_cost,
              d_color, d_qin, d_qout, d_num_t,
              d_head, d_tail, d_overflow, d_gray_shade, d_iter);

          hipDeviceSynchronize();

          //CUDA_ERR();
          if(rep >= p.n_warmup)
            timer.stop("Kernel on Device");

          if(rep >= p.n_warmup)
            timer.start("Copy Back and Merge");
          hipMemcpy(
              h_tail, d_tail, sizeof(int), hipMemcpyDeviceToHost);
          hipMemcpy(
              h_iter, d_iter, sizeof(int), hipMemcpyDeviceToHost);
          hipDeviceSynchronize();
          //CUDA_ERR();
          if(rep >= p.n_warmup)
            timer.stop("Copy Back and Merge");

          h_num_t[0] = h_tail[0].load(); // Number of elements in output queue
          h_tail[0].store(0);
          h_head[0].store(0);
          if(h_iter[0].load() % 2 == 0)
            h_gray_shade[0].store(GRAY0);
          else
            h_gray_shade[0].store(GRAY1);

        }

        if(rep >= p.n_warmup)
          timer.start("Copy Back and Merge");
        hipMemcpy(
            h_cost, d_cost, sizeof(int) * n_nodes, hipMemcpyDeviceToHost);
        hipMemcpy(
            h_color, d_color, sizeof(int) * n_nodes, hipMemcpyDeviceToHost);
        hipMemcpy(
            h_threads_run, d_threads_run, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(
            h_threads_end, d_threads_end, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(
            h_overflow, d_overflow, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(
            h_q1, d_q1, sizeof(int) * n_nodes, hipMemcpyDeviceToHost);
        hipMemcpy(
            h_q2, d_q2, sizeof(int) * n_nodes, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        //CUDA_ERR();
        if(rep >= p.n_warmup)
          timer.stop("Copy Back and Merge");
      }
    }
  } // end of iteration
  timer.print("Host/Device Allocation", 1);
  timer.print("Copy To Device", p.n_reps);
  timer.print("Kernel on Host", p.n_reps);
  timer.print("Kernel on Device", p.n_reps);
  timer.print("Copy Back and Merge", p.n_reps);

  // Verify answer
  bool ok = verify(h_cost, n_nodes, p.comparison_file);

  // Free memory
  timer.start("Host/Device Deallocation");
  free(h_nodes);
  free(h_edges);
  free(h_color);
  free(h_cost);
  free(h_q1);
  free(h_q2);
  hipFree(d_nodes);
  hipFree(d_edges);
  hipFree(d_cost);
  hipFree(d_color);
  hipFree(d_q1);
  hipFree(d_q2);
  hipFree(d_num_t);
  hipFree(d_head);
  hipFree(d_tail);
  hipFree(d_threads_end);
  hipFree(d_threads_run);
  hipFree(d_overflow);
  hipFree(d_iter);
  hipFree(d_gray_shade);
  //CUDA_ERR();
  timer.stop("Host/Device Deallocation");
  timer.print("Host/Device Deallocation", 1);

  printf("%s\n", ok ? "PASS" : "FAIL");
  return ok ? 0 : 1;
}
