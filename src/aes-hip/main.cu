#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <chrono>
#include <hip/hip_runtime.h>

#include "SDKBitMap.h"
#include "aes.h"
#include "kernels.cu"
#include "reference.cu"
#include "utils.cu"

int main(int argc, char * argv[])
{
  if (argc != 4) {
    printf("Usage: %s <iterations> <0 or 1> <path to bitmap image file>\n", argv[0]);
    printf("0=encrypt, 1=decrypt\n");
    return 1;
  }

  const unsigned int keySizeBits = 128;
  const unsigned int rounds = 10;
  const unsigned int seed = 123;

  const int iterations = atoi(argv[1]);
  const bool decrypt = atoi(argv[2]);
  const char* filePath = argv[3];

  SDKBitMap image;
  image.load(filePath);
  const int width  = image.getWidth();
  const int height = image.getHeight();

  /* check condition for the bitmap to be initialized */
  if (width <= 0 || height <= 0) return 1;

  std::cout << "Image width and height: " 
            << width << " " << height << std::endl;

  uchar4 *pixels = image.getPixels();

  unsigned int sizeBytes = width*height*sizeof(uchar);
  uchar *input = (uchar*)malloc(sizeBytes); 

  /* initialize the input array, do NOTHING but assignment when decrypt*/
  if (decrypt)
    convertGrayToGray(pixels, input, height, width);
  else
    convertColorToGray(pixels, input, height, width);

  unsigned int keySize = keySizeBits/8; // 1 Byte = 8 bits

  unsigned int keySizeBytes = keySize*sizeof(uchar);

  uchar *key = (uchar*)malloc(keySizeBytes);

  fillRandom<uchar>(key, keySize, 1, 0, 255, seed); 

  // expand the key
  unsigned int explandedKeySize = (rounds+1)*keySize;

  unsigned int explandedKeySizeBytes = explandedKeySize*sizeof(uchar);

  uchar *expandedKey = (uchar*)malloc(explandedKeySizeBytes);
  uchar *roundKey    = (uchar*)malloc(explandedKeySizeBytes);

  keyExpansion(key, expandedKey, keySize, explandedKeySize);
  for(unsigned int i = 0; i < rounds+1; ++i)
  {
    createRoundKey(expandedKey + keySize*i, roundKey + keySize*i);
  }

  // save device result
  uchar* output = (uchar*)malloc(sizeBytes);

  uchar *inputBuffer;
  hipMalloc((void**)&inputBuffer, sizeBytes);
  hipMemcpy(inputBuffer, input, width * height, hipMemcpyHostToDevice);

  uchar *outputBuffer;
  hipMalloc((void**)&outputBuffer, sizeBytes);

  uchar *rKeyBuffer;
  hipMalloc((void**)&rKeyBuffer, explandedKeySizeBytes);
  hipMemcpy(rKeyBuffer, roundKey, explandedKeySizeBytes, hipMemcpyHostToDevice);

  uchar *sBoxBuffer;
  hipMalloc((void**)&sBoxBuffer, 256);
  hipMemcpy(sBoxBuffer, sbox, 256, hipMemcpyHostToDevice);

  uchar *rsBoxBuffer;
  hipMalloc((void**)&rsBoxBuffer, 256);
  hipMemcpy(rsBoxBuffer, rsbox, 256, hipMemcpyHostToDevice);

  std::cout << "Executing kernel for " << iterations 
            << " iterations" << std::endl;
  std::cout << "-------------------------------------------" << std::endl;

  dim3 grid (width/4, height/4);
  dim3 block (1, 4);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for(int i = 0; i < iterations; i++)
  {
    if (decrypt) 
      hipLaunchKernelGGL(AESDecrypt, dim3(grid), dim3(block ), 0, 0, 
        (uchar4*)outputBuffer,
        (uchar4*)inputBuffer,
        (uchar4*)rKeyBuffer,
        rsBoxBuffer,
        width, rounds);
    else
      hipLaunchKernelGGL(AESEncrypt, dim3(grid), dim3(block ), 0, 0, 
        (uchar4*)outputBuffer,
        (uchar4*)inputBuffer,
        (uchar4*)rKeyBuffer,
        sBoxBuffer,
        width, rounds);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "Average kernel execution time " << (time * 1e-9f) / iterations << " (s)\n";

  hipMemcpy(output, outputBuffer, width * height, hipMemcpyDeviceToHost);

  // Verify
  uchar *verificationOutput = (uchar *) malloc(sizeBytes);

  reference(verificationOutput, input, roundKey, explandedKeySize, 
      width, height, decrypt, rounds, keySize);

  /* compare the results and see if they match */
  if(memcmp(output, verificationOutput, sizeBytes) == 0)
    std::cout<<"Pass\n";
  else {
    std::cout<<"Fail\n";
    exit(1);
  }

  /* release program resources (input memory etc.) */
  hipFree(inputBuffer);
  hipFree(outputBuffer);
  hipFree(rKeyBuffer);
  hipFree(sBoxBuffer);
  hipFree(rsBoxBuffer);

  if(input) free(input);

  if(key) free(key);

  if(expandedKey) free(expandedKey);

  if(roundKey) free(roundKey);

  if(output) free(output);

  if(verificationOutput) free(verificationOutput);

  return 0;
}
