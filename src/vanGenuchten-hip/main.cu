#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "reference.h"

__global__ 
void vanGenuchten(
  const double *__restrict__ Ksat,
  const double *__restrict__ psi,
        double *__restrict__ C,
        double *__restrict__ theta,
        double *__restrict__ K,
  const int size)
{
  double Se, _theta, _psi, lambda, m, t;

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < size)
  {
    lambda = n - 1.0;
    m = lambda/n;

    // Compute the volumetric moisture content [eqn 21]
    _psi = psi[i] * 100.0;
    if ( _psi < 0.0 )
      _theta = (theta_S - theta_R) / pow(1.0 + pow((alpha*(-_psi)),n), m) + theta_R;
    else
      _theta = theta_S;

    theta[i] = _theta;

    // Compute the effective saturation [eqn 2]
    Se = (_theta - theta_R)/(theta_S - theta_R);

    // Compute the hydraulic conductivity [eqn 8]
    t = 1.0 - pow(1.0-pow(Se,1.0/m), m);
    K[i] = Ksat[i] * sqrt(Se) * t * t;

    // Compute the specific moisture storage derivative of eqn (21).
    // So we have to calculate C = d(theta)/dh. Then the unit is converted into [1/m].
    if (_psi < 0.0)
      C[i] = 100.0 * alpha * n * (1.0/n-1.0)*pow(alpha*abs(_psi), n-1.0)
        * (theta_R-theta_S) * pow(pow(alpha*abs(_psi), n)+1.0, 1.0/n-2.0);
    else
      C[i] = 0.0;
  }
}

int main(int argc, char* argv[])
{
  if (argc != 5) {
    printf("Usage: ./%s <dimX> <dimY> <dimZ> <repeat>\n", argv[0]);
    return 1;
  }

  const int dimX = atoi(argv[1]);
  const int dimY = atoi(argv[2]);
  const int dimZ = atoi(argv[3]);
  const int repeat = atoi(argv[4]);

  const int size = dimX * dimY * dimZ;
  const int size_byte = size * sizeof(double);

  double *Ksat, *psi, *C, *theta, *K;
  double *C_ref, *theta_ref, *K_ref;
  
  Ksat = new double[size];
  psi = new double[size];
  C = new double[size];
  theta = new double[size];
  K = new double[size];

  C_ref = new double[size];
  theta_ref = new double[size];
  K_ref = new double[size];

  // arbitrary numbers
  for (int i = 0; i < size; i++) {
    Ksat[i] = 1e-6 +  (1.0 - 1e-6) * i / size; 
    psi[i] = -100.0 + 101.0 * i / size;
  }

  // for verification
  reference(Ksat, psi, C_ref, theta_ref, K_ref, size);

  double *d_Ksat, *d_psi, *d_C, *d_theta, *d_K;
  hipMalloc((void**)&d_Ksat, size_byte); 
  hipMalloc((void**)&d_psi, size_byte); 
  hipMalloc((void**)&d_C, size_byte); 
  hipMalloc((void**)&d_theta, size_byte); 
  hipMalloc((void**)&d_K, size_byte); 

  hipMemcpy(d_Ksat, Ksat, size_byte, hipMemcpyHostToDevice);
  hipMemcpy(d_psi, psi, size_byte, hipMemcpyHostToDevice);

  dim3 grids ((size+255)/256);
  dim3 blocks (256);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++)
    hipLaunchKernelGGL(vanGenuchten, grids, blocks , 0, 0, d_Ksat, d_psi, d_C, d_theta, d_K, size);

  hipDeviceSynchronize();

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time: %f (s)\n", (time * 1e-9f) / repeat);

  hipMemcpy(C, d_C, size_byte, hipMemcpyDeviceToHost);
  hipMemcpy(theta, d_theta, size_byte, hipMemcpyDeviceToHost);
  hipMemcpy(K, d_K, size_byte, hipMemcpyDeviceToHost);

  bool ok = true;
  for (int i = 0; i < size; i++) {
    if (fabs(C[i] - C_ref[i]) > 1e-3 || 
        fabs(theta[i] - theta_ref[i]) > 1e-3 ||
        fabs(K[i] - K_ref[i]) > 1e-3) {
      ok = false;
      break;
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");

  hipFree(d_Ksat);
  hipFree(d_psi);
  hipFree(d_C);
  hipFree(d_theta);
  hipFree(d_K);

  delete(Ksat);
  delete(psi);
  delete(C);
  delete(theta);
  delete(K);
  delete(C_ref);
  delete(theta_ref);
  delete(K_ref);

  return ok ? 0 : 1;
}
