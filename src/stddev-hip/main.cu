#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "reference.h"

// final step for the deviation of a sample
template <typename Type, typename IdxType>
__global__ void sampleKernel (Type *std, IdxType D, IdxType N) {
  IdxType i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < D) std[i] = sqrtf(std[i] / N);
}

// sum of products using atomics
template <typename Type, typename IdxType, int TPB, int ColsPerBlk = 32>
__global__ void sopKernel(
        Type *__restrict__ std, 
  const Type *__restrict__ data, 
  IdxType D, 
  IdxType N) 
{
  const int RowsPerBlkPerIter = TPB / ColsPerBlk;
  IdxType thisColId = threadIdx.x % ColsPerBlk;
  IdxType thisRowId = threadIdx.x / ColsPerBlk;
  IdxType colId = thisColId + ((IdxType)blockIdx.y * ColsPerBlk);
  IdxType rowId = thisRowId + ((IdxType)blockIdx.x * RowsPerBlkPerIter);
  Type thread_data = Type(0);
  const IdxType stride = RowsPerBlkPerIter * gridDim.x;
  for (IdxType i = rowId; i < N; i += stride) {
    Type val = (colId < D) ? data[i * D + colId] : Type(0);
    thread_data += val * val;
  }
  __shared__ Type sstd[ColsPerBlk];
  if (threadIdx.x < ColsPerBlk) sstd[threadIdx.x] = Type(0);
  __syncthreads();

  atomicAdd(sstd + thisColId, thread_data);
  __syncthreads();

  if (threadIdx.x < ColsPerBlk) atomicAdd(std + colId, sstd[thisColId]);
}

/**
 * @brief Compute stddev of the input matrix
 *
 * Stddev operation is assumed to be performed on a given column.
 *
 * @tparam Type the data type
 * @tparam IdxType Integer type used to for addressing
 * @param std the output stddev vector
 * @param data the input matrix
 * @param D number of columns of data
 * @param N number of rows of data
 * @param sample whether to evaluate sample stddev or not. In other words,
 * whether
 *  to normalize the output using N-1 or N, for true or false, respectively
 */
template <typename Type, typename IdxType = int>
void stddev(Type *std, const Type *data, IdxType D, IdxType N, bool sample) {
  static const int TPB = 256;
  static const int RowsPerThread = 4;
  static const int ColsPerBlk = 32;
  static const int RowsPerBlk = (TPB / ColsPerBlk) * RowsPerThread;
  dim3 grid((N + (IdxType)RowsPerBlk - 1) / (IdxType)RowsPerBlk, 
            (D + (IdxType)ColsPerBlk - 1) / (IdxType)ColsPerBlk);
  dim3 block(TPB);

  hipMemset(std, 0, sizeof(Type) * D); // required for atomics

  hipLaunchKernelGGL(HIP_KERNEL_NAME(sopKernel<Type, IdxType, TPB, ColsPerBlk>), grid, block, 0, 0, std, data, D, N);

  IdxType sampleSize = sample ? N-1 : N;
  sampleKernel<Type, IdxType> <<<(D+TPB-1)/TPB, TPB>>>(std, D, sampleSize);
}

int main(int argc, char* argv[]) {
  if (argc != 4) {
    printf("Usage: %s <D> <N> <repeat>\n", argv[0]);
    printf("D: number of columns of data (must be a multiple of 32)\n");
    printf("N: number of rows of data (at least one row)\n");
    return 1;
  }
  int D = atoi(argv[1]); // columns must be a multiple of 32
  int N = atoi(argv[2]); // at least one row
  int repeat = atoi(argv[3]);

  bool sample = true;
  long inputSize = D * N;
  long inputSizeByte = inputSize * sizeof(float);
  float *data = (float*) malloc (inputSizeByte);

  // input data 
  srand(123);
  for (int i = 0; i < N; i++)
    for (int j = 0; j < D; j++) 
      data[i*D + j] = rand() / (float)RAND_MAX; 

  float *d_data;
  hipMalloc((void**)&d_data, inputSizeByte);
  hipMemcpy(d_data, data, inputSizeByte, hipMemcpyHostToDevice);

  // host and device results
  long outputSize = D;
  long outputSizeByte = outputSize * sizeof(float);
  float *std  = (float*) malloc (outputSizeByte);
  float *std_ref  = (float*) malloc (outputSizeByte);
  float *d_std;
  hipMalloc((void**)&d_std, outputSizeByte);

  // warmup
  stddev(d_std, d_data, D, N, sample);

  // execute kernels on a device
  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++)
    stddev(d_std, d_data, D, N, sample);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of stddev kernels: %f (s)\n", (time * 1e-9f) / repeat);

  hipMemcpy(std, d_std, outputSizeByte, hipMemcpyDeviceToHost);

  // verify
  stddev_ref(std_ref, data, D, N, sample);

  bool ok = true;
  for (int i = 0; i < D; i++) {
    if (fabsf(std_ref[i] - std[i]) > 1e-3) {
      ok = false;
      break;
    }
  }

  printf("%s\n", ok ? "PASS" : "FAIL");
  free(std_ref);
  free(std);
  free(data);
  hipFree(d_std);
  hipFree(d_data);
  return ok ? 0 : 1;
}
