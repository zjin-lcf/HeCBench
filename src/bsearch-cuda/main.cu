#include <cstdlib>
#include <chrono>
#include <iostream>
#include <hip/hip_runtime.h>

#ifndef Real_t 
#define Real_t float
#endif

template <typename T>
__global__ void
kernel_BS (const T* __restrict__ acc_a,
           const T* __restrict__ acc_z,
            size_t* __restrict__ acc_r,
           const size_t zSize,
           const size_t n)
{ 
  size_t i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= zSize) return;
  T z = acc_z[i];
  size_t low = 0;
  size_t high = n;
  while (high - low > 1) {
    size_t mid = low + (high - low)/2;
    if (z < acc_a[mid])
      high = mid;
    else
      low = mid;
  }
  acc_r[i] = low;
}

template <typename T>
__global__ void
kernel_BS2 (const T* __restrict__ acc_a,
            const T* __restrict__ acc_z,
             size_t* __restrict__ acc_r,
            const size_t zSize,
            const size_t n)
{
  size_t i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= zSize) return;
  unsigned  nbits = 0;
  while (n >> nbits) nbits++;
  size_t k = 1ULL << (nbits - 1);
  T z = acc_z[i];
  size_t idx = (acc_a[k] <= z) ? k : 0;
  while (k >>= 1) {
    size_t r = idx | k;
    if (r < n && z >= acc_a[r]) { 
      idx = r;
    }
  }
  acc_r[i] = idx;
}

template <typename T>
__global__ void
kernel_BS3 (const T* __restrict__ acc_a,
            const T* __restrict__ acc_z,
             size_t* __restrict__ acc_r,
           const size_t zSize,
            const size_t n)
{
  size_t i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= zSize) return;
  unsigned nbits = 0;
  while (n >> nbits) nbits++;
  size_t k = 1ULL << (nbits - 1);
  T z = acc_z[i];
  size_t idx = (acc_a[k] <= z) ? k : 0;
  while (k >>= 1) {
    size_t r = idx | k;
    size_t w = r < n ? r : n; 
    if (z >= acc_a[w]) { 
      idx = r;
    }
  }
  acc_r[i] = idx;
}

template <typename T>
__global__ void
kernel_BS4 (const T* __restrict__ acc_a,
            const T* __restrict__ acc_z,
             size_t* __restrict__ acc_r,
            const size_t zSize,
            const size_t n)
{
  __shared__  size_t k;

  size_t i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i >= zSize) return;
  size_t lid = threadIdx.x; 

  if (lid == 0) {
    unsigned nbits = 0;
    while (n >> nbits) nbits++;
    k = 1ULL << (nbits - 1);
  }
  __syncthreads();

  size_t p = k;
  T z = acc_z[i];
  size_t idx = (acc_a[p] <= z) ? p : 0;
  while (p >>= 1) {
    size_t r = idx | p;
    size_t w = r < n ? r : n;
    if (z >= acc_a[w]) { 
      idx = r;
    }
  }
  acc_r[i] = idx;
}

template <typename T>
void bs ( const size_t aSize,
    const size_t zSize,
    const T *d_a,  // N+1
    const T *d_z,  // T
    size_t *d_r,   // T
    const size_t n,
    const int repeat )
{
  dim3 grids ((zSize + 255) / 256);
  dim3 blocks (256);
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++)
    kernel_BS<<<grids, blocks>>>(d_a, d_z, d_r, zSize, n);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "Average kernel execution time (bs1) " << (time * 1e-9f) / repeat << " (s)\n";
}

template <typename T>
void bs2 ( const size_t aSize,
    const size_t zSize,
    const T *d_a,  // N+1
    const T *d_z,  // T
    size_t *d_r,   // T
    const size_t n,
    const int repeat )
{
  dim3 grids ((zSize + 255) / 256);
  dim3 blocks (256);
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++)
    kernel_BS2<<<grids, blocks>>>(d_a, d_z, d_r, zSize, n);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "Average kernel execution time (bs2) " << (time * 1e-9f) / repeat << " (s)\n";
}

template <typename T>
void bs3 ( const size_t aSize,
    const size_t zSize,
    const T *d_a,  // N+1
    const T *d_z,  // T
    size_t *d_r,   // T
    const size_t n,
    const int repeat )
{
  dim3 grids ((zSize + 255) / 256);
  dim3 blocks (256);
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++)
    kernel_BS3<<<grids, blocks>>>(d_a, d_z, d_r, zSize, n);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "Average kernel execution time (bs3) " << (time * 1e-9f) / repeat << " (s)\n";
}

template <typename T>
void bs4 ( const size_t aSize,
    const size_t zSize,
    const T *d_a,  // N+1
    const T *d_z,  // T
    size_t *d_r,   // T
    const size_t n,
    const int repeat )
{
  dim3 grids ((zSize + 255) / 256);
  dim3 blocks (256);
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++)
    kernel_BS4<<<grids, blocks>>>(d_a, d_z, d_r, zSize, n);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "Average kernel execution time (bs4) " << (time * 1e-9f) / repeat << " (s)\n";
}

#ifdef DEBUG
void verify(Real_t *a, Real_t *z, size_t *r, size_t aSize, size_t zSize, std::string msg)
{
  for (size_t i = 0; i < zSize; ++i)
  {
    // check result
    if (!(r[i]+1 < aSize && a[r[i]] <= z[i] && z[i] < a[r[i] + 1]))
    {
      std::cout << msg << ": incorrect result:" << std::endl;
      std::cout << "index = " << i << " r[index] = " << r[i] << std::endl;
      std::cout << a[r[i]] << " <= " << z[i] << " < " << a[r[i] + 1] << std::endl;
      break;
    }
    // clear result
    r[i] = 0xFFFFFFFF;
  }
}
#endif

int main(int argc, char* argv[])
{
  if (argc != 3) {
    std::cout << "Usage ./main <number of elements> <repeat>\n";
    return 1;
  }

  size_t numElem = atol(argv[1]);
  uint repeat = atoi(argv[2]);

  srand(2);
  size_t aSize = numElem;
  size_t zSize = 2*aSize;
  Real_t *a = NULL;
  Real_t *z = NULL;
  size_t *r = NULL;
  posix_memalign((void**)&a, 1024, aSize * sizeof(Real_t));
  posix_memalign((void**)&z, 1024, zSize * sizeof(Real_t));
  posix_memalign((void**)&r, 1024, zSize * sizeof(size_t));

  size_t N = aSize-1;

  // strictly ascending
  for (size_t i = 0; i < aSize; i++) a[i] = i;

  // lower = 0, upper = n-1
  for (size_t i = 0; i < zSize; i++) z[i] = rand() % N;

  Real_t* d_a;
  Real_t* d_z;
  size_t *d_r;
  hipMalloc((void**)&d_a, sizeof(Real_t)*aSize);
  hipMalloc((void**)&d_z, sizeof(Real_t)*zSize);
  hipMalloc((void**)&d_r, sizeof(size_t)*zSize);
  hipMemcpy(d_a, a, sizeof(Real_t)*aSize, hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, sizeof(Real_t)*zSize, hipMemcpyHostToDevice);

  bs(aSize, zSize, d_a, d_z, d_r, N, repeat);

#ifdef DEBUG
  cudaMemcpy(r, d_r, sizeof(size_t)*zSize, cudaMemcpyDeviceToHost);
  verify(a, z, r, aSize, zSize, "bs");
#endif

  bs2(aSize, zSize, d_a, d_z, d_r, N, repeat);

#ifdef DEBUG
  cudaMemcpy(r, d_r, sizeof(size_t)*zSize, cudaMemcpyDeviceToHost);
  verify(a, z, r, aSize, zSize, "bs2");
#endif

  bs3(aSize, zSize, d_a, d_z, d_r, N, repeat);

#ifdef DEBUG
  cudaMemcpy(r, d_r, sizeof(size_t)*zSize, cudaMemcpyDeviceToHost);
  verify(a, z, r, aSize, zSize, "bs3");
#endif

  bs4(aSize, zSize, d_a, d_z, d_r, N, repeat);

#ifdef DEBUG
  cudaMemcpy(r, d_r, sizeof(size_t)*zSize, cudaMemcpyDeviceToHost);
  verify(a, z, r, aSize, zSize, "bs4");
#endif

  hipFree(d_a);
  hipFree(d_z);
  hipFree(d_r);
  free(a);
  free(z);
  free(r);
  return 0;
}
