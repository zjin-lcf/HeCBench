#include "hip/hip_runtime.h"
/*  Copyright (c) 2021-2022 Intel Corporation

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "reference.h"

__inline__ __device__
float exponent (float x) { return __expf(x); }

__inline__ __device__
double exponent (double x) { return exp(x); }

template <typename scalar_t, typename gscalar_t>
__global__ void loss_bwd (
    const scalar_t* __restrict__ log_softmax,
    const gscalar_t* __restrict__ grad_output,
    const gscalar_t* __restrict__ grad_output_neg,
    const int64_t* __restrict__ target,
    const scalar_t* __restrict__ weight,
    const int64_t* __restrict__ mask,
          gscalar_t* __restrict__ grad_predict)
{
  int local_id_x = threadIdx.x;
  int group_id_bs = blockIdx.y;
  int group_id_x = blockIdx.x;

  int linear_x_id = group_id_x * threadX + local_id_x;

  if (linear_x_id >= H) return;

  int offset2d = group_id_bs * H + linear_x_id;
  int idx = target[offset2d];
  int sum_offset = group_id_bs * W * H + idx * H + linear_x_id;

  gscalar_t tmp_grad;
  if (mask[offset2d])
    tmp_grad = -(grad_output[offset2d] + grad_output_neg[offset2d]);
  else
    tmp_grad = -grad_output[offset2d];

  tmp_grad = tmp_grad * weight[offset2d];

  float sum_value = tmp_grad * log_softmax[sum_offset];

  #pragma unroll
  for (int i = 0; i < W; ++i) {
    int in_offset = group_id_bs * W * H + i * H + linear_x_id;
    float tmp_sfm = exponent(log_softmax[in_offset]) * sum_value;
    float res = 0.f;
    if (i == idx) {
      res = (float)tmp_grad - tmp_sfm;
    }
    else {
      res = -tmp_sfm;
    }
    grad_predict[in_offset] = res;
  }
}

template <>
__global__ void loss_bwd<__half, __half> (
    const __half* __restrict__ log_softmax,
    const __half* __restrict__ grad_output,
    const __half* __restrict__ grad_output_neg,
    const int64_t* __restrict__ target,
    const __half* __restrict__ weight,
    const int64_t* __restrict__ mask,
          __half* __restrict__ grad_predict)
{
  int local_id_x = threadIdx.x;
  int group_id_bs = blockIdx.y;
  int group_id_x = blockIdx.x;

  int linear_x_id = group_id_x * threadX + local_id_x;

  if (linear_x_id >= H) return;

  int offset2d = group_id_bs * H + linear_x_id;
  int idx = target[offset2d];
  int sum_offset = group_id_bs * W * H + idx * H + linear_x_id;

  __half tmp_grad;
  if (mask[offset2d])
    tmp_grad = __hneg(__hadd(grad_output[offset2d] , grad_output_neg[offset2d]));
  else
    tmp_grad = __hneg(grad_output[offset2d]);

  tmp_grad = __hmul(tmp_grad , weight[offset2d]);

  float sum_value = h2f(__hmul(tmp_grad , log_softmax[sum_offset]));

  #pragma unroll
  for (int i = 0; i < W; ++i) {
    int in_offset = group_id_bs * W * H + i * H + linear_x_id;
    float tmp_sfm = h2f(hexp(log_softmax[in_offset])) * sum_value;
    float res = 0.f;
    if (i == idx) {
      res = h2f(tmp_grad) - tmp_sfm;
    }
    else {
      res = -tmp_sfm;
    }
    grad_predict[in_offset] = res;
  }
}

template <typename scalar_t, typename gscalar_t>
float loss_bwd_kernel(
  scalar_t* log_softmax,
  gscalar_t* grad_output,
  gscalar_t* grad_output_neg,
  int64_t* target, scalar_t* weight,
  int64_t* mask, gscalar_t* grad_predict)
{
  dim3 blocks ( threadX, threadBS );
  dim3 grids ( (H + threadX - 1) / threadX, bs );

  auto start = std::chrono::high_resolution_clock::now();

  hipLaunchKernelGGL(loss_bwd, grids, blocks, 0, 0, log_softmax, grad_output, grad_output_neg,
                              target, weight, mask, grad_predict);
  hipDeviceSynchronize();

  auto end = std::chrono::high_resolution_clock::now();
  float time = std::chrono::duration<float, std::milli>(end - start).count(); // ms
  return time;
}

// compute cross entropy in the backward phase
template <typename scalar_t, typename gscalar_t>
void LossNLL_BWD(int iterations) {

  vector<double> durations(3, 0.0); // timing

  const size_t grad_predict_size = sizeof(gscalar_t) * PredictShape;
  const size_t log_softmax_size = sizeof(scalar_t) * PredictShape;
  const size_t weight_size = sizeof(scalar_t) * OutputShape;
  const size_t target_size = sizeof(int64_t) * TargetShape;
  const size_t mask_size = sizeof(int64_t) * TargetShape;
  const size_t grad_output_size = sizeof(gscalar_t) * OutputShape;

  gscalar_t* grad_predict;
  hipHostMalloc((void**)&grad_predict, grad_predict_size);

  gscalar_t* grad_predict_device_host;
  hipHostMalloc((void**)&grad_predict_device_host, grad_predict_size);

  scalar_t* weight;
  hipHostMalloc((void**)&weight, weight_size);

  int64_t* target;
  hipHostMalloc((void**)&target, target_size);
  
  int64_t* mask;
  hipHostMalloc((void**)&mask, mask_size);

  scalar_t* log_softmax;
  hipHostMalloc((void**)&log_softmax, log_softmax_size);

  gscalar_t* grad_output;
  hipHostMalloc((void**)&grad_output, grad_output_size);

  gscalar_t* grad_output_neg;
  hipHostMalloc((void**)&grad_output_neg, grad_output_size);

  for (int i = 0; i < PredictShape; ++i)
    log_softmax[i] = static_cast<scalar_t>(random_float());
  for (int i = 0; i < OutputShape; ++i)
    grad_output[i] = static_cast<gscalar_t>(random_float());
  for (int i = 0; i < OutputShape; ++i)
    grad_output_neg[i] = static_cast<gscalar_t>(random_float());
  for (int i = 0; i < OutputShape; ++i)
    weight[i] = static_cast<scalar_t>(random_float());
  for (int i = 0; i < TargetShape; ++i)
    target[i] = static_cast<int64_t>(random_int(0, W - 1));
  for (int i = 0; i < TargetShape; ++i)
    mask[i] = static_cast<int64_t>(random_int(0, 1));

  // malloc device memory
  auto start = std::chrono::high_resolution_clock::now();

  gscalar_t* grad_predict_device;
  hipMalloc((void**)&grad_predict_device, grad_predict_size);

  scalar_t* weight_device;
  hipMalloc((void**)&weight_device, weight_size);

  int64_t* target_device;
  hipMalloc((void**)&target_device, target_size);
  
  int64_t* mask_device;
  hipMalloc((void**)&mask_device, mask_size);

  scalar_t* log_softmax_device;
  hipMalloc((void**)&log_softmax_device, log_softmax_size);

  gscalar_t* grad_output_device;
  hipMalloc((void**)&grad_output_device, grad_output_size);

  gscalar_t* grad_output_neg_device;
  hipMalloc((void**)&grad_output_neg_device, grad_output_size);

  hipMemcpy(grad_predict_device, grad_predict, grad_predict_size, hipMemcpyHostToDevice);
  hipMemcpy(target_device, target, target_size, hipMemcpyHostToDevice);
  hipMemcpy(mask_device, mask, mask_size, hipMemcpyHostToDevice);
  hipMemcpy(weight_device, weight, weight_size, hipMemcpyHostToDevice);
  hipMemcpy(grad_output_device, grad_output, grad_output_size, hipMemcpyHostToDevice);
  hipMemcpy(grad_output_neg_device, grad_output_neg, grad_output_size, hipMemcpyHostToDevice);
  hipMemcpy(log_softmax_device, log_softmax, log_softmax_size, hipMemcpyHostToDevice);

  auto end = std::chrono::high_resolution_clock::now();
  float time = std::chrono::duration<float, std::milli>(end - start).count(); // ms
  durations[0] += time;

  int warmup = 10;
  for (int k = 0; k < warmup + iterations; ++k) {

    float duration_gpu = loss_bwd_kernel<scalar_t, gscalar_t>(
      log_softmax_device, grad_output_device, grad_output_neg_device,
      target_device, weight_device, mask_device, grad_predict_device);

    if (k >= warmup) {
      durations[1] += duration_gpu;
    }
  }

  start = std::chrono::high_resolution_clock::now();
  hipMemcpy(grad_predict_device_host, grad_predict_device, grad_predict_size, hipMemcpyDeviceToHost);
  end = std::chrono::high_resolution_clock::now();
  time = std::chrono::duration<float, std::milli>(end - start).count(); // ms
  durations[0] += time;

  start = std::chrono::high_resolution_clock::now();

  loss_bwd_cpu<scalar_t, gscalar_t>(log_softmax, target, weight, mask, grad_output, grad_output_neg, grad_predict);

  end = std::chrono::high_resolution_clock::now();
  time = std::chrono::duration<float, std::milli>(end - start).count(); // ms
  durations[2] = time;

  verify<gscalar_t>(grad_predict, grad_predict_device_host, bs * W * H);

  std::cout << "GPU device memory allocation and data transfer time (ms) : "
            << (durations[0] / iterations) << std::endl;
  std::cout << "Average GPU kernel time (ms) : "
            << (durations[1] / iterations) << std::endl;
  std::cout << "CPU serial time (ms) : "
            << (durations[2]) << std::endl;

  double allBytes = static_cast<double>(sizeof(scalar_t)) * static_cast<double>(PredictShape * 2.0 + OutputShape * 3.0)
                  + static_cast<double>(sizeof(int64_t)) * static_cast<double>(TargetShape * 2.0);

  printf("BandWidth = %lf (GB / s)\n", allBytes / (durations[1] / iterations / 1000) / 1e9);

  hipFree(grad_predict_device);
  hipFree(weight_device);
  hipFree(target_device);
  hipFree(mask_device);
  hipFree(log_softmax_device);
  hipFree(grad_output_device);
  hipFree(grad_output_neg_device);

  hipHostFree(grad_predict);
  hipHostFree(grad_predict_device_host);
  hipHostFree(weight);
  hipHostFree(target);
  hipHostFree(mask);
  hipHostFree(log_softmax);
  hipHostFree(grad_output);
  hipHostFree(grad_output_neg);
}

int main(int argc, char** argv) {

  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  printf("Tensor size (BatchSize * Width * Height) = %d * %d * %d \n", bs, W, H);

  printf("=========== Data type is FP16 ==========\n");

  LossNLL_BWD<__half, __half>(repeat);
  LossNLL_BWD<__half, __half>(repeat);

  printf("%s\n", (errors == 0) ? "PASS" : "FAIL");
  if (errors)
    exit(1);

  printf("=========== Data type is FP32 ==========\n");

  LossNLL_BWD<float, float>(repeat);
  LossNLL_BWD<float, float>(repeat);

  printf("%s\n", (errors == 0) ? "PASS" : "FAIL");
  if (errors)
    exit(1);

  printf("=========== Data type is FP64 ==========\n");

  LossNLL_BWD<double, double>(repeat);
  LossNLL_BWD<double, double>(repeat);

  printf("%s\n", (errors == 0) ? "PASS" : "FAIL");
  if (errors)
    exit(1);

  return 0;
}
