/*
 * Copyright (c) <2017 - 2020>, ETH Zurich and Bilkent University
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * Redistributions of source code must retain the above copyright notice, this list
 * of conditions and the following disclaimer.
 * - Redistributions in binary form must reproduce the above copyright notice, this
 *   list of conditions and the following disclaimer in the documentation and/or other
 *   materials provided with the distribution.
 * - Neither the names of the ETH Zurich, Bilkent University,
 *   nor the names of its contributors may be
 *   used to endorse or promote products derived from this software without specific
 *   prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
 * LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stddef.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std::chrono;

#define warp_size 32
#define NBytes 8

__host__ __device__
inline uint lsr(uint x, int sa) {
  if(sa > 0 && sa < 32) return (x >> sa);
  return x;
}

__host__ __device__
inline uint lsl(uint x, int sa) {
  if (sa > 0 && sa < 32) return (x << sa);
  return x;
}

__host__ __device__
inline uint set_bit(uint &data, int y) {
  data |= lsl(1, y);
  return data;
}

#include "kernel.h"
#include "reference.h"

int main(int argc, const char * const argv[])
{
  if (argc != 5) {
    printf("Usage: ./%s [ReadLength] [ReadandRefFile] [#reads] [repeat]\n", argv[0]);
    exit(-1);
  }

  int ReadLength = atoi(argv[1]);//in my inputs, it is always 100. Just for the generality we keep it as a variable
  int NumReads = atoi(argv[3]); // Number of reads
  int repeat = atoi(argv[4]);
  int Size_of_int_in_Bit = 32; //in Bits 

  FILE * fp;
  char * line = NULL;
  size_t len = 0;
  ssize_t read;
  char *p;//when reading each char_basepair from the file, we read it into the p.

  int Number_of_warps_inside_each_block = 8; 
  int Concurrent_threads_In_Block = warp_size * Number_of_warps_inside_each_block;
  int Number_of_blocks_inside_each_kernel = (NumReads + Concurrent_threads_In_Block - 1) / 
                                            Concurrent_threads_In_Block;

  int F_ErrorThreshold =0;

  uint* ReadSeq = (uint *) calloc(NumReads * 8, sizeof(uint));
  uint* RefSeq = (uint *) calloc(NumReads * 8, sizeof(uint));
  int* DFinal_Results = (int *) calloc(NumReads, sizeof(int));
  int* HFinal_Results = (int *) calloc(NumReads, sizeof(int));

  int tokenIndex=1;
  fp = fopen(argv[2], "r");
  if (!fp){
    printf("The file %s does not exist or you do not have access permission\n", argv[2]);
    return 0;
  }
  for(int this_read = 0; this_read < NumReads; this_read++) {
    read = getline(&line, &len, fp);
    tokenIndex=1;
    for (p = strtok(line, "\t"); p != NULL; p = strtok(NULL, "\t"))
    {
      if (tokenIndex==1)
      {
        for (int j = 0; j < ReadLength; j++)
        {
          if(p[j] == 'A')
          {
            //do nothing (this is like storing 00)
          }
          else if (p[j] == 'C')
          {
            ReadSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)] = set_bit(ReadSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_int_in_Bit/2)) * 2 + 1));
          }
          else if (p[j] == 'G')
          {
            ReadSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)] = set_bit(ReadSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_int_in_Bit/2)) * 2));
          }
          else if (p[j] == 'T')
          {
            ReadSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)] = set_bit(ReadSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_int_in_Bit/2)) * 2));

            ReadSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)] = set_bit(ReadSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_int_in_Bit/2)) * 2 + 1));
          }
        }
      }
      else if(tokenIndex==2)
      {
        for (int j = 0; j < ReadLength; j++)
        {
          if(p[j] == 'A')
          {
            //do nothing (this is like storing 00)
          }
          else if (p[j] == 'C')
          {
            RefSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)] = set_bit(RefSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_int_in_Bit/2)) * 2 + 1));
          }
          else if (p[j] == 'G')
          {
            RefSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)] = set_bit(RefSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_int_in_Bit/2)) * 2));
          }
          else if (p[j] == 'T')
          {
            RefSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)] = set_bit(RefSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_int_in_Bit/2)) * 2));

            RefSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)] = set_bit(RefSeq[((j*2/Size_of_int_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_int_in_Bit/2)) * 2 + 1));
          }
        }
      }
      tokenIndex=tokenIndex+1;
    }
  }
  fclose(fp);

  uint* Dev_ReadSeq;
  uint* Dev_RefSeq;
  int* Dev_Results;
  hipMalloc((void**)&Dev_ReadSeq, sizeof(uint) * NumReads * 8);
  hipMalloc((void**)&Dev_RefSeq, sizeof(uint) * NumReads * 8);
  hipMalloc((void**)&Dev_Results, sizeof(int) * NumReads);

  dim3 grid (Number_of_blocks_inside_each_kernel);
  dim3 block (Concurrent_threads_In_Block);

  hipMemcpy(Dev_ReadSeq, ReadSeq, sizeof(int) * NumReads * 8, hipMemcpyHostToDevice);
  hipMemcpy(Dev_RefSeq, RefSeq, sizeof(int) * NumReads * 8, hipMemcpyHostToDevice);

  bool error = false;
  for (int loopPar = 0; loopPar <= 25; loopPar++) {

    F_ErrorThreshold = (loopPar*ReadLength)/100;

    auto t1 = high_resolution_clock::now();

    for (int n = 0; n < repeat; n++) {
      hipLaunchKernelGGL(sneaky_snake, grid, block, 0, 0, Dev_ReadSeq, Dev_RefSeq, Dev_Results, NumReads, F_ErrorThreshold);
    }

    hipDeviceSynchronize();
    auto t2 = high_resolution_clock::now();
    double elapsed_time = duration_cast<microseconds>(t2 - t1).count();
    
    hipMemcpy(DFinal_Results, Dev_Results, sizeof(int) * NumReads, hipMemcpyDeviceToHost);

    // verify
    sneaky_snake_ref(ReadSeq, RefSeq, HFinal_Results, NumReads, F_ErrorThreshold);
    error = memcmp(DFinal_Results, HFinal_Results, NumReads * sizeof(int));
    if (error) break;

    // stats
    int D_accepted = 0;
    for(int i = 0; i < NumReads; i++) if(DFinal_Results[i] == 1) D_accepted++;

    printf("Error threshold: %2d | Average kernel time (us): %5.4f | Accepted: %10d | Rejected: %10d\n", 
          F_ErrorThreshold, elapsed_time / repeat, D_accepted, NumReads - D_accepted);
  }
  printf("%s\n", error ? "FAIL" : "PASS");
  
  free(ReadSeq);
  free(RefSeq);
  free(DFinal_Results);
  free(HFinal_Results);
  hipFree(Dev_ReadSeq);
  hipFree(Dev_RefSeq);
  hipFree(Dev_Results);
  return error ? 1 : 0;
}
