#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "utils.h"
#include "kernels.cu"

int main(int argc, char *argv[])
{
  double t_start = 0;
  double dt = 0.02E-3;

  int num_timesteps = 1000000;
  int num_nodes = 1; 

  if (argc > 1) {
    num_timesteps = atoi(argv[1]);
    printf("num_timesteps set to %d\n", num_timesteps);

    num_nodes = atoi(argv[2]);
    printf("num_nodes set to %d\n", num_nodes);

    if(num_timesteps <= 0 || num_nodes <= 0)
      exit(EXIT_FAILURE);
  }

  unsigned int num_states = NUM_STATES;
  size_t total_num_states = num_nodes * num_states;
  size_t states_size = total_num_states * sizeof(double);
  double *states = (double*) malloc(states_size);
  init_state_values(states, num_nodes);

  double *states2 = (double*) malloc(states_size);
  memcpy(states2, states, states_size);

  unsigned int num_parameters = NUM_PARAMS;
  size_t parameters_size = num_nodes * num_parameters * sizeof(double);
  double *parameters = (double*) malloc(parameters_size);
  init_parameters_values(parameters, num_nodes);

  double t = t_start;

  struct timespec timestamp_start, timestamp_now;
  double time_elapsed;

  printf("Host: Rush Larsen (exp integrator on all gates)\n");
  for (int it = 0; it < num_timesteps; it++) {
    forward_rush_larsen(states, t, dt, parameters, num_nodes);
    t += dt;
  }

  printf("Device: Rush Larsen (exp integrator on all gates)\n");
  double* d_states;
  hipMalloc((void**)&d_states, states_size);
  hipMemcpy(d_states, states2, states_size, hipMemcpyHostToDevice);

  double* d_parameters;
  hipMalloc((void**)&d_parameters, parameters_size);
  hipMemcpy(d_parameters, parameters, parameters_size, hipMemcpyHostToDevice);

  // All nodes run the same kernel
  dim3 grid ((num_nodes + 255)/256);
  dim3 block (256);

  t = t_start;

  hipDeviceSynchronize();
  clock_gettime(CLOCK_MONOTONIC_RAW, &timestamp_start);

  for (int it = 0; it < num_timesteps; it++) {
    k_forward_rush_larsen<<<grid, block>>>(d_states, t, dt, d_parameters, num_nodes);  // run with a single node 
    t += dt;
  }

  hipDeviceSynchronize();
  clock_gettime(CLOCK_MONOTONIC_RAW, &timestamp_now);
  time_elapsed = timestamp_now.tv_sec - timestamp_start.tv_sec + 1E-9 * (timestamp_now.tv_nsec - timestamp_start.tv_nsec);
  printf("Device: computed %d time steps in %g s. Time steps per second: %g\n\n",
      num_timesteps, time_elapsed, num_timesteps/time_elapsed);

  hipMemcpy(states2, d_states, states_size, hipMemcpyDeviceToHost);

  double rmse = 0.0;
  for (size_t i = 0; i < total_num_states; i++) {
    rmse += (states2[i] - states[i]) * (states2[i] - states[i]);
#ifdef VERBOSE
    printf("state[%d] = %lf\n", i, states[i]);
#endif
  }
  printf("RMSE = %lf\n", sqrt(rmse / (total_num_states)));
 
  free(states);
  free(states2);
  free(parameters);
  hipFree(d_states);
  hipFree(d_parameters);

  return 0;
}
