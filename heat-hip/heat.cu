#include "hip/hip_runtime.h"
/*
** PROGRAM: heat equation solve
**
** PURPOSE: This program will explore use of an explicit
**          finite difference method to solve the heat
**          equation under a method of manufactured solution (MMS)
**          scheme. The solution has been set to be a simple 
**          function based on exponentials and trig functions.
**
**          A finite difference scheme is used on a 1000x1000 cube.
**          A total of 0.5 units of time are simulated.
**
**          The MMS solution has been adapted from
**          G.W. Recktenwald (2011). Finite difference approximations
**          to the Heat Equation. Portland State University.
**
**
** USAGE:   Run with two arguments:
**          First is the number of cells.
**          Second is the number of timesteps.
**
**          For example, with 100x100 cells and 10 steps:
**
**          ./heat 100 10
**
**
** HISTORY: Written by Tom Deakin, Oct 2018
**          Ported to SYCL by Tom Deakin, Nov 2019
**          Ported to OpenCL by Tom Deakin, Jan 2020
**
*/

#include <iostream>
#include <chrono>
#include <cmath>
#include <fstream>

#include <hip/hip_runtime.h>

// Key constants used in this program
#define PI acos(-1.0) // Pi
#define LINE "--------------------" // A line for fancy output

__global__ void initial_value(const unsigned int n, const double dx, const double length, double * u);
__global__ void zero(const unsigned int n, double * u);
__global__ void solve(const unsigned int n, const double alpha, const double dx, const double dt, const double r, const double r2,
		double * __restrict__ u, double * __restrict__ u_tmp);
double solution(const double t, const double x, const double y, const double alpha, const double length);
double l2norm(const int n, const double * u, const int nsteps, const double dt, const double alpha, const double dx, const double length);

int main(int argc, char *argv[]) {

  // Start the total program runtime timer
  auto start = std::chrono::high_resolution_clock::now();

  // Problem size, forms an nxn grid
  int n = 1000;

  // Number of timesteps
  int nsteps = 10;

  // Check for the correct number of arguments
  // Print usage and exits if not correct
  if (argc == 3) {

    // Set problem size from first argument
    n = atoi(argv[1]);
    if (n < 0) {
      std::cerr << "Error: n must be positive" << std::endl;
      exit(EXIT_FAILURE);
    }

    // Set number of timesteps from second argument
    nsteps = atoi(argv[2]);
    if (nsteps < 0) {
      std::cerr << "Error: nsteps must be positive" << std::endl;
      exit(EXIT_FAILURE);
    }
  }

  //
  // Set problem definition
  //
  double alpha = 0.1;          // heat equation coefficient
  double length = 1000.0;      // physical size of domain: length x length square
  double dx = length / (n+1);  // physical size of each cell (+1 as don't simulate boundaries as they are given)
  double dt = 0.5 / nsteps;    // time interval (total time of 0.5s)

  // Stability requires that dt/(dx^2) <= 0.5,
  double r = alpha * dt / (dx * dx);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  char *device_name = prop.name;

  // Print message detailing runtime configuration
  std::cout
    << std::endl
    << " MMS heat equation" << std::endl << std::endl
    << LINE << std::endl
    << "Problem input" << std::endl << std::endl
    << " Grid size: " << n << " x " << n << std::endl
    << " Cell width: " << dx << std::endl
    << " Grid length: " << length << "x" << length << std::endl
    << std::endl
    << " Alpha: " << alpha << std::endl
    << std::endl
    << " Steps: " <<  nsteps << std::endl
    << " Total time: " << dt*(double)nsteps << std::endl
    << " Time step: " << dt << std::endl
    << " GPU device: " << device_name << std::endl
    << LINE << std::endl;

  // Stability check
  std::cout << "Stability" << std::endl << std::endl;
  std::cout << " r value: " << r << std::endl;
  if (r > 0.5)
    std::cout << " Warning: unstable" << std::endl;
  std::cout << LINE << std::endl;

  // Allocate two nxn grids
  double *u;
  double *u_tmp;
  hipMalloc((void**)&u,     sizeof(double)*n*n);
  hipMalloc((void**)&u_tmp, sizeof(double)*n*n);

  // Set the initial value of the grid under the MMS scheme
  const int block_size = 256;
  int n_ceil = (n*n+block_size-1) / block_size;
  dim3 grid(n_ceil);
  dim3 block(block_size);
  hipLaunchKernelGGL(initial_value, grid, block, 0, 0, n, dx, length, u);
  hipLaunchKernelGGL(zero, grid, block, 0, 0, n, u_tmp);

  // Ensure everything is initalised on the device
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    std::cerr << "CUDA error after initalisation" << std::endl;
    exit(EXIT_FAILURE);
  }

  //
  // Run through timesteps under the explicit scheme
  //
  // Finite difference constant multiplier
  const double r2 = 1.0 - 4.0*r;

  // Start the solve timer
  auto tic = std::chrono::high_resolution_clock::now();

  for (int t = 0; t < nsteps; ++t) {

    // Call the solve kernel
    // Computes u_tmp at the next timestep
    // given the value of u at the current timestep
    hipLaunchKernelGGL(solve, grid, block, 0, 0, n, alpha, dx, dt, r, r2, u, u_tmp);

    // Pointer swap
    auto tmp = u;
    u = u_tmp;
    u_tmp = tmp;
  }

  // Stop solve timer
  hipDeviceSynchronize();
  auto toc = std::chrono::high_resolution_clock::now();

  // Get access to u on the host
  double *u_host = new double[n*n];
  err = hipMemcpy(u_host, u, sizeof(double)*n*n, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    std::cerr << "CUDA error on copying back data" << std::endl;
    exit(EXIT_FAILURE);
  }

  //
  // Check the L2-norm of the computed solution
  // against the *known* solution from the MMS scheme
  //
  double norm = l2norm(n, u_host, nsteps, dt, alpha, dx, length);

  // Stop total timer
  auto stop = std::chrono::high_resolution_clock::now();

  // Print results
  std::cout
    << "Results" << std::endl << std::endl
    << "Error (L2norm): " << norm << std::endl
    << "Solve time (s): " << std::chrono::duration_cast<std::chrono::duration<double>>(toc-tic).count() << std::endl
    << "Total time (s): " << std::chrono::duration_cast<std::chrono::duration<double>>(stop-start).count() << std::endl
    << "Bandwidth (GB/s): " << 1.0E-9*2.0*n*n*nsteps*sizeof(double)/std::chrono::duration_cast<std::chrono::duration<double>>(toc-tic).count() << std::endl
    << LINE << std::endl;

  delete[] u_host;
  hipFree(u);
  hipFree(u_tmp);
}

// Sets the mesh to an initial value, determined by the MMS scheme
__global__ void initial_value(const unsigned int n, const double dx, const double length, double * u) {

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < n*n) {
    int i = idx % n;
    int j = idx / n;
    double y = dx * (j+1); // Physical y position
    double x = dx * (i+1); // Physical x position
    u[i+j*n] = sin(PI * x / length) * sin(PI * y / length);
  }
}


// Zero the array u
__global__ void zero(const unsigned int n, double * u) {

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < n*n) u[idx] = 0.0;
}


// Compute the next timestep, given the current timestep
// Loop over the nxn grid
__global__ void solve(const unsigned int n, const double alpha, const double dx, const double dt, 
		const double r, const double r2,
		double * __restrict__ u, double * __restrict__ u_tmp) {

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < n * n) {
    int i = idx % n;
    int j = idx / n;
    // Boundaries are zero because the MMS solution is zero there.
    u_tmp[i+j*n] =  r2 * u[i+j*n] +
    r * ((i < n-1) ? u[i+1+j*n] : 0.0) +
    r * ((i > 0)   ? u[i-1+j*n] : 0.0) +
    r * ((j < n-1) ? u[i+(j+1)*n] : 0.0) +
    r * ((j > 0)   ? u[i+(j-1)*n] : 0.0);
  }
}


// True answer given by the manufactured solution
double solution(const double t, const double x, const double y, const double alpha, const double length) {

  return exp(-2.0*alpha*PI*PI*t/(length*length)) * sin(PI*x/length) * sin(PI*y/length);

}


// Computes the L2-norm of the computed grid and the MMS known solution
// The known solution is the same as the boundary function.
double l2norm(const int n, const double * u, const int nsteps, const double dt, const double alpha, const double dx, const double length) {

  // Final (real) time simulated
  double time = dt * (double)nsteps;

  // L2-norm error
  double l2norm = 0.0;

  // Loop over the grid and compute difference of computed and known solutions as an L2-norm
  double y = dx;
  for (int j = 0; j < n; ++j) {
    double x = dx;
    for (int i = 0; i < n; ++i) {
      double answer = solution(time, x, y, alpha, length);
      l2norm += (u[i+j*n] - answer) * (u[i+j*n] - answer);

      x += dx;
    }
    y += dx;
  }

  return sqrt(l2norm);
}
