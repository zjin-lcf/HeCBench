#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>

#define CHUNK_S 4096

typedef struct {
  float x, y, z;
} kdata;

__constant__ kdata k[CHUNK_S];

__global__
void cmpfhd(const float*__restrict__ rmu, 
            const float*__restrict__ imu,
                  float*__restrict__ rfhd,
                  float*__restrict__ ifhd,
            const float*__restrict__ x, 
            const float*__restrict__ y,
            const float*__restrict__ z,
            const int samples,
            const int voxels) 
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;

  if (n < samples) {
    float xn = x[n], yn = y[n], zn = z[n];
    float rfhdn = rfhd[n], ifhdn = ifhd[n];
    for (int m = 0; m < voxels; m++) {
      float e = 2.f * (float)M_PI * (k[m].x * xn + k[m].y * yn + k[m].z * zn);
      float c = __cosf(e);
      float s = __sinf(e);
      rfhdn += rmu[m] * c - imu[m] * s;
      ifhdn += imu[m] * c + rmu[m] * s;
    }
    rfhd[n] = rfhdn, ifhd[n] = ifhdn;
  }
}

int main(int argc, char* argv[]) {
  if (argc != 4) {
    printf("Usage: %s <#samples> <#voxels> <verify>\n", argv[0]);
    exit(1);
  }
  const int samples = atoi(argv[1]); // in the order of 100000
  const int voxels = atoi(argv[2]);  // cube(128)/2097152
  const int verify = atoi(argv[3]);
  const int sampleSize = samples * sizeof(float);
  const int voxelSize = voxels * sizeof(float);

  float *h_rmu = (float*) malloc (voxelSize);
  float *h_imu = (float*) malloc (voxelSize);
  float *h_kx = (float*) malloc (voxelSize);
  float *h_ky = (float*) malloc (voxelSize);
  float *h_kz = (float*) malloc (voxelSize);
  kdata *h_k = (kdata*) malloc (voxels * sizeof(kdata));

  float *h_rfhd = (float*) malloc (sampleSize);
  float *h_ifhd = (float*) malloc (sampleSize);
  float *h_x = (float*) malloc (sampleSize);
  float *h_y = (float*) malloc (sampleSize);
  float *h_z = (float*) malloc (sampleSize);

  // For device results
  float *rfhd = (float*) malloc (sampleSize);
  float *ifhd = (float*) malloc (sampleSize);

  srand(2);
  for (int i = 0; i < samples; i++) {
    h_rfhd[i] = (float)i/samples;
    h_ifhd[i] = (float)i/samples;
    h_x[i] = 0.3f + (rand()%2 ? 0.1 : -0.1);
    h_y[i] = 0.2f + (rand()%2 ? 0.1 : -0.1);
    h_z[i] = 0.1f + (rand()%2 ? 0.1 : -0.1);
  }

  for (int i = 0; i < voxels; i++) {
    h_rmu[i] = (float)i/voxels;
    h_imu[i] = (float)i/voxels;
    h_k[i].x = h_kx[i] = 0.1f + (rand()%2 ? 0.1 : -0.1);
    h_k[i].y = h_ky[i] = 0.2f + (rand()%2 ? 0.1 : -0.1);
    h_k[i].z = h_kz[i] = 0.3f + (rand()%2 ? 0.1 : -0.1);
  }

  printf("Run FHd on a device\n");
  float *d_rmu, *d_imu;
  float *d_rfhd, *d_ifhd;
  float *d_x, *d_y, *d_z;

  hipMalloc((void**)&d_rmu, voxelSize);
  hipMemcpy(d_rmu, h_rmu, voxelSize, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_imu, voxelSize);
  hipMemcpy(d_imu, h_imu, voxelSize, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_rfhd, sampleSize);
  hipMemcpy(d_rfhd, h_rfhd, sampleSize, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_ifhd, sampleSize);
  hipMemcpy(d_ifhd, h_ifhd, sampleSize, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_x, sampleSize);
  hipMemcpy(d_x, h_x, sampleSize, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_y, sampleSize);
  hipMemcpy(d_y, h_y, sampleSize, hipMemcpyHostToDevice);

  hipMalloc((void**)&d_z, sampleSize);
  hipMemcpy(d_z, h_z, sampleSize, hipMemcpyHostToDevice);
  
  const int ntpb = 256;
  const int nblks = (samples + ntpb - 1) / ntpb;
  dim3 grid (nblks);
  dim3 block (ntpb);

  int c = CHUNK_S;
  int s = sizeof(kdata) * c;
  int nchunks = (voxels + c - 1) / c;

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < nchunks; i++) {
    if (i == nchunks - 1) {
      c = voxels - CHUNK_S * i;
      s = sizeof(kdata) * c;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(k), &h_k[i * CHUNK_S], s);

    cmpfhd<<<grid, block>>>(d_rmu + i*CHUNK_S,
                            d_imu + i*CHUNK_S, 
                            d_rfhd, d_ifhd, 
                            d_x, d_y, d_z, 
                            samples, c);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Device execution time %f (s)\n", time * 1e-9f);

  hipMemcpy(rfhd, d_rfhd, sampleSize, hipMemcpyDeviceToHost);
  hipMemcpy(ifhd, d_ifhd, sampleSize, hipMemcpyDeviceToHost);

  if (verify) {
    printf("Computing root mean square error between host and device results.\n");
    printf("This will take a while..\n");

    #pragma omp parallel for 
    for (int n = 0; n < samples; n++) {
      float r = h_rfhd[n];
      float i = h_ifhd[n];
      #pragma omp parallel for simd reduction(+:r,i)
      for (int m = 0; m < voxels; m++) {
        float e = 2.f * (float)M_PI * 
                  (h_kx[m] * h_x[n] + h_ky[m] * h_y[n] + h_kz[m] * h_z[n]);
        float c = cosf(e);
        float s = sinf(e);
        r += h_rmu[m] * c - h_imu[m] * s;
        i += h_imu[m] * c + h_rmu[m] * s;
      }
      h_rfhd[n] = r;
      h_ifhd[n] = i;   
    }

    float err = 0.f;
    for (int i = 0; i < samples; i++) {
      err += (h_rfhd[i] - rfhd[i]) * (h_rfhd[i] - rfhd[i]) +
             (h_ifhd[i] - ifhd[i]) * (h_ifhd[i] - ifhd[i]) ;
    }
    printf("RMSE = %f\n", sqrtf(err / (2*samples)));
  }
 
  hipFree(d_rmu);
  hipFree(d_imu);
  hipFree(d_rfhd);
  hipFree(d_ifhd);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  free(h_rmu);
  free(h_imu);
  free(h_kx);
  free(h_ky);
  free(h_kz);
  free(h_k);
  free(h_rfhd);
  free(h_ifhd);
  free(rfhd);
  free(ifhd);
  free(h_x);
  free(h_y);
  free(h_z);

  return 0;
}
