#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "./../main.h"                // (in main directory)            needed to recognized input parameters
#include "./../util/avi/avilib.h"          // (in directory)              needed by avi functions
#include "./../util/avi/avimod.h"          // (in directory)              needed by avi functions

// CUDA kernel
#include "kernel.h"

  void 
kernel_gpu_wrapper(  params_common common,
    int* endoRow,
    int* endoCol,
    int* tEndoRowLoc,
    int* tEndoColLoc,
    int* epiRow,
    int* epiCol,
    int* tEpiRowLoc,
    int* tEpiColLoc,
    avi_t* frames)
{

  // common
  //printf("tSize is %d, sSize is %d\n", common.tSize, common.sSize);
  common.in_rows = common.tSize + 1 + common.tSize;
  common.in_cols = common.in_rows;
  common.in_elem = common.in_rows * common.in_cols;
  common.in_mem = sizeof(fp) * common.in_elem;

  //==================================================50
  // endo points templates
  //==================================================50

  fp* d_endoT;
  hipMalloc((void**)&d_endoT, common.in_mem * common.endoPoints);
  //printf("%d\n", common.in_elem * common.endoPoints);

  //==================================================50
  // epi points templates
  //==================================================50

  fp* d_epiT;
  hipMalloc((void**)&d_epiT, common.in_mem * common.epiPoints);

  //====================================================================================================100
  //   AREA AROUND POINT    FROM  FRAME  (LOCAL)
  //====================================================================================================100

  // common
  common.in2_rows = common.sSize + 1 + common.sSize;
  common.in2_cols = common.in2_rows;
  common.in2_elem = common.in2_rows * common.in2_cols;
  common.in2_mem = sizeof(fp) * common.in2_elem;

  fp* d_in2;
  hipMalloc((void**)&d_in2, common.in2_mem * common.allPoints);
  //printf("%d\n", common.in2_elem * common.allPoints);

  //====================================================================================================100
  //   CONVOLUTION  (LOCAL)
  //====================================================================================================100

  // common
  common.conv_rows = common.in_rows + common.in2_rows - 1;                        // number of rows in I
  common.conv_cols = common.in_cols + common.in2_cols - 1;                        // number of columns in I
  common.conv_elem = common.conv_rows * common.conv_cols;                          // number of elements
  common.conv_mem = sizeof(fp) * common.conv_elem;
  common.ioffset = 0;
  common.joffset = 0;

  // unique
  fp* d_conv;
  hipMalloc((void**)&d_conv, common.conv_mem * common.allPoints);

  //====================================================================================================100
  //   CUMULATIVE SUM  (LOCAL)
  //====================================================================================================100

  //==================================================50
  //   PADDING OF ARRAY, VERTICAL CUMULATIVE SUM
  //==================================================50

  // common
  common.in2_pad_add_rows = common.in_rows;
  common.in2_pad_add_cols = common.in_cols;

  common.in2_pad_cumv_rows = common.in2_rows + 2*common.in2_pad_add_rows;
  common.in2_pad_cumv_cols = common.in2_cols + 2*common.in2_pad_add_cols;
  common.in2_pad_cumv_elem = common.in2_pad_cumv_rows * common.in2_pad_cumv_cols;
  common.in2_pad_cumv_mem = sizeof(fp) * common.in2_pad_cumv_elem;

  // unique
  //buffer<fp,1> d_in2_pad_cumv(common.in2_pad_cumv_elem * common.allPoints);
  //printf("%d\n", common.in2_pad_cumv_elem * common.allPoints);
  fp* d_in2_pad_cumv;
  hipMalloc((void**)&d_in2_pad_cumv, common.in2_pad_cumv_mem * common.allPoints);

  //==================================================50
  //   SELECTION
  //==================================================50

  // common
  common.in2_pad_cumv_sel_rowlow = 1 + common.in_rows;                          // (1 to n+1)
  common.in2_pad_cumv_sel_rowhig = common.in2_pad_cumv_rows - 1;
  common.in2_pad_cumv_sel_collow = 1;
  common.in2_pad_cumv_sel_colhig = common.in2_pad_cumv_cols;
  common.in2_pad_cumv_sel_rows = common.in2_pad_cumv_sel_rowhig - common.in2_pad_cumv_sel_rowlow + 1;
  common.in2_pad_cumv_sel_cols = common.in2_pad_cumv_sel_colhig - common.in2_pad_cumv_sel_collow + 1;
  common.in2_pad_cumv_sel_elem = common.in2_pad_cumv_sel_rows * common.in2_pad_cumv_sel_cols;
  common.in2_pad_cumv_sel_mem = sizeof(fp) * common.in2_pad_cumv_sel_elem;

  // unique
  //buffer<fp,1> d_in2_pad_cumv_sel(common.in2_pad_cumv_sel_elem * common.allPoints);
  //printf("%d\n", common.in2_pad_cumv_sel_elem * common.allPoints);
  fp* d_in2_pad_cumv_sel;
  hipMalloc((void**)&d_in2_pad_cumv_sel, common.in2_pad_cumv_sel_mem * common.allPoints);

  //==================================================50
  //   SELECTION  2, SUBTRACTION, HORIZONTAL CUMULATIVE SUM
  //==================================================50

  // common
  common.in2_pad_cumv_sel2_rowlow = 1;
  common.in2_pad_cumv_sel2_rowhig = common.in2_pad_cumv_rows - common.in_rows - 1;
  common.in2_pad_cumv_sel2_collow = 1;
  common.in2_pad_cumv_sel2_colhig = common.in2_pad_cumv_cols;
  common.in2_sub_cumh_rows = common.in2_pad_cumv_sel2_rowhig - common.in2_pad_cumv_sel2_rowlow + 1;
  common.in2_sub_cumh_cols = common.in2_pad_cumv_sel2_colhig - common.in2_pad_cumv_sel2_collow + 1;
  common.in2_sub_cumh_elem = common.in2_sub_cumh_rows * common.in2_sub_cumh_cols;
  common.in2_sub_cumh_mem = sizeof(fp) * common.in2_sub_cumh_elem;

  // unique
  //buffer<fp,1> d_in2_sub_cumh(common.in2_sub_cumh_elem * common.allPoints);
  //printf("%d\n", common.in2_sub_cumh_elem * common.allPoints);
  fp* d_in2_sub_cumh;
  hipMalloc((void**)&d_in2_sub_cumh, common.in2_sub_cumh_mem * common.allPoints);

  //==================================================50
  //   SELECTION
  //==================================================50

  // common
  common.in2_sub_cumh_sel_rowlow = 1;
  common.in2_sub_cumh_sel_rowhig = common.in2_sub_cumh_rows;
  common.in2_sub_cumh_sel_collow = 1 + common.in_cols;
  common.in2_sub_cumh_sel_colhig = common.in2_sub_cumh_cols - 1;
  common.in2_sub_cumh_sel_rows = common.in2_sub_cumh_sel_rowhig - common.in2_sub_cumh_sel_rowlow + 1;
  common.in2_sub_cumh_sel_cols = common.in2_sub_cumh_sel_colhig - common.in2_sub_cumh_sel_collow + 1;
  common.in2_sub_cumh_sel_elem = common.in2_sub_cumh_sel_rows * common.in2_sub_cumh_sel_cols;
  common.in2_sub_cumh_sel_mem = sizeof(fp) * common.in2_sub_cumh_sel_elem;

  // unique
  //buffer<fp,1> d_in2_sub_cumh_sel(common.in2_sub_cumh_sel_elem * common.allPoints);
  //printf("%d\n", common.in2_sub_cumh_sel_elem * common.allPoints);
  fp* d_in2_sub_cumh_sel;
  hipMalloc((void**)&d_in2_sub_cumh_sel, common.in2_sub_cumh_sel_mem * common.allPoints);

  //==================================================50
  //  SELECTION 2, SUBTRACTION
  //==================================================50

  // common
  common.in2_sub_cumh_sel2_rowlow = 1;
  common.in2_sub_cumh_sel2_rowhig = common.in2_sub_cumh_rows;
  common.in2_sub_cumh_sel2_collow = 1;
  common.in2_sub_cumh_sel2_colhig = common.in2_sub_cumh_cols - common.in_cols - 1;
  common.in2_sub2_rows = common.in2_sub_cumh_sel2_rowhig - common.in2_sub_cumh_sel2_rowlow + 1;
  common.in2_sub2_cols = common.in2_sub_cumh_sel2_colhig - common.in2_sub_cumh_sel2_collow + 1;
  common.in2_sub2_elem = common.in2_sub2_rows * common.in2_sub2_cols;
  common.in2_sub2_mem = sizeof(fp) * common.in2_sub2_elem;

  // unique
  //buffer<fp,1> d_in2_sub2(common.in2_sub2_elem * common.allPoints);
  //printf("%d\n", common.in2_sub2_elem * common.allPoints);
  fp* d_in2_sub2;
  hipMalloc((void**)&d_in2_sub2, common.in2_sub2_mem * common.allPoints);

  //====================================================================================================100
  //  CUMULATIVE SUM 2  (LOCAL)
  //====================================================================================================100

  //==================================================50
  //  MULTIPLICATION
  //==================================================50

  // common
  common.in2_sqr_rows = common.in2_rows;
  common.in2_sqr_cols = common.in2_cols;
  common.in2_sqr_elem = common.in2_elem;
  common.in2_sqr_mem = common.in2_mem;

  // unique
  //buffer<fp,1> d_in2_sqr(common.in2_elem * common.allPoints);
  //printf("%d\n", common.in2_elem * common.allPoints);
  fp* d_in2_sqr;
  hipMalloc((void**)&d_in2_sqr, common.in2_sqr_mem * common.allPoints);

  //==================================================50
  //  SELECTION 2, SUBTRACTION
  //==================================================50

  // common
  common.in2_sqr_sub2_rows = common.in2_sub2_rows;
  common.in2_sqr_sub2_cols = common.in2_sub2_cols;
  common.in2_sqr_sub2_elem = common.in2_sub2_elem;
  common.in2_sqr_sub2_mem = common.in2_sub2_mem;

  // unique
  //buffer<fp,1> d_in2_sqr_sub2(common.in2_sub2_elem * common.allPoints);
  //printf("%d\n", common.in2_sub2_elem * common.allPoints);
  fp* d_in2_sqr_sub2;
  hipMalloc((void**)&d_in2_sqr_sub2, common.in2_sqr_sub2_mem * common.allPoints);

  //====================================================================================================100
  //  FINAL  (LOCAL)
  //====================================================================================================100

  // common
  common.in_sqr_rows = common.in_rows;
  common.in_sqr_cols = common.in_cols;
  common.in_sqr_elem = common.in_elem;
  common.in_sqr_mem = common.in_mem;

  // unique
  //buffer<fp,1> d_in_sqr(common.in_elem * common.allPoints);
  //printf("%d\n", common.in_elem * common.allPoints);
  fp* d_in_sqr;
  hipMalloc((void**)&d_in_sqr, common.in_sqr_mem * common.allPoints);

  //====================================================================================================100
  //  TEMPLATE MASK CREATE  (LOCAL)
  //====================================================================================================100

  // common
  common.tMask_rows = common.in_rows + (common.sSize+1+common.sSize) - 1;
  common.tMask_cols = common.tMask_rows;
  common.tMask_elem = common.tMask_rows * common.tMask_cols;
  common.tMask_mem = sizeof(fp) * common.tMask_elem;

  // unique
  //buffer<fp,1> d_tMask(common.tMask_elem * common.allPoints);
  //printf("%d\n", common.tMask_elem * common.allPoints);
  fp* d_tMask;
  hipMalloc((void**)&d_tMask, common.tMask_mem * common.allPoints);

  //====================================================================================================100
  //  POINT MASK INITIALIZE  (LOCAL)
  //====================================================================================================100

  // common
  common.mask_rows = common.maxMove;
  common.mask_cols = common.mask_rows;
  common.mask_elem = common.mask_rows * common.mask_cols;
  common.mask_mem = sizeof(fp) * common.mask_elem;

  //====================================================================================================100
  //  MASK CONVOLUTION  (LOCAL)
  //====================================================================================================100

  // common
  common.mask_conv_rows = common.tMask_rows;                        // number of rows in I
  common.mask_conv_cols = common.tMask_cols;                        // number of columns in I
  common.mask_conv_elem = common.mask_conv_rows * common.mask_conv_cols;                        // number of elements
  common.mask_conv_mem = sizeof(fp) * common.mask_conv_elem;
  common.mask_conv_ioffset = (common.mask_rows-1)/2;
  if((common.mask_rows-1) % 2 > 0.5){
    common.mask_conv_ioffset = common.mask_conv_ioffset + 1;
  }
  common.mask_conv_joffset = (common.mask_cols-1)/2;
  if((common.mask_cols-1) % 2 > 0.5){
    common.mask_conv_joffset = common.mask_conv_joffset + 1;
  }


  //printf("common.endPoints=%d\n", common.endoPoints); // 20 
  //printf("common.epiPoints=%d\n", common.epiPoints); // 31
  //printf("common.in_elem=%d\n", common.in_elem);
  //printf("common.endo_mem=%d\n", common.endo_mem); // 80
  //printf("common.epi_mem=%d\n", common.epi_mem); // 124
  //
  //buffer<params_common,1> d_common(&common, 1, props); // range is 1 ?
  //buffer<int,1> d_endoRow(endoRow, common.endoPoints, props);
  //d_endoRow.set_final_data(nullptr);
  //buffer<int,1> d_endoCol(endoCol, common.endoPoints, props);
  //d_endoCol.set_final_data(nullptr);
  //buffer<int,1> d_tEndoRowLoc(tEndoRowLoc, common.endoPoints * common.no_frames, props);
  //buffer<int,1> d_tEndoColLoc(tEndoColLoc, common.endoPoints * common.no_frames, props);
  //buffer<int,1> d_epiRow(epiRow, common.epiPoints, props);
  //d_epiRow.set_final_data(nullptr);
  //buffer<int,1> d_epiCol(epiCol, common.epiPoints, props);
  //d_epiCol.set_final_data(nullptr);
  //buffer<int,1> d_tEpiRowLoc(tEpiRowLoc, common.epiPoints * common.no_frames, props);
  //buffer<int,1> d_tEpiColLoc(tEpiColLoc, common.epiPoints * common.no_frames, props);

  int* d_endoRow;
  hipMalloc((void**)&d_endoRow, common.endo_mem);
  hipMemcpy(d_endoRow, endoRow, common.endo_mem, hipMemcpyHostToDevice);

  int* d_endoCol;
  hipMalloc((void**)&d_endoCol, common.endo_mem);
  hipMemcpy(d_endoCol, endoCol, common.endo_mem, hipMemcpyHostToDevice);

  int* d_tEndoRowLoc;
  int* d_tEndoColLoc;
  hipMalloc((void**)&d_tEndoRowLoc, common.endo_mem*common.no_frames);
  hipMemcpy(d_tEndoRowLoc, tEndoRowLoc, common.endo_mem*common.no_frames, hipMemcpyHostToDevice);
  hipMalloc((void**)&d_tEndoColLoc, common.endo_mem*common.no_frames);
  hipMemcpy(d_tEndoColLoc, tEndoColLoc, common.endo_mem*common.no_frames, hipMemcpyHostToDevice);

  int* d_epiRow;
  int* d_epiCol;
  hipMalloc((void**)&d_epiRow, common.epi_mem);
  hipMemcpy(d_epiRow, epiRow, common.epi_mem, hipMemcpyHostToDevice);
  hipMalloc((void**)&d_epiCol, common.epi_mem);
  hipMemcpy(d_epiCol, epiCol, common.epi_mem, hipMemcpyHostToDevice);

  int* d_tEpiRowLoc;
  int* d_tEpiColLoc;
  hipMalloc((void**)&d_tEpiRowLoc, common.epi_mem*common.no_frames);
  hipMemcpy(d_tEpiRowLoc, tEpiRowLoc, common.epi_mem*common.no_frames, hipMemcpyHostToDevice);
  hipMalloc((void**)&d_tEpiColLoc, common.epi_mem*common.no_frames);
  hipMemcpy(d_tEpiColLoc, tEpiColLoc, common.epi_mem*common.no_frames, hipMemcpyHostToDevice);

  //buffer<fp,1> d_mask_conv(common.mask_conv_elem * common.allPoints);
  //d_mask_conv.set_final_data(nullptr);
  fp* d_mask_conv;
  hipMalloc((void**)&d_mask_conv, common.mask_conv_mem * common.allPoints);

  //printf("%d\n", common.mask_conv_elem * common.allPoints);
  //buffer<fp,1> d_in_mod_temp(common.in_elem * common.allPoints);
  //d_in_mod_temp.set_final_data(nullptr);
  fp* d_in_mod_temp;
  hipMalloc((void**)&d_in_mod_temp, common.in_mem * common.allPoints);

  //printf("%d\n", common.in_elem * common.allPoints);
  //buffer<fp,1> d_in_partial_sum(common.in_cols * common.allPoints);
  //d_in_partial_sum.set_final_data(nullptr);

  fp* d_in_partial_sum;
  hipMalloc((void**)&d_in_partial_sum, sizeof(fp)*common.in_cols * common.allPoints);

  //printf("%d\n", common.in_cols * common.allPoints);
  //buffer<fp,1> d_in_sqr_partial_sum(common.in_sqr_rows * common.allPoints);
  //d_in_sqr_partial_sum.set_final_data(nullptr);

  fp* d_in_sqr_partial_sum;
  hipMalloc((void**)&d_in_sqr_partial_sum, sizeof(fp)*common.in_sqr_rows * common.allPoints);


  //printf("%d\n", common.in_sqr_rows * common.allPoints);
  //buffer<fp,1> d_par_max_val(common.mask_conv_rows * common.allPoints);
  //d_par_max_val.set_final_data(nullptr);

  fp* d_par_max_val;
  hipMalloc((void**)&d_par_max_val, sizeof(fp)*common.mask_conv_rows * common.allPoints);

  //printf("%d\n", common.mask_conv_rows * common.allPoints);
  //buffer<int,1> d_par_max_coo( common.mask_conv_rows * common.allPoints);
  //d_par_max_coo.set_final_data(nullptr);

  fp* d_par_max_coo;
  hipMalloc((void**)&d_par_max_coo, sizeof(fp)*common.mask_conv_rows * common.allPoints);

  //buffer<fp,1> d_in_final_sum(common.allPoints);
  //d_in_final_sum.set_final_data(nullptr);

  fp* d_in_final_sum;
  hipMalloc((void**)&d_in_final_sum, sizeof(fp)*common.allPoints);

  //buffer<fp,1> d_in_sqr_final_sum(common.allPoints);
  //d_in_sqr_final_sum.set_final_data(nullptr);
  fp* d_in_sqr_final_sum;
  hipMalloc((void**)&d_in_sqr_final_sum, sizeof(fp)*common.allPoints);

  //buffer<fp,1> d_denomT(common.allPoints);
  //d_denomT.set_final_data(nullptr);

  fp* d_denomT;
  hipMalloc((void**)&d_denomT, sizeof(fp)*common.allPoints);

#ifdef TEST_CHECKSUM
  //buffer<fp,1> d_checksum(CHECK);
  //d_checksum.set_final_data(nullptr);
  //printf("%d\n", CHECK);
  fp* checksum = (fp*) malloc (sizeof(fp)*CHECK);
  fp* d_checksum;
  hipMalloc((void**)&d_checksum, sizeof(fp)*CHECK);
#endif

  //====================================================================================================100
  //  EXECUTION PARAMETERS
  //====================================================================================================100

  // All kernels operations within kernel use same max size of threads. Size of block size is set to the size appropriate for max size operation (on padded matrix). Other use subsets of that.
  dim3 threads(NUMBER_THREADS);
  dim3 grids(common.allPoints);


  printf("frame progress: ");
  fflush(NULL);

  //====================================================================================================100
  //  LAUNCH
  //====================================================================================================100

  // variables
  fp* frame;
  int frame_no;

  //buffer<fp,1> d_frame(common.frame_elem);
  fp* d_frame;
  hipMalloc((void**)&d_frame, sizeof(fp)*common.frame_elem);

  for(frame_no=0; frame_no<common.frames_processed; frame_no++) {

    //==================================================50
    //  get and write current frame to GPU buffer
    //==================================================50

    // Extract a cropped version of the first frame from the video file
    frame = get_frame(  frames,                // pointer to video file
        frame_no,              // number of frame that needs to be returned
        0,                  // cropped?
        0,                  // scaled?
        1);                  // converted

    // copy frame to GPU memory
    hipMemcpy(d_frame, frame, sizeof(fp)*common.frame_elem, hipMemcpyHostToDevice);

    //==================================================50
    //  launch kernel
    //==================================================50
    hw<<<grids, threads>>>(
        frame_no,
        common,
        d_frame, 
        d_endoRow, 
        d_endoCol, 
        d_tEndoRowLoc, 
        d_tEndoColLoc,
        d_epiRow, 
        d_epiCol, 
        d_tEpiRowLoc, 
        d_tEpiColLoc,
        d_endoT,
        d_epiT,
        d_in2,
        d_conv,
        d_in2_pad_cumv,
        d_in2_pad_cumv_sel,
        d_in2_sub_cumh,
        d_in2_sub_cumh_sel,
        d_in2_sub2,
        d_in2_sqr,
        d_in2_sqr_sub2,
        d_in_sqr,
        d_tMask,
        d_mask_conv,
        d_in_mod_temp,
        d_in_partial_sum,
        d_in_sqr_partial_sum,
        d_par_max_val,
        d_par_max_coo,
        d_in_final_sum,
        d_in_sqr_final_sum,
        d_denomT
#ifdef TEST_CHECKSUM
          ,d_checksum
#endif
          );

    // free frame after each loop iteration, since AVI library allocates memory for every frame fetched
    free(frame);

    //==================================================50
    //  print frame progress
    //==================================================50

    // print frame progress
    printf("%d ", frame_no);
    fflush(NULL);

    //==================================================50
    //  DISPLAY CHECKSUM (TESTING)
    //==================================================50

#ifdef TEST_CHECKSUM
    hipMemcpy(checksum, d_checksum, sizeof(fp)*CHECK, hipMemcpyDeviceToHost);
    printf("CHECKSUM:\n");
    for(int i=0; i<CHECK; i++){
      printf("i=%d checksum=%f\n", i, checksum[i]);
    }
    printf("\n\n");
#endif

  }

  hipMemcpy(tEndoRowLoc, d_tEndoRowLoc, common.endo_mem * common.no_frames, hipMemcpyDeviceToHost);
  hipMemcpy(tEndoColLoc, d_tEndoColLoc, common.endo_mem * common.no_frames, hipMemcpyDeviceToHost);
  hipMemcpy(tEpiRowLoc, d_tEpiRowLoc, common.epi_mem * common.no_frames, hipMemcpyDeviceToHost);
  hipMemcpy(tEpiColLoc, d_tEpiColLoc, common.epi_mem * common.no_frames, hipMemcpyDeviceToHost);


  //====================================================================================================100
  //  PRINT FRAME PROGRESS END
  //====================================================================================================100
#ifdef TEST_CHECKSUM
  free(checksum);
  hipFree(d_checksum);
#endif
  hipFree(d_epiT);
  hipFree(d_endoT);
  hipFree(d_in2);
  hipFree(d_conv);
  hipFree(d_in2_pad_cumv);
  hipFree(d_in2_pad_cumv_sel);
  hipFree(d_in2_sub_cumh);
  hipFree(d_in2_sub_cumh_sel);
  hipFree(d_in2_sub2);
  hipFree(d_in2_sqr);
  hipFree(d_in2_sqr_sub2);
  hipFree(d_in_sqr);
  hipFree(d_tMask);
  hipFree(d_endoRow);
  hipFree(d_endoCol);
  hipFree(d_tEndoRowLoc);
  hipFree(d_tEndoColLoc);
  hipFree(d_epiRow);
  hipFree(d_epiCol);
  hipFree(d_tEpiRowLoc);
  hipFree(d_tEpiColLoc);
  hipFree(d_mask_conv);
  hipFree(d_in_mod_temp);
  hipFree(d_in_partial_sum);
  hipFree(d_in_sqr_partial_sum);
  hipFree(d_par_max_val);
  hipFree(d_par_max_coo);
  hipFree(d_in_final_sum);
  hipFree(d_in_sqr_final_sum);
  hipFree(d_denomT);
  hipFree(d_frame);

  printf("\n");
  fflush(NULL);

}

