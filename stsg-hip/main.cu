#include "hip/hip_runtime.h"
//******************************************************************
//cuSTSG is used to reconstruct high-quality NDVI time series data(MODIS/SPOT) based on STSG
//
//This procedure cuSTSG is the source code for the first version of cuSTSG.
//This is a parallel computing code using GPU.
//
//Coded by Yang Xue
// Reference:Xue Yang, Jin Chen, Qingfeng Guan, Huan Gao, and Wei Xia.
// Enhanced Spatial-Temporal Savitzky-Golay Method for Reconstructing High-quality NDVI Time Series: Reduced Sensitivity
// to Quality Flags and Improved Computational Efficiency.Transactions on Geoscience and Remote Sensing
//******************************************************************

#include <iostream>
#include <algorithm>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <gdal/gdal_priv.h>
#include "Filter.h"

using namespace std;

int main(int argc, char *argv[])
{
  GDALAllRegister();

  //parameters
  if (argc != 2)
  {
    cout << "No parameter file!" << endl;
    return 1;
  }

  ifstream parameter(argv[1]);
  if (!parameter)
  {
    cout << "Can't open parameter file!" << endl;
    return 1;
  }

  int* Years = nullptr;
  string NDVI_path, Reliability_path, STSG_Test_path;
  float cosyear, sampcorr;
  int win_year, win, snow_address, n_Years;
  string par;
  while (getline(parameter, par))
  {
    if (par.substr(0, 2) == "//" || par == "")
      continue;

    for (int i = 0; i < par.size(); )
    {
      if (isspace(par[i]))
        par.erase(i,1);
      else
        i++;
    }
    if (par.substr(0, par.find("=")) == "Years")
    {
      vector<int> year;
      while (par.rfind(",") < par.size())
      {
        year.push_back(stoi(par.substr(par.rfind(",") + 1)));
        par = par.substr(0, par.rfind(","));
      }
      year.push_back(stoi(par.substr(par.rfind("=") + 1)));

      n_Years = year.size();
      Years = new int[n_Years];
      for (int i = 0; i < n_Years; i++)
        Years[i] = year[n_Years - i - 1];
    }
    else if (par.substr(0, par.find("=")) == "NDVI_path")
      NDVI_path = par.substr(par.find("=") + 1);
    else if (par.substr(0, par.find("=")) == "Reliability_path")
      Reliability_path = par.substr(par.find("=") + 1);
    else if (par.substr(0, par.find("=")) == "STSG_Test_path")
      STSG_Test_path = par.substr(par.find("=") + 1);
    else if (par.substr(0, par.find("=")) == "cosyear")
      cosyear = stof(par.substr(par.find("=") + 1));
    else if (par.substr(0, par.find("=")) == "win_year")
      win_year = stoi(par.substr(par.find("=") + 1));
    else if (par.substr(0, par.find("=")) == "win")
      win = stoi(par.substr(par.find("=") + 1));
    else if (par.substr(0, par.find("=")) == "sampcorr")
      sampcorr = stof(par.substr(par.find("=") + 1));
    else if (par.substr(0, par.find("=")) == "snow_address")
      snow_address = stoi(par.substr(par.find("=") + 1));
  }
  parameter.close();

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  cout << "Device name:" << prop.name << endl;
  size_t const totalGlobalMem = std::min(prop.totalGlobalMem, 4UL*1024*1024*1024);
  cout << "Device global memory used: " << totalGlobalMem / 1024 / 1024 << " MB" << endl;

  vector<GDALDataset*> NDVI(n_Years);
  vector<GDALDataset*> QA(n_Years);
  int n_X, n_Y, n_B;
  GDALDataType type_NDVI, type_QA;
  for (int i = 0; i < n_Years; i++)
  {
    string FileName = NDVI_path + to_string(Years[i]);
    NDVI[i] = (GDALDataset*)GDALOpen(FileName.c_str(), GA_ReadOnly);
    if (i == 0)
    {
      n_X = NDVI[i]->GetRasterXSize();
      n_Y = NDVI[i]->GetRasterYSize();
      n_B = NDVI[i]->GetRasterCount();
      type_NDVI = NDVI[i]->GetRasterBand(1)->GetRasterDataType();
    }

    FileName = Reliability_path + to_string(Years[i]);
    QA[i] = (GDALDataset*)GDALOpen(FileName.c_str(), GA_ReadOnly);
    if (i == 0)
      type_QA = QA[i]->GetRasterBand(1)->GetRasterDataType();
  }

  cout << "Execution start " << endl;
  size_t PerYSize = n_X*n_B *(n_Years * sizeof(short) + n_Years * sizeof(unsigned char) + 2 * n_Years * sizeof(float) + sizeof(int) + sizeof(float) + n_Years * sizeof(float)) + n_X*(2 * win + 1)*(2 * win + 1) *(7 * sizeof(float) + 3 * sizeof(int));
  if (totalGlobalMem <= 2 * win*n_X*n_B* (n_Years * sizeof(short) + n_Years * sizeof(unsigned char) + 2 * n_Years * sizeof(float) + sizeof(float)) + n_X*n_Y*n_B*n_Years*sizeof(float))
  {
    cout << "Size of vector_out is larger than total device global memory. Exit!" << endl;
    return 1;
  }

  size_t PerStep = (totalGlobalMem - 2 * win*n_X*n_B* (n_Years * sizeof(short) + n_Years * sizeof(unsigned char) + 2 * n_Years * sizeof(float) + sizeof(float)) - n_X*n_Y*n_B*n_Years*sizeof(float)) / PerYSize;
  int Loops = 1;
  if (PerStep < n_Y)
  {
    Loops = n_Y / PerStep + 1;
    PerStep = n_Y / Loops + 1;
  }

  float *d_vector_out;
  size_t nBytes = n_X*n_Y*n_B*n_Years * sizeof(float);
  hipMalloc((void**)&d_vector_out, nBytes);
  hipMemset((void*)d_vector_out, 0, nBytes);
  nBytes = win*n_X*(2 * win + 1)*(2 * win + 1) * 4 * sizeof(float);
  float *res = (float*)malloc(nBytes);
  memset((void*)res, 0, nBytes);
  int last_Buffer_Dn = 0;

  printf("Number of loops: %d\n", Loops);
  for (int i = 1, StartY = 0; i <= Loops && StartY < n_Y; i++, StartY += PerStep)
  {
    cout << "Loops " << i << endl;
    if (i == Loops)
      PerStep = n_Y - StartY;

    int Buffer_Up = 0;
    int Buffer_Dn = 0;
    if (StartY + PerStep < n_Y - win)
      Buffer_Dn = win;
    else
      Buffer_Dn = n_Y - PerStep - StartY;
    if (StartY >= win)
      Buffer_Up = win;
    else
      Buffer_Up = StartY;

    int blkwidth = 16;
    int blkheight = 16;
    dim3 blocks(blkwidth, blkheight);
    dim3 grids(n_X % blkwidth == 0 ? n_X / blkwidth : n_X / blkwidth + 1, (PerStep + Buffer_Up + Buffer_Dn) % blkheight == 0 ? (PerStep + Buffer_Up + Buffer_Dn) / blkheight : (PerStep + Buffer_Up + Buffer_Dn) / blkheight + 1);

    short *img_NDVI = new short[(PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B*n_Years];
    unsigned char *img_QA = new unsigned char[(PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B*n_Years];
    for (int i = 0; i < n_Years; i++)
    {
      NDVI[i]->RasterIO(GF_Read, 0, StartY - Buffer_Up, n_X, (PerStep + Buffer_Up + Buffer_Dn), &img_NDVI[i*(PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B], n_X, (PerStep + Buffer_Up + Buffer_Dn), type_NDVI, n_B, nullptr, 0, 0, 0);
      QA[i]->RasterIO(GF_Read, 0, StartY - Buffer_Up, n_X, (PerStep + Buffer_Up + Buffer_Dn), &img_QA[i*(PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B], n_X, (PerStep + Buffer_Up + Buffer_Dn), type_QA, n_B, nullptr, 0, 0, 0);
    }

    short *d_imgNDVI;
    nBytes = (PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B*n_Years * sizeof(short);
    hipMalloc((void**)&d_imgNDVI, nBytes);
    hipMemcpy((void*)d_imgNDVI, (void*)img_NDVI, nBytes, hipMemcpyHostToDevice);
    unsigned char *d_imgQA;
    nBytes = (PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B*n_Years * sizeof(unsigned char);
    hipMalloc((void**)&d_imgQA, nBytes);
    hipMemcpy((void*)d_imgQA, (void*)img_QA, nBytes, hipMemcpyHostToDevice);
    float *d_img_NDVI, *d_img_QA;
    nBytes = (PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B*n_Years * sizeof(float);
    hipMalloc((void**)&d_img_NDVI, nBytes);
    hipMalloc((void**)&d_img_QA, nBytes);
    hipMemset((void*)d_img_NDVI, 0, nBytes);
    hipMemset((void*)d_img_QA, 0, nBytes);
    float *d_NDVI_Reference, *d_res;
    nBytes = (PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B * sizeof(float);
    hipMalloc((void**)&d_NDVI_Reference, nBytes);
    hipMemset((void*)d_NDVI_Reference, 0, nBytes);
    nBytes = (PerStep + Buffer_Dn)*n_X*(2 * win + 1)*(2 * win + 1) * 4 * sizeof(float);
    hipMalloc((void**)&d_res, nBytes);
    hipMemset((void*)d_res, 0, nBytes);
    nBytes = last_Buffer_Dn*n_X*(2 * win + 1)*(2 * win + 1) * 4 * sizeof(float);
    hipMemcpy((void*)d_res, (void*)res, nBytes, hipMemcpyHostToDevice);

    int *d_res_vec_res1;
    nBytes = (PerStep + Buffer_Up + Buffer_Dn)*n_X*n_B * sizeof(int);
    hipMalloc((void**)&d_res_vec_res1, nBytes);
    hipMemset((void*)d_res_vec_res1, 0, nBytes);
    float *d_vector_in, *d_res_3;
    nBytes = PerStep*n_X* n_B * sizeof(float);
    hipMalloc((void**)&d_vector_in, nBytes);
    hipMemset((void*)d_vector_in, 0, nBytes);
    nBytes = PerStep*n_X*(2 * win + 1)*(2 * win + 1) * 3 * sizeof(float);
    hipMalloc((void**)&d_res_3, nBytes);//(slope_intercept(2);corr_similar;)
    hipMemset((void*)d_res_3, 0, nBytes);
    int *d_index;
    nBytes = PerStep*n_X*(2 * win + 1)*(2 * win + 1) * 3 * sizeof(int);
    hipMalloc((void**)&d_index, nBytes);//(similar_index(2);new_corr;)
    hipMemset((void*)d_index, 0, nBytes);
    hipDeviceSynchronize();

    auto start = std::chrono::steady_clock::now();
    Short_to_Float <<<grids, blocks >>>(d_imgNDVI, d_imgQA, n_X, (PerStep + Buffer_Up + Buffer_Dn), n_B, n_Years, d_img_NDVI, d_img_QA);

    Generate_NDVI_reference <<<grids, blocks >>>(cosyear, win_year, d_img_NDVI, d_img_QA, n_X, (PerStep + Buffer_Up + Buffer_Dn), n_B, n_Years, d_NDVI_Reference, d_res_3, d_res_vec_res1);

    nBytes = PerStep*n_X*(2 * win + 1)*(2 * win + 1) * 3 * sizeof(float);
    hipMemset((void*)d_res_3, 0, nBytes);

    Compute_d_res <<<grids, blocks >>>(d_img_NDVI, d_img_QA, d_NDVI_Reference, StartY, n_Y, Buffer_Up, Buffer_Dn, n_X, (PerStep + Buffer_Up + Buffer_Dn), n_B, n_Years, win, d_res);

    STSG_filter <<<grids, blocks >>>(d_img_NDVI, d_img_QA, d_NDVI_Reference, StartY, n_Y, Buffer_Up, Buffer_Dn, n_X, PerStep, n_B, n_Years, win, sampcorr, snow_address, d_vector_out, d_vector_in, d_res, d_res_3, d_index);

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    cout << "Total kernel time: " << time * 1e-9f  << " s" << endl;

    nBytes = win*n_X*(2 * win + 1)*(2 * win + 1) * 4 * sizeof(float);
    memset((void*)res, 0, nBytes);
    nBytes = Buffer_Dn*n_X*(2 * win + 1)*(2 * win + 1) * 4 * sizeof(float);
    hipMemcpy((void*)res, (void*)&d_res[(PerStep + Buffer_Dn - win)*n_X*(2 * win + 1)*(2 * win + 1) * 4], nBytes, hipMemcpyDeviceToHost);
    last_Buffer_Dn = Buffer_Dn;

    delete[] img_NDVI;
    delete[] img_QA;
    hipFree((void*)d_imgNDVI);
    hipFree((void*)d_imgQA);
    hipFree((void*)d_img_NDVI);
    hipFree((void*)d_img_QA);
    hipFree((void*)d_NDVI_Reference);
    hipFree((void*)d_res);
    hipFree((void*)d_res_vec_res1);
    hipFree((void*)d_vector_in);
    hipFree((void*)d_res_3);
    hipFree((void*)d_index);
  }
  free((void*)res);

  float *vector_out = new float[n_X*n_Y*n_B*n_Years];
  nBytes = n_X*n_Y*n_B*n_Years* sizeof(float);
  hipMemcpy((void*)vector_out, (void*)d_vector_out, nBytes, hipMemcpyDeviceToHost);
  hipFree((void*)d_vector_out);

  long cnt = 0;
  double sum = 0;
  for (int i = 0; i < n_X*n_Y*n_B*n_Years; i++) {
    if (vector_out[i] < 1.f || vector_out[i] > 0.f) {
      sum += vector_out[i];
      cnt++;
    }
  }
  cout << "Checksum: " << sum << " " << cnt << " " << sum / cnt << endl;

  delete[] vector_out;

  return 0;
}
