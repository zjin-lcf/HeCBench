/**********************************************************************
  Copyright �2013 Advanced Micro Devices, Inc. All rights reserved.

  Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

  Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
  Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or
  other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY
  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS
  OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ********************************************************************/

#include <chrono>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#include "reference.h"
#include "utils.cu"
#include "kernels.cu"

void runKernels(
    const float *diagonalBuffer,
           uint *numEigenValuesIntervalBuffer,
    const float *offDiagonalBuffer,
    float **eigenIntervalBuffer,

    // reset the eigenvalue intervals buffer
    float **eigenIntervals,

    const int length,
    const float tolerance,
    // index of the two eigenInterval buffers
    uint &in )
{
  dim3 grid (length/256);
  dim3 block (256);

  for (int i = 0; i < 2; i++)
    hipMemcpy(eigenIntervalBuffer[i], eigenIntervals[i], length*2*sizeof(float),
               hipMemcpyHostToDevice);

  in = 0;
  while (isComplete(eigenIntervals[in], length, tolerance)) {

    hipLaunchKernelGGL(calNumEigenValueInterval, grid, block, 0, 0, 
            numEigenValuesIntervalBuffer,
            eigenIntervalBuffer[in],
            diagonalBuffer,
            offDiagonalBuffer,
            length);

    hipLaunchKernelGGL(recalculateEigenIntervals, grid, block, 0, 0, 
          eigenIntervalBuffer[1-in],
          eigenIntervalBuffer[in],
          numEigenValuesIntervalBuffer,
          diagonalBuffer,
          offDiagonalBuffer,
          length,
          tolerance);

    in = 1 - in;

    hipMemcpy(eigenIntervals[in], eigenIntervalBuffer[in], length*2*sizeof(float),
               hipMemcpyDeviceToHost);
  }
}

int main(int argc, char * argv[])
{
  if (argc != 3) {
    printf("Usage: %s <length of the diagonal of the square matrix> <repeat>\n", argv[0]);
    return 1;
  }

  // Length of the diagonal of the square matrix
  int length = atoi(argv[1]);
  // Number of iterations for kernel execution
  int iterations = atoi(argv[2]);
  // Seed value for random number generation 
  uint seed = 123;
  float tolerance;
  // diagonal elements of the matrix
  float *diagonal;
  // off-diagonal elements of the matrix
  float *offDiagonal;
  // calculated eigen values of the matrix
  float *eigenIntervals[2];
  // index to one of the two eigen interval buffers
  uint  in;
  // eigen values using reference implementation
  float *verificationEigenIntervals[2];
  // index to one of the two eigen interval arrays
  uint   verificationIn;

  // allocate memory for diagonal elements of the matrix  of size lengthxlength

  if(isPowerOf2(length))
  {
    length = roundToPowerOf2(length);
  }

  if(length < 256)
  {
    length = 256;
  }

  uint diagonalSizeBytes = length * sizeof(float);
  diagonal = (float *) malloc(diagonalSizeBytes);
  CHECK_ALLOCATION(diagonal, "Failed to allocate host memory. (diagonal)");

  // allocate memory for offdiagonal elements of the matrix of length (length-1)
  uint offDiagonalSizeBytes = (length - 1) * sizeof(float);
  offDiagonal = (float *) malloc(offDiagonalSizeBytes);
  CHECK_ALLOCATION(offDiagonal, "Failed to allocate host memory. (offDiagonal)");

  /*
   * allocate memory to store the eigenvalue intervals interleaved with upperbound followed
   * by the lower bound interleaved
   * An array of two is used for using it for two different passes
   */
  uint eigenIntervalsSizeBytes = (2*length) * sizeof(float);
  for(int i = 0; i < 2; ++i)
  {
    eigenIntervals[i] = (float *) malloc(eigenIntervalsSizeBytes);
    CHECK_ALLOCATION(eigenIntervals[i],
        "Failed to allocate host memory. (eigenIntervals)");
  }

  // random initialisation of input using a seed
  fillRandom<float>(diagonal   , length  , 1, 0, 255, seed);
  fillRandom<float>(offDiagonal, length-1, 1, 0, 255, seed+10);

  // calculate the upperbound and the lowerbound of the eigenvalues of the matrix
  float lowerLimit;
  float upperLimit;
  computeGerschgorinInterval(&lowerLimit, &upperLimit, diagonal, offDiagonal, length);

  // initialize the eigenvalue intervals
  eigenIntervals[0][0]= lowerLimit;
  eigenIntervals[0][1]= upperLimit;

  // the following intervals have no eigenvalues
  for (int i = 2 ; i < 2*length ; i++)
  {
    eigenIntervals[0][i] = upperLimit;
  }

  tolerance = 0.001f;
  /*
   * Unless quiet mode has been enabled, print the INPUT array.
   */
#ifdef DEBUG
    printArray<float>("Diagonal", diagonal, length, 1);
    printArray<float>("offDiagonal", offDiagonal, length-1, 1);
#endif

  // store the diagonal elements of the matrix
  float *diagonalBuffer;
  hipMalloc((void**)&diagonalBuffer, sizeof(float) * length);
  hipMemcpy(diagonalBuffer, diagonal, sizeof(float) * length, hipMemcpyHostToDevice); 

  // store the number of eigenvalues in each interval
  uint *numEigenValuesIntervalBuffer;
  hipMalloc((void**)&numEigenValuesIntervalBuffer, sizeof(uint) * length);

  // store the offDiagonal elements of the matrix
  float *offDiagonalBuffer;
  hipMalloc((void**)&offDiagonalBuffer, sizeof(float) * (length-1));
  hipMemcpy(offDiagonalBuffer, offDiagonal, sizeof(float) * (length-1), hipMemcpyHostToDevice); 

  // store the eigenvalue intervals
  float *eigenIntervalBuffer[2];
  for(int i = 0 ; i < 2 ; ++ i)
  {
    hipMalloc((void**)&(eigenIntervalBuffer[i]), sizeof(float) * (length * 2));
  }

  // Warm up
  for(int i = 0; i < 2 && iterations != 1; i++)
  {
    // Arguments are set and execution call is enqueued on command buffer
    runKernels(
        diagonalBuffer,
        numEigenValuesIntervalBuffer,
        offDiagonalBuffer,
        eigenIntervalBuffer,
        eigenIntervals,   // reset eigenIntervals
        length,
        tolerance,
        in);
  }

  std::cout << "Executing kernel for " << iterations
            << " iterations" << std::endl;
  std::cout << "-------------------------------------------" << std::endl;
  
  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();
  
  for(int i = 0; i < iterations; i++)
  {
    runKernels(
        diagonalBuffer,
        numEigenValuesIntervalBuffer,
        offDiagonalBuffer,
        eigenIntervalBuffer,
        eigenIntervals,   // reset eigenIntervals
        length,
        tolerance,
        in);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "Average kernel execution time " << (time * 1e-3f) / iterations << " (us)\n";

  // Verify results
  for(int i = 0 ; i < 2; ++i)
  {
    verificationEigenIntervals[i] = (float *) malloc(eigenIntervalsSizeBytes);

    if(verificationEigenIntervals[i] == NULL)
    {
      error("Failed to allocate host memory. (verificationEigenIntervals)");
      return 1;
    }
  }

  computeGerschgorinInterval(&lowerLimit, &upperLimit, diagonal, offDiagonal, length);

  verificationIn = 0;
  verificationEigenIntervals[verificationIn][0]= lowerLimit;
  verificationEigenIntervals[verificationIn][1]= upperLimit;

  for(int i = 2 ; i < 2*length ; i++)
  {
    verificationEigenIntervals[verificationIn][i] = upperLimit;
  }

  while(isComplete(verificationEigenIntervals[verificationIn], length, tolerance))
  {
    eigenValueCPUReference(diagonal,offDiagonal, length,
        verificationEigenIntervals[verificationIn],
        verificationEigenIntervals[1-verificationIn],
        tolerance);
    verificationIn = 1 - verificationIn;
  }

  // select the buffers for comparison
  if(compare(eigenIntervals[in], 
             verificationEigenIntervals[verificationIn], 2*length))
  {
    std::cout<<"PASS\n" << std::endl;
  }
  else
  {
    std::cout<<"FAIL\n" << std::endl;
  }

  // release program resources
  hipFree(diagonalBuffer);
  hipFree(offDiagonalBuffer);
  hipFree(numEigenValuesIntervalBuffer);
  hipFree(eigenIntervalBuffer[0]);
  hipFree(eigenIntervalBuffer[1]);
  free(diagonal);
  free(offDiagonal);
  free(eigenIntervals[0]);
  free(eigenIntervals[1]);
  free(verificationEigenIntervals[0]);
  free(verificationEigenIntervals[1]);

  return 0;
}
