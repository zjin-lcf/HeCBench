#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <chrono>
#include <hip/hip_runtime.h>

__host__ __device__
double Fresnel_Sine_Integral(double);

void reference (const double *__restrict input,
                      double *__restrict output, const int n)
{
  for (int i = 0; i < n; i++)
    output[i] = Fresnel_Sine_Integral(input[i]);
}

__global__ void 
kernel (const double *__restrict__ input,
              double *__restrict__ output, const int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    output[i] = Fresnel_Sine_Integral(input[i]);
}

int main(int argc, char *argv[])
{
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  // range [0, 8], interval 1e-7
  const double interval = 1e-7;
  const int points = (int)(8.0 / interval);
  const size_t points_size = points * sizeof(double);
  double *x = (double*) malloc (points_size);
  double *output = (double*) malloc (points_size);
  double *h_output = (double*) malloc (points_size);
  for (int i = 0; i < points; i++)
    x[i] = (double)i * interval;
	   
  double *d_x;
  hipMalloc((void**)&d_x, points_size);
  double *d_output;
  hipMalloc((void**)&d_output, points_size);

  dim3 grids = (points + 255)/256;
  dim3 blocks = 256;

  hipMemcpy(d_x, x, points_size, hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++)
    hipLaunchKernelGGL(kernel, grids, blocks, 0, 0, d_x, d_output, points);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time %f (s)\n", (time * 1e-9f) / repeat);

  hipMemcpy(output, d_output, points_size, hipMemcpyDeviceToHost);

  // verify
  reference(x, h_output, points);
  bool ok = true;
  for (int i = 0; i < points; i++) {
    if (fabs(h_output[i] - output[i]) > 1e-6) {
      printf("%lf %lf\n", h_output[i], output[i]);
      ok = false;
      break;
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");
  
  hipFree(d_x);
  hipFree(d_output);
  free(x);
  free(output);
  free(h_output);
  return 0;
}
