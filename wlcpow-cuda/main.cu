#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include "utils.h"

__global__ 
void bond_wlcpowallvisc(
             r64* __restrict__ force_x,
             r64* __restrict__ force_y,
             r64* __restrict__ force_z,
    const float4* __restrict__ coord_merged,
    const float4* __restrict__ veloc,
    const int*  __restrict__ nbond,
    const int2* __restrict__ bonds,
    const r64* __restrict__ bond_r0,
    const r32* __restrict__ temp_global,
    const r32* __restrict__ r0_global,
    const r32* __restrict__ mu_targ_global,
    const r32* __restrict__ qp_global,
    const r32* __restrict__ gamc_global,
    const r32* __restrict__ gamt_global,
    const r32* __restrict__ sigc_global,
    const r32* __restrict__ sigt_global,
    const float3 period,
    const int padding,
    const int n_type,
    const int n_local )
{
  extern __shared__ r32 shared_data[];
  r32* temp    = &shared_data[0];
  r32* r0      = &shared_data[1*(n_type+1)];
  r32* mu_targ = &shared_data[2*(n_type+1)];
  r32* qp      = &shared_data[3*(n_type+1)];
  r32* gamc    = &shared_data[4*(n_type+1)];
  r32* gamt    = &shared_data[5*(n_type+1)];
  r32* sigc    = &shared_data[6*(n_type+1)];
  r32* sigt    = &shared_data[7*(n_type+1)];

  for ( int i = threadIdx.x; i < n_type + 1; i += blockDim.x ) {
    temp[i]    = temp_global[i];
    r0[i]      = r0_global[i];
    mu_targ[i] = mu_targ_global[i];
    qp[i]      = qp_global[i];
    gamc[i]    = gamc_global[i];
    gamt[i]    = gamt_global[i];
    sigc[i]    = sigc_global[i];
    sigt[i]    = sigt_global[i];
  }
  __syncthreads();

  for( int i = blockIdx.x * blockDim.x + threadIdx.x;
           i < n_local ; i += gridDim.x * blockDim.x ) {
    int n = nbond[i];
    float4 coord1 = coord_merged[i];
    float4 veloc1 = veloc[i];
    r32 fxi = 0.f, fyi = 0.f, fzi = 0.f;

    for( int p = 0; p < n; p++ ) {
      int j = bonds[ i + p*padding ].x;
      int type = bonds[ i + p*padding ].y;
      float4 coord2 = coord_merged[j];
      r32 delx = minimum_image( coord1.x - coord2.x, period.x );
      r32 dely = minimum_image( coord1.y - coord2.y, period.y );
      r32 delz = minimum_image( coord1.z - coord2.z, period.z );
      float4 veloc2 = veloc[j];
      r32 dvx = veloc1.x - veloc2.x;
      r32 dvy = veloc1.y - veloc2.y;
      r32 dvz = veloc1.z - veloc2.z;

      r32 l0 = bond_r0[ i + p*padding ];
      r32 ra = sqrtf(delx*delx + dely*dely + delz*delz);
      r32 lmax = l0*r0[type];
      r32 rr = 1.0f/r0[type];
      r32 sr = (1.0f-rr)*(1.0f-rr);
      r32 kph = powf(l0,qp[type])*temp[type]*(0.25f/sr-0.25f+rr);
      // mu is described in the papers
      r32 mu = 0.433f*(   // 0.25 * sqrt(3)
	       temp[type]*(-0.25f/sr + 0.25f + 
               0.5f*rr/(sr*(1.0f-rr)))/(lmax*rr) +
               kph*(qp[type]+1.0f)/powf(l0,qp[type]+1.0f));
      r32 lambda = mu/mu_targ[type];
      kph = kph/lambda;
      rr = ra/lmax;
      r32 rlogarg = powf(ra,qp[type]+1.0f);
      r32 vv = (delx*dvx + dely*dvy + delz*dvz)/ra;

      if (rr >= 0.99) rr = 0.99f;
      if (rlogarg < 0.01) rlogarg = 0.01f;

      float4 wrr;
      r32 ww[3][3];

      for (int tes=0; tes<3; tes++) {
        for (int see=0; see<3; see++) {
          int v1 = __float_as_int(veloc1.w);
          int v2 = __float_as_int(veloc2.w);
          ww[tes][see] = gaussian_TEA_fast<4>(v1 > v2, v1+tes, v2+see);
        }
      }

      wrr.w = (ww[0][0]+ww[1][1]+ww[2][2])/3.0f;
      wrr.x = (ww[0][0]-wrr.w)*delx + 0.5f*(ww[0][1]+ww[1][0])*dely + 0.5f*(ww[0][2]+ww[2][0])*delz;
      wrr.y = 0.5f*(ww[1][0]+ww[0][1])*delx + (ww[1][1]-wrr.w)*dely + 0.5f*(ww[1][2]+ww[2][1])*delz;
      wrr.z = 0.5f*(ww[2][0]+ww[0][2])*delx + 0.5f*(ww[2][1]+ww[1][2])*dely + (ww[2][2]-wrr.w)*delz;

      r32 fforce = - temp[type]*(0.25f/(1.0f-rr)/(1.0f-rr)-0.25f+rr)/lambda/ra + kph/rlogarg + (sigc[type]*wrr.w - gamc[type]*vv)/ra;
      r32 fxij = delx*fforce - gamt[type]*dvx + sigt[type]*wrr.x/ra;
      r32 fyij = dely*fforce - gamt[type]*dvy + sigt[type]*wrr.y/ra;
      r32 fzij = delz*fforce - gamt[type]*dvz + sigt[type]*wrr.z/ra;

      fxi += fxij;
      fyi += fyij;
      fzi += fzij;
    }
    force_x[i] += fxi;
    force_y[i] += fyi;
    force_z[i] += fzi;
  }
}

template <typename T>
T* resize (int n) {
  return (T*) malloc (sizeof(T) * n);
}

template <typename T>
T* grow (int n) {
  T* p;
  hipMalloc((void**)&p, sizeof(T) * n);
  return p;
}

template <typename T>
void upload(T* d, T* h, int n) {
  hipMemcpy(d, h, sizeof(T) * n, hipMemcpyHostToDevice); 
}

template <typename T>
void reset(T* d, int n) {
  hipMemset(d, (T)0, sizeof(T) * n);
}

template <typename T>
void download(T* h, T* d, int n) {
  hipMemcpy(h, d, sizeof(T) * n, hipMemcpyDeviceToHost); 
}

int main(int argc, char* argv[]) {
  if (argc != 2) {
    printf("Usage: ./%s <repeat>\n", argv[0]);
    return 1;
  }

  const int repeat = atoi(argv[1]);

  int i;

  // all the values are randomly initialized

  float3 period = {0.5f, 0.5f, 0.5f};
  int padding = 1;
  int n_type = 32;
  int n = 1e6;  // problem size

  float4 *coord_merged = resize<float4>(n+1);
  float4 *veloc = resize<float4>(n+1);
  int *nbond = resize<int>(n+1);

  // set the sizes properly
  int2 *bonds = resize<int2>(n+n+1);
  r64 *bond_r0 = resize<r64>(n+n+1);

  r64 *force_x = resize<r64>(n+1);
  r64 *force_y = resize<r64>(n+1);
  r64 *force_z = resize<r64>(n+1);

  r32 *bond_l0 = resize<r32>(n+1);
  r32 *temp = resize<r32>(n+1);
  r32 *mu_targ = resize<r32>(n+1);
  r32 *qp = resize<r32>(n+1);
  r32 *gamc = resize<r32>(n+1);
  r32 *gamt = resize<r32>(n+1);
  r32 *sigc = resize<r32>(n+1);
  r32 *sigt = resize<r32>(n+1);

  std::mt19937 g (19937);
  std::uniform_real_distribution<r64> dist_r64(0.1, 0.9);
  std::uniform_real_distribution<r32> dist_r32(0.1, 0.9);
  std::uniform_int_distribution<i32> dist_i32(0, n_type);

  for (i = 0; i < n + n + 1; i++) {
    bond_r0[i] = dist_r64(g) + 0.001;
    // select two distinct atoms in the kernel to evaluate their forces
    bonds[i] = { (i+1)%(n+1), 
                 dist_i32(g) };
  }

  for (i = 0; i < n + 1; i++) {
    nbond[i] = dist_i32(g);
    coord_merged[i] = {dist_r32(g), dist_r32(g), dist_r32(g), 0};
    r32 vx = dist_r32(g), vy = dist_r32(g), vz = dist_r32(g);
    veloc[i] = {vx, vy, vz, sqrtf(vx*vx+vy*vy+vz*vz)};
    bond_l0[i] = dist_r32(g);
    gamt[i] = dist_r32(g);
    gamc[i] = ((dist_i32(g) % 4) + 4) * gamt[i]; // gamt[i] <= 3.0*gamc[i]
    temp[i] = dist_r32(g);
    mu_targ[i] = dist_r32(g);
    qp[i] = dist_r32(g);
    sigc[i] = sqrt(2.0*temp[i]*(3.0*gamc[i]-gamt[i]));
    sigt[i] = 2.0*sqrt(gamt[i]*temp[i]);
  }

  float4 *dev_coord_merged = grow<float4>(n + 1);
  float4 *dev_veloc = grow<float4>(n + 1);
  int *dev_nbond = grow<int>(n + 1);
  int2 *dev_bonds = grow<int2>(n + n + 1);
  r64 *dev_bond_r0 = grow<r64>(n + n + 1);

  r64 *dev_force_x = grow<r64>(n + 1);
  r64 *dev_force_y = grow<r64>(n + 1);
  r64 *dev_force_z = grow<r64>(n + 1);

  r32 *dev_bond_l0 = grow<r32>(n + 1);
  r32 *dev_temp = grow<r32>(n + 1);
  r32 *dev_mu_targ = grow<r32>(n + 1);
  r32 *dev_qp = grow<r32>(n + 1);
  r32 *dev_gamc = grow<r32>(n + 1);
  r32 *dev_gamt = grow<r32>(n + 1);
  r32 *dev_sigc = grow<r32>(n + 1);
  r32 *dev_sigt = grow<r32>(n + 1);

  reset (dev_force_x, n+1);
  reset (dev_force_y, n+1);
  reset (dev_force_z, n+1);
  upload (dev_coord_merged, coord_merged, n+1);
  upload (dev_veloc, veloc, n+1);
  upload (dev_nbond, nbond, n+1);
  upload (dev_bonds, bonds, n+n+1);
  upload (dev_bond_r0, bond_r0, n+n+1);
  upload (dev_temp, temp, n+1);
  upload (dev_bond_l0, bond_l0, n+1);
  upload (dev_mu_targ, mu_targ, n+1);
  upload (dev_qp, qp, n+1);
  upload (dev_gamc, gamc, n+1);
  upload (dev_gamt, gamt, n+1);
  upload (dev_sigc, sigc, n+1);
  upload (dev_sigt, sigt, n+1);

  dim3 grids ((n+127)/128);
  dim3 blocks (128);

  const int sm_size = (n_type+1) * 8 * sizeof(r32);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  // note the outputs are not reset for each run
  for (i = 0; i < repeat; i++) {
    bond_wlcpowallvisc <<<grids, blocks, sm_size, 0>>> (
      dev_force_x,
      dev_force_y,
      dev_force_z,
      dev_coord_merged,
      dev_veloc,
      dev_nbond,
      dev_bonds,
      dev_bond_r0,
      dev_temp,
      dev_bond_l0,
      dev_mu_targ,
      dev_qp,
      dev_gamc,
      dev_gamt,
      dev_sigc,
      dev_sigt,
      period,
      padding,
      n_type,
      n);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average kernel execution time: %f (us)\n", time * 1e-3f / repeat);

  download (force_x, dev_force_x, n+1);
  download (force_y, dev_force_y, n+1);
  download (force_z, dev_force_z, n+1);

  // no NaN values in the outputs
  for (i = 0; i < n+1; i++) {
    bool r = (isnan(force_x[i]) || isnan(force_y[i]) || isnan(force_z[i]));
    if (r) printf("There are NaN numbers at index %d\n", i);
  }

  double force_x_sum = 0, force_y_sum = 0, force_z_sum = 0;
  for (i = 0; i < n+1; i++) {
    force_x_sum += force_x[i];
    force_y_sum += force_y[i];
    force_z_sum += force_z[i];
  }
  // values are meaningless, but they should be consistent across devices
  printf("checksum: forceX=%lf forceY=%lf forceZ=%lf\n",
         force_x_sum/(n+1), force_y_sum/(n+1), force_z_sum/(n+1));
  
#ifdef DEBUG
  for (i = 0; i < 16; i++) {
    printf("%d %lf %lf %lf\n", i, force_x[i], force_y[i], force_z[i]);
  }
#endif

  free(coord_merged);
  free(veloc);
  free(force_x);
  free(force_y);
  free(force_z);
  free(nbond);
  free(bonds);
  free(bond_r0);
  free(bond_l0);
  free(temp);
  free(mu_targ);
  free(qp);
  free(gamc);
  free(gamt);
  free(sigc);
  free(sigt);

  hipFree(dev_coord_merged);
  hipFree(dev_veloc);
  hipFree(dev_force_x);
  hipFree(dev_force_y);
  hipFree(dev_force_z);
  hipFree(dev_nbond);
  hipFree(dev_bonds);
  hipFree(dev_bond_r0);
  hipFree(dev_bond_l0);
  hipFree(dev_temp);
  hipFree(dev_mu_targ);
  hipFree(dev_qp);
  hipFree(dev_gamc);
  hipFree(dev_gamt);
  hipFree(dev_sigc);
  hipFree(dev_sigt);
  return 0;
}
