#include "hip/hip_runtime.h"
/*****************************************************************************
 This file is part of the XLQC program.                                      
 Copyright (C) 2015 Xin Li <lixin.reco@gmail.com>                            
                                                                           
 Filename:  main.cu                                                      
 License:   BSD 3-Clause License

 This software is provided by the copyright holders and contributors "as is"
 and any express or implied warranties, including, but not limited to, the
 implied warranties of merchantability and fitness for a particular purpose are
 disclaimed. In no event shall the copyright holder or contributors be liable
 for any direct, indirect, incidental, special, exemplary, or consequential
 damages (including, but not limited to, procurement of substitute goods or
 services; loss of use, data, or profits; or business interruption) however
 caused and on any theory of liability, whether in contract, strict liability,
 or tort (including negligence or otherwise) arising in any way out of the use
 of this software, even if advised of the possibility of such damage.
 *****************************************************************************/

#include <chrono>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cassert>
#include <string>
#include <iostream>

#include <gsl/gsl_math.h>
#include <gsl/gsl_blas.h>
#include <gsl/gsl_eigen.h>
#include <gsl/gsl_matrix.h>
#include <gsl/gsl_linalg.h>

#include "int_lib/cints.h"
#include "int_lib/crys.h"

#include "typedef.h"
#include "basis.h"
#include "scf.h"

#include "cuda_rys_sp.h"
#include "cuda_rys_dp.h"

int main(int argc, char* argv[])
{
    // use spherical harmonic d function?
    int use_5d = 1;
    // use double precision?
    int use_dp = 1;

    if (argc > 1) {
        for (int i = 1; i < argc; ++ i) {
            if (0 == strcmp(argv[i],"sp")) { use_dp = 0; }
            if (0 == strcmp(argv[i],"dp")) { use_dp = 1; }
            if (0 == strcmp(argv[i],"6d")) { use_5d = 0; }
            if (0 == strcmp(argv[i],"5d")) { use_5d = 1; }
        }
    }

    // initialize timer
    double time_in_usec, time_total, time_mat_J, time_mat_K;

    auto start = std::chrono::steady_clock::now();
    std::string time_txt ("");
    time_total = 0.0;
    time_mat_J = 0.0;
    time_mat_K = 0.0;

    Atom   *p_atom   = (Atom *)my_malloc(sizeof(Atom) * 1);
    Basis  *p_basis  = (Basis *)my_malloc(sizeof(Basis) * 1);


    //====== parse geom.dat ========

    // get number of atoms
    p_atom->num = get_natoms();
    fprintf(stdout, "Natoms = %d\n", p_atom->num);

    // atomic coordinates and atom name
    p_atom->pos = (double **)my_malloc(sizeof(double *) * p_atom->num);
    p_atom->name = (char **)my_malloc(sizeof(char *) * p_atom->num);

    for (int iatom = 0; iatom < p_atom->num; ++ iatom)
    {
        p_atom->pos[iatom] = (double *)my_malloc(sizeof(double) * CART_DIM);
        p_atom->name[iatom] = (char *)my_malloc(sizeof(char) * 5);
    }

    // nuclear charge
    p_atom->nuc_chg = (int *)my_malloc(sizeof(int) * p_atom->num);

    // read atomic positions, nuclear charge and atom name
    read_geom(p_atom);

    fprintf(stdout, "Coordinates in atomic unit:\n");
    for (int iatom = 0; iatom < p_atom->num; ++ iatom)
    {
        fprintf(stdout, "%s (%.1f)  %.10f  %.10f  %.10f\n", 
                p_atom->name[iatom], (double)p_atom->nuc_chg[iatom],
                p_atom->pos[iatom][0], p_atom->pos[iatom][1], p_atom->pos[iatom][2]);
    }
    
    // nuclear repulsion energy
    double ene_nucl = calc_ene_nucl(p_atom);
    fprintf(stdout, "Nuclear repulsion = %-20.10f\n", ene_nucl);


    //====== parse basis.dat ========

    // parse basis functions
    parse_basis(p_atom, p_basis, use_5d);

    fprintf(stdout, "System Nbasis = %d\n", p_basis->num);

    // basis function exponents, coefficients, and normalization factors
    p_basis->expon = (double **)my_malloc(sizeof(double *) * p_basis->num);
    p_basis->coef  = (double **)my_malloc(sizeof(double *) * p_basis->num);
    p_basis->norm  = (double **)my_malloc(sizeof(double *) * p_basis->num);

    // number of primitive functions in each contracted funciton
    p_basis->nprims = (int *)my_malloc(sizeof(int) * p_basis->num);

    // Cartesian coordinates and l,m,n numbers
    p_basis->xbas  = (double *)my_malloc(sizeof(double) * p_basis->num);
    p_basis->ybas  = (double *)my_malloc(sizeof(double) * p_basis->num);
    p_basis->zbas  = (double *)my_malloc(sizeof(double) * p_basis->num);

    p_basis->lx = (int **)my_malloc(sizeof(int *) * p_basis->num);
    p_basis->ly = (int **)my_malloc(sizeof(int *) * p_basis->num);
    p_basis->lz = (int **)my_malloc(sizeof(int *) * p_basis->num);

    // read basis set (also calculate normalization factors)
    read_basis(p_atom, p_basis, use_5d);

#ifdef DEBUG
    print_basis(p_basis);
#endif

    auto end = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    time_in_usec = time * 1e-3f;
    time_txt += "Time_Basis    = " + std::to_string(time_in_usec) + " usec\n";
    time_total += time_in_usec;


    //====== one-electron integrals ========

    start = std::chrono::steady_clock::now();

    // overlap, kinetic energy and nuclear attraction integral
    gsl_matrix *S = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *T = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *V = gsl_matrix_alloc(p_basis->num, p_basis->num);

    for (int a = 0; a < p_basis->num; ++ a)
    {
        for (int b = 0; b <= a; ++ b)
        {
            // overlap
            double s = calc_int_overlap(p_basis, a, b);

            // kinetic energy
            double t = calc_int_kinetic(p_basis, a, b);

            // nuclear repulsion
            double v = calc_int_nuc_attr(p_basis, a, b, p_atom);

            // save one-electron integrals in matrices
            gsl_matrix_set(S, a, b, s);
            gsl_matrix_set(T, a, b, t);
            gsl_matrix_set(V, a, b, v);
            if (a != b)
            {
                gsl_matrix_set(S, b, a, s);
                gsl_matrix_set(T, b, a, t);
                gsl_matrix_set(V, b, a, v);
            }
        }
    }

    end = std::chrono::steady_clock::now();
    time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    time_in_usec = time * 1e-3f;
    time_txt += "Time_1e_Ints  = " + std::to_string(time_in_usec) + " usec\n";
    time_total += time_in_usec;

    //====== allocate memory for arrays on host ========

    start = std::chrono::steady_clock::now();

    // number of primitive basis functions (pbf)
    int n_pbf = 0;
    for (int a = 0; a < p_basis->num; ++ a) {
        n_pbf += p_basis->nprims[a];
    }

    // number of unique pbf pairs
    int n_pbf_combi = n_pbf * (n_pbf + 1) / 2;

    size_t n_PBF_bytes  = sizeof(double) * n_pbf;
    size_t n_PBF_bytes_int = sizeof(int) * n_pbf;
    size_t n_PI_bytes   = sizeof(double) * n_pbf_combi;

    // pbf_xlec contains information of each pbf: x,y,z, lx,ly,lz, expon, coef*norm
    // pbf_to_cbf returns index of contracted basis function (cbf) of each pbf
    double *h_pbf_xlec = (double *)my_malloc(n_PBF_bytes * 8);
    int  *h_pbf_to_cbf = (int *)my_malloc(n_PBF_bytes_int);

    // mat_J_PI and mat_K_PI are primitive J and K matrices
    double *h_mat_J_PI = (double *)my_malloc(n_PI_bytes);
    double *h_mat_K_PI = (double *)my_malloc(n_PI_bytes);

    // counter for pbf_xlec; i_pbf for pbf_to_cbf
    int counter = 0;
    int i_pbf = 0;
    for (int a = 0; a < p_basis->num; ++ a) 
    {
        for (int i = 0; i < p_basis->nprims[a]; ++ i)
        {
            h_pbf_to_cbf[i_pbf] = a; ++ i_pbf;

            h_pbf_xlec[counter] = p_basis->xbas[a]; ++ counter;
            h_pbf_xlec[counter] = p_basis->ybas[a]; ++ counter;
            h_pbf_xlec[counter] = p_basis->zbas[a]; ++ counter;

            h_pbf_xlec[counter] = (double)p_basis->lx[a][i]; ++ counter;
            h_pbf_xlec[counter] = (double)p_basis->ly[a][i]; ++ counter;
            h_pbf_xlec[counter] = (double)p_basis->lz[a][i]; ++ counter;

            h_pbf_xlec[counter] = p_basis->expon[a][i]; ++ counter;
            h_pbf_xlec[counter] = p_basis->coef[a][i] * p_basis->norm[a][i]; ++ counter;
            // note that 'norm' is absorbed into 'coef'
        }
    }
    assert(counter == n_pbf * 8);


    // number of unique pairs of contracted basis functions
    int n_combi = p_basis->num * (p_basis->num + 1) / 2;

    size_t n_CI_bytes = sizeof(double) * n_combi;

    // D: density matrix
    // J: Coulomb matrix
    // K: exchange matrix
    // Q: Schwartz pre-screening matrix
    double *h_mat_D = (double *)my_malloc(n_CI_bytes);
    double *h_mat_J = (double *)my_malloc(n_CI_bytes);
    double *h_mat_K = (double *)my_malloc(n_CI_bytes);
    double *h_mat_Q = (double *)my_malloc(n_CI_bytes);


    // memory usage on device
    size_t mem_on_dev = n_PBF_bytes*8 + n_PBF_bytes_int + n_PI_bytes*2 + n_CI_bytes*2;
    fprintf(stdout, "Mem_on_Device = ");
    if   (mem_on_dev > 1000000000) { fprintf(stdout, "%zu GB\n", mem_on_dev / 1000000000); }
    else if (mem_on_dev > 1000000) { fprintf(stdout, "%zu MB\n", mem_on_dev / 1000000); }
    else if (mem_on_dev > 1000)    { fprintf(stdout, "%zu KB\n", mem_on_dev / 1000); }
    else                           { fprintf(stdout, "%zu B\n",  mem_on_dev); }


    end = std::chrono::steady_clock::now();
    time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    time_in_usec = time * 1e-3f;
    time_txt += "Time_2e_Prep  = " + std::to_string(time_in_usec) + " usec\n";
    time_total += time_in_usec;

    //====== start SCF calculation ========

    start = std::chrono::steady_clock::now();

    // NOTE: assume zero charge and closed-shell electronics structure
    int n_elec = 0;
    for (int iatom = 0; iatom < p_atom->num; ++ iatom) {
        n_elec += p_atom->nuc_chg[iatom];
    }

    if (n_elec % 2 != 0) {
        fprintf(stderr, "Error: Number of electrons (%d) is not even!\n", n_elec);
    }

    int n_occ = n_elec / 2;


    // get core Hamiltonian
    gsl_matrix *H_core = gsl_matrix_alloc(p_basis->num, p_basis->num);
    sum_H_core(p_basis->num, H_core, T, V);

    // get S^-1/2
    gsl_matrix *S_invsqrt = gsl_matrix_alloc(p_basis->num, p_basis->num);
    diag_overlap(p_basis->num, S, S_invsqrt);

#ifdef DEBUG
    printf("S:\n"); my_print_matrix(S);
    printf("T:\n"); my_print_matrix(T);
    printf("V:\n"); my_print_matrix(V);
    printf("H_core:\n"); my_print_matrix(H_core);
    printf("S^-1/2:\n"); my_print_matrix(S_invsqrt);
#endif

    // matrices, vector and variables to be used in SCF
    gsl_matrix *D      = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *D_prev = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *D_diff = gsl_matrix_alloc(p_basis->num, p_basis->num);

    gsl_matrix *Fock      = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *Fock_prev = gsl_matrix_alloc(p_basis->num, p_basis->num);

    gsl_matrix *Coef   = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_vector *emo    = gsl_vector_alloc(p_basis->num);

    double ene_elec, ene_total, ene_prev;

    // Coulomb(J) and exchange(K) matrices
    gsl_matrix *J = gsl_matrix_alloc(p_basis->num, p_basis->num);
    gsl_matrix *K = gsl_matrix_alloc(p_basis->num, p_basis->num);

    // initialize density matrix
    gsl_matrix_set_zero(D_prev);
    gsl_matrix_set_zero(D);
    ene_prev = 0.0;


    // Generalized Wolfsberg-Helmholtz initial guess
    init_guess_GWH(p_basis, H_core, S, Fock);
    Fock_to_Coef(p_basis->num, Fock, S_invsqrt, Coef, emo);
    Coef_to_Dens(p_basis->num, n_occ, Coef, D_prev);

    gsl_matrix_memcpy(D_diff, D_prev);
    gsl_matrix_memcpy(Fock_prev, Fock);


    // DIIS error and Fock matrices
    double ***diis_err  = (double ***)my_malloc(sizeof(double **) * MAX_DIIS_DIM);
    double ***diis_Fock = (double ***)my_malloc(sizeof(double **) * MAX_DIIS_DIM);
    int idiis, ibasis;
    for (idiis = 0; idiis < MAX_DIIS_DIM; ++ idiis)
    {
        diis_err[idiis]  = (double **)my_malloc(sizeof(double *) * p_basis->num);
        diis_Fock[idiis] = (double **)my_malloc(sizeof(double *) * p_basis->num);
        for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
        {
            diis_err[idiis][ibasis]  = (double *)my_malloc(sizeof(double) * p_basis->num);
            diis_Fock[idiis][ibasis] = (double *)my_malloc(sizeof(double) * p_basis->num);
        }
    }

    // DIIS index and dimension
    int diis_index = 0;
    int diis_dim = 0;
    double delta_DIIS;

    // mat_Q: sqrt(ab|ab) for prescreening of two-electron integrals
    for (int a = 0; a < p_basis->num; ++ a) {
        for (int b = 0; b <= a; ++ b) {
            h_mat_Q[ij2intindex(a,b)] = calc_int_eri_rys(p_basis, a, b, a, b);
        }
    }

    //====== allocate memory for arrays on device ========

    // initialize arrays on device
    double *dev_pbf_xlec;
    int    *dev_pbf_to_cbf;
    double *dev_mat_D, *dev_mat_Q, *dev_mat_J_PI, *dev_mat_K_PI;

    // allocate memories for arrays on device
    my_cuda_safe(hipMalloc((void**)&dev_pbf_xlec,    n_PBF_bytes * 8),"alloc_pbf_xlec");
    my_cuda_safe(hipMalloc((void**)&dev_pbf_to_cbf,  n_PBF_bytes_int),"alloc_pbf_to_cbf");
    my_cuda_safe(hipMalloc((void**)&dev_mat_J_PI, n_PI_bytes),"alloc_mat_J_PI");
    my_cuda_safe(hipMalloc((void**)&dev_mat_K_PI, n_PI_bytes),"alloc_mat_K_PI");
    my_cuda_safe(hipMalloc((void**)&dev_mat_D, n_CI_bytes),"alloc_D");
    my_cuda_safe(hipMalloc((void**)&dev_mat_Q, n_CI_bytes),"alloc_Q");

    // copy data from host to device
    my_cuda_safe(hipMemcpy(dev_pbf_xlec,   h_pbf_xlec,   n_PBF_bytes * 8, hipMemcpyHostToDevice),"mem_pbf_xlec");
    my_cuda_safe(hipMemcpy(dev_pbf_to_cbf, h_pbf_to_cbf, n_PBF_bytes_int, hipMemcpyHostToDevice),"mem_pbf_to_cbf");
    my_cuda_safe(hipMemcpy(dev_mat_Q, h_mat_Q, n_CI_bytes, hipMemcpyHostToDevice),"mem_Q");

    fprintf(stdout, "%5s %20s %20s %20s %20s\n",
            "Iter", "E_total", "delta_E", "rms_D", "delta_DIIS");

    end = std::chrono::steady_clock::now();
    time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    time_in_usec = time * 1e-3f;
    time_txt += "Time_SCF_Init = " + std::to_string(time_in_usec) + " usec\n";
    time_total += time_in_usec;


    // start SCF iterations

    start = std::chrono::steady_clock::now();

    int iter = 0;
    while (1)
    {
        /*------------------------------------*
         * SCF procedure:
         * Form new Fock matrix
         * F' = S^-1/2 * F * S^-1/2
         * diagonalize F' matrix to get C'
         * C = S^-1/2 * C'
         * compute new density matrix 
         *------------------------------------*/

        // when iter > 0, use incremental Fock matrix formation and DIIS
        int use_incr_fock = iter;
        int use_diis = 0;
        if (p_basis->num > 5) { use_diis = iter; } // sometimes DIIS does not work well


        // copy density matrix to device
        for (int a = 0; a < p_basis->num; ++ a) {
            for (int b = 0; b <= a; ++ b) {
                if (use_incr_fock) { h_mat_D[ij2intindex(a,b)] = gsl_matrix_get(D_diff,a,b); }
                else               { h_mat_D[ij2intindex(a,b)] = gsl_matrix_get(D_prev,a,b); }
            }
        }
        my_cuda_safe(hipMemcpy(dev_mat_D, h_mat_D, n_CI_bytes, hipMemcpyHostToDevice),"mem_D");


        // create 8x8 thread blocks
        dim3 block_size(BLOCKSIZE,BLOCKSIZE);
 
        // configure a two dimensional grid
        dim3 grid_size(n_pbf,n_pbf);


        // timer for J and K matrices
        auto kstart = std::chrono::steady_clock::now();

        // use 1T1PI for J and K matrices
        if (use_dp) {
            hipLaunchKernelGGL(cuda_mat_J_PI_dp, grid_size, block_size, 0, 0, dev_pbf_xlec, dev_pbf_to_cbf, n_pbf, dev_mat_D, dev_mat_J_PI, dev_mat_Q);
        } else {
            hipLaunchKernelGGL(cuda_mat_J_PI, grid_size, block_size, 0, 0, dev_pbf_xlec, dev_pbf_to_cbf, n_pbf, dev_mat_D, dev_mat_J_PI, dev_mat_Q);
        }

        hipDeviceSynchronize();
        auto kend = std::chrono::steady_clock::now();
        auto ktime = std::chrono::duration_cast<std::chrono::nanoseconds>(kend - kstart).count();
        time_in_usec = ktime * 1e-3f;
        time_mat_J += time_in_usec;

        my_cuda_safe(hipMemcpy(h_mat_J_PI, dev_mat_J_PI, n_PI_bytes, hipMemcpyDeviceToHost),"mem_mat_J_PI");

        kstart = std::chrono::steady_clock::now();

        if (use_dp) {
            hipLaunchKernelGGL(cuda_mat_K_PI_dp, grid_size, block_size, 0, 0, dev_pbf_xlec, dev_pbf_to_cbf, n_pbf, dev_mat_D, dev_mat_K_PI, dev_mat_Q);
        } else {
            hipLaunchKernelGGL(cuda_mat_K_PI, grid_size, block_size, 0, 0, dev_pbf_xlec, dev_pbf_to_cbf, n_pbf, dev_mat_D, dev_mat_K_PI, dev_mat_Q);
        }

        hipDeviceSynchronize();
        kend = std::chrono::steady_clock::now();
        ktime = std::chrono::duration_cast<std::chrono::nanoseconds>(kend - kstart).count();
        time_in_usec = ktime * 1e-3f;
        time_mat_K += time_in_usec;

        my_cuda_safe(hipMemcpy(h_mat_K_PI, dev_mat_K_PI, n_PI_bytes, hipMemcpyDeviceToHost),"mem_mat_K_PI");

        // sum up primitive J and K matrices to contracted ones
        for (int a = 0; a < p_basis->num; ++ a) {
            for (int b = 0; b <= a; ++ b) {
                int ab = ij2intindex(a,b);
                h_mat_J[ab] = 0.0;
                h_mat_K[ab] = 0.0;
            }
        }

        for (int i = 0; i < n_pbf; ++ i)
        {
            int a = h_pbf_to_cbf[i];
            for (int j = 0; j < n_pbf; ++ j)
            {
                int b = h_pbf_to_cbf[j];
                if (a < b) { continue; }

                int ab = ij2intindex(a,b);
                int ij = ij2intindex(i,j);
                h_mat_J[ab] += h_mat_J_PI[ij];
                h_mat_K[ab] += h_mat_K_PI[ij];
            }
        }


        // use J and K matrix from GPU
        for (int a = 0; a < p_basis->num; ++ a) {
            for (int b = 0; b < p_basis->num; ++ b) {
                int ab = ij2intindex(a,b);
                gsl_matrix_set(J,a,b,h_mat_J[ab]);
                gsl_matrix_set(K,a,b,h_mat_K[ab]);
            }
        }


#ifdef DEBUG
        printf("J:\n"); my_print_matrix(J);
        printf("K:\n"); my_print_matrix(K);
#endif

        if (use_incr_fock) { form_Fock(p_basis->num, Fock_prev, J, K, Fock); }
        else               { form_Fock(p_basis->num, H_core, J, K, Fock); }

        // save Fock_prev at this point, so as not to mix with DIIS
        gsl_matrix_memcpy(Fock_prev, Fock);


        // DIIS
        if (use_diis)
        {
            update_Fock_DIIS(&diis_dim, &diis_index, &delta_DIIS, 
                Fock, D_prev, S, p_basis, diis_err, diis_Fock);
        }

        // update density matrix and energies
        Fock_to_Coef(p_basis->num, Fock, S_invsqrt, Coef, emo);
        Coef_to_Dens(p_basis->num, n_occ, Coef, D);

        ene_elec = get_elec_ene(p_basis->num, D, H_core, Fock);
        ene_total = ene_nucl + ene_elec;

#ifdef DEBUG
        printf("F:\n"); my_print_matrix(Fock);
        printf("C:\n"); my_print_matrix(Coef);
        printf("P:\n"); my_print_matrix(D);
#endif

        // check convergence
        double delta_E = ene_total - ene_prev;

        double rms_D = 0.0;
        double dd_max = 0.0;
        int mu, nu;
        for (mu = 0; mu < p_basis->num; ++ mu)
        {
            for (nu = 0; nu < p_basis->num; ++ nu)
            {
                double dd = gsl_matrix_get(D, mu, nu) - 
                            gsl_matrix_get(D_prev, mu, nu);

                gsl_matrix_set(D_diff, mu, nu, dd);

                rms_D += dd * dd;
                if (fabs(dd) > dd_max) { dd_max = fabs(dd); }
            }
        }
        rms_D = sqrt(rms_D) / p_basis->num;

        fprintf(stdout, "%5d %20.10f", iter, ene_total);
        if (iter > 0) { fprintf(stdout, " %20.10f %20.10f", delta_E, rms_D); }
        if (use_diis && iter > 1) { fprintf(stdout, " %20.10f", delta_DIIS); }
        fprintf(stdout, "\n");

        // convergence criteria
        if (fabs(delta_E/ene_total) < 1.0e-9 && rms_D < 1.0e-7 && dd_max < 1.0e-6) { break; }

        // update energy and density matrix for the next iteration
        ene_prev = ene_total;
        gsl_matrix_memcpy(D_prev, D);

        // count iterations
        ++ iter;
    }

    // SCF converged
    fprintf(stdout, "SCF converged! E_total = %20.10f\n", ene_total);


    end = std::chrono::steady_clock::now();
    time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    time_in_usec = time * 1e-3f;
    time_txt += "Time_SCF_Conv = " + std::to_string(time_in_usec) + " usec\n";
    time_total += time_in_usec;


    // print MO information
    start = std::chrono::steady_clock::now();

    fprintf(stdout, "%5s %10s %15s %12s\n", "MO", "State", "E(Eh)", "E(eV)");
    for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
    {
        char occ[10];
        if (ibasis < n_occ) { strcpy(occ, "occ."); }
        else { strcpy(occ, "virt."); }

        double ener = gsl_vector_get(emo, ibasis);
        fprintf(stdout, "%5d %10s %15.5f %12.2f\n",
                ibasis + 1, occ, ener, ener * HARTREE2EV);
    }


    //====== free device memories ========

    hipFree(dev_pbf_xlec);
    hipFree(dev_pbf_to_cbf);
    hipFree(dev_mat_D);
    hipFree(dev_mat_Q);
    hipFree(dev_mat_J_PI);
    hipFree(dev_mat_K_PI);


    //====== free host memories ========

    free(h_mat_D);
    free(h_mat_Q);
    free(h_mat_J);
    free(h_mat_K);

    // free DIIS error and Fock matrices
    for (idiis = 0; idiis < MAX_DIIS_DIM; ++ idiis)
    {
        for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
        {
            free(diis_err[idiis][ibasis]);
            free(diis_Fock[idiis][ibasis]);
        }
        free(diis_err[idiis]);
        free(diis_Fock[idiis]);
    }
    free(diis_err);
    free(diis_Fock);

    // free arrays for one- and two-electron integral
    gsl_matrix_free(S);
    gsl_matrix_free(T);
    gsl_matrix_free(V);

    // free matrices and vector for SCF
    gsl_matrix_free(H_core);
    gsl_matrix_free(S_invsqrt);
    gsl_matrix_free(D_prev);
    gsl_matrix_free(Fock);
    gsl_matrix_free(Coef);
    gsl_matrix_free(D);
    gsl_vector_free(emo);

    gsl_matrix_free(D_diff);
    gsl_matrix_free(Fock_prev);

    gsl_matrix_free(J);
    gsl_matrix_free(K);

    // free arrays for geometry
    for (int iatom = 0; iatom < p_atom->num; ++ iatom)
    {
        free(p_atom->pos[iatom]);
        free(p_atom->name[iatom]);
    }
    free(p_atom->pos);
    free(p_atom->name);

    free(p_atom->nuc_chg);

    free(p_atom);

    // free arrays for basis set
    for (ibasis = 0; ibasis < p_basis->num; ++ ibasis)
    {
        free(p_basis->expon[ibasis]);
        free(p_basis->coef[ibasis]);
        free(p_basis->lx[ibasis]);
        free(p_basis->ly[ibasis]);
        free(p_basis->lz[ibasis]);
        free(p_basis->norm[ibasis]);
    }
    free(p_basis->expon);
    free(p_basis->coef);
    free(p_basis->lx);
    free(p_basis->ly);
    free(p_basis->lz);
    free(p_basis->norm);

    free(p_basis->xbas);
    free(p_basis->ybas);
    free(p_basis->zbas);

    free(p_basis->nprims);

    free(p_basis);

    end = std::chrono::steady_clock::now();
    time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    time_in_usec = time * 1e-3f;
    time_txt += "Time_Finalize = " + std::to_string(time_in_usec) + " usec\n";
    time_total += time_in_usec;

    std::cout << time_txt;
    std::cout << "Total time: " << time_total << " usec\n";

    std::cout << "Total kernel Mat_J time: " << time_mat_J << " usec\n";
    std::cout << "Total kernel Mat_K time: " << time_mat_K << " usec\n";


    //====== the end of program ========

    return 0;
}
