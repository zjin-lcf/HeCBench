//bondsEngine.cu
//Scott Grauer-Gray sgrauerg@gmail.com
//Contains main function for running bonds application on a GPU

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h> 
#include "bondsStructs.h"
#include "bondsKernelsGpu.cu"
#include "bondsKernelsCpu.cu"

#define MIN(a, b)  (((a) < (b)) ? (a) : (b))
#define MAX(a, b)  (((a) > (b)) ? (a) : (b))


int monthLengthCpu(int month, bool leapYear) 
{
  int MonthLength[] = {
    31, 28, 31, 30, 31, 30, 31, 31, 30, 31, 30, 31
  };

  int MonthLeapLength[] = {
    31, 29, 31, 30, 31, 30, 31, 31, 30, 31, 30, 31
  };

  return (leapYear? MonthLeapLength[month-1] : MonthLength[month-1]);
}


int monthOffsetCpu(int m, bool leapYear) 
{
  int MonthOffset[] = {
    0,  31,  59,  90, 120, 151,   // Jan - Jun
    181, 212, 243, 273, 304, 334,   // Jun - Dec
    365     // used in dayOfMonth to bracket day
  };

  int MonthLeapOffset[] = {
    0,  31,  60,  91, 121, 152,   // Jan - Jun
    182, 213, 244, 274, 305, 335,   // Jun - Dec
    366     // used in dayOfMonth to bracket day
  };

  return (leapYear? MonthLeapOffset[m-1] : MonthOffset[m-1]);
}


int yearOffsetCpu(int y)
{
  // the list of all December 31st in the preceding year
  // e.g. for 1901 yearOffset[1] is 366, that is, December 31 1900
  int YearOffset[] = {
    // 1900-1909
    0,  366,  731, 1096, 1461, 1827, 2192, 2557, 2922, 3288,
    // 1910-1919
    3653, 4018, 4383, 4749, 5114, 5479, 5844, 6210, 6575, 6940,
    // 1920-1929
    7305, 7671, 8036, 8401, 8766, 9132, 9497, 9862,10227,10593,
    // 1930-1939
    10958,11323,11688,12054,12419,12784,13149,13515,13880,14245,
    // 1940-1949
    14610,14976,15341,15706,16071,16437,16802,17167,17532,17898,
    // 1950-1959
    18263,18628,18993,19359,19724,20089,20454,20820,21185,21550,
    // 1960-1969
    21915,22281,22646,23011,23376,23742,24107,24472,24837,25203,
    // 1970-1979
    25568,25933,26298,26664,27029,27394,27759,28125,28490,28855,
    // 1980-1989
    29220,29586,29951,30316,30681,31047,31412,31777,32142,32508,
    // 1990-1999
    32873,33238,33603,33969,34334,34699,35064,35430,35795,36160,
    // 2000-2009
    36525,36891,37256,37621,37986,38352,38717,39082,39447,39813,
    // 2010-2019
    40178,40543,40908,41274,41639,42004,42369,42735,43100,43465,
    // 2020-2029
    43830,44196,44561,44926,45291,45657,46022,46387,46752,47118,
    // 2030-2039
    47483,47848,48213,48579,48944,49309,49674,50040,50405,50770,
    // 2040-2049
    51135,51501,51866,52231,52596,52962,53327,53692,54057,54423,
    // 2050-2059
    54788,55153,55518,55884,56249,56614,56979,57345,57710,58075,
    // 2060-2069
    58440,58806,59171,59536,59901,60267,60632,60997,61362,61728,
    // 2070-2079
    62093,62458,62823,63189,63554,63919,64284,64650,65015,65380,
    // 2080-2089
    65745,66111,66476,66841,67206,67572,67937,68302,68667,69033,
    // 2090-2099
    69398,69763,70128,70494,70859,71224,71589,71955,72320,72685,
    // 2100-2109
    73050,73415,73780,74145,74510,74876,75241,75606,75971,76337,
    // 2110-2119
    76702,77067,77432,77798,78163,78528,78893,79259,79624,79989,
    // 2120-2129
    80354,80720,81085,81450,81815,82181,82546,82911,83276,83642,
    // 2130-2139
    84007,84372,84737,85103,85468,85833,86198,86564,86929,87294,
    // 2140-2149
    87659,88025,88390,88755,89120,89486,89851,90216,90581,90947,
    // 2150-2159
    91312,91677,92042,92408,92773,93138,93503,93869,94234,94599,
    // 2160-2169
    94964,95330,95695,96060,96425,96791,97156,97521,97886,98252,
    // 2170-2179
    98617,98982,99347,99713,100078,100443,100808,101174,101539,101904,
    // 2180-2189
    102269,102635,103000,103365,103730,104096,104461,104826,105191,105557,
    // 2190-2199
    105922,106287,106652,107018,107383,107748,108113,108479,108844,109209,
    // 2200
    109574
  };

  return YearOffset[y-1900];
}


bool isLeapCpu(int y) 
{
  bool YearIsLeap[] = {
    // 1900 is leap in agreement with Excel's bug
    // 1900 is out of valid date range anyway
    // 1900-1909
    true,false,false,false, true,false,false,false, true,false,
    // 1910-1919
    false,false, true,false,false,false, true,false,false,false,
    // 1920-1929
    true,false,false,false, true,false,false,false, true,false,
    // 1930-1939
    false,false, true,false,false,false, true,false,false,false,
    // 1940-1949
    true,false,false,false, true,false,false,false, true,false,
    // 1950-1959
    false,false, true,false,false,false, true,false,false,false,
    // 1960-1969
    true,false,false,false, true,false,false,false, true,false,
    // 1970-1979
    false,false, true,false,false,false, true,false,false,false,
    // 1980-1989
    true,false,false,false, true,false,false,false, true,false,
    // 1990-1999
    false,false, true,false,false,false, true,false,false,false,
    // 2000-2009
    true,false,false,false, true,false,false,false, true,false,
    // 2010-2019
    false,false, true,false,false,false, true,false,false,false,
    // 2020-2029
    true,false,false,false, true,false,false,false, true,false,
    // 2030-2039
    false,false, true,false,false,false, true,false,false,false,
    // 2040-2049
    true,false,false,false, true,false,false,false, true,false,
    // 2050-2059
    false,false, true,false,false,false, true,false,false,false,
    // 2060-2069
    true,false,false,false, true,false,false,false, true,false,
    // 2070-2079
    false,false, true,false,false,false, true,false,false,false,
    // 2080-2089
    true,false,false,false, true,false,false,false, true,false,
    // 2090-2099
    false,false, true,false,false,false, true,false,false,false,
    // 2100-2109
    false,false,false,false, true,false,false,false, true,false,
    // 2110-2119
    false,false, true,false,false,false, true,false,false,false,
    // 2120-2129
    true,false,false,false, true,false,false,false, true,false,
    // 2130-2139
    false,false, true,false,false,false, true,false,false,false,
    // 2140-2149
    true,false,false,false, true,false,false,false, true,false,
    // 2150-2159
    false,false, true,false,false,false, true,false,false,false,
    // 2160-2169
    true,false,false,false, true,false,false,false, true,false,
    // 2170-2179
    false,false, true,false,false,false, true,false,false,false,
    // 2180-2189
    true,false,false,false, true,false,false,false, true,false,
    // 2190-2199
    false,false, true,false,false,false, true,false,false,false,
    // 2200
    false
  };

  return YearIsLeap[y-1900];
}


bondsDateStruct intializeDateCpu(int d, int m, int y) 
{
  bondsDateStruct currDate;

  currDate.day = d;
  currDate.month = m;
  currDate.year = y;

  bool leap = isLeapCpu(y);
  int offset = monthOffsetCpu(m,leap);

  currDate.dateSerialNum = d + offset + yearOffsetCpu(y);

  return currDate;
}

void runBoundsEngine(const int repeat)
{
  //can run multiple times with different number of bonds by uncommenting these lines
  int nBondsArray[] = {1000000};

  for (int numTime=0; numTime < 1; numTime++)
  {
    int numBonds = nBondsArray[numTime];  
    printf("\nNumber of Bonds: %d\n\n", numBonds);

    inArgsStruct inArgsHost;
    inArgsHost.discountCurve = (bondsYieldTermStruct*)malloc(numBonds*sizeof(bondsYieldTermStruct));
    inArgsHost.repoCurve = (bondsYieldTermStruct*)malloc(numBonds*sizeof(bondsYieldTermStruct));
    inArgsHost.currDate = (bondsDateStruct*)malloc(numBonds*sizeof(bondsDateStruct));
    inArgsHost.maturityDate = (bondsDateStruct*)malloc(numBonds*sizeof(bondsDateStruct));
    inArgsHost.bondCleanPrice = (dataType*)malloc(numBonds*sizeof(dataType));
    inArgsHost.bond = (bondStruct*)malloc(numBonds*sizeof(bondStruct));
    inArgsHost.dummyStrike = (dataType*)malloc(numBonds*sizeof(dataType));

    srand (123);

    int numBond;
    for (numBond = 0; numBond < numBonds; numBond++)
    {
      dataType repoRate = 0.07;

      //int repoSettlementDays = 0;
      int repoCompounding = SIMPLE_INTEREST;
      dataType repoCompoundFreq = 1;

      // assume a ten year bond- this is irrelevant
      bondsDateStruct bondIssueDate =  intializeDateCpu(rand() % 28 + 1, rand() % 12 + 1, 1999 - (rand() % 2));
      bondsDateStruct bondMaturityDate = intializeDateCpu(rand() % 28 + 1, rand() % 12 + 1, 2000 + (rand() % 2));

      bondsDateStruct todaysDate = intializeDateCpu(bondMaturityDate.day-1,bondMaturityDate.month,bondMaturityDate.year);

      bondStruct bond;
      bond.startDate = bondIssueDate;
      bond.maturityDate = bondMaturityDate;
      bond.rate = 0.08 + ((float)rand()/(float)RAND_MAX - 0.5)*0.1;

      dataType bondCouponFrequency = 2;

      dataType bondCleanPrice = 89.97693786;

      bondsYieldTermStruct bondCurve;

      bondCurve.refDate = todaysDate;
      bondCurve.calDate = todaysDate;
      bondCurve.forward = -0.1f;  // dummy rate
      bondCurve.compounding = COMPOUNDED_INTEREST;
      bondCurve.frequency = bondCouponFrequency;
      bondCurve.dayCounter = USE_EXACT_DAY;

      bondCurve.refDate = todaysDate;
      bondCurve.calDate = todaysDate;
      bondCurve.compounding = COMPOUNDED_INTEREST;
      bondCurve.frequency = bondCouponFrequency;

      dataType dummyStrike = 91.5745;

      bondsYieldTermStruct repoCurve;
      repoCurve.refDate = todaysDate;
      repoCurve.calDate = todaysDate;
      repoCurve.forward = repoRate;
      repoCurve.compounding = repoCompounding;
      repoCurve.frequency = repoCompoundFreq;
      repoCurve.dayCounter = USE_SERIAL_NUMS;

      inArgsHost.discountCurve[numBond] = bondCurve;
      inArgsHost.repoCurve[numBond] = repoCurve;
      inArgsHost.currDate[numBond] = todaysDate;
      inArgsHost.maturityDate[numBond] = bondMaturityDate;
      inArgsHost.bondCleanPrice[numBond] = bondCleanPrice;
      inArgsHost.bond[numBond] = bond;
      inArgsHost.dummyStrike[numBond] = dummyStrike;
    }
    printf("Inputs for bond with index %d\n", numBonds/2);
    printf("Bond Issue Date: %d-%d-%d\n", inArgsHost.bond[numBonds/2].startDate.month, 
                                          inArgsHost.bond[numBonds/2].startDate.day, 
                                          inArgsHost.bond[numBonds/2].startDate.year);
    printf("Bond Maturity Date: %d-%d-%d\n", inArgsHost.bond[numBonds/2].maturityDate.month, 
                                          inArgsHost.bond[numBonds/2].maturityDate.day, 
                                          inArgsHost.bond[numBonds/2].maturityDate.year);
    printf("Bond rate: %f\n\n", inArgsHost.bond[numBonds/2].rate);

    resultsStruct resultsHost;
    resultsStruct resultsFromGpu;

    resultsHost.dirtyPrice = (dataType*)malloc(numBonds*sizeof(dataType));
    resultsHost.accruedAmountCurrDate = (dataType*)malloc(numBonds*sizeof(dataType));;
    resultsHost.cleanPrice = (dataType*)malloc(numBonds*sizeof(dataType));;
    resultsHost.bondForwardVal = (dataType*)malloc(numBonds*sizeof(dataType));;

    resultsFromGpu.dirtyPrice = (dataType*)malloc(numBonds*sizeof(dataType));
    resultsFromGpu.accruedAmountCurrDate = (dataType*)malloc(numBonds*sizeof(dataType));;
    resultsFromGpu.cleanPrice = (dataType*)malloc(numBonds*sizeof(dataType));;
    resultsFromGpu.bondForwardVal = (dataType*)malloc(numBonds*sizeof(dataType));;

    long ktimeGpu = 0;
    double timeCpu;
    double timeGpu;

    struct timeval start;
    struct timeval end;

    gettimeofday(&start, NULL);

    for (int i = 0; i < repeat; i++)
      ktimeGpu += getBondsResultsGpu(inArgsHost, resultsFromGpu, numBonds);

    gettimeofday(&end, NULL);
    timeGpu = (end.tv_sec - start.tv_sec) * 1e6 + end.tv_usec - start.tv_usec;

    printf("Run on GPU\n");
    printf("Average kernel execution time on GPU: %lf (ms)  \n\n", ktimeGpu * 1e-3 / repeat);
    printf("Average processing time on GPU: %f (ms)  \n\n", timeGpu * 1e-3 / repeat);

    double totPrice = 0.0;
    int numBond1;
    for (numBond1= 0; numBond1< numBonds; numBond1++)
    {
      totPrice += resultsFromGpu.dirtyPrice[numBond1];
    }

    printf("Sum of output dirty prices on GPU: %f\n", totPrice);
    printf("Outputs on GPU for bond with index %d: \n", numBonds/2);
    printf("Dirty Price: %f\n", resultsFromGpu.dirtyPrice[numBonds/2]);
    printf("Accrued Amount: %f\n", resultsFromGpu.accruedAmountCurrDate[numBonds/2]);
    printf("Clean Price: %f\n", resultsFromGpu.cleanPrice[numBonds/2]);
    printf("Bond Forward Val: %f\n\n", resultsFromGpu.bondForwardVal[numBonds/2]);

    gettimeofday(&start, NULL);

    for (int i = 0; i < 2; i++)
      getBondsResultsCpu(inArgsHost, resultsHost, numBonds);

    gettimeofday(&end, NULL);
    timeCpu = (end.tv_sec - start.tv_sec) * 1e6 + end.tv_usec - start.tv_usec;

    printf("Run on CPU\n");
    printf("Average processing time on CPU: %lf (ms)  \n\n", timeCpu * 1e-3 / 2);

    totPrice = 0.0;
    for (numBond1= 0; numBond1< numBonds; numBond1++)
    {
      totPrice += resultsHost.dirtyPrice[numBond1];
    }
    printf("Sum of output dirty prices on CPU: %f\n", totPrice);
    printf("Outputs on CPU for bond with index %d: \n", numBonds/2);
    printf("Dirty Price: %f\n", resultsHost.dirtyPrice[numBonds/2]);
    printf("Accrued Amount: %f\n", resultsHost.accruedAmountCurrDate[numBonds/2]);
    printf("Clean Price: %f\n", resultsHost.cleanPrice[numBonds/2]);
    printf("Bond Forward Val: %f\n\n", resultsHost.bondForwardVal[numBonds/2]);

    printf("Speedup using GPU: %f\n", (timeCpu / 2) / (timeGpu / repeat) );

    free(resultsHost.dirtyPrice);
    free(resultsHost.accruedAmountCurrDate);;
    free(resultsHost.cleanPrice);;
    free(resultsHost.bondForwardVal);;

    free(resultsFromGpu.dirtyPrice);
    free(resultsFromGpu.accruedAmountCurrDate);;
    free(resultsFromGpu.cleanPrice);;
    free(resultsFromGpu.bondForwardVal);

    free(inArgsHost.discountCurve);
    free(inArgsHost.repoCurve);
    free(inArgsHost.currDate);
    free(inArgsHost.maturityDate);
    free(inArgsHost.bondCleanPrice);
    free(inArgsHost.bond);
    free(inArgsHost.dummyStrike);
  }
}

int main(int argc, char* argv[])
{
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);
  runBoundsEngine(repeat);
  return 0;
}
