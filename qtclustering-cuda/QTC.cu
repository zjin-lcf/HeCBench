#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <chrono>
#include <sstream>
#include <iostream>
#include <fstream>
#include "tuningParameters.h"
#include "qtclib.h"
#include "OptionParser.h"
#include "libdata.h"

#include "cudacommon.h"
#define _USE_MATH_DEFINES
#include <float.h>
#include "comm.h"


using namespace std;

#include "kernels_common.h"
#include "kernels_compact_storage.h"

// ****************************************************************************
// Function: addBenchmarkSpecOptions
//
// Purpose:
//   Add benchmark specific options parsing.  The user is allowed to specify
//   the size of the input data in megabytes if they are not using a
//   predefined size (i.e. the -s option).
//
// Arguments:
//   op: the options parser / parameter database
//
// Programmer: Anthony Danalis
// Creation: February 04, 2011
// Returns:  nothing
//
// ****************************************************************************
void addBenchmarkSpecOptions(OptionParser &op){
  op.addOption("PointCount", OPT_INT, "4096", "point count (default: 4096)");
  op.addOption("Threshold", OPT_FLOAT, "1", "cluster diameter threshold (default: 1)");
  op.addOption("SaveOutput", OPT_BOOL, "", "Save output results in files (default: false)");
  op.addOption("Verbose", OPT_BOOL, "", "Print cluster cardinalities (default: false)");
}

// ****************************************************************************
// Function: RunBenchmark
//
// Purpose:
//   Calls single precision and, if viable, double precision QT-Clustering
//   benchmark.
//
// Arguments:
//  resultDB: the benchmark stores its results in this ResultDatabase
//  op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Anthony Danalis
// Creation: February 04, 2011
//
// ****************************************************************************
void runTest(const string& name, OptionParser& op);

void RunBenchmark(OptionParser &op){
  runTest("QTC", op);
}

// ****************************************************************************
// Function: calculate_participants
//
// Purpose:
//   This function decides how many GPUs (up to the maximum requested by the user)
//   and threadblocks per GPU will be used. It also returns the total number of
//   thread-blocks across all GPUs and the number of thread-blocks that are in nodes
//   before the current one.
//   In the future, the behavior of this function should be decided based on
//   auto-tuning instead of arbitrary decisions.
//
// Arguments:
//   The number of nodes requested by the user and the four
//   variables that the function computes (passed by reference)
//
//
// Returns:  nothing
//
// Programmer: Anthony Danalis
// Creation: May 25, 2011
//
// ****************************************************************************
void calculate_participants(int point_count, int node_count, int cwrank,
                            int *thread_block_count,
                            int *total_thread_block_count,
                            int *active_node_count){

  int ac_nd_cnt, thr_blc_cnt, total_thr_blc_cnt;

  ac_nd_cnt = node_count;
  if( point_count <= (node_count-1) * SM_COUNT * GPU_MIN_SATURATION_FACTOR ){
    int K = SM_COUNT * GPU_MIN_SATURATION_FACTOR;
    ac_nd_cnt = (point_count+K-1) / K;
  }

  if( point_count >= ac_nd_cnt * SM_COUNT * OVR_SBSCR_FACTOR ){
    thr_blc_cnt = SM_COUNT * OVR_SBSCR_FACTOR;
    total_thr_blc_cnt = thr_blc_cnt * ac_nd_cnt;
  }else{
    thr_blc_cnt = point_count/ac_nd_cnt;
    if( cwrank < point_count%ac_nd_cnt ){
      thr_blc_cnt++;
    }
    total_thr_blc_cnt = point_count;
  }

  *active_node_count  = ac_nd_cnt;
  *thread_block_count = thr_blc_cnt;
  *total_thread_block_count = total_thr_blc_cnt;

  return;
}

// ****************************************************************************
// Function: runTest
//
// Purpose:
//   This benchmark measures the performance of applying QT-clustering on
//   single precision data.
//
// Arguments:
//  resultDB: the benchmark stores its results in this ResultDatabase
//  op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Anthony Danalis
// Creation: February 04, 2011
//
// ****************************************************************************

void runTest(const string& name, OptionParser& op)
{
  int matrix_type = 0x0;
  if( 0 == comm_get_rank() ){
    matrix_type |= GLOBAL_MEMORY;

    // find out what type of distance matrix we will be using.
    matrix_type |= COMPACT_STORAGE_MATRIX;
  }
  comm_broadcast ( &matrix_type, 1, COMM_TYPE_INT, 0);

  QTC(name, op, matrix_type);

}

void QTC(const string& name, OptionParser& op, int matrix_type){
  ofstream debug_out, seeds_out;
  void *Ai_mask, *cardnl, *ungrpd_pnts_indr, *clustered_pnts_mask, *result, *dist_to_clust;
  void *indr_mtrx, *degrees;
  int *indr_mtrx_host, *ungrpd_pnts_indr_host, *cardinalities, *output;
  bool save_clusters = false;
  bool be_verbose = false;
  void *distance_matrix_gmem, *distance_matrix;
  float *dist_source, *pnts = NULL;
  float threshold = 1.0f;
  unsigned long int i;
  int max_degree, thread_block_count, total_thread_block_count, active_node_count;
  int cwrank=0, node_count=1, tpb, max_card, iter=0;
  unsigned long int dst_matrix_elems, point_count, max_point_count;

  point_count = op.getOptionInt("PointCount");
  threshold = op.getOptionFloat("Threshold");
  save_clusters = op.getOptionBool("SaveOutput");
  be_verbose = op.getOptionBool("Verbose");


  // TODO - only deal with this size-switch once
  int def_size = op.getOptionInt("size");
  switch( def_size ) {
    case 1:
      // size == 1 should match default values of PointCount,
      // Threshold, TextureMem, and CompactStorage parameters.
      // (i.e., -s 1 is the default)
      point_count    = 4*1024;
      break;
    case 2:
      point_count    = 8*1024;
      break;
    case 3:
      point_count    = 16*1024;
      break;
    case 4:
      point_count    = 16*1024;
      break;
    case 5:
      point_count    = 26*1024;
      break;
    default:
      fprintf( stderr, "unsupported size %d given; terminating\n", def_size );
      return;
  }

  cwrank = comm_get_rank();
  node_count = comm_get_size();

  if( cwrank == 0 ){
    pnts = generate_synthetic_data(&dist_source, &indr_mtrx_host, &max_degree, threshold, point_count, matrix_type);
  }

  comm_broadcast ( &point_count, 1, COMM_TYPE_INT, 0);
  comm_broadcast ( &max_degree, 1, COMM_TYPE_INT, 0);

  dst_matrix_elems = point_count*max_degree;

  if( cwrank != 0 ){ // For all nodes except zero, in a distributed run.
    dist_source = (float*) malloc (sizeof(float)*dst_matrix_elems);
    indr_mtrx_host = (int*) malloc (sizeof(int)*point_count*max_degree);
  }
  // If we need to print the actual clusters later on, we'll need to have all points in all nodes.
  if( save_clusters ){
    if( cwrank != 0 ){
      pnts = (float *)malloc( 2*point_count*sizeof(float) );
    }
    comm_broadcast ( pnts, 2*point_count, COMM_TYPE_FLOAT, 0);
  }

  comm_broadcast ( dist_source, dst_matrix_elems, COMM_TYPE_FLOAT, 0);
  comm_broadcast ( indr_mtrx_host, point_count*max_degree, COMM_TYPE_INT, 0);

  assert( max_degree > 0 );

  calculate_participants(point_count, node_count, cwrank, &thread_block_count, &total_thread_block_count, &active_node_count);

  ungrpd_pnts_indr_host = (int*) malloc (sizeof(int)*point_count);
  for(i=0; i<point_count; i++){
    ungrpd_pnts_indr_host[i] = i;
  }

  cardinalities = (int*) malloc (sizeof(int)*2);
  output = (int*) malloc (sizeof(int)*max_degree);

  allocDeviceBuffer(&distance_matrix_gmem, dst_matrix_elems*sizeof(float));
  CHECK_CUDA_ERROR();

  // This is the N*Delta indirection matrix
  allocDeviceBuffer(&indr_mtrx, point_count*max_degree*sizeof(int));

  allocDeviceBuffer(&degrees,             point_count*sizeof(int));
  allocDeviceBuffer(&ungrpd_pnts_indr,    point_count*sizeof(int));
  allocDeviceBuffer(&Ai_mask,             thread_block_count*point_count*sizeof(char));
  allocDeviceBuffer(&dist_to_clust,       thread_block_count*max_degree*sizeof(float));
  allocDeviceBuffer(&clustered_pnts_mask, point_count*sizeof(char));
  allocDeviceBuffer(&cardnl,              thread_block_count*2*sizeof(int));
  allocDeviceBuffer(&result,              point_count*sizeof(int));
#ifdef DEBUG
    int* cardinalities_debug = (int*) malloc (sizeof(int)*thread_block_count*2);
#endif

  // Copy to device, and record transfer time
  copyToDevice(distance_matrix_gmem, dist_source, dst_matrix_elems*sizeof(float));
  copyToDevice(indr_mtrx, indr_mtrx_host, point_count*max_degree*sizeof(int));
  copyToDevice(ungrpd_pnts_indr, ungrpd_pnts_indr_host, point_count*sizeof(int));
  hipMemset(clustered_pnts_mask, 0, point_count*sizeof(char));
  hipMemset(dist_to_clust, 0, max_degree*thread_block_count*sizeof(float));

  tpb = ( point_count > THREADSPERBLOCK )? THREADSPERBLOCK : point_count;
  compute_degrees<<<thread_block_count, tpb>>>((int *)indr_mtrx, (int *)degrees, point_count, max_degree);
  hipDeviceSynchronize();
  CHECK_CUDA_ERROR();

  // The names of the saved outputs, if enabled, are "p", "p_seeds", and "p."
  if( 0 == cwrank ){
    if( save_clusters ){
      debug_out.open("p");
      for(i=0; i<point_count; i++){
        debug_out << pnts[2*i] << " " << pnts[2*i+1] << endl;
      }
      debug_out.close();
      seeds_out.open("p_seeds");
    }

    cout << "\nInitial ThreadBlockCount: " << thread_block_count;
    cout << " PointCount: " << point_count;
    cout << " Max degree: " << max_degree << "\n" << endl;
    cout.flush();
  }

  max_point_count = point_count;

  tpb = THREADSPERBLOCK;

  distance_matrix = distance_matrix_gmem;

  // Kernel execution
  double qtc_time = 0.0, trim_time = 0.0, update_time = 0.0;

  do{
    stringstream ss;
    int winner_node=-1;
    int winner_index=-1;
    bool this_node_participates = true;

    ++iter;

    calculate_participants(point_count, node_count, cwrank, &thread_block_count, &total_thread_block_count, &active_node_count);

    // If there are only a few elements left to cluster, reduce the number of participating nodes (GPUs).
    if( cwrank >= active_node_count ){
      this_node_participates = false;
    }
    comm_update_communicator(cwrank, active_node_count);
    if( !this_node_participates )
      break;
    cwrank = comm_get_rank();

    auto start = std::chrono::steady_clock::now();

    // Main kernel
    QTC_device<<<thread_block_count, tpb>>>((float*)distance_matrix, (char *)Ai_mask, (char *)clustered_pnts_mask,
        (int *)indr_mtrx, (int *)cardnl, (int *)ungrpd_pnts_indr,
        (float *)dist_to_clust, (int *)degrees, point_count, max_point_count,
        max_degree, threshold, cwrank, active_node_count,
        total_thread_block_count);

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    qtc_time += std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

    CHECK_CUDA_ERROR();

#ifdef DEBUG
    printf("cardinalities\n");
    copyFromDevice( cardinalities_debug, cardnl, 2*576*sizeof(int) );
    for (int i = 0; i < 576*2; i++)
     printf("%d %d\n", i, cardinalities_debug[i]);
#endif

    if( thread_block_count > 1 ){
      // We are reducing 128 numbers or less, so one thread should be sufficient.
      reduce_card_device<<<1, 1>>>((int *)cardnl, thread_block_count);
      hipDeviceSynchronize();
      CHECK_CUDA_ERROR();
    }

    copyFromDevice( cardinalities, cardnl, 2*sizeof(int) );
    max_card     = cardinalities[0];
    winner_index = cardinalities[1];

    comm_barrier();

    comm_find_winner(&max_card, &winner_node, &winner_index, cwrank, max_point_count+1);

    if( be_verbose && cwrank == winner_node){ // for non-parallel cases, both "cwrank" and "winner_node" should be zero.
      cout << "[" << cwrank << "] Cluster Cardinality: " << max_card << " (Node: " << cwrank << ", index: " << winner_index << ")" << endl;
    }

    start = std::chrono::steady_clock::now();

    trim_ungrouped_pnts_indr_array<<<1, tpb>>>(winner_index, (int*)ungrpd_pnts_indr, (float*)distance_matrix,
        (int *)result, (char *)Ai_mask, (char *)clustered_pnts_mask,
        (int *)indr_mtrx, (int *)cardnl, (float *)dist_to_clust, (int *)degrees,
        point_count, max_point_count, max_degree, threshold);
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR();

    end = std::chrono::steady_clock::now();
    trim_time += std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

    if( cwrank == winner_node){ // for non-parallel cases, these should both be zero.
      if( save_clusters ){
        ss << "p." << iter;
        debug_out.open(ss.str().c_str());
      }

      copyFromDevice(output, (void *)result, max_card*sizeof(int) );

      if( save_clusters ){
        for(int i=0; i<max_card; i++){
          debug_out << pnts[2*output[i]] << " " << pnts[2*output[i]+1] << endl;
        }
        seeds_out << pnts[2*winner_index] << " " << pnts[2*winner_index+1] << endl;
        debug_out.close();
      }
    }

    start = std::chrono::steady_clock::now();

    update_clustered_pnts_mask<<<1, tpb>>>((char *)clustered_pnts_mask, (char *)Ai_mask, max_point_count);
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR();

    end = std::chrono::steady_clock::now();
    update_time += std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

    point_count -= max_card;

  }while( max_card > 1 && point_count );


  if( save_clusters ){
    seeds_out.close();
  }

  if( cwrank == 0){
    cout << "QTC is complete. Clustering iteration count: " << iter << endl;
    cout << "\nKernel execution time\n";
    cout << "qtc: " << qtc_time * 1e-9f << " (s)\n";
    cout << "trim: " << trim_time * 1e-9f << " (s)\n";
    cout << "update: " << update_time * 1e-9f << " (s)\n";
    cout << "total: " << (qtc_time + trim_time + update_time) * 1e-9f << " (s)\n";
    cout.flush();
  }

  free(dist_source);
  free(indr_mtrx_host);
  free(output);
  free(cardinalities);
#ifdef DEBUG
  free(cardinalities_debug);
#endif

  freeDeviceBuffer(distance_matrix_gmem);
  freeDeviceBuffer(indr_mtrx);
  freeDeviceBuffer(degrees);
  freeDeviceBuffer(ungrpd_pnts_indr);
  freeDeviceBuffer(Ai_mask);
  freeDeviceBuffer(dist_to_clust);
  freeDeviceBuffer(clustered_pnts_mask);
  freeDeviceBuffer(cardnl);
  freeDeviceBuffer(result);

  return;
}

////////////////////////////////////////////////////////////////////////////////

void allocDeviceBuffer(void** bufferp, unsigned long bytes)
{
  hipMalloc(bufferp, bytes);
  CHECK_CUDA_ERROR();
}

void freeDeviceBuffer(void* buffer)
{
  hipFree(buffer);
}

void copyToDevice(void* to_device, void* from_host, unsigned long bytes)
{
  hipMemcpy(to_device, from_host, bytes, hipMemcpyHostToDevice);
  CHECK_CUDA_ERROR();
}

void copyFromDevice(void* to_host, void* from_device, unsigned long bytes)
{
  hipMemcpy(to_host, from_device, bytes, hipMemcpyDeviceToHost);
  CHECK_CUDA_ERROR();
}
