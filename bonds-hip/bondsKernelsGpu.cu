#include "hip/hip_runtime.h"
//bondsKernelsGpu.cu
//Scott Grauer-Gray
//Bonds kernels to run on the GPU

#include <hip/hip_runtime.h>
#include "bondsKernelsGpu.h"


__device__ int monthLengthKernelGpu(int month, bool leapYear) 
{
  int MonthLength[12];
  MonthLength[0]=31;
  MonthLength[1]=28;
  MonthLength[2]=31;
  MonthLength[3]=30;
  MonthLength[4]=31;
  MonthLength[5]=30;
  MonthLength[6]=31;
  MonthLength[7]=31;
  MonthLength[8]=30;
  MonthLength[9]=31;
  MonthLength[10]=30;
  MonthLength[11]=31;

  int MonthLeapLength[12];
  MonthLeapLength[0]=31;
  MonthLeapLength[1]=29;
  MonthLeapLength[2]=31;
  MonthLeapLength[3]=30;
  MonthLeapLength[4]=31;
  MonthLeapLength[5]=30;
  MonthLeapLength[6]=31;
  MonthLeapLength[7]=31;
  MonthLeapLength[8]=30;
  MonthLeapLength[9]=31;
  MonthLeapLength[10]=30;
  MonthLeapLength[11]=31;

  return (leapYear? MonthLeapLength[month-1] : MonthLength[month-1]);
}


__device__ int monthOffsetKernelGpu(int m, bool leapYear) 
{
  int MonthOffset[13];
  MonthOffset[0]=0;
  MonthOffset[1]=31;
  MonthOffset[2]=59;
  MonthOffset[3]=90;
  MonthOffset[4]=120;
  MonthOffset[5]=151;
  MonthOffset[6]=181;
  MonthOffset[7]=212;
  MonthOffset[8]=243;
  MonthOffset[9]=273;
  MonthOffset[10]=304;
  MonthOffset[11]=334;
  MonthOffset[12]=365;

  int MonthLeapOffset[13];
  MonthLeapOffset[0]=0;
  MonthLeapOffset[1]=31;
  MonthLeapOffset[2]=60;
  MonthLeapOffset[3]=91;
  MonthLeapOffset[4]=121;
  MonthLeapOffset[5]=152;
  MonthLeapOffset[6]=182;
  MonthLeapOffset[7]=213;
  MonthLeapOffset[8]=244;
  MonthLeapOffset[9]=274;
  MonthLeapOffset[10]=305;
  MonthLeapOffset[11]=335;
  MonthLeapOffset[12]=366;

  return (leapYear? MonthLeapOffset[m-1] : MonthOffset[m-1]);
}


__device__ int yearOffsetKernelGpu(int y)
{

  int YearOffset[121];
  YearOffset[0] = 0;;
  YearOffset[1] = 366;;
  YearOffset[2] = 731;
  YearOffset[3] = 1096;
  YearOffset[4] = 1461;
  YearOffset[5] = 1827;
  YearOffset[6] = 2192;
  YearOffset[7] = 2557;
  YearOffset[8] = 2922;
  YearOffset[9] = 3288;
  YearOffset[10] = 3653;
  YearOffset[11] = 4018;
  YearOffset[12] = 4383;
  YearOffset[13] = 4749;
  YearOffset[14] = 5114;
  YearOffset[15] = 5479;
  YearOffset[16] = 5844;
  YearOffset[17] = 6210;
  YearOffset[18] = 6575;
  YearOffset[19] = 6940;
  YearOffset[20] = 7305;
  YearOffset[21] = 7671;
  YearOffset[22] = 8036;
  YearOffset[23] = 8401;
  YearOffset[24] = 8766;
  YearOffset[25] = 9132;
  YearOffset[26] = 9497;
  YearOffset[27] = 9862;
  YearOffset[28] = 10227;
  YearOffset[29] = 10593;
  YearOffset[30] = 10958;
  YearOffset[31] = 11323;
  YearOffset[32] = 11688;
  YearOffset[33] = 12054;
  YearOffset[34] = 12419;
  YearOffset[35] = 12784;
  YearOffset[36] = 13149;
  YearOffset[37] = 13515;
  YearOffset[38] = 13880;
  YearOffset[39] = 14245;
  YearOffset[40] = 14610;
  YearOffset[41] = 14976;
  YearOffset[42] = 15341;
  YearOffset[43] = 15706;
  YearOffset[44] = 16071;
  YearOffset[45] = 16437;
  YearOffset[46] = 16802;
  YearOffset[47] = 17167;
  YearOffset[48] = 17532;
  YearOffset[49] = 17898;
  YearOffset[50] = 18263;
  YearOffset[51] = 18628;
  YearOffset[52] = 18993;
  YearOffset[53] = 19359;
  YearOffset[54] = 19724;
  YearOffset[55] = 20089;
  YearOffset[56] = 20454;
  YearOffset[57] = 20820;
  YearOffset[58] = 21185;
  YearOffset[59] = 21550;
  YearOffset[60] = 21915;
  YearOffset[61] = 22281;
  YearOffset[62] = 22646;
  YearOffset[63] = 23011;
  YearOffset[64] = 23376;
  YearOffset[65] = 23742;
  YearOffset[66] = 24107;
  YearOffset[67] = 24472;
  YearOffset[68] = 24837;
  YearOffset[69] = 25203;
  YearOffset[70] = 25568;
  YearOffset[71] = 25933;
  YearOffset[72] = 26298;
  YearOffset[73] = 26664;
  YearOffset[74] = 27029;
  YearOffset[75] = 27394;
  YearOffset[76] = 27759;
  YearOffset[77] = 28125;
  YearOffset[78] = 28490;
  YearOffset[79] = 28855;
  YearOffset[80] = 29220;
  YearOffset[81] = 29586;
  YearOffset[82] = 29951;
  YearOffset[83] = 30316;
  YearOffset[84] = 30681;
  YearOffset[85] = 31047;
  YearOffset[86] = 31412;
  YearOffset[87] = 31777;
  YearOffset[88] = 32142;
  YearOffset[89] = 32508;
  YearOffset[90] = 32873;
  YearOffset[91] = 33238;
  YearOffset[92] = 33603;
  YearOffset[93] = 33969;
  YearOffset[94] = 34334;
  YearOffset[95] = 34699;
  YearOffset[96] = 35064;
  YearOffset[97] = 35430;
  YearOffset[98] = 35795;
  YearOffset[99] = 36160;
  YearOffset[100] = 36525;
  YearOffset[101] = 36891;
  YearOffset[102] = 37256;
  YearOffset[103] = 37621;
  YearOffset[104] = 37986;
  YearOffset[105] = 38352;
  YearOffset[106] = 38717;
  YearOffset[107] = 39082;
  YearOffset[108] = 39447;
  YearOffset[109] = 39813;
  YearOffset[110] = 40178;
  YearOffset[111] = 40543;
  YearOffset[112] = 40908;
  YearOffset[113] = 41274;
  YearOffset[114] = 41639;
  YearOffset[115] = 42004;
  YearOffset[116] = 42369;
  YearOffset[117] = 42735;
  YearOffset[118] = 43100;
  YearOffset[119] = 42735;
  YearOffset[120] = 43830;

  return YearOffset[y-1900];
}


__device__ bool isLeapKernelGpu(int y) 
{
  bool YearIsLeap[121];

  YearIsLeap[0] = 1;;
  YearIsLeap[1] = 0;;
  YearIsLeap[2] = 0;
  YearIsLeap[3] = 0;//1096;
  YearIsLeap[4] = 1;//1461;
  YearIsLeap[5] = 0;//1827;
  YearIsLeap[6] = 0;//2192;
  YearIsLeap[7] = 0;//2557;
  YearIsLeap[8] = 1;//2922;
  YearIsLeap[9] = 0;//3288;
  YearIsLeap[10] = 0;//3653;
  YearIsLeap[11] = 0;//4018;
  YearIsLeap[12] = 1;//4383;
  YearIsLeap[13] = 0;//4749;
  YearIsLeap[14] = 0;//5114;
  YearIsLeap[15] = 0;//5479;
  YearIsLeap[16] = 1;//5844;
  YearIsLeap[17] = 0;//6210;
  YearIsLeap[18] = 0;//6575;
  YearIsLeap[19] = 0;//6940;
  YearIsLeap[20] = 1;//7305;
  YearIsLeap[21] = 0;//7671;
  YearIsLeap[22] = 0;//8036;
  YearIsLeap[23] = 0;//8401;
  YearIsLeap[24] = 1;//8766;
  YearIsLeap[25] = 0;//9132;
  YearIsLeap[26] = 0;//9497;
  YearIsLeap[27] = 0;//9862;
  YearIsLeap[28] = 1;//10227;
  YearIsLeap[29] = 0;//10593;
  YearIsLeap[30] = 0;//10958;
  YearIsLeap[31] = 0;//11323;
  YearIsLeap[32] = 1;//11688;
  YearIsLeap[33] = 0;//12054;
  YearIsLeap[34] = 0;//12419;
  YearIsLeap[35] = 0;//12784;
  YearIsLeap[36] = 1;//13149;
  YearIsLeap[37] = 0;//13515;
  YearIsLeap[38] = 0;//13880;
  YearIsLeap[39] = 0;//14245;
  YearIsLeap[40] = 1;//14610;
  YearIsLeap[41] = 0;//14976;
  YearIsLeap[42] = 0;//15341;
  YearIsLeap[43] = 0;//15706;
  YearIsLeap[44] = 1;//16071;
  YearIsLeap[45] = 0;//16437;
  YearIsLeap[46] = 0;//16802;
  YearIsLeap[47] = 0;//17167;
  YearIsLeap[48] = 1;//17532;
  YearIsLeap[49] = 0;//17898;
  YearIsLeap[50] = 0;//18263;
  YearIsLeap[51] = 0;//18628;
  YearIsLeap[52] = 1;//18993;
  YearIsLeap[53] = 0;//19359;
  YearIsLeap[54] = 0;//19724;
  YearIsLeap[55] = 0;//20089;
  YearIsLeap[56] = 1;//20454;
  YearIsLeap[57] = 0;//20820;
  YearIsLeap[58] = 0;//21185;
  YearIsLeap[59] = 0;//21550;
  YearIsLeap[60] = 1;//21915;
  YearIsLeap[61] = 0;//22281;
  YearIsLeap[62] = 0;//22646;
  YearIsLeap[63] = 0;//23011;
  YearIsLeap[64] = 1;//23376;
  YearIsLeap[65] = 0;//23742;
  YearIsLeap[66] = 0;//24107;
  YearIsLeap[67] = 0;//24472;
  YearIsLeap[68] = 1;//24837;
  YearIsLeap[69] = 0;//25203;
  YearIsLeap[70] = 0;//25568;
  YearIsLeap[71] = 0;//25933;
  YearIsLeap[72] = 1;//26298;
  YearIsLeap[73] = 0;//26664;
  YearIsLeap[74] = 0;//27029;
  YearIsLeap[75] = 0;//27394;
  YearIsLeap[76] = 1;//27759;
  YearIsLeap[77] = 0;//28125;
  YearIsLeap[78] = 0;//28490;
  YearIsLeap[79] = 0;//28855;
  YearIsLeap[80] = 1;//29220;
  YearIsLeap[81] = 0;//29586;
  YearIsLeap[82] = 0;//29951;
  YearIsLeap[83] = 0;//30316;
  YearIsLeap[84] = 1;//30681;
  YearIsLeap[85] = 0;//31047;
  YearIsLeap[86] = 0;//31412;
  YearIsLeap[87] = 0;//31777;
  YearIsLeap[88] = 1;//32142;
  YearIsLeap[89] = 0;//32508;
  YearIsLeap[90] = 0;//32873;
  YearIsLeap[91] = 0;//33238;
  YearIsLeap[92] = 1;//33603;
  YearIsLeap[93] = 0;//33969;
  YearIsLeap[94] = 0;//34334;
  YearIsLeap[95] = 0;//34699;
  YearIsLeap[96] = 1;//35064;
  YearIsLeap[97] = 0;//35430;
  YearIsLeap[98] = 0;//35795;
  YearIsLeap[99] = 0;//36160;
  YearIsLeap[100] = 1;// 36525;
  YearIsLeap[101] = 0;// 36891;
  YearIsLeap[102] = 0;// 37256;
  YearIsLeap[103] = 0;// 37621;
  YearIsLeap[104] = 1;// 37986;
  YearIsLeap[105] = 0;// 38352;
  YearIsLeap[106] = 0;//38717;
  YearIsLeap[107] = 0;//39082;
  YearIsLeap[108] = 1;//39447;
  YearIsLeap[109] = 0;//39813;
  YearIsLeap[110] = 0;//40178;
  YearIsLeap[111] = 0;//40543;
  YearIsLeap[112] = 1;//40908;
  YearIsLeap[113] = 0;//41274;
  YearIsLeap[114] = 0;//41639;
  YearIsLeap[115] = 0;//42004;
  YearIsLeap[116] = 1;//42369;
  YearIsLeap[117] = 0;//42735;
  YearIsLeap[118] = 0;//43100;
  YearIsLeap[119] = 0;//42735;
  YearIsLeap[120] = 1;//43830;

  return YearIsLeap[y-1900];
}


__device__ bondsDateStruct intializeDateKernelGpu(int d, int m, int y) 
{
  bondsDateStruct currDate;

  currDate.day = d;
  currDate.month = m;
  currDate.year = y;

  bool leap = isLeapKernelGpu(y);
  int offset = monthOffsetKernelGpu(m,leap);

  currDate.dateSerialNum = d + offset + yearOffsetKernelGpu(y);

  return currDate;
}


__device__ dataType yearFractionGpu(bondsDateStruct d1,
    bondsDateStruct d2, int dayCounter)
{
  return dayCountGpu(d1, d2, dayCounter) / (dataType)360.0; 
}


__device__ int dayCountGpu(bondsDateStruct d1, bondsDateStruct d2, int dayCounter) 
{
  if (dayCounter == USE_EXACT_DAY)
  {
    int dd1 = d1.day, dd2 = d2.day;
    int mm1 = d1.month, mm2 = d2.month;
    int yy1 = d1.year, yy2 = d2.year;

    if (dd2 == 31 && dd1 < 30) 
    { 
      dd2 = 1; mm2++; 
    }

    return 360*(yy2-yy1) + 30*(mm2-mm1-1) + MAX(0, 30-dd1) + MIN(30, dd2);
  }
  else
  {
    return (d2.dateSerialNum - d1.dateSerialNum);
  }
}


__device__ dataType couponNotionalGpu()
{
  return (dataType)100.0;
}

__device__ dataType bondNotionalGpu()
{
  return (dataType)100.0;
}


__device__ dataType fixedRateCouponNominalGpu()
{
  return (dataType)100.0;
}

__device__ bool eventHasOccurredGpu(bondsDateStruct currDate, bondsDateStruct eventDate)
{
  return eventDate.dateSerialNum > currDate.dateSerialNum;
}


__device__ bool cashFlowHasOccurredGpu(bondsDateStruct refDate, bondsDateStruct eventDate)
{
  return eventHasOccurredGpu(refDate, eventDate);
}


__device__ bondsDateStruct advanceDateGpu(bondsDateStruct date, int numMonthsAdvance) 
{
  int d = date.day;
  int m = date.month+numMonthsAdvance;
  int y = date.year;

  while (m > 12) 
  {
    m -= 12;
    y += 1;
  }

  while (m < 1) 
  {
    m += 12;
    y -= 1;
  }

  int length = monthLengthKernelGpu(m, isLeapKernelGpu(y));
  if (d > length)
    d = length;

  bondsDateStruct newDate = intializeDateKernelGpu(d, m, y);

  return newDate;
}

__device__ int getNumCashFlowsGpu(inArgsStruct inArgs, int bondNum)
{
  int numCashFlows = 0;

  //bondsDateStruct endDate = inArgs.bond[bondNum].maturityDate;
  bondsDateStruct currCashflowDate = inArgs.bond[bondNum].maturityDate;

  while (currCashflowDate.dateSerialNum > inArgs.bond[bondNum].startDate.dateSerialNum)
  {
    numCashFlows++;
    currCashflowDate = advanceDateGpu(currCashflowDate, -6); 
  }

  return numCashFlows+1;
}


__device__ dataType getDirtyPriceGpu(inArgsStruct inArgs, int bondNum, cashFlowsStruct cashFlows, int numLegs)
{
  dataType currentNotional = bondNotionalGpu();
  return discountingBondEngineCalculateSettlementValueGpu(inArgs, bondNum, cashFlows, numLegs) * (dataType)100.0 / currentNotional;
}


__device__ dataType getAccruedAmountGpu(inArgsStruct inArgs, bondsDateStruct date, int bondNum, cashFlowsStruct cashFlows, int numLegs)
{
  return bondAccruedAmountGpu(inArgs, date, bondNum, cashFlows, numLegs);
}


__device__ dataType discountingBondEngineCalculateSettlementValueGpu(inArgsStruct inArgs, int bondNum, cashFlowsStruct cashFlows, int numLegs)  
{

  bondsDateStruct currDate = inArgs.currDate[bondNum];

  if (currDate.dateSerialNum < inArgs.bond[bondNum].startDate.dateSerialNum)
  {
    currDate = inArgs.bond[bondNum].startDate;
  }

  return cashFlowsNpvGpu(cashFlows,
      inArgs.discountCurve[bondNum],
      false,
      currDate,
      currDate, 
      numLegs);
}


__device__ dataType bondAccruedAmountGpu(inArgsStruct inArgs, bondsDateStruct date, int bondNum, cashFlowsStruct cashFlows, int numLegs)
{
  dataType currentNotional = bondNotionalGpu();
  if (currentNotional == (dataType)0.0)
    return (dataType)0.0;

  return bondFunctionsAccruedAmountGpu(inArgs, date, bondNum, cashFlows, numLegs);
}


__device__ dataType bondFunctionsAccruedAmountGpu(inArgsStruct inArgs, bondsDateStruct date, int bondNum, cashFlowsStruct cashFlows, int numLegs) 
{
  return cashFlowsAccruedAmountGpu(cashFlows,
      false, date, numLegs, inArgs, bondNum) * (dataType)100.0 / bondNotionalGpu();
}


__device__ dataType cashFlowsAccruedAmountGpu(cashFlowsStruct cashFlows,
    bool includecurrDateFlows,
    bondsDateStruct currDate,
    int numLegs, inArgsStruct inArgs, int bondNum) 
{
  int legComputeNum = cashFlowsNextCashFlowNumGpu(cashFlows,
      currDate, numLegs); 

  dataType result = 0.0;

  int i;

  for (i = legComputeNum; i < (numLegs); ++i)
  {
    result += fixedRateCouponAccruedAmountGpu(cashFlows, i, currDate, inArgs, bondNum);
  }

  return result;
}


__device__ dataType fixedRateCouponAccruedAmountGpu(cashFlowsStruct cashFlows, int numLeg, bondsDateStruct d, inArgsStruct inArgs, int bondNum) 
{
  if (d.dateSerialNum <= cashFlows.legs[numLeg].accrualStartDate.dateSerialNum || d.dateSerialNum > inArgs.maturityDate[bondNum].dateSerialNum) 
  {
    return (dataType)0.0;
  }
  else
  {
    bondsDateStruct endDate = cashFlows.legs[numLeg].accrualEndDate;
    if (d.dateSerialNum < cashFlows.legs[numLeg].accrualEndDate.dateSerialNum)
    {
      endDate = d;
    }

    return fixedRateCouponNominalGpu()*(interestRateCompoundFactorGpu(cashFlows.intRate, 
           cashFlows.legs[numLeg].accrualStartDate, endDate, cashFlows.dayCounter) - (dataType)1.0);
  }
}


__device__ dataType cashFlowsNpvGpu(cashFlowsStruct cashFlows,
    bondsYieldTermStruct discountCurve,
    bool includecurrDateFlows,
    bondsDateStruct currDate,
    bondsDateStruct npvDate,
    int numLegs) 
{
  npvDate = currDate;

  dataType totalNPV = 0.0;

  int i;

  for (i=0; i<numLegs; ++i) {
    if (!(cashFlowHasOccurredGpu(cashFlows.legs[i].paymentDate, currDate)))
      totalNPV += fixedRateCouponAmountGpu(cashFlows, i) *
        bondsYieldTermStructureDiscountGpu(discountCurve, cashFlows.legs[i].paymentDate);
  }

  return totalNPV/bondsYieldTermStructureDiscountGpu(discountCurve, npvDate);
}


__device__ dataType bondsYieldTermStructureDiscountGpu(bondsYieldTermStruct ytStruct, bondsDateStruct t)
{
  ytStruct.intRate.rate = ytStruct.forward;
  ytStruct.intRate.freq = ytStruct.frequency;
  ytStruct.intRate.comp = ytStruct.compounding;
  return flatForwardDiscountImplGpu(ytStruct.intRate, yearFractionGpu(ytStruct.refDate, t, ytStruct.dayCounter));
}


__device__ dataType flatForwardDiscountImplGpu(intRateStruct intRate, dataType t) 
{
  return interestRateDiscountFactorGpu(intRate, t);
}


__device__ dataType interestRateDiscountFactorGpu(intRateStruct intRate, dataType t) 
{
  return (dataType)1.0/interestRateCompoundFactorGpuTwoArgs(intRate, t);
}


__device__ dataType interestRateCompoundFactorGpuTwoArgs(intRateStruct intRate, dataType t) 
{
  if (intRate.comp == SIMPLE_INTEREST)
    return (dataType)1.0 + intRate.rate*t;
  else if (intRate.comp == COMPOUNDED_INTEREST)
    return pow((dataType)1.0+intRate.rate/intRate.freq, intRate.freq*t);
  else if (intRate.comp == CONTINUOUS_INTEREST)
    return exp(intRate.rate*t);
  return (dataType)0.0;
}


__device__ dataType fixedRateCouponAmountGpu(cashFlowsStruct cashFlows, int numLeg) 
{
  if (cashFlows.legs[numLeg].amount == COMPUTE_AMOUNT)
  {
    return fixedRateCouponNominalGpu()*(interestRateCompoundFactorGpu(cashFlows.intRate, cashFlows.legs[numLeg].accrualStartDate,
          cashFlows.legs[numLeg].accrualEndDate, cashFlows.dayCounter) - (dataType)1.0);
  }
  else
  {
    return cashFlows.legs[numLeg].amount;
  }
}

__device__ dataType interestRateCompoundFactorGpu(intRateStruct intRate, bondsDateStruct d1,
    bondsDateStruct d2, int dayCounter)
{
  dataType t = yearFractionGpu(d1, d2, dayCounter);
  return interestRateCompoundFactorGpuTwoArgs(intRate, t);
}


__device__ dataType interestRateImpliedRateGpu(dataType compound,                                        
    int comp,
    dataType freq,
    dataType t) 
{
  dataType r = 0.0f;
  if (compound==(dataType)1.0) 
  {
    r = 0.0;
  } 
  else 
  {
    switch (comp) 
    {
      case SIMPLE_INTEREST:
        r = (compound - (dataType)1.0)/t;
        break;
      case COMPOUNDED_INTEREST:
        r = (pow((dataType)compound, (dataType)1.0/((freq)*t))-(dataType)1.0)*(freq);
        break;
    }
  }

  return r;
}


__device__ couponStruct cashFlowsNextCashFlowGpu(cashFlowsStruct cashFlows,
    bondsDateStruct currDate,
    int numLegs) 
{
  int i;
  for (i = 0; i < numLegs; ++i) 
  {
    if ( ! (cashFlowHasOccurredGpu(cashFlows.legs[i].paymentDate, currDate) ))
      return cashFlows.legs[i];
  }
  return cashFlows.legs[numLegs-1];
}


__device__ int cashFlowsNextCashFlowNumGpu(cashFlowsStruct cashFlows,
    bondsDateStruct currDate,
    int numLegs) 
{
  int i;
  for (i = 0; i < numLegs; ++i) 
  {
    if ( ! (cashFlowHasOccurredGpu(cashFlows.legs[i].paymentDate, currDate) ))
      return i;
  }

  return (numLegs-1);
}


__device__ dataType getBondYieldGpu(dataType cleanPrice,
    int dc,
    int comp,
    dataType freq,
    bondsDateStruct settlement,
    dataType accuracy,
    int maxEvaluations,
    inArgsStruct currInArgs, int bondNum, cashFlowsStruct cashFlows, int numLegs)
{
  dataType currentNotional = bondNotionalGpu();

  if (currentNotional == (dataType)0.0)
    return (dataType)0.0;

  if (currInArgs.bond[bondNum].startDate.dateSerialNum > settlement.dateSerialNum)
  {
    settlement = currInArgs.bond[bondNum].startDate;
  }

  return getBondFunctionsYieldGpu(cleanPrice, dc, comp, freq,
      settlement, accuracy, maxEvaluations,
      currInArgs, bondNum, cashFlows, numLegs);
}


__device__ dataType getBondFunctionsYieldGpu(dataType cleanPrice,
    int dc,
    int comp,
    dataType freq,
    bondsDateStruct settlement,
    dataType accuracy,
    int maxEvaluations,
    inArgsStruct currInArgs, int bondNum, cashFlowsStruct cashFlows, int numLegs)
{
  dataType dirtyPrice = cleanPrice + bondFunctionsAccruedAmountGpu(currInArgs, settlement, bondNum, cashFlows, numLegs); 
  dirtyPrice /= (dataType)100.0 / bondNotionalGpu();

  return getCashFlowsYieldGpu(cashFlows, dirtyPrice,
      dc, comp, freq,
      false, settlement, settlement, numLegs,
      accuracy, maxEvaluations, (dataType)0.05);
}


__device__ dataType getCashFlowsYieldGpu(cashFlowsStruct leg,
    dataType npv,
    int dayCounter,
    int compounding,
    dataType frequency,
    bool includecurrDateFlows,
    bondsDateStruct currDate,
    bondsDateStruct npvDate,
    int numLegs,
    dataType accuracy,
    int maxIterations,
    dataType guess)
{
  //Brent solver;
  solverStruct solver;
  solver.maxEvaluations_ = maxIterations;
  irrFinderStruct objFunction;

  objFunction.npv = npv;
  objFunction.dayCounter = dayCounter;
  objFunction.comp = compounding;
  objFunction.freq = frequency;
  objFunction.includecurrDateFlows = includecurrDateFlows;
  objFunction.currDate = currDate;
  objFunction.npvDate = npvDate;

  return solverSolveGpu(solver, objFunction, accuracy, guess, guess/(dataType)10.0, leg, numLegs);
}


__device__ dataType solverSolveGpu(solverStruct solver,
    irrFinderStruct f,
    dataType accuracy,
    dataType guess,
    dataType step,
    cashFlowsStruct cashFlows,
    int numLegs)
{
  // check whether we really want to use epsilon
  accuracy = MAX(accuracy, QL_EPSILON_GPU);

  dataType growthFactor = (dataType)1.6;
  int flipflop = -1;

  solver.root_ = guess;
  solver.fxMax_ = fOpGpu(f, solver.root_, cashFlows, numLegs);

  // monotonically crescent bias, as in optionValue(volatility)
  if (closeGpu(solver.fxMax_,(dataType)0.0))
  {
    return solver.root_;
  }
  else if (closeGpu(solver.fxMax_, (dataType)0.0)) 
  {
    solver.xMin_ = /*enforceBounds*/(solver.root_ - step);
    solver.fxMin_ = fOpGpu(f, solver.xMin_, cashFlows, numLegs);
    solver.xMax_ = solver.root_;
  } 
  else 
  {
    solver.xMin_ = solver.root_;
    solver.fxMin_ = solver.fxMax_;
    solver.xMax_ = /*enforceBounds*/(solver.root_+step);
    solver.fxMax_ = fOpGpu(f, solver.xMax_, cashFlows, numLegs);
  }

  solver.evaluationNumber_ = 2;
  while (solver.evaluationNumber_ <= solver.maxEvaluations_) 
  {
    if (solver.fxMin_*solver.fxMax_ <= (dataType)0.0) 
    {
      if (closeGpu(solver.fxMin_, (dataType)0.0))
        return solver.xMin_;
      if (closeGpu(solver.fxMax_, (dataType)0.0))
        return solver.xMax_;
      solver.root_ = (solver.xMax_+solver.xMin_)/(dataType)2.0;
      return solveImplGpu(solver, f, accuracy, cashFlows, numLegs);
    }
    if (fabs(solver.fxMin_) < fabs(solver.fxMax_)) 
    {
      solver.xMin_ = /*enforceBounds*/(solver.xMin_+growthFactor*(solver.xMin_ - solver.xMax_));
      solver.fxMin_= fOpGpu(f, solver.xMin_, cashFlows, numLegs);
    } 
    else if (fabs(solver.fxMin_) > fabs(solver.fxMax_)) 
    {
      solver.xMax_ = /*enforceBounds*/(solver.xMax_+growthFactor*(solver.xMax_ - solver.xMin_));
      solver.fxMax_= fOpGpu(f, solver.xMax_, cashFlows, numLegs);
    } 
    else if (flipflop == -1) 
    {
      solver.xMin_ = /*enforceBounds*/(solver.xMin_+growthFactor*(solver.xMin_ - solver.xMax_));
      solver.fxMin_= fOpGpu(f, solver.xMin_, cashFlows, numLegs);
      solver.evaluationNumber_++;
      flipflop = 1;
    } 
    else if (flipflop == 1) 
    {
      solver.xMax_ = /*enforceBounds*/(solver.xMax_+growthFactor*(solver.xMax_ - solver.xMin_));
      solver.fxMax_= fOpGpu(f, solver.xMax_, cashFlows, numLegs);
      flipflop = -1;
    }
    solver.evaluationNumber_++;
  }

  return (dataType)0.0;
}


__device__ dataType cashFlowsNpvYieldGpu(cashFlowsStruct cashFlows,
    intRateStruct y,
    bool includecurrDateFlows,
    bondsDateStruct currDate,
    bondsDateStruct npvDate,
    int numLegs) 
{
  dataType npv = 0.0;
  dataType discount = 1.0;
  bondsDateStruct lastDate;
  bool first = true;

  int i;
  for (i=0; i<numLegs; ++i) 
  {
    if (cashFlowHasOccurredGpu(cashFlows.legs[i].paymentDate, currDate))
      continue;

    bondsDateStruct couponDate = cashFlows.legs[i].paymentDate;
    dataType amount = fixedRateCouponAmountGpu(cashFlows, i);
    if (first) 
    {
      first = false;
      if (i > 0) {
        lastDate = advanceDateGpu(cashFlows.legs[i].paymentDate, -1*6); 
      } else {
        lastDate = cashFlows.legs[i].accrualStartDate;
      }
      discount *= interestRateDiscountFactorGpu(y, yearFractionGpu(npvDate, couponDate, y.dayCounter));
    } 
    else  
    {
      discount *= interestRateDiscountFactorGpu(y, yearFractionGpu(lastDate, couponDate, y.dayCounter));
    }

    lastDate = couponDate;

    npv += amount * discount;
  }

  return npv;
}

__device__ dataType fOpGpu(irrFinderStruct f, dataType y, cashFlowsStruct cashFlows, int numLegs)
{
  intRateStruct yield;

  yield.rate = y;
  yield.comp = f.comp;
  yield.freq = f.freq;
  yield.dayCounter = f.dayCounter;

  dataType NPV = cashFlowsNpvYieldGpu(cashFlows,
      yield,
      false,
      f.currDate,
      f.npvDate, numLegs);

  return (f.npv - NPV);
}



__device__ dataType fDerivativeGpu(irrFinderStruct f, dataType y, cashFlowsStruct cashFlows, int numLegs)
{
  intRateStruct yield;
  yield.rate = y;
  yield.dayCounter = f.dayCounter;
  yield.comp = f.comp;
  yield.freq = f.freq;

  return modifiedDurationGpu(cashFlows, yield,
      f.includecurrDateFlows,
      f.currDate, f.npvDate, numLegs);
}


__device__ bool closeGpu(dataType x, dataType y)
{
  return closeGpuThreeArgs(x,y,42);
}


__device__ bool closeGpuThreeArgs(dataType x, dataType y, int n)
{
  dataType diff = fabs(x-y);
  dataType tolerance = n*QL_EPSILON_GPU;

  return diff <= tolerance*fabs(x) &&
    diff <= tolerance*fabs(y);
}


__device__ dataType enforceBoundsGpu(dataType x)
{
  return x;
}


__device__ dataType solveImplGpu(solverStruct solver, irrFinderStruct f,
    dataType xAccuracy, cashFlowsStruct cashFlows, int numLegs)
{
  dataType froot, dfroot, dx, dxold;
  dataType xh, xl;

  // Orient the search so that f(xl) < 0
  if (solver.fxMin_ < (dataType)0.0) 
  {
    xl = solver.xMin_;
    xh = solver.xMax_;
  } 
  else 
  {
    xh = solver.xMin_;
    xl = solver.xMax_;
  }

  // the "stepsize before last"
  dxold = solver.xMax_ - solver.xMin_;
  // it was dxold=std::fabs(xMax_-xMin_); in Numerical Recipes
  // here (xMax_-xMin_ > 0) is verified in the constructor

  // and the last step
  dx = dxold;

  froot = fOpGpu(f, solver.root_, cashFlows, numLegs);
  dfroot = fDerivativeGpu(f, solver.root_, cashFlows, numLegs);

  ++solver.evaluationNumber_;

  while (solver.evaluationNumber_<=solver.maxEvaluations_) 
  {
    // Bisect if (out of range || not decreasing fast enough)
    if ((((solver.root_-xh)*dfroot-froot)*
          ((solver.root_-xl)*dfroot-froot) > (dataType)0.0)
        || (fabs((dataType)2.0*froot) > fabs(dxold*dfroot))) 
    {
      dxold = dx;
      dx = (xh-xl)/(dataType)2.0;
      solver.root_=xl+dx;
    } 
    else 
    {
      dxold = dx;
      dx = froot/dfroot;
      solver.root_ -= dx;
    }

    // Convergence criterion
    if (fabs(dx) < xAccuracy)
      return solver.root_;
    froot = fOpGpu(f, solver.root_, cashFlows, numLegs);
    dfroot = fDerivativeGpu(f, solver.root_, cashFlows, numLegs);
    ++solver.evaluationNumber_;
    if (froot < (dataType)0.0)
      xl=solver.root_;
    else
      xh=solver.root_;
  }

  return solver.root_;
}


__device__ dataType modifiedDurationGpu(cashFlowsStruct cashFlows,
    intRateStruct y,
    bool includecurrDateFlows,
    bondsDateStruct currDate,
    bondsDateStruct npvDate,
    int numLegs)
{
  dataType P = 0.0;
  dataType dPdy = 0.0;
  dataType r = y.rate;
  dataType N = y.freq;
  int dc = y.dayCounter;

  int i;
  for (i=0; i<numLegs; ++i) 
  {
    if (!cashFlowHasOccurredGpu(cashFlows.legs[i].paymentDate, currDate)) 
    {
      dataType t = yearFractionGpu(npvDate,
          cashFlows.legs[i].paymentDate, dc);
      dataType c = fixedRateCouponAmountGpu(cashFlows, i);  
      dataType B = interestRateDiscountFactorGpu(y, t); 

      P += c * B;
      if (y.comp == SIMPLE_INTEREST)
        dPdy -= c * B*B * t;
      if (y.comp == COMPOUNDED_INTEREST)
        dPdy -= c * t * B/(1+r/N);
      if (y.comp == CONTINUOUS_INTEREST)
        dPdy -= c * B * t;
      if (y.comp == SIMPLE_THEN_COMPOUNDED_INTEREST)
      {
        if (t<=(dataType)1.0/N)
          dPdy -= c * B*B * t;
        else
          dPdy -= c * t * B/((dataType)1+r/N);
      }
    }
  }

  if (P == (dataType)0.0) // no cashflows
  {
    return (dataType)0.0;
  }
  return (-1*dPdy)/P; // reverse derivative sign
}


__global__ void bonds(inArgsStruct inArgs, resultsStruct results, int n)
{
  int bondNum = blockIdx.x*blockDim.x + threadIdx.x;
  if (bondNum < n)
  {
    int numLegs;

    int numCashFlows = 0;

    //bondsDateStruct endDate = inArgs.bond[bondNum].maturityDate;
    bondsDateStruct currCashflowDate = inArgs.bond[bondNum].maturityDate;

    while (currCashflowDate.dateSerialNum > inArgs.bond[bondNum].startDate.dateSerialNum)
    {
      numCashFlows++;
      currCashflowDate = advanceDateGpu(currCashflowDate, -6); 
    }

    numLegs = numCashFlows+1;

    cashFlowsStruct cashFlows; 
    couponStruct cashLegs[9];
    cashFlows.legs = cashLegs;

    cashFlows.intRate.dayCounter = USE_EXACT_DAY;
    cashFlows.intRate.rate  = inArgs.bond[bondNum].rate;
    cashFlows.intRate.freq  = ANNUAL_FREQ;
    cashFlows.intRate.comp  = SIMPLE_INTEREST;
    cashFlows.dayCounter  = USE_EXACT_DAY;
    cashFlows.nominal  = (dataType)100.0;

    //bondsDateStruct currPaymentDate;
    bondsDateStruct currStartDate = advanceDateGpu(inArgs.bond[bondNum].maturityDate, (numLegs - 1)*-6);
    bondsDateStruct currEndDate = advanceDateGpu(currStartDate, 6); 

    int cashFlowNum;
    for (cashFlowNum = 0; cashFlowNum < numLegs-1; cashFlowNum++)
    {
      cashFlows.legs[cashFlowNum].paymentDate = currEndDate;


      cashFlows.legs[cashFlowNum].accrualStartDate  = currStartDate;
      cashFlows.legs[cashFlowNum].accrualEndDate  = currEndDate;

      cashFlows.legs[cashFlowNum].amount = COMPUTE_AMOUNT;

      currStartDate = currEndDate;
      currEndDate = advanceDateGpu(currEndDate, 6); 
    }

    cashFlows.legs[numLegs-1].paymentDate  = inArgs.bond[bondNum].maturityDate;
    cashFlows.legs[numLegs-1].accrualStartDate = inArgs.currDate[bondNum];
    cashFlows.legs[numLegs-1].accrualEndDate  = inArgs.currDate[bondNum];
    cashFlows.legs[numLegs-1].amount = (dataType)100.0;

    results.bondForwardVal[bondNum] = getBondYieldGpu(inArgs.bondCleanPrice[bondNum],
        USE_EXACT_DAY,
        COMPOUNDED_INTEREST,
        (dataType)2.0,
        inArgs.currDate[bondNum],
        ACCURACY,
        100,
        inArgs, bondNum, cashFlows, numLegs);
    inArgs.discountCurve[bondNum].forward = results.bondForwardVal[bondNum];
    results.dirtyPrice[bondNum] = getDirtyPriceGpu(inArgs, bondNum, cashFlows, numLegs);
    results.accruedAmountCurrDate[bondNum] = getAccruedAmountGpu(inArgs, inArgs.currDate[bondNum], bondNum, cashFlows, numLegs);
    results.cleanPrice[bondNum] = results.dirtyPrice[bondNum] - results.accruedAmountCurrDate[bondNum];
  }
}

long getBondsResultsGpu(inArgsStruct inArgsHost, resultsStruct resultsFromGpu, int numBonds)
{
  bondsYieldTermStruct* discountCurveGpu;
  bondsYieldTermStruct* repoCurveGpu;
  bondsDateStruct* currDateGpu;
  bondsDateStruct* maturityDateGpu;
  dataType* bondCleanPriceGpu;
  bondStruct* bondGpu;
  dataType* dummyStrikeGpu;

  dataType* dirtyPriceGpu;
  dataType* accruedAmountCurrDateGpu;
  dataType* cleanPriceGpu;
  dataType* bondForwardValGpu;

  hipMalloc((void**)&discountCurveGpu, numBonds*sizeof(bondsYieldTermStruct));
  hipMalloc((void**)&repoCurveGpu, numBonds*sizeof(bondsYieldTermStruct));
  hipMalloc((void**)&currDateGpu, numBonds*sizeof(bondsDateStruct));
  hipMalloc((void**)&maturityDateGpu, numBonds*sizeof(bondsDateStruct));
  hipMalloc((void**)&bondCleanPriceGpu, numBonds*sizeof(dataType));
  hipMalloc((void**)&bondGpu, numBonds*sizeof(bondStruct));
  hipMalloc((void**)&dummyStrikeGpu, numBonds*sizeof(dataType));

  hipMalloc((void**)&dirtyPriceGpu, numBonds*sizeof(dataType));
  hipMalloc((void**)&accruedAmountCurrDateGpu, numBonds*sizeof(dataType));
  hipMalloc((void**)&cleanPriceGpu, numBonds*sizeof(dataType));
  hipMalloc((void**)&bondForwardValGpu, numBonds*sizeof(dataType));

  hipMemcpy(discountCurveGpu, inArgsHost.discountCurve, numBonds*sizeof(bondsYieldTermStruct), hipMemcpyHostToDevice);
  hipMemcpy(repoCurveGpu, inArgsHost.repoCurve, numBonds*sizeof(bondsYieldTermStruct), hipMemcpyHostToDevice);
  hipMemcpy(currDateGpu, inArgsHost.currDate, numBonds*sizeof(bondsDateStruct), hipMemcpyHostToDevice);
  hipMemcpy(maturityDateGpu, inArgsHost.maturityDate, numBonds*sizeof(bondsDateStruct), hipMemcpyHostToDevice);
  hipMemcpy(bondCleanPriceGpu, inArgsHost.bondCleanPrice, numBonds*sizeof(dataType), hipMemcpyHostToDevice);
  hipMemcpy(bondGpu, inArgsHost.bond, numBonds*sizeof(bondStruct), hipMemcpyHostToDevice);
  hipMemcpy(dummyStrikeGpu, inArgsHost.dummyStrike, numBonds*sizeof(dataType), hipMemcpyHostToDevice);

  inArgsStruct inArgs;
  inArgs.discountCurve    = discountCurveGpu;
  inArgs.repoCurve        = repoCurveGpu;
  inArgs.currDate   = currDateGpu;
  inArgs.maturityDate     = maturityDateGpu;
  inArgs.bondCleanPrice   = bondCleanPriceGpu;
  inArgs.bond             = bondGpu;
  inArgs.dummyStrike      = dummyStrikeGpu;

  resultsStruct results;
  results.dirtyPrice                = dirtyPriceGpu;
  results.accruedAmountCurrDate  = accruedAmountCurrDateGpu;
  results.cleanPrice                = cleanPriceGpu;
  results.bondForwardVal         = bondForwardValGpu;

  dim3  grid((ceil(((float)numBonds)/((float)256.0f))), 1, 1);
  dim3  threads(256, 1, 1);

  struct timeval start;
  struct timeval end;
  gettimeofday(&start, NULL);

  bonds <<< dim3(grid), dim3(threads ) >>> (inArgs, results, numBonds);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  long seconds  = end.tv_sec  - start.tv_sec;
  long useconds = end.tv_usec - start.tv_usec;
  long ktime = seconds * 1e6 + useconds;

  hipMemcpy(resultsFromGpu.dirtyPrice, dirtyPriceGpu, numBonds*sizeof(dataType), hipMemcpyDeviceToHost);
  hipMemcpy(resultsFromGpu.accruedAmountCurrDate, accruedAmountCurrDateGpu, numBonds*sizeof(dataType), hipMemcpyDeviceToHost);
  hipMemcpy(resultsFromGpu.cleanPrice, cleanPriceGpu, numBonds*sizeof(dataType), hipMemcpyDeviceToHost);
  hipMemcpy(resultsFromGpu.bondForwardVal, bondForwardValGpu, numBonds*sizeof(dataType), hipMemcpyDeviceToHost);

  hipFree(discountCurveGpu);
  hipFree(repoCurveGpu);
  hipFree(currDateGpu);
  hipFree(maturityDateGpu);
  hipFree(bondCleanPriceGpu);
  hipFree(bondGpu);
  hipFree(dummyStrikeGpu);

  hipFree(dirtyPriceGpu);
  hipFree(accruedAmountCurrDateGpu);
  hipFree(cleanPriceGpu);
  hipFree(bondForwardValGpu);
  return ktime;
}
